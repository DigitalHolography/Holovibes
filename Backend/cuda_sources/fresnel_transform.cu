#include "fresnel_transform.cuh"

#include <hipfft/hipfftXt.h>

#include "apply_mask.cuh"
#include "common.cuh"
#include "cuda_memory.cuh"
#include "frame_desc.hh"
#include "masks.cuh"
#include "unique_ptr.hh"

using camera::FrameDescriptor;

void fresnel_transform_lens(hipComplex* lens,
                            const uint lens_side_size,
                            const uint frame_height,
                            const uint frame_width,
                            const float lambda,
                            const float z,
                            const float pixel_size,
                            const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(lens_side_size * lens_side_size, threads);

    hipComplex* square_lens;
    // In anamorphic mode, the lens is initally a square, it's then cropped to
    // be the same dimension as the frame
    if (frame_height != frame_width)
        cudaXMalloc(&square_lens, lens_side_size * lens_side_size * sizeof(hipComplex));
    else
        square_lens = lens;

    kernel_quadratic_lens<<<blocks, threads, 0, stream>>>(square_lens, lens_side_size, lambda, z, pixel_size);
    cudaCheckError();

    if (frame_height != frame_width)
    {
        // Data is contiguous for a horizontal frame so a simple memcpy with an
        // offset and a limited size works
        if (frame_width > frame_height)
            cudaXMemcpyAsync(lens,
                             square_lens + ((lens_side_size - frame_height) / 2) * frame_width,
                             frame_width * frame_height * sizeof(hipComplex),
                             hipMemcpyDeviceToDevice,
                             stream);
        else
        {
            // For a vertical frame we need memcpy 2d to copy row by row, taking
            // the offset into account every time
            cudaSafeCall(hipMemcpy2DAsync(lens,                            // Destination (frame)
                                           frame_width * sizeof(hipComplex), // Destination width in byte
                                           square_lens + ((lens_side_size - frame_width) / 2), // Source (lens)
                                           lens_side_size * sizeof(hipComplex),                 // Source width in byte
                                           frame_width * sizeof(hipComplex), // Destination width in byte
                                                                            // (yes it's redoundant)
                                           frame_height,                    // Destination height (not in byte)
                                           hipMemcpyDeviceToDevice,
                                           stream));
        }
        cudaXFree(square_lens);
    }
}

void fresnel_transform(hipComplex* input,
                       hipComplex* output,
                       const uint batch_size,
                       const hipComplex* lens,
                       const hipfftHandle plan2D,
                       const size_t frame_resolution,
                       const hipStream_t stream)
{
    apply_mask(input, lens, output, frame_resolution, batch_size, stream);

    // No sync needed between kernel call and cufft call
    cudaCheckError();
    // FFT

    cufftSafeCall(hipfftXtExec(plan2D, input, output, HIPFFT_FORWARD));
    // Same, no sync needed since everything is executed on the stream 0

    cudaCheckError();
}
