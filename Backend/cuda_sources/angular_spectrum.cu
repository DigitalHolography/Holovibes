#include "angular_spectrum.cuh"

#include <hipfft/hipfftXt.h>

#include "apply_mask.cuh"
#include "cuda_memory.cuh"
#include "frame_desc.hh"
#include "masks.cuh"
#include "shift_corners.cuh"
#include "tools_compute.cuh"

using camera::FrameDescriptor;

void angular_spectrum_lens(hipFloatComplex* output,
                           const int Nx,
                           const int Ny,
                           const float z,
                           const float lambda,
                           const float x_step,
                           const float y_step,
                           const hipStream_t stream)
{

    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(1 + (Nx - 1) / threads_2d, 1 + (Ny - 1) / threads_2d);

    kernel_spectral_lens<<<lblocks, lthreads, 0, stream>>>(output, Nx, Ny, z, lambda, x_step, y_step);
    cudaXStreamSynchronize(stream);
    cudaCheckError();
}

void angular_spectrum(hipComplex* input,
                      hipComplex* output,
                      const uint batch_size,
                      const hipComplex* lens,
                      hipComplex* mask_output,
                      bool store_frame,
                      const hipfftHandle plan2d,
                      const FrameDescriptor& fd,
                      const hipStream_t stream)
{
    const uint frame_res = fd.get_frame_res();
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_res, threads);

    cufftSafeCall(hipfftXtExec(plan2d, input, input, HIPFFT_FORWARD));

    // Lens and Mask already shifted
    // thus we don't have to shift the 'input' buffer each time
    apply_mask(input, lens, output, frame_res, batch_size, stream);
    if (store_frame)
    {
        cudaXMemcpyAsync(mask_output, input, frame_res * sizeof(hipComplex), hipMemcpyDeviceToDevice, stream);
    }

    cudaCheckError();

    cufftSafeCall(hipfftXtExec(plan2d, input, input, HIPFFT_BACKWARD));

    kernel_complex_divide<<<blocks, threads, 0, stream>>>(input, frame_res, static_cast<float>(frame_res), batch_size);
    cudaCheckError();
}
