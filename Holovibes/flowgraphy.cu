#include "hip/hip_runtime.h"
#include <>

#include "flowgraphy.cuh"
#include "hardware_limits.hh"
#include "tools.hh"

__global__ void kernel_flowgraphy(
	hipfftComplex* input,
	const hipfftComplex* gpu_special_queue,
	const unsigned int gpu_special_queue_buffer_length,
	const hipfftComplex* gpu_special_queue_end,
	const unsigned int start_index,
	const unsigned int max_index,
	const unsigned int frame_resolution,
	const unsigned int i_width,
	const unsigned int nsamples,
	const unsigned int n_i)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int size = frame_resolution * nsamples;

	while (index < frame_resolution)
	{
		hipfftComplex M = make_hipComplex(0, 0);
		hipfftComplex D = make_hipComplex(0, 0);
		int deplacement = (index  + (1 + i_width + ((1 + start_index) % max_index) *  frame_resolution) * (nsamples / 2)) % gpu_special_queue_buffer_length;
		hipfftComplex b = gpu_special_queue[deplacement];

		for (int k = 0; k < nsamples; ++k)
		for (int j = 0; j < nsamples; ++j)
		for (int i = 0; i < nsamples; ++i)
		{
			deplacement = (index + i + (j * i_width) + (((k + start_index) % max_index) * frame_resolution)) % gpu_special_queue_buffer_length; // while x while y, on peut virer le modulo
			hipfftComplex a = gpu_special_queue[deplacement];
			M.x += a.x;
			M.y += a.y;
			D.x += std::sqrt(pow((a.x - b.x), 2) + pow((a.y - b.y), 2)); // |a - b|
		}
		M.x += (n_i * b.x);
		M.y += (n_i * b.y);
		M.x /= D.x;
		M.y /= D.x;
		M.x = pow(M.x, 2);
		M.y = pow(M.y, 2);
	/*	float tmp = pow(M.x, 2) + pow(M.y, 2);
		M.x = (M.x * D.x) / tmp;
		M.y = (M.y * D.x) / tmp;
		tmp = M.x;
		M.x = pow(M.x, 2) - pow(M.y, 2);
		M.y = 2 * tmp * M.y;*/
		input[index] = M;
		index += blockDim.x * gridDim.x;
	}
}


void convolution_flowgraphy(
	hipfftComplex* input,
	hipfftComplex* gpu_special_queue,
	unsigned int &gpu_special_queue_start_index,
	const unsigned int gpu_special_queue_max_index,
	const unsigned int frame_resolution,
	const unsigned int frame_width,
	const unsigned int nframes,
	hipStream_t stream)
{
	// const unsigned int n_frame_resolution = frame_resolution * nframes;
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);


	hipStreamSynchronize(stream);

	if (gpu_special_queue_start_index == 0)
		gpu_special_queue_start_index = gpu_special_queue_max_index - 1;
	else
		--gpu_special_queue_start_index;
	hipMemcpy(
		gpu_special_queue + frame_resolution * gpu_special_queue_start_index,
		input,
		sizeof(hipfftComplex) * frame_resolution,
		hipMemcpyDeviceToDevice);

	unsigned int n = pow(nframes, 3) - 3;
	unsigned int  gpu_special_queue_buffer_length = gpu_special_queue_max_index * frame_resolution;
	hipfftComplex* gpu_special_queue_end = gpu_special_queue + gpu_special_queue_buffer_length;

	kernel_flowgraphy << <blocks, threads, 0, stream >> >(
		input,
		gpu_special_queue,
		gpu_special_queue_buffer_length,
		gpu_special_queue_end,
		gpu_special_queue_start_index,
		gpu_special_queue_max_index,
		frame_resolution,
		frame_width,
		nframes,
		n
		);

	hipStreamSynchronize(stream);
}