#include "hip/hip_runtime.h"
#include "focus.cuh"

static __global__ void kernel_complex_2_to_pow(hipfftComplex* input,unsigned int size, int power)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    input[index].x = pow((double)input[index].x, power);
    input[index].y = input[index].x;
    index += blockDim.x * gridDim.x;
  }
}

void complex_to_pow(hipfftComplex *input, unsigned int size, int power)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;
  kernel_complex_2_to_pow<<<blocks,threads>>>(input, size, power);
}

static __global__  void kernel_average_complex(hipfftComplex* input, float *output, unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    atomicAdd(output, input[index].x);
  }
}

float average_complex_2d(hipfftComplex* input, unsigned int size) // <>
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;
  float sum_value;
  kernel_average_complex<<<blocks,threads>>>(input, &sum_value, size);
  return (sum_value / (float)size);
}


/*hipfftComplex *output; // should be done once
hipMalloc(&output, 2048 * 2048 * sizeof(hipfftComplex)); //should be done once
hipMemset(output, 0, 2048 * 2048 * sizeof(hipfftComplex)); // should be done everytime the s zone*/

void extract_s(hipfftComplex* input, hipfftComplex *output,
  unsigned int size, holovibes::Rectangle& s_coord,
  camera::FrameDescriptor input_fd)
{
  int start_x = s_coord.top_left.x;
  int start_y = s_coord.top_left.y;
  int end_x = s_coord.top_right.x;
  int end_y = s_coord.bottom_left.y;
  int s_size_x = end_x - start_x;
  int y_output = 0;
  for (int y = start_y; y < end_y; y++)
  {
    hipMemcpy(output + y_output * input_fd.width,
      input + y * input_fd.width + start_x,
      s_size_x * sizeof(hipfftComplex),
      hipMemcpyDeviceToDevice);
    y_output++;
  }
}

float compute_v(hipfftComplex *s, unsigned int size) //extracted s
{
  hipfftComplex *square_s;
  hipMalloc(&square_s, size * sizeof(hipfftComplex));
  hipMemcpy(square_s, s, size * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
  complex_to_pow(square_s, size, 2);
  float s_av = average_complex_2d(s, size);// <I>
  float s_square_av = average_complex_2d(square_s, size);//<I^2>
  hipFree(square_s);
  return (s_square_av - (s_av * s_av));
}

static __global__ void kernel_multiply_matrixes(hipfftComplex *input1, hipfftComplex *input2,
  hipfftComplex *output, unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    output[index].x = input1[index].x * input2[index].x;
    output[index].y = input1[index].y * input2[index].y;
  }
}
// plan should be allocated at size of
void convolution(hipfftComplex *x, hipfftComplex *k, hipfftHandle plan2d, unsigned int size)
{
  hipfftComplex *to_invert;
  hipMalloc(&to_invert, size);
  hipfftExecC2C(plan2d, x, x, HIPFFT_FORWARD);
  hipfftExecC2C(plan2d, k, k, HIPFFT_FORWARD);
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;
  kernel_multiply_matrixes<<<blocks,threads>>>(x, k, to_invert, size);
  hipfftExecC2C(plan2d, to_invert, to_invert, HIPFFT_BACKWARD);
  hipFree(to_invert);
}



