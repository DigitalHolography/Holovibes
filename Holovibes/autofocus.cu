#include "hip/hip_runtime.h"
#include "autofocus.cuh"

# include <hip/hip_runtime.h>
# include ""
# include "hardware_limits.hh"
# include "tools.cuh"
# include "average.cuh"

/* -- REMOVE THIS -- */
# include <iostream>

static float global_variance_intensity(
  const float* input,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  float* squared_input;
  hipMalloc<float>(&squared_input, size * sizeof(float));

  kernel_multiply_frames_float <<<blocks, threads>>>(input, input, squared_input, size);

  float average_squared_input = average_operator(squared_input, size);
  float average_input = average_operator(input, size);
  float average_input_squared = average_input * average_input;

  float global_variance = average_squared_input - average_input_squared;

  hipFree(squared_input);

  return global_variance;
}

static __global__ void kernel_float_to_complex(
  const float* input,
  hipfftComplex* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    output[index].x = input[index];
    output[index].y = input[index];
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_minus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    output[index] = input_left[index] - input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static float average_local_variance(
  const float* input,
  const unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* ke matrix with same size than input */
  hipfftComplex* ke_gpu_frame;
  size_t ke_gpu_frame_pitch;

  /* Allocates memory for ke_gpu_frame. */
  hipMallocPitch(&ke_gpu_frame,
    &ke_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ke_gpu_frame,
    ke_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    hipfftComplex ke_19;
    ke_19.x = sqrtf(1.0f / 9.0f);
    ke_19.y = sqrtf(1.0f / 9.0f);

    /* Build the ke 3x3 matrix */
    hipfftComplex ke_cpu[9];
    for (int i = 0; i < 9; ++i)
      ke_cpu[i] = ke_19;

    /* Copy the ke matrix to ke_gpu_frame. */
    hipMemcpy2D(
      ke_gpu_frame,
      ke_gpu_frame_pitch,
      ke_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex <<<blocks, threads>>>(input, input_complex, size);

  /* Allocation of convolution i * ke output */
  float* i_ke_convolution;
  hipMalloc(&i_ke_convolution, size * sizeof(float));

  /* Allocation of convolution i^2 * ke output */
  float* i2_ke_convolution;
  hipMalloc(&i2_ke_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ke. */
  convolution_operator(
    input_complex,
    ke_gpu_frame,
    i_ke_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * ke)^2 */
  kernel_multiply_frames_float<<<blocks, threads>>>(
    i_ke_convolution,
    i_ke_convolution,
    i_ke_convolution,
    size);

  /* Compute i^2 * ke. */
  kernel_multiply_frames_complex<<<blocks, threads>>>(
    input_complex,
    input_complex,
    input_complex,
    size);

  convolution_operator(
    input_complex,
    ke_gpu_frame,
    i2_ke_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i^2 * ke) - (i * ke)^2 */
  kernel_minus_operator<<<blocks, threads>>>(
    i2_ke_convolution,
    i_ke_convolution,
    i_ke_convolution,
    size);

  hipDeviceSynchronize();

  float average_local_variance = average_operator(i_ke_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i2_ke_convolution);
  hipFree(i_ke_convolution);

  hipFree(input_complex);
  hipFree(ke_gpu_frame);

  return average_local_variance;
}

/* TODO: Fix me */
float focus_metric(
  float* input,
  unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  float global_variance = global_variance_intensity(input, size);
  float avr_local_variance = average_local_variance(input, square_size);

  std::cout << "global variance: " << global_variance << std::endl;
  std::cout << "avr_local_variance: " << avr_local_variance << std::endl;

  return global_variance * avr_local_variance;
}
