#include "hip/hip_runtime.h"
#include "autofocus.cuh"

# include <hip/hip_runtime.h>
# include ""
# include "hardware_limits.hh"
# include "tools.cuh"
# include "average.cuh"

/* -- REMOVE THIS -- */
# include <iostream>

static float global_variance_intensity(
  const float* input,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  float* squared_input;
  hipMalloc<float>(&squared_input, size * sizeof(float));

  kernel_multiply_frames_float <<<blocks, threads>>>(input, input, squared_input, size);

  float average_squared_input = average_operator(squared_input, size);
  float average_input = average_operator(input, size);
  float average_input_squared = average_input * average_input;

  float global_variance = average_squared_input - average_input_squared;

  hipFree(squared_input);

  return global_variance;
}

static __global__ void kernel_float_to_complex(
  const float* input,
  hipfftComplex* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    output[index].x = input[index];
    output[index].y = input[index];
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_minus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    output[index] = input_left[index] - input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static float average_local_variance(
  const float* input,
  const unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* ke matrix with same size than input */
  hipfftComplex* ke_gpu_frame;
  size_t ke_gpu_frame_pitch;

  /* Allocates memory for ke_gpu_frame. */
  hipMallocPitch(&ke_gpu_frame,
    &ke_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ke_gpu_frame,
    ke_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    hipfftComplex ke_19;
    ke_19.x = sqrtf(1.0f / 9.0f);
    ke_19.y = sqrtf(1.0f / 9.0f);

    /* Build the ke 3x3 matrix */
    hipfftComplex ke_cpu[9];
    for (int i = 0; i < 9; ++i)
      ke_cpu[i] = ke_19;

    /* Copy the ke matrix to ke_gpu_frame. */
    hipMemcpy2D(
      ke_gpu_frame,
      ke_gpu_frame_pitch,
      ke_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex <<<blocks, threads>>>(input, input_complex, size);

  /* Allocation of convolution i * ke output */
  float* i_ke_convolution;
  hipMalloc(&i_ke_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ke. */
  convolution_operator(
    input_complex,
    ke_gpu_frame,
    i_ke_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute i - i * ke. */
  kernel_minus_operator<<<blocks, threads>>>(
    input,
    i_ke_convolution,
    i_ke_convolution,
    size);

  /* Compute (i - i * ke)^2 */
  kernel_multiply_frames_float<<<blocks, threads>>>(
    i_ke_convolution,
    i_ke_convolution,
    i_ke_convolution,
    size);

  hipDeviceSynchronize();

  float average_local_variance = average_operator(i_ke_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i_ke_convolution);

  hipFree(input_complex);
  hipFree(ke_gpu_frame);

  return average_local_variance;
}

static __global__ void kernel_plus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    output[index] = input_left[index] + input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_sqrt_operator(
  const float* input,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    output[index] = sqrtf(input[index]);
    index += blockDim.x * gridDim.x;
  }
}

static float sobel_operator(
  const float* input,
  unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* ks matrix with same size than input */
  hipfftComplex* ks_gpu_frame;
  size_t ks_gpu_frame_pitch;

  /* Allocates memory for ks_gpu_frame. */
  hipMallocPitch(&ks_gpu_frame,
    &ks_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ks_gpu_frame,
    ks_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    const float coeff = 1.0f / (2.0f * sqrtf(3.0f));

    /* Build the ks 3x3 matrix */
    float ks_cpu[9] =
    {
      1.0f, 0.0f, -1.0f,
      2.0f, 0.0f, -2.0f,
      1.0f, 0.0f, -1.0f
    };

    hipfftComplex ks_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      ks_cpu[i] *= coeff;
      ks_complex_cpu[i].x = ks_cpu[i];
      ks_complex_cpu[i].y = ks_cpu[i];
    }

    /* Copy the ks matrix to ks_gpu_frame. */
    hipMemcpy2D(
      ks_gpu_frame,
      ks_gpu_frame_pitch,
      ks_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  /* kst matrix with same size than input */
  hipfftComplex* kst_gpu_frame;
  size_t kst_gpu_frame_pitch;

  /* Allocates memory for kst_gpu_frame. */
  hipMallocPitch(&kst_gpu_frame,
    &kst_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    kst_gpu_frame,
    kst_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    const float coeff = 1.0f / (2.0f * sqrtf(3.0f));

    /* Build the kst 3x3 matrix */
    float kst_cpu[9] =
    {
      1.0f, 2.0f, 1.0f,
      0.0f, 0.0f, 0.0f,
      -1.0f, -2.0f, -1.0f
    };

    hipfftComplex kst_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      kst_cpu[i] *= coeff;
      kst_complex_cpu[i].x = kst_cpu[i];
      kst_complex_cpu[i].y = kst_cpu[i];
    }

    /* Copy the kst matrix to kst_gpu_frame. */
    hipMemcpy2D(
      kst_gpu_frame,
      kst_gpu_frame_pitch,
      kst_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex <<<blocks, threads>>>(input, input_complex, size);

  /* Allocation of convolution i * ks output */
  float* i_ks_convolution;
  hipMalloc(&i_ks_convolution, size * sizeof(float));

  /* Allocation of convolution i * kst output */
  float* i_kst_convolution;
  hipMalloc(&i_kst_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ks. */
  convolution_operator(
    input_complex,
    ks_gpu_frame,
    i_ks_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * ks)^2 */
  kernel_multiply_frames_float<<<blocks, threads>>>(
    i_ks_convolution,
    i_ks_convolution,
    i_ks_convolution,
    size);

  /* Compute i * kst. */
  convolution_operator(
    input_complex,
    kst_gpu_frame,
    i_kst_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * kst)^2 */
  kernel_multiply_frames_float<<<blocks, threads>>>(
    i_kst_convolution,
    i_kst_convolution,
    i_kst_convolution,
    size);

  /* Compute (i * ks)^2 - (i * kst)^2 */
  kernel_plus_operator<<<blocks, threads>>>(
    i_ks_convolution,
    i_kst_convolution,
    i_ks_convolution,
    size);

  kernel_sqrt_operator<<<blocks, threads>>>(
    i_ks_convolution,
    i_ks_convolution,
    size);

  hipDeviceSynchronize();

  float average_magnitude = average_operator(i_ks_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i_ks_convolution);
  hipFree(i_kst_convolution);

  hipFree(input_complex);

  hipFree(kst_gpu_frame);
  hipFree(ks_gpu_frame);

  return average_magnitude;
}

float focus_metric(
  float* input,
  unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* Divide each pixels to avoid higher values than float can contains. */
  kernel_float_divide<<<blocks, threads>>>(input, size, size);

  float global_variance = global_variance_intensity(input, size);
  float avr_local_variance = average_local_variance(input, square_size);
  float avr_magnitude = sobel_operator(input, square_size);

  return global_variance * avr_local_variance * avr_magnitude;
}
