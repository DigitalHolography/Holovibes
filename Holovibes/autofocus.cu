#include "hip/hip_runtime.h"
# include <stdio.h>
# include <hip/hip_runtime.h>

# include "autofocus.cuh"
# include ""
# include "hardware_limits.hh"
# include "tools.cuh"
# include "tools.hh"
# include "tools_compute.cuh"
# include "average.cuh"

static __global__ void kernel_minus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input_left[index] - input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static float global_variance_intensity(
  const float* input,
  const unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  // <I>
  const float average_input = average_operator(input, size);

  // We create a matrix of <I> in order to do the substraction
  float* matrix_average;
  hipMalloc(&matrix_average, size * sizeof(float));

  float* cpu_average_matrix = (float *)malloc(sizeof(float)* size);
  for (unsigned int i = 0; i < size; ++i)
    cpu_average_matrix[i] = average_input;

  hipMemcpy(matrix_average, cpu_average_matrix, size * sizeof(float), hipMemcpyHostToDevice);

  // I - <I>
  kernel_minus_operator << <blocks, threads >> >(input, matrix_average, matrix_average, size);

  // We take it to the power of 2
  kernel_multiply_frames_float << <blocks, threads >> >(matrix_average, matrix_average, matrix_average, size);

  // And we take the average
  const float global_variance = average_operator(matrix_average, size);

  hipFree(matrix_average);

  return global_variance;
}

static __global__ void kernel_float_to_complex(
  const float* input,
  hipfftComplex* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index].x = input[index];
    output[index].y = input[index];
    index += blockDim.x * gridDim.x;
  }
}

#include <iostream>
static float average_local_variance(
  const float* input,
  const unsigned int square_size,
  const unsigned int mat_size)
{
  const unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  /* ke matrix with same size than input */
  hipfftComplex* ke_gpu_frame;
  size_t ke_gpu_frame_pitch;

  /* Allocates memory for ke_gpu_frame. */
  hipMallocPitch(&ke_gpu_frame,
    &ke_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ke_gpu_frame,
    ke_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    const unsigned square_mat_size = mat_size * mat_size;
    hipfftComplex* ke_complex_cpu = (hipfftComplex*) malloc(sizeof(hipfftComplex) * square_mat_size);
    for (int i = 0; i < square_mat_size; ++i)
    {
      ke_complex_cpu[i].x = 1 / square_mat_size;
      ke_complex_cpu[i].y = 1 / square_mat_size;
    }

    /* Copy the ke matrix to ke_gpu_frame. */
    hipMemcpy2D(
      ke_gpu_frame,
      ke_gpu_frame_pitch,
      ke_complex_cpu,
      mat_size * sizeof(hipfftComplex),
      mat_size * sizeof(hipfftComplex),
      mat_size,
      hipMemcpyHostToDevice);

    free(ke_complex_cpu);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex << <blocks, threads >> >(input, input_complex, size);

  /* Allocation of convolution i * ke output */
  float* i_ke_convolution;
  hipMalloc(&i_ke_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ke. */
  convolution_operator(
    input_complex,
    ke_gpu_frame,
    i_ke_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute i - i * ke. */
  kernel_minus_operator << <blocks, threads >> >(
    input,
    i_ke_convolution,
    i_ke_convolution,
    size);

  /* Compute (i - i * ke)^2 */
  kernel_multiply_frames_float << <blocks, threads >> >(
    i_ke_convolution,
    i_ke_convolution,
    i_ke_convolution,
    size);

  hipDeviceSynchronize();

  const float average_local_variance = average_operator(i_ke_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i_ke_convolution);

  hipFree(input_complex);
  hipFree(ke_gpu_frame);

  return average_local_variance;
}

static __global__ void kernel_plus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input_left[index] + input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_sqrt_operator(
  const float* input,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = sqrtf(input[index]);
    index += blockDim.x * gridDim.x;
  }
}

static float sobel_operator(
  const float* input,
  const unsigned int square_size)
{
  const unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  /* ks matrix with same size than input */
  hipfftComplex* ks_gpu_frame;
  size_t ks_gpu_frame_pitch;

  /* Allocates memory for ks_gpu_frame. */
  hipMallocPitch(&ks_gpu_frame,
    &ks_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ks_gpu_frame,
    ks_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    /* Build the ks 3x3 matrix */
    float ks_cpu[9] =
    {
      1.0f, 0.0f, -1.0f,
      2.0f, 0.0f, -2.0f,
      1.0f, 0.0f, -1.0f
    };

    hipfftComplex ks_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      ks_complex_cpu[i].x = ks_cpu[i];
	  ks_complex_cpu[i].y = 0;//ks_cpu[i];
    }

    /* Copy the ks matrix to ks_gpu_frame. */
    hipMemcpy2D(
      ks_gpu_frame,
      ks_gpu_frame_pitch,
      ks_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  /* kst matrix with same size than input */
  hipfftComplex* kst_gpu_frame;
  size_t kst_gpu_frame_pitch;

  /* Allocates memory for kst_gpu_frame. */
  hipMallocPitch(&kst_gpu_frame,
    &kst_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    kst_gpu_frame,
    kst_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    /* Build the kst 3x3 matrix */
    float kst_cpu[9] =
    {
      1.0f, 2.0f, 1.0f,
      0.0f, 0.0f, 0.0f,
      -1.0f, -2.0f, -1.0f
    };

    hipfftComplex kst_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      kst_complex_cpu[i].x = kst_cpu[i];
      kst_complex_cpu[i].y = kst_cpu[i];
    }

    /* Copy the kst matrix to kst_gpu_frame. */
    hipMemcpy2D(
      kst_gpu_frame,
      kst_gpu_frame_pitch,
      kst_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex << <blocks, threads >> >(input, input_complex, size);

  /* Allocation of convolution i * ks output */
  float* i_ks_convolution;
  hipMalloc(&i_ks_convolution, size * sizeof(float));

  /* Allocation of convolution i * kst output */
  float* i_kst_convolution;
  hipMalloc(&i_kst_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ks. */
  convolution_operator(
    input_complex,
    ks_gpu_frame,
    i_ks_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * ks)^2 */
  kernel_multiply_frames_float << <blocks, threads >> >(
    i_ks_convolution,
    i_ks_convolution,
    i_ks_convolution,
    size);

  /* Compute i * kst. */
  convolution_operator(
    input_complex,
    kst_gpu_frame,
    i_kst_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * kst)^2 */
  kernel_multiply_frames_float << <blocks, threads >> >(
    i_kst_convolution,
    i_kst_convolution,
    i_kst_convolution,
    size);

  /* Compute (i * ks)^2 - (i * kst)^2 */
  kernel_plus_operator << <blocks, threads >> >(
    i_ks_convolution,
    i_kst_convolution,
    i_ks_convolution,
    size);

  kernel_sqrt_operator << <blocks, threads >> >(
    i_ks_convolution,
    i_ks_convolution,
    size);

  hipDeviceSynchronize();

  const float average_magnitude = average_operator(i_ks_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i_ks_convolution);
  hipFree(i_kst_convolution);

  hipFree(input_complex);

  hipFree(kst_gpu_frame);
  hipFree(ks_gpu_frame);

  // HEHEHEHEHEHEHEHEH
  return 1.0f / average_magnitude;
}

float focus_metric(
  float* input,
  const unsigned int square_size,
  hipStream_t stream,
  const unsigned int local_var_size)
{
  const unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  /* Divide each pixels to avoid higher values than float can contains. */
  kernel_float_divide << <blocks, threads, 0, stream >> >(input, size, static_cast<float>(size));

  const float global_variance = global_variance_intensity(input, size);
  const float avr_local_variance = average_local_variance(input, square_size, local_var_size);
  const float avr_magnitude = sobel_operator(input, square_size);

  return global_variance * avr_local_variance * avr_magnitude;
}