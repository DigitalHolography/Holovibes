#include "hip/hip_runtime.h"
/*! \file */
#include "autofocus.cuh"

# include <hip/hip_runtime.h>
# include ""
# include "hardware_limits.hh"
# include "tools.cuh"
# include "tools_multiply.cuh"
# include "tools_divide.cuh"
# include "average.cuh"
# include <stdio.h>

static __global__ void kernel_minus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input_left[index] - input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static float global_variance_intensity(
  const float* input,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  // <I>
  const float average_input = average_operator(input, size);

  // We create a matrix of <I> in order to do the substraction
  float* matrix_average;
  hipMalloc(&matrix_average, size * sizeof(float));

  float* cpu_average_matrix = (float *)malloc(sizeof(float) * size);
  for (unsigned int i = 0; i < size; ++i)
    cpu_average_matrix[i] = average_input;

  hipMemcpy(matrix_average, cpu_average_matrix, size * sizeof(float), hipMemcpyHostToDevice);

  // I - <I>
  kernel_minus_operator << <blocks, threads >> >(input, matrix_average, matrix_average, size);

  // We take it to the power of 2
  kernel_multiply_frames_float << <blocks, threads >> >(matrix_average, matrix_average, matrix_average, size);

  // And we take the average
  const float global_variance = average_operator(matrix_average, size);

  hipFree(matrix_average);

  return global_variance;
}

static __global__ void kernel_float_to_complex(
  const float* input,
  hipfftComplex* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index].x = input[index];
    output[index].y = input[index];
    index += blockDim.x * gridDim.x;
  }
}

static float average_local_variance(
  const float* input,
  const unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* ke matrix with same size than input */
  hipfftComplex* ke_gpu_frame;
  size_t ke_gpu_frame_pitch;

  /* Allocates memory for ke_gpu_frame. */
  hipMallocPitch(&ke_gpu_frame,
    &ke_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ke_gpu_frame,
    ke_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    /* Build the 3x3 matrix */
    float ke_cpu[9] =
    {
      1.0f, 1.0f, 1.0f,
      1.0f, 1.0f, 1.0f,
      1.0f, 1.0f, 1.0f
    };

    hipfftComplex ke_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      ke_complex_cpu[i].x = ke_cpu[i];
      ke_complex_cpu[i].y = ke_cpu[i];
    }

    /* Copy the ke matrix to ke_gpu_frame. */
    hipMemcpy2D(
      ke_gpu_frame,
      ke_gpu_frame_pitch,
      ke_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex << <blocks, threads >> >(input, input_complex, size);

  /* Allocation of convolution i * ke output */
  float* i_ke_convolution;
  hipMalloc(&i_ke_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ke. */
  convolution_operator(
    input_complex,
    ke_gpu_frame,
    i_ke_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute i - i * ke. */
  kernel_minus_operator << <blocks, threads >> >(
    input,
    i_ke_convolution,
    i_ke_convolution,
    size);

  /* Compute (i - i * ke)^2 */
  kernel_multiply_frames_float << <blocks, threads >> >(
    i_ke_convolution,
    i_ke_convolution,
    i_ke_convolution,
    size);

  hipDeviceSynchronize();

  const float average_local_variance = average_operator(i_ke_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i_ke_convolution);

  hipFree(input_complex);
  hipFree(ke_gpu_frame);


  return average_local_variance;
}

static __global__ void kernel_plus_operator(
  const float* input_left,
  const float* input_right,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input_left[index] + input_right[index];
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_sqrt_operator(
  const float* input,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = sqrtf(input[index]);
    index += blockDim.x * gridDim.x;
  }
}

static float sobel_operator(
  const float* input,
  unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* ks matrix with same size than input */
  hipfftComplex* ks_gpu_frame;
  size_t ks_gpu_frame_pitch;

  /* Allocates memory for ks_gpu_frame. */
  hipMallocPitch(&ks_gpu_frame,
    &ks_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    ks_gpu_frame,
    ks_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    /* Build the ks 3x3 matrix */
    float ks_cpu[9] =
    {
      1.0f, 0.0f, -1.0f,
      2.0f, 0.0f, -2.0f,
      1.0f, 0.0f, -1.0f
    };

    hipfftComplex ks_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      ks_complex_cpu[i].x = ks_cpu[i];
      ks_complex_cpu[i].y = ks_cpu[i];
    }

    /* Copy the ks matrix to ks_gpu_frame. */
    hipMemcpy2D(
      ks_gpu_frame,
      ks_gpu_frame_pitch,
      ks_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  /* kst matrix with same size than input */
  hipfftComplex* kst_gpu_frame;
  size_t kst_gpu_frame_pitch;

  /* Allocates memory for kst_gpu_frame. */
  hipMallocPitch(&kst_gpu_frame,
    &kst_gpu_frame_pitch,
    square_size * sizeof(hipfftComplex),
    square_size);
  hipMemset2D(
    kst_gpu_frame,
    kst_gpu_frame_pitch,
    0,
    square_size * sizeof(hipfftComplex),
    square_size);

  {
    /* Build the kst 3x3 matrix */
    float kst_cpu[9] =
    {
      1.0f, 2.0f, 1.0f,
      0.0f, 0.0f, 0.0f,
      -1.0f, -2.0f, -1.0f
    };

    hipfftComplex kst_complex_cpu[9];
    for (int i = 0; i < 9; ++i)
    {
      kst_complex_cpu[i].x = kst_cpu[i];
      kst_complex_cpu[i].y = kst_cpu[i];
    }

    /* Copy the kst matrix to kst_gpu_frame. */
    hipMemcpy2D(
      kst_gpu_frame,
      kst_gpu_frame_pitch,
      kst_complex_cpu,
      3 * sizeof(hipfftComplex),
      3 * sizeof(hipfftComplex),
      3,
      hipMemcpyHostToDevice);
  }

  hipfftComplex* input_complex;
  hipMalloc(&input_complex, size * sizeof(hipfftComplex));

  /* Convert input float frame to complex frame. */
  kernel_float_to_complex << <blocks, threads >> >(input, input_complex, size);

  /* Allocation of convolution i * ks output */
  float* i_ks_convolution;
  hipMalloc(&i_ks_convolution, size * sizeof(float));

  /* Allocation of convolution i * kst output */
  float* i_kst_convolution;
  hipMalloc(&i_kst_convolution, size * sizeof(float));

  hipfftHandle plan2d_x;
  hipfftHandle plan2d_k;
  hipfftPlan2d(&plan2d_x, square_size, square_size, HIPFFT_C2C);
  hipfftPlan2d(&plan2d_k, square_size, square_size, HIPFFT_C2C);

  /* Compute i * ks. */
  convolution_operator(
    input_complex,
    ks_gpu_frame,
    i_ks_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * ks)^2 */
  kernel_multiply_frames_float << <blocks, threads >> >(
    i_ks_convolution,
    i_ks_convolution,
    i_ks_convolution,
    size);

  /* Compute i * kst. */
  convolution_operator(
    input_complex,
    kst_gpu_frame,
    i_kst_convolution,
    size,
    plan2d_x,
    plan2d_k);

  /* Compute (i * kst)^2 */
  kernel_multiply_frames_float << <blocks, threads >> >(
    i_kst_convolution,
    i_kst_convolution,
    i_kst_convolution,
    size);

  /* Compute (i * ks)^2 - (i * kst)^2 */
  kernel_plus_operator << <blocks, threads >> >(
    i_ks_convolution,
    i_kst_convolution,
    i_ks_convolution,
    size);

  kernel_sqrt_operator << <blocks, threads >> >(
    i_ks_convolution,
    i_ks_convolution,
    size);

  hipDeviceSynchronize();

  const float average_magnitude = average_operator(i_ks_convolution, size);

  /* -- Free ressources -- */
  hipfftDestroy(plan2d_x);
  hipfftDestroy(plan2d_k);

  hipFree(i_ks_convolution);
  hipFree(i_kst_convolution);

  hipFree(input_complex);

  hipFree(kst_gpu_frame);
  hipFree(ks_gpu_frame);

  // HEHEHEHEHEHEHEHEH
  return 1.0f / average_magnitude;
}



float focus_metric(
  float* input,
  unsigned int square_size)
{
  unsigned int size = square_size * square_size;
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* Divide each pixels to avoid higher values than float can contains. */
  kernel_float_divide << <blocks, threads >> >(input, size, static_cast<float>(size));

  const float global_variance = global_variance_intensity(input, size);
  const float avr_local_variance = average_local_variance(input, square_size);
  const float avr_magnitude = sobel_operator(input, square_size);

  return global_variance * avr_local_variance * avr_magnitude;
}
