#include "hip/hip_runtime.h"
#include <>
#include <cmath>
# ifndef _USE_MATH_DEFINES
/* Enables math constants. */
#  define _USE_MATH_DEFINES
# endif /* !_USE_MATH_DEFINES */

#include "tools_unwrap.cuh"

__global__ void kernel_extract_angle(
  const hipfftComplex* input,
  float* output,
  const size_t size)
{
  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= size)
    return;

  // We use std::atan2 in order to obtain results in [-pi; pi].
  output[index] = std::atan2(input[index].y, input[index].x);
}

__global__ void kernel_unwrap(
  float* pred,
  float* cur,
  float* output,
  const size_t size)
{
  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= size)
    return;
  const float pi = M_PI;

  float local_diff = cur[index] - pred[index];
  // Unwrapping //
  float local_adjust;
  if (local_diff > pi)
    local_adjust = -2.f * pi;
  else if (local_diff < -pi)
    local_adjust = 2.f * pi;
  else
    local_adjust = 0.f;

  // Cumulating each angle with its correction
  output[index] = cur[index] + local_adjust;
}

__global__ void kernel_compute_angle_mult(
  const hipfftComplex* pred,
  const hipfftComplex* cur,
  float* output,
  const size_t size)
{
  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= size)
    return;

  hipfftComplex conj_prod;
  conj_prod = cur[index];

  conj_prod.x *= pred[index].x;
  conj_prod.x += cur[index].y * pred[index].y;

  conj_prod.y *= pred[index].x;
  conj_prod.y -= cur[index].x * pred[index].y;

  output[index] = std::atan2(conj_prod.y, conj_prod.x);
}

__global__ void kernel_compute_angle_diff(
  const hipfftComplex* pred,
  const hipfftComplex* cur,
  float* output,
  const size_t size)
{
  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= size)
    return;

  hipfftComplex diff;
  diff = cur[index];
  diff.x -= pred[index].x;
  diff.y -= pred[index].y;

  output[index] = std::atan2(diff.y, diff.x);
}

__global__ void kernel_correct_angles(
  float* data,
  const float* corrections,
  const size_t image_size,
  const size_t history_size)
{
  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= image_size)
    return;

  for (auto correction_idx = index;
    correction_idx < history_size * image_size;
    correction_idx += image_size)
  {
    data[index] += corrections[correction_idx];
  }
}

__global__ void kernel_init_unwrap_2d(
	unsigned int width,
	unsigned int height,
	unsigned int frame_res,
	float *input,
	float *fx,
	float *fy,
	hipfftComplex *z)
{
	const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < frame_res)
	{
		const unsigned int j = index / width;
		const unsigned int i = index % width;
		
		/*fx and fy init*/

		fx[index] = i - roundf(width / 2);
		fy[index] = j - roundf(width / 2);

		/*z init*/

		z[index].x = cosf(input[index]);
		z[index].y = sinf(input[index]);
	/*	const float modulus = sqrtf(input[index].x * input[index].x + input[index].y * input[index].y);
		if (modulus == 0)
		{
			z[index].x = 0;
			z[index].y = 0;
		}
		else
		{
			z[index].x = input[index].x / modulus;
			z[index].y = input[index].y / modulus;
		}*/
	}
}