#include "demodulation.cuh"
#include "hardware_limits.hh"
#include "tools.cuh"
#include "tools.hh"


void demodulation(
	hipfftComplex* input,
	hipfftComplex*                   stft_buf,
	hipfftComplex*                   stft_dup_buf,
	const hipfftHandle  plan1d,
	const unsigned int frame_resolution,
	const unsigned int nframes,
	hipStream_t stream)
{
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);

	hipMemcpyAsync(stft_buf,
		&(stft_buf[1]),
		sizeof(hipfftComplex)* (nframes * frame_resolution - 1),
		hipMemcpyDeviceToDevice,
		stream);

	// Do the ROI
	kernel_bursting << <blocks, threads, 0, stream >> >(
		input,
		frame_resolution,
		nframes,
		stft_buf
		);

	// FFT 1D
	hipfftExecC2C(plan1d, stft_buf, stft_dup_buf, HIPFFT_FORWARD);
	hipStreamSynchronize(stream);
}