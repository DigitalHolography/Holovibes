#include "hip/hip_runtime.h"
#include "vibrometry.cuh"
#include <>


__global__ void kernel_vibro(hipfftComplex *image_p, hipfftComplex *image_q, hipfftComplex *output, unsigned int nb_pixels)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < nb_pixels)
  {
    output[index].x = image_p[index].x / image_q[index].x;
    output[index].y = image_p[index].y / image_q[index].y;
    index += blockDim.x * gridDim.x;
  }
}

hipfftComplex *vibrometry(unsigned int p, unsigned int q, hipfftComplex *images, const camera::FrameDescriptor fd)
{
  unsigned int pixels = fd.frame_res();
  unsigned int size = fd.frame_size();
  hipfftComplex *output;
  hipMalloc(&output, size);
  hipfftComplex *image_p = images + pixels * p;
  hipfftComplex *image_q = images + pixels * q;

  unsigned int threads = get_max_threads_1d();
  unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks - 1;
  kernel_vibro<<<blocks,threads>>>(image_p, image_q, output, pixels);
  return output;
}

