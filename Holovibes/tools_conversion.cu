#include "hip/hip_runtime.h"
#include <algorithm>
#include <>

#include "tools_conversion.cuh"
#include "hardware_limits.hh"
#include "tools.hh"
#include <iostream>

__global__ void img8_to_complex(
  hipfftComplex* output,
  const unsigned char* input,
  const unsigned int size,
  const float* sqrt_array)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    // Image rescaling on 2^16 colors (65535 / 255 = 257)
    float val  = sqrt(static_cast<float>(2 * (input[index] * 257)));
	output[index].x = val;
	output[index].y = val;
    index += blockDim.x * gridDim.x;
  }
}

//TODO:   removed the sqrt_array from computation and it seems more effective without.
//        need to do proper benchmarks.

__global__ void img16_to_complex(
  hipfftComplex* output,
  const unsigned short* input,
  const unsigned int size,
  const float* sqrt_array)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
	  float val = sqrt(2 * static_cast<float>(input[index]));
	  output[index].x = val;
	  output[index].y = val;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void float_to_complex(
	hipfftComplex* output,
	const float* input,
	const unsigned int size,
	const float* sqrt_array)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		float val = input[index];
		if (val < 0)
		{
			val = abs(val);
			val = sqrtf(val);
			val *= -1;
		}
		else
			val = sqrtf(val);
		output[index].x = val;
		output[index].y = val;
		index += blockDim.x * gridDim.x;
	}
}

/* Kernel function wrapped by complex_to_modulus. */
static __global__ void kernel_complex_to_modulus(
  const hipfftComplex* input,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = hypotf(input[index].x, input[index].y);

    index += blockDim.x * gridDim.x;
  }
}

void complex_to_modulus(
  const hipfftComplex* input,
  float* output,
  const unsigned int size,
  hipStream_t stream)
{
  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(size, threads);

  kernel_complex_to_modulus << <blocks, threads, 0, stream >> >(input, output, size);
}

/* Kernel function wrapped in complex_to_squared_modulus. */
static __global__ void kernel_complex_to_squared_modulus(
  const hipfftComplex* input,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
	  output[index] = pow(hypotf(input[index].x, input[index].y), 2);//input[index].x * input[index].x + input[index].y * input[index].y;
    index += blockDim.x * gridDim.x;
  }
}

void complex_to_squared_modulus(
  const  hipfftComplex* input,
  float* output,
  const unsigned int size,
  hipStream_t stream)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  kernel_complex_to_squared_modulus << <blocks, threads, 0, stream >> >(input, output, size);
}

/* Kernel function wrapped in complex_to_argument. */
static __global__ void kernel_complex_to_argument(
  const hipfftComplex* input,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  const float pi_div_2 = M_PI / 2.0f;
  const float c = 65535.0f / M_PI;

  while (index < size)
  {
    output[index] = (atanf(input[index].y / input[index].x) + pi_div_2) * c;

    index += blockDim.x * gridDim.x;
  }
}

void complex_to_argument(
  const hipfftComplex* input,
  float* output,
  const unsigned int size,
  hipStream_t stream)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  kernel_complex_to_argument << <blocks, threads, 0, stream >> >(input, output, size);
}

/* Find the minimum and the maximum of a floating-point array.
 *
 * The minimum and maximum can't be computed directly, because blocks
 * cannot communicate. Hence we compute local minima and maxima and
 * put them in two arrays.
 *
 * \param Size Number of threads in a block for this kernel.
 * Also, it's the size of min and max.
 * \param min Array of Size floats, which will contain local minima.
 * \param max Array of Size floats, which will contain local maxima.
 */
template <unsigned Size>
static __global__ void kernel_minmax(
  const float* data,
  const size_t size,
  float* min,
  float* max)
{
  __shared__ float local_min[Size];
  __shared__ float local_max[Size];

  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index > size)
    return;
  local_min[threadIdx.x] = data[index];
  local_max[threadIdx.x] = data[index];

  __syncthreads();

  if (threadIdx.x == 0)
  {
    /* Accumulate the results of the neighbors, computing min-max values,
     * and store them in the first element of local arrays. */
    for (auto i = 1; i < Size; ++i)
    {
      if (local_min[i] < local_min[0])
        local_min[0] = local_min[i];
      if (local_max[i] > local_max[0])
        local_max[0] = local_max[i];
    }
    min[blockIdx.x] = local_min[0];
    max[blockIdx.x] = local_max[0];
  }
}

template <typename T>
static __global__ void kernel_rescale(T* data,
  const size_t size,
  const T min,
  const T max,
  const T new_max)
{
  const unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index > size)
    return;

  data[index] = (data[index] + fabsf(min)) * new_max / (fabsf(max) + fabsf(min));
}

void rescale_float(
  const float* input,
  float* output,
  const unsigned int size,
  hipStream_t stream)
{
  const unsigned threads = 128;
  unsigned blocks = map_blocks_to_problem(size, threads);

  // TODO : See if gpu_float_buffer_ could be used directly.
  hipMemcpy(output, input, sizeof(float)* size, hipMemcpyDeviceToDevice);

  // Computing minimum and maximum values, in order to rescale properly.
  float* gpu_local_mins;
  float* gpu_local_maxs;
  hipMalloc(&gpu_local_mins, sizeof(float)* blocks);
  hipMalloc(&gpu_local_maxs, sizeof(float)* blocks);

  /* We have to hardcode the template parameter, unfortunately.
   * It must be equal to the number of threads per block. */
  kernel_minmax <128> << <blocks, threads, threads * 2, stream >> > (output,
    size,
    gpu_local_mins,
    gpu_local_maxs);

  float* cpu_local_mins = new float[blocks];
  float* cpu_local_maxs = new float[blocks];
  hipMemcpy(cpu_local_mins, gpu_local_mins, sizeof(float)* blocks, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_local_maxs, gpu_local_maxs, sizeof(float)* blocks, hipMemcpyDeviceToHost);

  const float max_intensity = 65535.f;
  kernel_rescale << <blocks, threads, 0, stream >> >(
    output,
    size,
    *(std::min_element(cpu_local_mins, cpu_local_mins + threads)),
    *(std::max_element(cpu_local_maxs, cpu_local_maxs + threads)),
    max_intensity);

  hipFree(gpu_local_mins);
  hipFree(gpu_local_maxs);
}

/*! \brief Kernel function wrapped in endianness_conversion, making
 ** the call easier
 **/
static __global__ void kernel_endianness_conversion(
  const unsigned short* input,
  unsigned short* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = (input[index] << 8) | (input[index] >> 8);

    index += blockDim.x * gridDim.x;
  }
}

void endianness_conversion(
  const unsigned short* input,
  unsigned short* output,
  const unsigned int size,
  hipStream_t stream)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  kernel_endianness_conversion << <blocks, threads, 0, stream >> >(input, output, size);
}

/*! \brief Kernel function wrapped in float_to_ushort, making
 ** the call easier
 **/
static __global__ void kernel_float_to_ushort(
  const float* input,
  unsigned short* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    if (input[index] > 65535.0f)
      output[index] = 65535;
    else if (input[index] < 0.0f)
      output[index] = 0;
    else
      output[index] = static_cast<unsigned short>(input[index]);

    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_complex_to_ushort(
	const hipfftComplex* input,
	unsigned int * output,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		unsigned short x = 0;
		unsigned short y = 0;
		if (input[index].x > 65535.0f)
			x = 65535;
		else if (input[index].x >= 1.0f)
		x = static_cast<unsigned short>(pow(input[index].x, 2));
		
		if (input[index].y > 65535.0f)
			y = 65535;
		else if (input[index].y >= 0.0f)
			y = static_cast<unsigned short>(pow(input[index].y, 2));
		auto& res = output[index];
		res ^ res;
		res = x << 16;
		res += y;
		index += blockDim.x * gridDim.x;
	}
}

void float_to_ushort(
  const float* input,
  unsigned short* output,
  const unsigned int size,
  hipStream_t stream)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(size, threads);

  kernel_float_to_ushort << <blocks, threads, 0, stream >> >(input, output, size);
}

void complex_to_ushort(
	const hipfftComplex* input,
	unsigned int* output,
	const unsigned int size,
	hipStream_t stream)
{
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(size, threads);

	kernel_complex_to_ushort << <blocks, threads, 0 >> >(input, output, size);
}

__global__ void kernel_accumulate_images(
	const hipfftComplex *input,
	hipfftComplex *output,
	const size_t start,
	const size_t max_elmt,
	const size_t nb_elmt,
	const size_t nb_pixel)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	size_t	i = 0;
	size_t	pos = start;

	if (index < nb_pixel)
	{
		output[index].x = 0;
		output[index].y = 0;
		while (i < nb_elmt)
		{
			output[index].x += input[index + pos * nb_pixel].x;
			output[index].y += input[index + pos * nb_pixel].y;
			i++;
			pos++;
			if (pos > max_elmt)
				pos = 0;
		}
	}
}

void accumulate_images(
	const hipfftComplex *input,
	hipfftComplex *output,
	const size_t start,
	const size_t max_elmt,
	const size_t nb_elmt,
	const size_t nb_pixel,
	hipStream_t stream)
{
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(nb_pixel, threads);

	kernel_accumulate_images << <blocks, threads, 0, stream >> >(
		input,
		output,
		start,
		max_elmt,
		nb_elmt,
		nb_pixel);
}