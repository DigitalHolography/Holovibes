#include <>

#include "stft.cuh"
#include "hardware_limits.hh"
#include "geometry.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "tools.cuh"
#include "geometry.hh"

void stft(
  hipfftComplex*                   input,
  hipfftComplex*                   gpu_queue,
  hipfftComplex*                   stft_buf,
  const hipfftHandle               plan1d,
  unsigned int                    stft_level,
  unsigned int                    p, 
  unsigned int                    q, 
  unsigned int                    frame_size,
  hipStream_t                    stream)
{
  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(frame_size, threads);


  // FFT 1D
  hipfftExecC2C(plan1d, gpu_queue, stft_buf, HIPFFT_FORWARD);
  hipStreamSynchronize(stream);

  hipMemcpy(
	  input,
	  stft_buf + p * frame_size,
	  sizeof(hipfftComplex)* frame_size,
	  hipMemcpyDeviceToDevice);

  if (p != q)
  {
	  hipMemcpy(
		  input + frame_size,
		  stft_buf + q * frame_size,
		  sizeof(hipfftComplex)* frame_size,
		  hipMemcpyDeviceToDevice);

  }

}