# include "stft.cuh"

# include <hip/hip_runtime.h>
# include "hardware_limits.hh"
# include "tools.cuh"

void stft(
  hipfftComplex*                   input,
  hipfftComplex*                   lens,
  hipfftComplex*                   stft_buf,
  hipfftComplex*                   stft_dup_buf,
  hipfftHandle                     plan2d,
  hipfftHandle                     plan1d,
  const holovibes::Rectangle&     r,
  unsigned int&                   curr_elt,
  camera::FrameDescriptor&        desc,
  unsigned int                    nsamples,
  unsigned int                    pindex)
{
  unsigned int threads = 128;
  unsigned int blocks = desc.frame_res() / threads;
  
  if (blocks > get_max_blocks())
    blocks = get_max_blocks();
  
  // Apply lens on multiple frames.
  kernel_apply_lens << <blocks, threads >> >(input, desc.frame_res(), lens, desc.frame_res());

  hipDeviceSynchronize();

  // FFT 2D
  hipfftExecC2C(plan2d, input, input, HIPFFT_FORWARD);

  hipDeviceSynchronize();

  if (curr_elt == nsamples)
  {
    // Remove first element and move all element on left
    hipMemcpy(stft_buf, &(stft_buf[1]), sizeof(hipfftComplex)* (nsamples * r.area() - 1), hipMemcpyDeviceToDevice);
    --curr_elt;
  }
  // Do the ROI
  kernel_bursting_roi << <blocks, threads >> >(
    input,
    r.top_left.x,
    r.top_left.y,
    r.bottom_right.x,
    r.bottom_right.y,
    curr_elt,
    nsamples,
    desc.width,
    stft_buf);
  ++curr_elt;

  // FFT 1D
  hipfftExecC2C(plan1d, stft_buf, stft_dup_buf, HIPFFT_FORWARD);
  hipDeviceSynchronize();

  // Reconstruct Roi
  kernel_reconstruct_roi << <blocks, threads >> >(
    stft_dup_buf,
    input,
    r.get_width(),
    r.get_height(),
    desc.width,
    pindex,
    nsamples);
}