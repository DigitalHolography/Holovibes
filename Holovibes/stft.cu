#include "stft.cuh"
#include "hardware_limits.hh"
#include "geometry.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "tools.cuh"

void stft(
  hipfftComplex*                   input,
  const hipfftComplex*             gpu_queue,
  hipfftComplex*                   stft_buf,
  hipfftComplex*                   stft_dup_buf,
  const hipfftHandle               plan2d,
  const hipfftHandle               plan1d,
  const holovibes::Rectangle&     r,
  unsigned int&                   curr_elt,
  const camera::FrameDescriptor&  desc,
  unsigned int                    nsamples,
  unsigned int                    stft_level,
  hipStream_t                    stream)
{
  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(desc.frame_res(), threads);

 /* // Apply lens on multiple frames.
  kernel_apply_lens << <blocks, threads, 0, stream >> >(input, desc.frame_res(), lens, desc.frame_res());

  hipStreamSynchronize(stream);

  // FFT 2D
  hipfftExecC2C(plan2d, input, input, HIPFFT_FORWARD);
  hipStreamSynchronize(stream);*/

  if (!r.area())
    return;

/*  if (curr_elt == nsamples)
  {
    // Remove first element and move all element on left
    hipMemcpyAsync(stft_buf,
      &(stft_buf[1]),
      sizeof(hipfftComplex)* (nsamples * r.area() - 1),
      hipMemcpyDeviceToDevice,
      stream);
    --curr_elt;
  }*/

  // Do the ROI
/*  kernel_bursting_roi << <blocks, threads, 0, stream >> >(
    input,
    r.top_left.x,
    r.top_left.y,
    r.bottom_right.x,
    r.bottom_right.y,
    curr_elt,
    nsamples,
    desc.width,
    desc.width * desc.height,
    stft_buf);
  ++curr_elt;*/

  // FFT 1D
  hipfftExecC2C(plan1d, stft_buf, stft_dup_buf, HIPFFT_FORWARD);
  hipStreamSynchronize(stream);
}

void stft_recontruct(
  hipfftComplex*                   output,
  hipfftComplex*                   stft_dup_buf,
  const holovibes::Rectangle      r,
  const camera::FrameDescriptor&  desc,
  const unsigned int              reconstruct_width,
  const unsigned int              reconstruct_height,
  const unsigned int              pindex,
  const unsigned int              nsamples,
  hipStream_t stream)
{
  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(desc.frame_res(), threads);

  if (!r.area())
    return;
  // Reconstruct Roi
/*  kernel_reconstruct_roi << <blocks, threads, 0, stream >> >(
    stft_dup_buf,
    output,
    r.get_width(),
    r.get_height(),
    desc.width,
    reconstruct_width,
    reconstruct_height,
    pindex,
    nsamples);*/
}