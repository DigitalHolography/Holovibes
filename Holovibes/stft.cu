#include "hip/hip_runtime.h"
#include <>

#include "stft.cuh"
#include "hardware_limits.hh"
#include "geometry.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "tools.cuh"
#include "geometry.hh"

void stft(
	hipfftComplex*                   input,
	hipfftComplex*                   gpu_queue,
	hipfftComplex*                   stft_buf,
	const hipfftHandle               plan1d,
	unsigned int                    stft_level,
	unsigned int                    p,
	unsigned int                    q,
	unsigned int                    frame_size,
	bool                            stft_activated,
	hipStream_t                    stream)
{
	//unsigned int threads = 128;
	//unsigned int blocks = map_blocks_to_problem(frame_size, threads);

	// FFT 1D
	if (stft_activated)
		hipfftExecC2C(plan1d, gpu_queue, stft_buf, HIPFFT_FORWARD);
	hipStreamSynchronize(stream);

	hipMemcpy(
		input,
		stft_buf + p * frame_size,
		sizeof(hipfftComplex)* frame_size,
		hipMemcpyDeviceToDevice);

	if (p != q)
	{
		hipMemcpy(
			input + frame_size,
			stft_buf + q * frame_size,
			sizeof(hipfftComplex)* frame_size,
			hipMemcpyDeviceToDevice);

	}

}

__global__	void	stft_view_xz(	hipfftComplex	*input,
									ushort			*output,
									uint			x0,
									uint			y0,
									uint			z0,
									uint			frame_size,
									uint			width,
									uint			height,
									uint			depth)
{
	uint id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < frame_size)
	{
		uint index_x = id;
		uint index_z = id % depth;
//		hipfftComplex pixel = input[(y0 * width) + (index_x / width) * frame_size + index_x % width];
		hipfftComplex pixel = input[(y0 * width) + (index_x / width) * frame_size + index_x % width];
		float res = hypotf(pixel.x, pixel.y);
		output[id] = static_cast<ushort>(pixel.x);
	}
}

void	stft_view_begin(	hipfftComplex	*input,
							ushort			*output,
							uint			x0,
							uint			y0,
							uint			z0,
							uint			frame_size,
							uint			width,
							uint			height,
							uint			depth)
{
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(frame_size, threads);

	stft_view_xz<<<blocks, threads, 0, 0 >>>(input, output, x0, y0, z0, frame_size, width, height, depth);
}
