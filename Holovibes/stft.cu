#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "stft.cuh"
#include "hardware_limits.hh"
#include "geometry.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "tools.cuh"
#include "geometry.hh"

void stft(	complex				*input,
			complex				*gpu_queue,
			complex				*stft_buf,
			const hipfftHandle	plan1d,
			uint				stft_level,
			uint				p,
			uint				q,
			uint				frame_size,
			bool				stft_activated,
			hipStream_t		stream)
{
	//uint threads = 128;
	//uint blocks = map_blocks_to_problem(frame_size, threads);

	// FFT 1D
	if (stft_activated)
		hipfftExecC2C(plan1d, gpu_queue, stft_buf, HIPFFT_FORWARD);
	hipStreamSynchronize(stream);
	uint complex_frame_size = sizeof(complex)* frame_size;
	hipMemcpy(
		input,
		stft_buf + p * frame_size,
		complex_frame_size,
		hipMemcpyDeviceToDevice);

	if (p != q)
	{
	hipMemcpy(	input + frame_size,
				stft_buf + q * frame_size,
				complex_frame_size,
				hipMemcpyDeviceToDevice);
	}
}

__global__	void	kernel_stft_view_xz(const complex	*input,
										ushort			*output,
										const uint		x0,
										const uint		y0,
										const uint		frame_size,
										const uint		output_size,
										const uint		width,
										const uint		height,
										const uint		depth)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	complex		pixel;
	if (id < output_size)
	{
		pixel = input[(y0 * width) + (id / width) * frame_size + id % width];
		output[output_size - id] = static_cast<ushort>(pixel.x);
	}
}

__global__	void	kernel_stft_view_yz(	const complex	*input,
										ushort			*output,
										const uint		x0,
										const uint		y0,
										const uint		frame_size,
										const uint		output_size,
										const uint		width,
										const uint		height,
										const uint		depth)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	complex		pixel;
	if (id < output_size)
	{
		pixel = input[x0 + id * width];
		output[output_size - id] = static_cast<ushort>(pixel.x);
	}
}

void	stft_view_begin(const complex	*input,
						ushort			*outputxz,
						ushort			*outputyz,
						const uint		x0,
						const uint		y0,
						const uint		width,
						const uint		height,
						const uint		depth)
{
	uint frame_size = width * height;
	uint output_size_xz = width * depth;
	uint output_size_yz = height * depth;
	uint threads = get_max_threads_1d();
	uint blocks_xz = map_blocks_to_problem(output_size_xz, threads);
	uint blocks_yz = map_blocks_to_problem(output_size_yz, threads);
	
	kernel_stft_view_xz << <blocks_xz, threads, 0, 0 >> >(input, outputxz, x0, y0, frame_size, output_size_xz, width, height, depth);
	kernel_stft_view_yz << <blocks_yz, threads, 0, 0 >> >(input, outputyz, x0, y0, frame_size, output_size_yz, width, height, depth);
}