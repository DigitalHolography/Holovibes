# include "stft.cuh"

# include <hip/hip_runtime.h>
# include "hardware_limits.hh"
# include "tools.cuh"

void stft(
  hipfftComplex*                   input,
  hipfftComplex*                   lens,
  hipfftComplex*                   stft_buf,
  hipfftComplex*                   stft_dup_buf,
  hipfftHandle                     plan2d,
  hipfftHandle                     plan1d,
  const holovibes::Rectangle&     r,
  unsigned int&                   curr_elt,
  camera::FrameDescriptor&        desc,
  unsigned int                    nsamples,
  unsigned int                    pindex)
{
  unsigned int threads = 128;
  unsigned int blocks = desc.frame_res() / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  // Apply lens on multiple frames.
  kernel_apply_lens << <blocks, threads >> >(input, desc.frame_res(), lens, desc.frame_res());

  hipDeviceSynchronize();

  // FFT 2D
  hipfftExecC2C(plan2d, input, input, HIPFFT_FORWARD);

  hipDeviceSynchronize();
  // Do the ROI
  kernel_bursting_roi << <blocks, threads >> >(
    input,
    r.top_left.x,
    r.top_left.y,
    r.bottom_right.x,
    r.bottom_right.y,
    curr_elt,
    nsamples,
    desc.width,
    stft_buf);

  // FFT 1D
  hipMemcpy(stft_dup_buf, stft_buf, sizeof(hipfftComplex)* r.area() * nsamples, hipMemcpyDeviceToDevice);
  std::cout << nsamples << " : " << curr_elt << std::endl;
  hipfftExecC2C(plan1d, stft_dup_buf, stft_dup_buf, HIPFFT_FORWARD);
  hipDeviceSynchronize();

  // Reconstruct Roi
  kernel_reconstruct_roi << <blocks, threads >> >(
    stft_dup_buf,
    input,
    r.get_width(),
    r.get_height(),
    desc.width,
    pindex,
    nsamples);

  curr_elt = ++curr_elt % nsamples;
}