#include "fft1.cuh"

#include <hip/hip_runtime.h>
#include "hardware_limits.hh"
#include "tools.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"

hipfftComplex* create_lens(
  const camera::FrameDescriptor& fd,
  float lambda,
  float z)
{
  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);
  hipfftComplex *lens;
  hipMalloc(&lens, fd.width * fd.height * sizeof(hipfftComplex));
  kernel_quadratic_lens <<<lblocks, lthreads>>>(lens, fd, lambda, z);

  return lens;
}

void fft_1(
  unsigned short *result_buffer,
  holovibes::Queue& q,
  hipfftComplex *lens,
  float *sqrt_vect,
  hipfftHandle plan,
  int nbimages)
{
  // Sizes
  unsigned int pixel_size = q.get_frame_desc().width * q.get_frame_desc().height * nbimages;

  // Loaded images --> complex
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (pixel_size + threads - 1) / threads;

  // Hardware limit !!
  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  hipfftComplex* complex_input = make_contiguous_complex(q, nbimages, sqrt_vect);

  // Apply lens
  apply_quadratic_lens <<<blocks, threads>>>(complex_input, pixel_size, lens, q.get_pixels());

  // FFT
  hipfftExecC2C(plan, complex_input, complex_input, HIPFFT_FORWARD);

  // Complex --> real (unsigned short)
  complex_2_argument <<<blocks, threads>>>(complex_input, result_buffer, pixel_size);

  // Free all
  hipFree(complex_input);
}