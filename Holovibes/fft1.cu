#include "fft1.cuh"
#include "hardware_limits.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "tools.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"

void fft1_lens(
  hipfftComplex* lens,
  const camera::FrameDescriptor& fd,
  const float lambda,
  const float z,
  hipStream_t stream)
{
  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(fd.frame_res(), threads);

  kernel_quadratic_lens << <blocks, threads, 0, stream >> >(lens, fd, lambda, z);
}

void fft_1(
	hipfftComplex* input,
	const hipfftComplex* lens,
	const hipfftHandle plan1D,
	const hipfftHandle plan2D,
	const unsigned int frame_resolution,
	const unsigned int nframes,
	const unsigned int p,
	const unsigned int q,
	hipStream_t stream)
{
	const unsigned int n_frame_resolution = frame_resolution * nframes;

	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);
	
	hipfftComplex* pframe = input + frame_resolution * p;
	hipfftComplex* qframe = input + frame_resolution * q;

	hipfftExecC2C(plan1D, input, input, HIPFFT_FORWARD);

	// Apply lens on multiple frames.
	kernel_apply_lens << <blocks, threads, 0, stream >> >(pframe, frame_resolution, lens, frame_resolution);
	hipStreamSynchronize(stream);
	// FFT

    hipfftExecC2C(plan2D, pframe, pframe, HIPFFT_FORWARD);

	if (p != q)
	{
		kernel_apply_lens << <blocks, threads, 0, stream >> >(qframe, frame_resolution, lens, frame_resolution);
		hipfftExecC2C(plan2D, qframe, qframe, HIPFFT_FORWARD);
	}

	hipStreamSynchronize(stream);
}
/*
void fft_1(
  hipfftComplex* input,
  const hipfftComplex* lens,
  const hipfftHandle plan,
  const unsigned int frame_resolution,
  const unsigned int nframes,
  hipStream_t stream)
{
  const unsigned int n_frame_resolution = frame_resolution * nframes;

  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);

  // Apply lens on multiple frames.
  kernel_apply_lens << <blocks, threads, 0, stream >> >(input, n_frame_resolution, lens, frame_resolution);

  hipStreamSynchronize(stream);
  // FFT
  hipfftResult res = hipfftExecC2C(plan, input, input, HIPFFT_FORWARD);
  
  hipStreamSynchronize(stream);
}*/
