#include "fft1.cuh"
#include "hardware_limits.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "tools.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"

void fft1_lens(
  hipfftComplex* lens,
  const camera::FrameDescriptor& fd,
  const float lambda,
  const float z,
  hipStream_t stream)
{
  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(fd.frame_res(), threads);

  kernel_quadratic_lens << <blocks, threads, 0, stream >> >(lens, fd, lambda, z);
}

void fft_1(
  hipfftComplex* input,
  const hipfftComplex* lens,
  const hipfftHandle plan,
  const unsigned int frame_resolution,
  const unsigned int nframes,
  hipStream_t stream)
{
  const unsigned int n_frame_resolution = frame_resolution * nframes;

  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);

  // Apply lens on multiple frames.
  kernel_apply_lens << <blocks, threads, 0, stream >> >(input, n_frame_resolution, lens, frame_resolution);

  hipStreamSynchronize(stream);

  // FFT
  hipfftExecC2C(plan, input, input, HIPFFT_FORWARD);

  hipStreamSynchronize(stream);
}