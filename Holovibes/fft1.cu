#include "fft1.cuh"

#include <hip/hip_runtime.h>

#include "hardware_limits.hh"
#include "tools.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"

void fft1_lens(
  hipfftComplex* lens,
  const camera::FrameDescriptor& fd,
  float lambda,
  float z)
{
  unsigned int threads = 128;
  unsigned int blocks = (fd.frame_res() + threads - 1) / threads;

  if (blocks > get_max_blocks())
	  blocks = get_max_blocks();
  kernel_quadratic_lens<<<blocks, threads>>>(lens, fd, lambda, z);
}

void fft_1(
  hipfftComplex* input,
  hipfftComplex* lens,
  hipfftHandle plan,
  unsigned int frame_resolution,
  unsigned int nframes)
{
  const unsigned int n_frame_resolution = frame_resolution * nframes;

  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = n_frame_resolution / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  // Apply lens on multiple frames.
  kernel_apply_lens <<<blocks, threads>>>(input, n_frame_resolution, lens, frame_resolution);

  hipDeviceSynchronize();

  // FFT
  hipfftExecC2C(plan, input, input, HIPFFT_FORWARD);

  hipDeviceSynchronize();
}
