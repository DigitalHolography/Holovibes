#include "fft1.cuh"

#include <hip/hip_runtime.h>
#include "hardware_limits.hh"
#include "tools.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"

void fft1_lens(
  hipfftComplex* lens,
  const camera::FrameDescriptor& fd,
  float lambda,
  float z)
{
  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);

  kernel_quadratic_lens <<<lblocks, lthreads>>>(lens, fd, lambda, z);
}

void fft_1(
  hipfftComplex* input_buffer,
  unsigned short *result_buffer,
  holovibes::Queue& q,
  hipfftComplex *lens,
  hipfftHandle plan,
  unsigned int nbimages)
{
  // Sizes
  unsigned int pixel_size = q.get_frame_desc().width * q.get_frame_desc().height * nbimages;

  // Loaded images --> complex
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (pixel_size + threads - 1) / threads;

  // Hardware limit !!
  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  // Apply lens
  apply_quadratic_lens <<<blocks, threads>>>(input_buffer, pixel_size, lens, q.get_pixels());

  // FFT
  hipfftExecC2C(plan, input_buffer, input_buffer, HIPFFT_FORWARD);

  // Complex --> real (unsigned short)
  complex_2_module <<<blocks, threads>>>(input_buffer, result_buffer, pixel_size);
}