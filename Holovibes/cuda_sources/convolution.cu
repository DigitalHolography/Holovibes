#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#pragma once
#include "convolution.cuh"
#include "fft1.cuh"
#include "tools.cuh"
#include "tools_compute.cuh"
#include "logger.hh"
using holovibes::cuda_tools::CufftHandle;

//the three next function are for test
__global__
void print_kernel(hipComplex *output)
{
	if (threadIdx.x < 32)
		printf("%d, %f, %f\n", threadIdx.x, output[threadIdx.x].x, output[threadIdx.x].y);
}

__global__
void print_float(float *output)
{
	if (threadIdx.x < 32)
		printf("%d, %f\n", threadIdx.x, output[threadIdx.x]);
}

__global__
void fill_output(float *out, unsigned size)
{
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = 65000.f;
	}
}

void normalize_kernel(float		*gpu_kernel_buffer_,
					  size_t	size)
{
	float sum = get_norm(gpu_kernel_buffer_, size);
}

void convolution_kernel(float				*gpu_input,
						float				*gpu_convolved_buffer,
						hipComplex	        *cuComplex_buffer,
						CufftHandle			*plan,
						const uint			frame_width,
						const uint			frame_height,
						const hipComplex		*gpu_kernel,
						const bool			divide_convolution_enabled,
						const bool			normalize_enabled)
{
	size_t size = frame_width * frame_height;

	uint	threads = get_max_threads_1d();
	uint	blocks = map_blocks_to_problem(size, threads);

	hipMemset(cuComplex_buffer, 0, size * sizeof(hipComplex));
	cudaCheckError();
	hipMemcpy2D(cuComplex_buffer, sizeof(hipComplex), gpu_input, sizeof(float), sizeof(float), size, hipMemcpyDeviceToDevice);
	//At this point, cuComplex_buffer is the same as the input

	hipfftExecC2C(plan->get(), cuComplex_buffer, cuComplex_buffer, HIPFFT_FORWARD);
	//At this point, cuComplex_buffer is the FFT of the input

	kernel_multiply_frames_complex << <blocks, threads >> > (cuComplex_buffer, gpu_kernel, cuComplex_buffer, static_cast<uint>(size));
	//At this point, cuComplex_buffer is the FFT of the input multiplied by the FFT of the kernel

	hipfftExecC2C(plan->get(), cuComplex_buffer, cuComplex_buffer, HIPFFT_BACKWARD);

	kernel_complex_to_modulus << <blocks, threads >> > (cuComplex_buffer, gpu_convolved_buffer, (uint)size);

	if (divide_convolution_enabled)
	{
		kernel_divide_frames_float << <blocks, threads >> > (gpu_input, gpu_convolved_buffer, gpu_input, static_cast<uint>(size));
	}
	else
	{
		hipMemcpy(gpu_input, gpu_convolved_buffer, size * sizeof(float), hipMemcpyDeviceToDevice);
	}
}
