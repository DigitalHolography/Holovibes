#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#pragma once
#include "convolution.cuh"
#include "fft1.cuh"
#include "tools.cuh"
#include "tools_compute.cuh"
#include "logger.hh"
using holovibes::cuda_tools::CufftHandle;

//the three next function are for test
__global__
void print_kernel(hipComplex *output)
{
	if (threadIdx.x < 32)
		printf("%d, %f, %f\n", threadIdx.x, output[threadIdx.x].x, output[threadIdx.x].y);
}

__global__
void print_float(float *output)
{
	if (threadIdx.x < 32)
		printf("%d, %f\n", threadIdx.x, output[threadIdx.x]);
}

__global__
void fill_output(float *out, unsigned size)
{
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		out[idx] = 65000.f;
	}
}

void normalize_kernel(float		*gpu_kernel_buffer_,
					  size_t	size)
{
	float sum = get_norm(gpu_kernel_buffer_, size);

	// Calling an empty function, why?
	// gpu_float_divide(gpu_kernel_buffer_, static_cast<uint>(size), sum);
}

void convolution_kernel(float		*gpu_input,
	float			*gpu_convolved_buffer,
	CufftHandle		*plan,
	const uint		frame_width,
	const uint		frame_height,
	const float		*gpu_kernel,
	const bool		divide_convolution_enabled,
	const bool		normalize_enabled)
{
	size_t size = frame_width * frame_height;

	float norm_input;
	if (normalize_enabled)
		norm_input = get_norm(gpu_input, size);

	uint	threads = get_max_threads_1d();
	uint	blocks = map_blocks_to_problem(size, threads);

	holovibes::cuda_tools::UniquePtr<hipComplex> output_fft(size);
	holovibes::cuda_tools::UniquePtr<hipComplex> output_kernel(size);
	if (!output_fft || !output_kernel)
	{
		LOG_ERROR("Couldn't allocate buffers for convolution.\n");
		return;
	}

	holovibes::cuda_tools::UniquePtr<hipComplex> tmp_complex(size);
	hipMemset(tmp_complex.get(), 0, size * sizeof(hipComplex));
	cudaCheckError();
	hipMemcpy2D(tmp_complex.get(), sizeof(hipComplex), gpu_input, sizeof(float), sizeof(float), size, hipMemcpyDeviceToDevice);
	hipfftExecC2C(plan->get(), tmp_complex.get(), output_fft.get(), HIPFFT_FORWARD);

	hipMemcpy2D(tmp_complex.get(), sizeof(hipComplex), gpu_kernel, sizeof(float), sizeof(float), size, hipMemcpyDeviceToDevice);
	hipfftExecC2C(plan->get(), tmp_complex.get(), output_kernel.get(), HIPFFT_FORWARD);

	kernel_multiply_frames_complex << <blocks, threads >> > (output_fft, output_kernel, output_fft, static_cast<uint>(size));

	hipfftExecC2C(plan->get(), output_fft, output_fft, HIPFFT_BACKWARD);

	kernel_complex_to_modulus << <blocks, threads >> > (output_fft, gpu_convolved_buffer, (uint)size);

	if (divide_convolution_enabled)
		kernel_divide_frames_float << <blocks, threads >> > (gpu_input, gpu_convolved_buffer, gpu_input, static_cast<uint>(size));
	else
		hipMemcpy(gpu_input, gpu_convolved_buffer, size * sizeof(float), hipMemcpyDeviceToDevice);

	if (normalize_enabled) {
		float norm_output = get_norm(gpu_input, size);
		gpu_multiply_const(gpu_input, static_cast<uint>(size), (norm_input / norm_output));
	}

}
