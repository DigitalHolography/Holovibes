#pragma once
#include "convolution.cuh"
#include "fresnel_transform.cuh"
#include "tools.cuh"
#include "tools_compute.cuh"
#include "complex_utils.cuh"
#include "logger.hh"
#include "common.cuh"
#include "cuda_memory.cuh"
#include <hipfft/hipfft.h>
#include "cuda_tools\unique_ptr.hh"
#include "cuda_tools\array.hh"
#include "cuda_tools\cufft_handle.hh"
#include <npp.h>

#include "matrix_operations.hh"
using holovibes::cuda_tools::CufftHandle;

void convolution_kernel(float* input_output,
                        float* gpu_convolved_buffer,
                        hipComplex* cuComplex_buffer,
                        CufftHandle* plan,
                        const size_t size,
                        const hipComplex* gpu_kernel,
                        const bool divide_convolution_enabled,
                        const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    /* Copy input_output (float*) to cuComplex_buffer (hipComplex*)
     * We only want to copy the float value as real part float number in the
     * cuComplex_buffer To skip the imaginary part, we use a pitch (skipped
     * data) of size sizeof(float)
     *
     * The value are first all set to 0 (real & imaginary)
     * Then value are copied 1 by 1 from input_output into the real part
     * Imaginary is skipped and thus left to its value
     */
    cudaXMemsetAsync(cuComplex_buffer, 0, size * sizeof(hipComplex), stream);
    cudaSafeCall(hipMemcpy2DAsync(cuComplex_buffer,  // Destination memory address
                                   sizeof(hipComplex), // Pitch of destination memory
                                   input_output,      // Source memory address
                                   sizeof(float),     // Pitch of source memory
                                   sizeof(float),     // Width of matrix transfer (columns in bytes)
                                   size,              // Height of matrix transfer (rows)
                                   hipMemcpyDeviceToDevice,
                                   stream));
    // At this point, cuComplex_buffer is the same as the input

    cufftSafeCall(hipfftExecC2C(plan->get(), cuComplex_buffer, cuComplex_buffer, HIPFFT_FORWARD));
    // At this point, cuComplex_buffer is the FFT of the input

    complex_hadamard_product(cuComplex_buffer, cuComplex_buffer, gpu_kernel, size, stream);
    // At this point, cuComplex_buffer is the FFT of the input multiplied by the
    // FFT of the kernel

    cufftSafeCall(hipfftExecC2C(plan->get(), cuComplex_buffer, cuComplex_buffer, HIPFFT_BACKWARD));

    if (divide_convolution_enabled)
    {
        kernel_complex_to_modulus<<<blocks, threads, 0, stream>>>(gpu_convolved_buffer, cuComplex_buffer, size);
        cudaCheckError();
        kernel_divide_frames_float<<<blocks, threads, 0, stream>>>(input_output,
                                                                   input_output,
                                                                   gpu_convolved_buffer,
                                                                   size);
    }
    else
    {
        kernel_complex_to_modulus<<<blocks, threads, 0, stream>>>(input_output, cuComplex_buffer, size);
    }
    cudaCheckError();
}

void xcorr2(float* output,
            float* input1,
            float* input2,
            hipfftComplex* d_freq_1,
            hipfftComplex* d_freq_2,
            hipfftHandle plan_2d,
            hipfftHandle plan_2dinv,
            const int freq_size,
            hipStream_t stream)
{
    cufftSafeCall(hipfftExecR2C(plan_2d, input1, d_freq_1));
    cufftSafeCall(hipfftExecR2C(plan_2d, input2, d_freq_2));

    conjugate_complex(d_freq_2, freq_size, stream);
    complex_hadamard_product(d_freq_1, d_freq_1, d_freq_2, freq_size, stream);

    cufftSafeCall(hipfftExecC2R(plan_2dinv, d_freq_1, output));
}