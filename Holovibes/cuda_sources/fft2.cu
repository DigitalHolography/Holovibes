#include "hip/hip_runtime.h"
/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include "fft2.cuh"
#include "transforms.cuh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"

#include <hipfft/hipfftXt.h>

using camera::FrameDescriptor;

enum mode
{
    APPLY_PHASE_FORWARD,
    APPLY_PHASE_INVERSE
};

__global__ static void kernel_fft2_dc(const hipComplex* const input,
                                      hipComplex* const output,
                                      const ushort width,
                                      const uint frame_res,
                                      const uint batch_size,
                                      const bool mode)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < frame_res)
    {
        for (uint i = 0; i < batch_size; ++i)
        {
            const uint batch_index = index + i * frame_res;

            const float pi_pxl = M_PI * (index / width + index % width);
            if (mode == APPLY_PHASE_FORWARD)
                output[batch_index] =
                    hipCmulf(input[batch_index],
                            make_hipComplex(cosf(pi_pxl), sinf(pi_pxl)));
            else if (mode == APPLY_PHASE_INVERSE)
                output[batch_index] =
                    hipCmulf(input[batch_index],
                            make_hipComplex(cosf(-pi_pxl), sinf(-pi_pxl)));
        }
    }
}

static void fft_2_dc(const ushort width,
                     const uint frame_res,
                     hipComplex* pframe,
                     const bool mode,
                     const uint batch_size,
                     const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_fft2_dc<<<blocks, threads, 0, stream>>>(pframe,
                                                   pframe,
                                                   width,
                                                   frame_res,
                                                   batch_size,
                                                   mode);
    cudaCheckError();
}

void fft2_lens(hipComplex* lens,
               const uint lens_side_size,
               const uint frame_height,
               const uint frame_width,
               const float lambda,
               const float z,
               const float pixel_size,
               const hipStream_t stream)
{
    const uint threads_2d = get_max_threads_2d();
    const dim3 lthreads(threads_2d, threads_2d);
    const dim3 lblocks(lens_side_size / threads_2d,
                       lens_side_size / threads_2d);

    hipComplex* square_lens;
    // In anamorphic mode, the lens is initally a square, it's then cropped to
    // be the same dimension as the frame
    if (frame_height != frame_width)
        cudaXMalloc(&square_lens,
                    lens_side_size * lens_side_size * sizeof(hipComplex));
    else
        square_lens = lens;

    kernel_spectral_lens<<<lblocks, lthreads, 0, stream>>>(square_lens,
                                                           lens_side_size,
                                                           lambda,
                                                           z,
                                                           pixel_size);
    cudaCheckError();

    if (frame_height != frame_width)
    {
        cudaXMemcpyAsync(lens,
                    square_lens +
                        ((lens_side_size - frame_height) / 2) * frame_width,
                    frame_width * frame_height * sizeof(hipComplex),
                    hipMemcpyDeviceToDevice,
                    stream);
        cudaXFree(square_lens);
    }
}

void fft_2(hipComplex* input,
           hipComplex* output,
           const uint batch_size,
           const hipComplex* lens,
           const hipfftHandle plan2d,
           const FrameDescriptor& fd,
           const hipStream_t stream)
{
    const uint frame_resolution = fd.frame_res();
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_resolution, threads);

    fft_2_dc(fd.width, frame_resolution, input, 0, batch_size, stream);

    cufftSafeCall(hipfftXtExec(plan2d, input, input, HIPFFT_FORWARD));

    kernel_apply_lens<<<blocks, threads, 0, stream>>>(input,
                                                      output,
                                                      batch_size,
                                                      frame_resolution,
                                                      lens,
                                                      frame_resolution);
    cudaCheckError();

    cufftSafeCall(hipfftXtExec(plan2d, input, input, HIPFFT_BACKWARD));

    fft_2_dc(fd.width, frame_resolution, input, 1, batch_size, stream);

    kernel_complex_divide<<<blocks, threads, 0, stream>>>(
        input,
        frame_resolution,
        static_cast<float>(frame_resolution),
        batch_size);
    cudaCheckError();
}