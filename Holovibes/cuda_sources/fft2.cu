#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "fft2.cuh"
#include "hardware_limits.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "transforms.cuh"
#include "preprocessing.cuh"
#include "tools_compute.cuh"
#include "tools.cuh"

enum mode
{
	APPLY_PHASE_FORWARD,
	APPLY_PHASE_INVERSE
};

__global__ static void kernel_fft2_dc(	const complex	*input,
										complex			*output,
										const ushort	width,
										const uint		frame_res,
										const bool		mode)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;;
	if (id < frame_res)
	{
		uint idx = id / width;
		uint idy = id % width;
		complex product;
		if (mode == APPLY_PHASE_FORWARD)
			product = make_hipComplex(cosf(M_PI * (idx + idy)), sinf(M_PI * (idx + idy)));
		else if (mode == APPLY_PHASE_INVERSE)
			product = make_hipComplex(cosf(-M_PI * (idx + idy)), sinf(-M_PI * (idx + idy)));
		output[id] = hipCmulf(input[id], product);
	}
}

void fft_2_dc(	const complex	*input,
				complex			*output,
				const ushort	width,
				const uint		frame_res,
				const uint		p,
				const bool		mode,
				hipStream_t	stream)
{
	const uint	threads = THREADS_128;
	const uint	blocks = map_blocks_to_problem(frame_res, threads);

	kernel_fft2_dc << <blocks, threads, 0, stream >> >(input + frame_res * p, output + frame_res * p, width, frame_res, mode);
	hipStreamSynchronize(stream);
}

void fft2_lens(	complex					*lens,
				const FrameDescriptor&	fd,
				const float				lambda,
				const float				z,
				hipStream_t			stream)
{
	uint threads_2d = get_max_threads_2d();
	dim3 lthreads(threads_2d, threads_2d);
	dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);
	//float test = fd.pixel_size;
	kernel_spectral_lens << <lblocks, lthreads, 0, stream >> >(lens, fd, lambda, z);
}

void fft_2(	complex				*input,
			const complex		*lens,
			const hipfftHandle	plan1d,
			const hipfftHandle	plan2d,
			const uint			frame_resolution,
			const uint			nframes,
			const uint			p,
			const uint			q,
			hipStream_t		stream)
{
	const uint		n_frame_resolution = frame_resolution * nframes;
	uint			threads = THREADS_128;
	uint			blocks = map_blocks_to_problem(frame_resolution, threads);

	hipfftExecC2C(plan1d, input, input, HIPFFT_FORWARD);

	hipStreamSynchronize(stream);

	complex* pframe = input + frame_resolution * p;

	hipfftExecC2C(plan2d, pframe, pframe, HIPFFT_FORWARD);

	kernel_apply_lens << <blocks, threads, 0, stream >> >(pframe, frame_resolution, lens, frame_resolution);

	hipStreamSynchronize(stream);


	hipfftExecC2C(plan2d, pframe, pframe, HIPFFT_BACKWARD);

	kernel_complex_divide << <blocks, threads, 0, stream >> >(pframe, frame_resolution, static_cast<float>(n_frame_resolution));

	if (p != q)
	{
		complex* qframe = input + frame_resolution * q;
		hipfftExecC2C(plan2d, qframe, qframe, HIPFFT_FORWARD);
		kernel_apply_lens << <blocks, threads, 0, stream >> >(qframe, frame_resolution, lens, frame_resolution);
		hipfftExecC2C(plan2d, qframe, qframe, HIPFFT_BACKWARD);
		kernel_complex_divide << <blocks, threads, 0, stream >> >(qframe, frame_resolution, static_cast<float>(n_frame_resolution));
	}
	hipStreamSynchronize(stream);
}