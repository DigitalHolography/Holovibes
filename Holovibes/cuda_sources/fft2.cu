#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "fft2.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"
#include "tools_compute.cuh"

using camera::FrameDescriptor;

enum mode
{
	APPLY_PHASE_FORWARD,
	APPLY_PHASE_INVERSE
};

__global__
static void kernel_fft2_dc(const hipComplex	*input,
						hipComplex			*output,
						const ushort		width,
						const uint			frame_res,
						const bool			mode)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;;
	if (id < frame_res)
	{
		const float	pi_pxl = M_PI * (id / width + id % width);
		if (mode == APPLY_PHASE_FORWARD)
			output[id] = hipCmulf(input[id], make_hipComplex(cosf(pi_pxl), sinf(pi_pxl)));
		else if (mode == APPLY_PHASE_INVERSE)
			output[id] = hipCmulf(input[id], make_hipComplex(cosf(-pi_pxl), sinf(-pi_pxl)));
	}
}

void fft_2_dc(	const ushort	width,
				const uint		frame_res,
				hipComplex		*pframe,
				const bool		mode,
				hipStream_t	stream)
{
	const uint	threads = get_max_threads_1d();
	const uint	blocks = map_blocks_to_problem(frame_res, threads);

	kernel_fft2_dc << <blocks, threads, 0, stream >> >(pframe, pframe, width, frame_res, mode);
}

void fft2_lens(hipComplex			*lens,
			const FrameDescriptor&	fd,
			const float				lambda,
			const float				z,
			const float				pixel_size,
			hipStream_t			stream)
{
	uint threads_2d = get_max_threads_2d();
	dim3 lthreads(threads_2d, threads_2d);
	dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);

	kernel_spectral_lens << <lblocks, lthreads, 0, stream >> >(lens, fd, lambda, z, pixel_size);
}

void fft_2(hipComplex			*input,
		const hipComplex			*lens,
		const hipfftHandle		plan2d,
		const FrameDescriptor&	fd,
		hipStream_t			stream)
{
	const uint	frame_resolution = fd.frame_res();
	uint		threads = get_max_threads_1d();
	uint		blocks = map_blocks_to_problem(frame_resolution, threads);

	hipStreamSynchronize(stream);

	fft_2_dc(fd.width, frame_resolution, input, 0, stream);

	hipfftExecC2C(plan2d, input, input, HIPFFT_FORWARD);

	kernel_apply_lens << <blocks, threads, 0, stream >> >(input, frame_resolution, lens, frame_resolution);

	hipStreamSynchronize(stream);


	hipfftExecC2C(plan2d, input, input, HIPFFT_BACKWARD);

	fft_2_dc(fd.width, frame_resolution, input, 1, stream);

	kernel_complex_divide << <blocks, threads, 0, stream >> >(input, frame_resolution, static_cast<float>(frame_resolution));

	hipStreamSynchronize(stream);
}
