#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "stft.cuh"

// Short-Time Fourier Transform
void stft(hipComplex			*input,
		hipComplex			*gpu_queue,
		hipComplex			*stft_buf,
		const hipfftHandle	plan1d,
		const uint			tft_level,
		const uint			p,
		const uint			q,
		const uint			frame_size,
		const bool			stft_activated,
		hipStream_t		stream)
{
	const uint complex_frame_size = sizeof(hipComplex) * frame_size;
	// FFT 1D
	if (stft_activated)
		hipfftExecC2C(plan1d, gpu_queue, stft_buf, HIPFFT_FORWARD);
	hipStreamSynchronize(stream);
	hipMemcpy(	input,
				stft_buf + p * frame_size,
				complex_frame_size,
				hipMemcpyDeviceToDevice);

	if (p != q)
	{
		hipMemcpy(	input + frame_size,
					stft_buf + q * frame_size,
					complex_frame_size,
					hipMemcpyDeviceToDevice);
	}
}

#pragma region moment
__global__
static void kernel_stft_moment(hipComplex	*input,
							hipComplex		*output,
							const uint		frame_res,
							ushort			pmin,
							ushort			pmax,
							const uint		nsamples)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < frame_res)
	{
		output[id] = make_hipComplex(0.f, 0.f);
		while (pmin <= pmax)
		{
			hipComplex *current_pframe = input + (frame_res * pmin);
			output[id].x += hypotf(current_pframe[id].x, current_pframe[id].y);
			++pmin;
		}
	}
}

void stft_moment(hipComplex		*input,
				hipComplex		*output,
				const uint		frame_res,
				ushort			pmin,
				const ushort	pmax,
				const uint		nsamples)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(frame_res, threads);

	kernel_stft_moment << <blocks, threads, 0, 0 >> > (input, output, frame_res, pmin, pmax, nsamples);
}
#pragma endregion


__global__
static void	fill_64bit_slices(const hipComplex	*input,
							hipComplex			*output_xz,
							hipComplex			*output_yz,
							const uint			start_x,
							const uint			start_y,
							const uint			frame_size,
							const uint			output_size,
							const uint			width,
							const uint			height,
							const uint			acc_level_xz,
							const uint			acc_level_yz)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < output_size)
	{
		output_xz[id] = input[start_x * width + (id / width) * frame_size + id % width];
		output_yz[id] = input[start_x + id * width];
	}
}

__global__
static void	fill_32bit_slices(const hipComplex	*input,
							float				*output_xz,
							float				*output_yz,
							const uint			x0,
							const uint			y0,
							const uint			frame_size,
							const uint			output_size,
							const uint			width,
							const uint			height,
							const uint			acc_level_xz,
							const uint			acc_level_yz)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < output_size)
	{
		hipComplex pixel = make_hipComplex(0, 0);
		for (int i = 0; i < acc_level_yz; ++i)
			pixel = hipCaddf(pixel, input[x0 + i + id * width]);
		output_yz[id] = hypotf(pixel.x, pixel.y) / static_cast<float>(acc_level_yz);
		/* ********** */
		pixel = make_hipComplex(0, 0);
		for (int i = 0; i < acc_level_xz; ++i)
			pixel = hipCaddf(pixel, input[((y0 + i) * width) + (id / width) * frame_size + id % width]);
		output_xz[id] = hypotf(pixel.x, pixel.y) / static_cast<float>(acc_level_xz);
	}
}

void stft_view_begin(const hipComplex	*input,
					void				*output_xz,
					void				*output_yz,
					const ushort		x0,
					const ushort		y0,
					const ushort		width,
					const ushort		height,
					const uint			viewmode,
					const ushort		nsamples,
					const uint			acc_level_xz,
					const uint			acc_level_yz)
{
	const uint frame_size = width * height;
	const uint output_size = width * nsamples;
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(output_size, threads); 

	if (static_cast<ComplexViewMode>(viewmode) == ComplexViewMode::Complex)
		fill_64bit_slices << <blocks, threads, 0, 0 >> >(
			input,
			reinterpret_cast<hipComplex *>(output_xz),
			reinterpret_cast<hipComplex *>(output_yz),
			x0, y0,
			frame_size,
			output_size,
			width, height,
			acc_level_xz, acc_level_yz);
	else
		fill_32bit_slices <<<blocks, threads, 0, 0>>>(
			input,
			reinterpret_cast<float *>(output_xz),
			reinterpret_cast<float *>(output_yz),
			x0, y0,
			frame_size,
			output_size,
			width, height,
			acc_level_xz, acc_level_yz);
}
