#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "aberration.cuh"
#include "unique_ptr.hh"
# include "tools_compute.cuh"
# include "cufft_handle.hh"
#include "operator_overload.cuh"

static __global__
void kernel_apply_aberration_phis(hipfftComplex*			lens,
								  const hipfftComplex*	phis,
								  const uint			frame_size,
								  const uint			frame_width,
								  const uint			chunks_per_row,
								  const uint			chunk_width,
								  const uint			chunk_height)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < frame_size)
	{
		unsigned int x = index % frame_width;
		unsigned int y = index / frame_width;

		unsigned int chunk_no = (y / chunk_height) * chunks_per_row + x / chunk_width;
		lens[index] = lens[index] * phis[chunk_no];
	}
}

void apply_aberration_phis(ComplexArray& lens,
						   std::vector<hipfftComplex> phis,
						   unsigned int nb_chunks_per_row,
						   unsigned int nb_chunks_per_column,
						   const camera::FrameDescriptor& fd)
{
	holovibes::cuda_tools::UniquePtr<hipfftComplex> gpu_phis(phis.size());
	hipMemcpy(gpu_phis, phis.data(), phis.size() * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(fd.frame_res(), threads);
	kernel_apply_aberration_phis << <threads, blocks, 0, 0 >> >
		(lens,
		gpu_phis,
		fd.frame_res(),
		fd.width,
		nb_chunks_per_row,
		fd.width / nb_chunks_per_row,
		fd.height / nb_chunks_per_column);
}