/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#pragma once

#include "cuda_memory.cuh"

void cudaXMemcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind)
{
    cudaSafeCall(hipMemcpy(dst, src, count, kind));
}

void cudaXMemcpyAsync(void* dst,
                      const void* src,
                      size_t count,
                      hipMemcpyKind kind,
                      const hipStream_t stream)
{
    cudaSafeCall(hipMemcpyAsync(dst, src, count, kind, stream));
}

void cudaXMemset(void* devPtr, int value, size_t count)
{
    cudaSafeCall(hipMemset(devPtr, value, count));
}

void cudaXMemsetAsync(void* devPtr,
                      int value,
                      size_t count,
                      const hipStream_t stream)
{
    cudaSafeCall(hipMemsetAsync(devPtr, value, count, stream));
}

void cudaXFree(void* devPtr) { cudaSafeCall(hipFree(devPtr)); }

void cudaXFreeHost(void* devPtr) { cudaSafeCall(hipHostFree(devPtr)); }

void cudaXStreamSynchronize(const hipStream_t stream,
                            const char* file,
                            const int line)
{
    std::cout << "File : " << file << " Line : " << line << std::endl;
    cudaSafeCall(hipStreamSynchronize(stream));
}