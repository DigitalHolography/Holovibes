#include "angular_spectrum.cuh"
#include "masks.cuh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"
#include "apply_mask.cuh"
#include "shift_corners.cuh"

#include <hipfft/hipfftXt.h>

using camera::FrameDescriptor;

void angular_spectrum_lens(hipComplex* lens,
                           const uint lens_side_size,
                           const uint frame_height,
                           const uint frame_width,
                           const float lambda,
                           const float z,
                           const float pixel_size,
                           const hipStream_t stream)
{
    const uint threads_2d = get_max_threads_2d();
    const dim3 lthreads(threads_2d, threads_2d);
    const dim3 lblocks(lens_side_size / threads_2d, lens_side_size / threads_2d);

    hipComplex* square_lens;
    // In anamorphic mode, the lens is initally a square, it's then cropped to
    // be the same dimension as the frame
    if (frame_height != frame_width)
        cudaXMalloc(&square_lens, lens_side_size * lens_side_size * sizeof(hipComplex));
    else
        square_lens = lens;

    kernel_spectral_lens<<<lblocks, lthreads, 0, stream>>>(square_lens, lens_side_size, lambda, z, pixel_size);
    cudaCheckError();

    if (frame_height != frame_width)
    {
        cudaXMemcpyAsync(lens,
                         square_lens + ((lens_side_size - frame_height) / 2) * frame_width,
                         frame_width * frame_height * sizeof(hipComplex),
                         hipMemcpyDeviceToDevice,
                         stream);
        cudaXFree(square_lens);
    }
}

void angular_spectrum(hipComplex* input,
                      hipComplex* output,
                      const uint batch_size,
                      const hipComplex* lens,
                      hipComplex* mask_output,
                      bool store_frame,
                      const hipfftHandle plan2d,
                      const FrameDescriptor& fd,
                      const hipStream_t stream)
{
    const uint frame_res = fd.get_frame_res();
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_res, threads);

    cufftSafeCall(hipfftXtExec(plan2d, input, input, HIPFFT_FORWARD));

    // Lens and Mask already shifted
    // thus we don't have to shift the 'input' buffer each time
    apply_mask(input, lens, output, frame_res, batch_size, stream);
    if (store_frame)
    {
        cudaXMemcpyAsync(mask_output, input, frame_res * sizeof(hipComplex), hipMemcpyDeviceToDevice, stream);
    }

    cudaCheckError();

    cufftSafeCall(hipfftXtExec(plan2d, input, input, HIPFFT_BACKWARD));

    kernel_complex_divide<<<blocks, threads, 0, stream>>>(input, frame_res, static_cast<float>(frame_res), batch_size);
    cudaCheckError();
}
