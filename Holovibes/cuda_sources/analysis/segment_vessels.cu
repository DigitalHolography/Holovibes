#include "hip/hip_runtime.h"
#include "cuda_memory.cuh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include "tools_analysis_debug.hh"

__global__ void kernel_minus_negation_times_2(float* R_vascular_pulse, float* mask_vesselnessClean, uint size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
        R_vascular_pulse[idx] = R_vascular_pulse[idx] - !mask_vesselnessClean[idx] * 2;
}

void minus_negation_times_2(float* R_vascular_pulse, float* mask_vesselnessClean, uint size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_minus_negation_times_2<<<blocks, threads, 0, stream>>>(R_vascular_pulse, mask_vesselnessClean, size);
    cudaCheckError();
}

__global__ void kernel_negation(float* input_output, uint size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
        input_output[idx] = !input_output[idx];
}

void negation(float* input_output, uint size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_negation<<<blocks, threads, 0, stream>>>(input_output, size);
    cudaCheckError();
}

__global__ void kernel_quantize(float* output, float* input, float* thresholds, int length_input, int lenght_threshold)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Si l'index est dans la plage du tableau d'entrée
    if (idx < length_input)
    {
        float value = input[idx];
        int quantized_level = 1;

        // Trouver le niveau de quantification en fonction des seuils
        for (int t = 0; t < lenght_threshold; ++t)
        {
            if (value > thresholds[t])
                quantized_level = t + 2;
            else
                break;
        }

        // Stocker le résultat
        output[idx] = quantized_level;
    }
}

void imquantize(
    float* output, float* input, float* thresholds, int length_input, int lenght_threshold, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(length_input, threads);
    kernel_quantize<<<blocks, threads, 0, stream>>>(output, input, thresholds, length_input, lenght_threshold);
    cudaCheckError();
}

void segment_vessels(float* output,
                     float* new_thresholds,
                     float* R_VascularPulse,
                     float* mask_vesselness_clean,
                     uint size,
                     float* thresholds,
                     hipStream_t stream)
{
    float minus_one = -1;
    cudaXMemcpyAsync(new_thresholds + 1, thresholds, sizeof(float) * 3, hipMemcpyHostToDevice, stream);
    cudaXMemcpyAsync(new_thresholds, &minus_one, sizeof(float), hipMemcpyHostToDevice, stream);

    minus_negation_times_2(R_VascularPulse, mask_vesselness_clean, size, stream);
    imquantize(output, R_VascularPulse, new_thresholds, size, 4, stream);
}

__global__ void kernel_is_both_value(float* output, float* input, uint size, float value1, float value2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
        output[idx] = input[idx] == value1 || input[idx] == value2;
}

void is_both_value(float* output, float* input, uint size, float value1, float value2, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_is_both_value<<<blocks, threads, 0, stream>>>(output, input, size, value1, value2);
    cudaCheckError();
}

void compute_first_mask_artery(float* output, float* input, uint size, hipStream_t stream)
{
    is_both_value(output, input, size, 5, 4, stream);
}

void compute_first_mask_vein(float* output, float* input, uint size, hipStream_t stream)
{
    is_both_value(output, input, size, 2, 3, stream);
}