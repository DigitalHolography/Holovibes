#include "hip/hip_runtime.h"
#include "otsu.cuh"
#include "common.cuh"
#include "hip/hip_complex.h"
#include "hip/hip_runtime.h"
#include "hardware_limits.hh"
using uint = unsigned int;

#define NUM_BINS 256

__global__ void histogram_kernel(float* image, int* hist, int imgSize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < imgSize)
        atomicAdd(&hist[(unsigned char)(image[idx] * NUM_BINS)], 1);
}

__global__ void _normalise(float* d_input, float min, float max, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
        d_input[tid] = (int) (((d_input[tid] - min) / (max - min)) * NUM_BINS);
}

void normalise(float* d_input, float min, float max, const size_t size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    _normalise<<<blocks, threads, 0, stream>>>(d_input, min, max, size);
    hipDeviceSynchronize();
}

__global__ void global_threshold_kernel(float* input, int size, float globalThreshold)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
        input[idx] = (input[idx] > globalThreshold) ? 1.0f : 0.0f;
}

__global__ void bradley_threshold_kernel(const float* image,
                                         float* output,
                                         int width,
                                         int height,
                                         int windowSize,
                                         float globalThreshold,
                                         float localThresholdFactor)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int x = idx % width;
    int y = idx / width;

    if (x >= width || y >= height)
        return;

    int halfWindow = windowSize / 2;
    int startX = max(x - halfWindow, 0);
    int startY = max(y - halfWindow, 0);
    int endX = min(x + halfWindow, width - 1);
    int endY = min(y + halfWindow, height - 1);

    float localSum = 0;
    int count = 0;

    for (int i = startX; i <= endX; i++)
    {
        for (int j = startY; j <= endY; j++)
        {
            localSum += image[j * width + i];
            count++;
        }
    }

    float localMean = localSum / count;
    float localThreshold = localMean * (1 - localThresholdFactor * globalThreshold);
    output[y * width + x] = (image[y * width + x] > localThreshold) ? 1.0f : 0.0f;
}

float otsu_threshold(float* d_image, int size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);

    // get histogram
    int h_hist[NUM_BINS];
    int* d_hist;
    hipMalloc(&d_hist, NUM_BINS * sizeof(int));
    hipMemset(d_hist, 0, NUM_BINS * sizeof(int));
    histogram_kernel<<<blocks, threads, 0, stream>>>(d_image, d_hist, size);
    hipDeviceSynchronize();
    hipMemcpy(h_hist, d_hist, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_hist);

    // Compute optimal threshold
    int total = size;
    float sum = 0, sumB = 0, varMax = 0;
    int wB = 0, wF = 0;
    float threshold = 0;

    for (int i = 0; i < NUM_BINS; i++)
        sum += i * h_hist[i];
    for (int t = 0; t < NUM_BINS; t++)
    {
        wB += h_hist[t];
        if (wB == 0)
            continue;
        wF = total - wB;
        if (wF == 0)
            break;

        sumB += t * h_hist[t];
        float mB = sumB / wB;
        float mF = (sum - sumB) / wF;
        float varBetween = wB * wF * (mB - mF) * (mB - mF);

        if (varBetween > varMax)
        {
            varMax = varBetween;
            threshold = t;
        }
    }
    return threshold / NUM_BINS;
}

void compute_binarise_otsu(float* d_image, const size_t width, const size_t height, const hipStream_t stream)
{
    size_t img_size = width * height;

    float global_threshold = otsu_threshold(d_image, img_size, stream);

    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(img_size, threads);

    global_threshold_kernel<<<blocks, threads, 0, stream>>>(d_image, img_size, global_threshold);
    hipDeviceSynchronize();
}

void compute_binarise_otsu_bradley(float* d_image,
                                   float*& d_output,
                                   const size_t width,
                                   const size_t height,
                                   const int window_size,
                                   const float local_threshold_factor,
                                   const hipStream_t stream)
{
    size_t img_size = width * height;

    float global_threshold = otsu_threshold(d_image, img_size, stream);

    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(img_size, threads);

    bradley_threshold_kernel<<<blocks, threads, 0, stream>>>(d_image,
                                                             d_output,
                                                             width,
                                                             height,
                                                             window_size,
                                                             global_threshold,
                                                             local_threshold_factor);
    hipDeviceSynchronize();
}
