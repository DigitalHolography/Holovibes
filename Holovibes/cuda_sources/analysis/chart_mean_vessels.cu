#include "hip/hip_runtime.h"
#include "chart_mean_vessels.cuh"

#include "hip/hip_runtime.h"
#include "cuda_memory.cuh"
#include "hardware_limits.hh"

__global__ void get_sum_with_mask_kernel(const float* input, const float* mask, size_t size, float* sum_res)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size)
    {
        atomicAdd(sum_res, input[idx] * mask[idx]);
    }
}

float get_sum_with_mask(const float* input, const float* mask, size_t size, float* sum_res, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    float res = 0.0f;

    cudaXMemset(sum_res, 0.0f, sizeof(float));
    get_sum_with_mask_kernel<<<blocks, threads, 0, stream>>>(input, mask, size, sum_res);
    hipStreamSynchronize(stream);
    cudaXMemcpy(&res, sum_res, sizeof(float), hipMemcpyDeviceToHost);
    return res;
}
