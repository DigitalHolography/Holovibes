#include "hip/hip_runtime.h"
#include "tools_unwrap.cuh"

#include "cuda_memory.cuh"
#include "frame_desc.hh"
#include "complex_utils.cuh"

using namespace holovibes;
using namespace camera;

__global__ void kernel_extract_angle(float* output, const hipComplex* input, const size_t size)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size)
    {
        // We use std::atan2 in order to obtain results in [-pi; pi].
        output[index] = std::atan2(input[index].y, input[index].x);
    }
}

__global__ void kernel_unwrap(float* output, const float* pred, const float* cur, const size_t size)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size)
    {
        const float local_diff = cur[index] - pred[index];
        // Unwrapping //
        float local_adjust;
        if (local_diff > M_PI)
            local_adjust = -M_2PI;
        else if (local_diff < -M_PI)
            local_adjust = M_2PI;
        else
            local_adjust = 0.f;
        // Cumulating each angle with its correction
        output[index] = cur[index] + local_adjust;
    }
}

__global__ void kernel_compute_angle_mult(float* output, const hipComplex* pred, const hipComplex* cur, const size_t size)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size)
    {
        hipComplex conj_prod;
        conj_prod = cur[index];

        conj_prod.x *= pred[index].x;
        conj_prod.x += cur[index].y * pred[index].y;

        conj_prod.y *= pred[index].x;
        conj_prod.y -= cur[index].x * pred[index].y;

        output[index] = std::atan2(conj_prod.y, conj_prod.x);
    }
}

__global__ void kernel_compute_angle_diff(float* output, const hipComplex* pred, const hipComplex* cur, const size_t size)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size)
    {
        hipComplex diff = cur[index];
        diff.x -= pred[index].x;
        diff.y -= pred[index].y;
        output[index] = std::atan2(diff.y, diff.x);
    }
}

__global__ void
kernel_correct_angles(float* data, const float* corrections, const size_t image_size, const size_t history_size)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t size = history_size * image_size;
    if (index < image_size)
        for (auto correction_idx = index; correction_idx < size; correction_idx += image_size)
            data[index] += corrections[correction_idx];
}

__global__ void kernel_init_unwrap_2d(
    const uint width, const uint height, const uint frame_res, const float* input, float* fx, float* fy, hipComplex* z)
{
    const uint i = blockIdx.x * blockDim.x + threadIdx.x;
    const uint j = blockIdx.y * blockDim.y + threadIdx.y;
    const uint index = j * blockDim.x * gridDim.x + i;

    if (index < frame_res)
    {
        fx[index] = (i - static_cast<float>(lrintf(static_cast<float>(width >> 1))));
        fy[index] = (j - static_cast<float>(lrintf(static_cast<float>(height >> 1))));

        /*z init*/
        z[index].x = cosf(input[index]);
        z[index].y = sinf(input[index]);
    }
}

__global__ void kernel_multiply_complexes_by_floats_(
    hipComplex* output1, hipComplex* output2, const float* input1, const float* input2, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        output1[index].x *= input1[index];
        output1[index].y *= input1[index];
        output2[index].x *= input2[index];
        output2[index].y *= input2[index];
    }
}

__global__ void kernel_multiply_complexes_by_single_complex(hipComplex* output1,
                                                            hipComplex* output2,
                                                            const hipComplex input,
                                                            const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        const hipComplex cpy_o1 = output1[index];
        const hipComplex cpy_o2 = output2[index];

        output1[index].x = cpy_o1.x * input.x - cpy_o1.y * input.y;
        output1[index].y = cpy_o1.x * input.y + cpy_o1.y * input.x;
        output2[index].x = cpy_o2.x * input.x - cpy_o2.y * input.y;
        output2[index].y = cpy_o2.x * input.y + cpy_o2.y * input.x;
    }
}

__global__ void kernel_multiply_complex_by_single_complex(hipComplex* output, const hipComplex input, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        const hipComplex cpy_o1 = output[index];

        output[index].x = cpy_o1.x * input.x - cpy_o1.y * input.y;
        output[index].y = cpy_o1.x * input.y + cpy_o1.y * input.x;
    }
}

__global__ void kernel_multiply_complex_frames_by_complex_frame(hipComplex* output1,
                                                                hipComplex* output2,
                                                                const hipComplex* input,
                                                                const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        const hipComplex cpy_o1 = output1[index];
        const hipComplex cpy_o2 = output2[index];

        output1[index].x = cpy_o1.x * input[index].x - cpy_o1.y * input[index].y;
        output1[index].y = cpy_o1.x * input[index].y + cpy_o1.y * input[index].x;
        output2[index].x = cpy_o2.x * input[index].x - cpy_o2.y * input[index].y;
        output2[index].y = cpy_o2.x * input[index].y + cpy_o2.y * input[index].x;
    }
}

__global__ void
kernel_norm_ratio(hipComplex* output1, hipComplex* output2, const float* input1, const float* input2, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        const float norm = input1[index] * input1[index] + input2[index] * input2[index];

        if (norm != 0)
        {
            const float coeff_x = input1[index] / norm;
            const float coeff_y = input2[index] / norm;

            output1[index].x = output1[index].x * coeff_x;
            output1[index].y = output1[index].y * coeff_x;
            output2[index].x = output2[index].x * coeff_y;
            output2[index].y = output2[index].y * coeff_y;
        }
        else
        {
            output1[index].x = 0;
            output1[index].y = 0;
            output2[index].x = 0;
            output2[index].y = 0;
        }
    }
}

__global__ void kernel_add_complex_frames(hipComplex* output, const hipComplex* input, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        output[index].x += input[index].x;
        output[index].y += input[index].y;
    }
}

__global__ void kernel_unwrap2d_last_step(float* output, const hipComplex* input, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
        output[index] = input[index].y / -M_2PI;
}

void phase_increase(const hipComplex* cur,
                    UnwrappingResources* resources,
                    const size_t image_size,
                    const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(image_size, threads);
    static bool first_time = true;
    if (first_time)
    {
        cudaXMemcpyAsync(resources->gpu_predecessor_,
                         cur,
                         sizeof(hipComplex) * image_size,
                         hipMemcpyDeviceToDevice,
                         stream);
        first_time = false;
    }

    // Compute the newest phase image, not unwrapped yet
    kernel_compute_angle_mult<<<blocks, threads, 0, stream>>>(resources->gpu_angle_current_,
                                                              resources->gpu_predecessor_,
                                                              cur,
                                                              image_size);
    cudaCheckError();
    // Updating predecessor (complex image) for the next iteration
    cudaXMemcpyAsync(resources->gpu_predecessor_,
                     cur,
                     sizeof(hipComplex) * image_size,
                     hipMemcpyDeviceToDevice,
                     stream);

    /* Copying in order to later enqueue the (not summed up with values
     * in gpu_unwrap_buffer_) phase image. */
    cudaXMemcpyAsync(resources->gpu_angle_copy_,
                     resources->gpu_angle_current_,
                     sizeof(float) * image_size,
                     hipMemcpyDeviceToDevice,
                     stream);

    // Applying history on the latest phase image
    kernel_correct_angles<<<blocks, threads, 0, stream>>>(resources->gpu_angle_current_,
                                                          resources->gpu_unwrap_buffer_,
                                                          image_size,
                                                          resources->size_);
    cudaCheckError();

    /* Store the new phase image in the next buffer position.
     * The buffer is handled as a circular buffer. */
    float* next_unwrap = resources->gpu_unwrap_buffer_ + image_size * resources->next_index_;
    cudaXMemcpyAsync(next_unwrap,
                     resources->gpu_angle_copy_,
                     sizeof(float) * image_size,
                     hipMemcpyDeviceToDevice,
                     stream);
    if (resources->size_ < resources->capacity_)
        ++resources->size_;
    resources->next_index_ = (resources->next_index_ + 1) % resources->capacity_;
}

void unwrap_2d(float* output,
               float* input,
               const hipfftHandle plan2d,
               UnwrappingResources_2d* res,
               const FrameDescriptor& fd,
               const hipStream_t stream)
{
    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);
    kernel_init_unwrap_2d<<<lblocks, lthreads, 0, stream>>>(fd.width,
                                                            fd.height,
                                                            fd.get_frame_res(),
                                                            input,
                                                            res->gpu_fx_,
                                                            res->gpu_fy_,
                                                            res->gpu_z_);
    cudaCheckError();

    ushort middlex = fd.width >> 1;
    ushort middley = fd.height >> 1;
    circ_shift(res->gpu_shift_fx_, res->gpu_fx_, fd.width, fd.height, -middlex, -middley, stream);
    circ_shift(res->gpu_shift_fy_, res->gpu_fy_, fd.width, fd.height, -middlex, -middley, stream);

    gradient_unwrap_2d(plan2d, res, fd, stream);
    eq_unwrap_2d(plan2d, res, fd, stream);
    phi_unwrap_2d(output, plan2d, res, fd, stream);
}

void gradient_unwrap_2d(const hipfftHandle plan2d,
                        UnwrappingResources_2d* res,
                        const FrameDescriptor& fd,
                        const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(res->image_resolution_, threads);
    hipComplex single_complex = make_hipComplex(0.f, static_cast<float>(M_2PI));

    hipfftExecC2C(plan2d, res->gpu_z_, res->gpu_grad_eq_x_, HIPFFT_FORWARD);
    hipfftExecC2C(plan2d, res->gpu_z_, res->gpu_grad_eq_y_, HIPFFT_FORWARD);
    kernel_multiply_complexes_by_floats_<<<blocks, threads, 0, stream>>>(res->gpu_grad_eq_x_,
                                                                         res->gpu_grad_eq_y_,
                                                                         res->gpu_shift_fx_,
                                                                         res->gpu_shift_fy_,
                                                                         fd.get_frame_res());
    cudaCheckError();
    hipfftExecC2C(plan2d, res->gpu_grad_eq_x_, res->gpu_grad_eq_x_, HIPFFT_BACKWARD);
    hipfftExecC2C(plan2d, res->gpu_grad_eq_y_, res->gpu_grad_eq_y_, HIPFFT_BACKWARD);
    kernel_multiply_complexes_by_single_complex<<<blocks, threads, 0, stream>>>(res->gpu_grad_eq_x_,
                                                                                res->gpu_grad_eq_y_,
                                                                                single_complex,
                                                                                fd.get_frame_res());
    cudaCheckError();
}

void eq_unwrap_2d(const hipfftHandle plan2d,
                  UnwrappingResources_2d* res,
                  const FrameDescriptor& fd,
                  const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(res->image_resolution_, threads);
    hipComplex single_complex = make_hipComplex(0, 1);

    kernel_multiply_complex_by_single_complex<<<blocks, threads, 0, stream>>>(res->gpu_z_,
                                                                              single_complex,
                                                                              fd.get_frame_res());
    cudaCheckError();
    conjugate_complex(res->gpu_z_, fd.get_frame_res(), stream);
    kernel_multiply_complex_frames_by_complex_frame<<<blocks, threads, 0, stream>>>(res->gpu_grad_eq_x_,
                                                                                    res->gpu_grad_eq_y_,
                                                                                    res->gpu_z_,
                                                                                    fd.get_frame_res());
    cudaCheckError();
    hipfftExecC2C(plan2d, res->gpu_grad_eq_x_, res->gpu_grad_eq_x_, HIPFFT_FORWARD);
    hipfftExecC2C(plan2d, res->gpu_grad_eq_y_, res->gpu_grad_eq_y_, HIPFFT_FORWARD);
    kernel_norm_ratio<<<blocks, threads, 0, stream>>>(res->gpu_grad_eq_x_,
                                                      res->gpu_grad_eq_y_,
                                                      res->gpu_shift_fx_,
                                                      res->gpu_shift_fy_,
                                                      fd.get_frame_res());
    cudaCheckError();
}

void phi_unwrap_2d(float* output,
                   const hipfftHandle plan2d,
                   UnwrappingResources_2d* res,
                   const FrameDescriptor& fd,
                   const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(res->image_resolution_, threads);

    kernel_add_complex_frames<<<blocks, threads, 0, stream>>>(res->gpu_grad_eq_x_,
                                                              res->gpu_grad_eq_y_,
                                                              fd.get_frame_res());
    cudaCheckError();
    hipfftExecC2C(plan2d, res->gpu_grad_eq_x_, res->gpu_grad_eq_x_, HIPFFT_BACKWARD);
    kernel_unwrap2d_last_step<<<blocks, threads, 0, stream>>>(output, res->gpu_grad_eq_x_, fd.get_frame_res());
    cudaCheckError();
}
