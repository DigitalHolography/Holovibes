#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "tools_hsv.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "percentile.cuh"
#include "cuda_memory.cuh"
#include "shift_corners.cuh"
#include "map.cuh"
#include "reduce.cuh"
#include "unique_ptr.hh"
#include "logger.hh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

__global__ void kernel_minus_negation_times_2(float* R_vascular_pulse, float* mask_vesselnessClean, uint size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        R_vascular_pulse[idx] = R_vascular_pulse[idx] - !mask_vesselnessClean[idx] * 2;
    }
}

void minus_negation_times_2(float* R_vascular_pulse, float* mask_vesselnessClean, uint size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_minus_negation_times_2<<<blocks, threads, 0, stream>>>(R_vascular_pulse, mask_vesselnessClean, size);
}

__global__ void kernel_quantize(float* output, float* input, float* thresholds, int length_input, int lenght_threshold)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Si l'index est dans la plage du tableau d'entrée
    if (idx < length_input) {
        float value = input[idx];
        int quantized_level = 1;

        // Trouver le niveau de quantification en fonction des seuils
        for (int t = 0; t < lenght_threshold; ++t) {
            if (value > thresholds[t]) {
                quantized_level = t + 2;
            } else {
                break;
            }
        }

        // Stocker le résultat
        output[idx] = quantized_level;
    }
}

void imquantize(float* output, float* input, float* thresholds, int length_input, int lenght_threshold, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(length_input, threads);
    kernel_quantize<<<blocks, threads, 0, stream>>>(output, input, thresholds, length_input, lenght_threshold);
}


void segment_vessels(float *output, float* R_VascularPulse, float* mask_vesselness_clean, uint size, hipStream_t stream)
{
    int numClassesVessels = 5;
    float* firstThresholds = new float[4] {
        -1.0f,
        -0.145349865260771f,
        0.225070673825605f,
        0.58226190794461f
    };
    float* firstThresholdsGPU;
    cudaXMalloc(&firstThresholdsGPU, sizeof(float) * 4);
    cudaXMemcpy(firstThresholdsGPU, firstThresholds, sizeof(float) * 4, hipMemcpyHostToDevice);
    delete[] firstThresholds;
    
    minus_negation_times_2(R_VascularPulse, mask_vesselness_clean, size, stream);
    imquantize(output, R_VascularPulse, firstThresholdsGPU, size, 4, stream);

    cudaXFree(firstThresholdsGPU);
}

__global__ void kernel_is_both_value(float* output, float* input, uint size, float value1, float value2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        output[idx] = input[idx] == value1 || input[idx] == value2;
    }
}

void is_both_value(float* output, float* input, uint size, float value1, float value2, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_is_both_value<<<blocks, threads, 0, stream>>>(output, input, size, value1, value2);
}

void compute_first_mask_artery(float* output, float* input, uint size, hipStream_t stream)
{
    is_both_value(output, input, size, 5, 4, stream);
}

void compute_first_mask_vein(float* output, float* input, uint size, hipStream_t stream)
{
    is_both_value(output, input, size, 2, 3, stream);
}