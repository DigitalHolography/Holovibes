#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "tools_hsv.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "percentile.cuh"
#include "cuda_memory.cuh"
#include "shift_corners.cuh"
#include "map.cuh"
#include "reduce.cuh"
#include "unique_ptr.hh"
#include "logger.hh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

__global__ void kernel_add_frame_to_sum(const float* const new_frame, const size_t frame_size, float* const sum_image)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < frame_size)
        sum_image[idx] += new_frame[idx];
}

void add_frame_to_sum(const float* const new_frame, const size_t size, float* const sum_image, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_add_frame_to_sum<<<blocks, threads, 0, stream>>>(new_frame, size, sum_image);
}

__global__ void kernel_subtract_frame_from_sum(const float* old_frame, const size_t frame_size, float* const sum_image)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < frame_size)
        sum_image[idx] -= old_frame[idx];
}

void subtract_frame_from_sum(const float* const new_frame,
                             const size_t size,
                             float* const sum_image,
                             hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_subtract_frame_from_sum<<<blocks, threads, 0, stream>>>(new_frame, size, sum_image);
}

__global__ void kernel_compute_mean(float* output, float* input, const size_t time_window, const size_t frame_size)
{
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < frame_size)
        output[index] = input[index] / time_window;
}

void compute_mean(float* output, float* input, const size_t time_window, const size_t frame_size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_size, threads);
    kernel_compute_mean<<<blocks, threads, 0, stream>>>(output, input, time_window, frame_size);
}

__global__ void
kernel_image_centering(float* output, const float* m0_video_frame, const float* m0_img, const uint frame_size)
{
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < frame_size)
        output[index] = m0_video_frame[index] - m0_img[index];
}

void image_centering(
    float* output, const float* m0_img, const float* m0_video_frame, const uint frame_size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_size, threads);
    kernel_image_centering<<<blocks, threads, 0, stream>>>(output, m0_video_frame, m0_img, frame_size);
    cudaXStreamSynchronize(stream);
}