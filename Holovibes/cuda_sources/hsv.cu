#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hsv.cuh"
#include "tools_hsv.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "percentile.cuh"
#include "cuda_memory.cuh"
#include "shift_corners.cuh"
#include "map.cuh"
#include "reduce.cuh"
#include "unique_ptr.hh"
#include "logger.hh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#define SAMPLING_FREQUENCY 1
static constexpr ushort max_ushort_value = (1 << (sizeof(ushort) * 8)) - 1;

__global__ void kernel_normalized_convert_hsv_to_rgb(const float* src, float* dst, size_t frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float nNormalizedH = src[frame_res * HSV::H + id];
        float nNormalizedS = src[frame_res * HSV::S + id];
        float nNormalizedV = src[frame_res * HSV::V + id];
        float nR;
        float nG;
        float nB;
        if (nNormalizedS == 0.0F)
        {
            nR = nG = nB = nNormalizedV;
        }
        else
        {
            if (nNormalizedH == 1.0F)
                nNormalizedH = 0.0F;
            else
                nNormalizedH = nNormalizedH * 6.0F; // / 0.1667F
        }
        float nI = floorf(nNormalizedH);
        float nF = nNormalizedH - nI;
        float nM = nNormalizedV * (1.0F - nNormalizedS);
        float nN = nNormalizedV * (1.0F - nNormalizedS * nF);
        float nK = nNormalizedV * (1.0F - nNormalizedS * (1.0F - nF));
        if (nI == 0.0F)
        {
            nR = nNormalizedV;
            nG = nK;
            nB = nM;
        }
        else if (nI == 1.0F)
        {
            nR = nN;
            nG = nNormalizedV;
            nB = nM;
        }
        else if (nI == 2.0F)
        {
            nR = nM;
            nG = nNormalizedV;
            nB = nK;
        }
        else if (nI == 3.0F)
        {
            nR = nM;
            nG = nN;
            nB = nNormalizedV;
        }
        else if (nI == 4.0F)
        {
            nR = nK;
            nG = nM;
            nB = nNormalizedV;
        }
        else if (nI == 5.0F)
        {
            nR = nNormalizedV;
            nG = nM;
            nB = nN;
        }
        dst[id * 3 + HSV::H] = nR * max_ushort_value;
        dst[id * 3 + HSV::S] = nG * max_ushort_value;
        dst[id * 3 + HSV::V] = nB * max_ushort_value;
    }
}

/// @brief Convert an array of HSV normalized float [0,1] to an array of RGB float [0,65536]
/// @param src Input hsv array (contiguous pixel on x: [h1,...,hn,s1,...,sn,v1,...,vn])
/// @param dst Output rgb array (contiguous rgb channels: [r1,g1,b1,...,rn,gn,bn])
/// @param frame_res Total number of pixels on one frame
/// @param stream The used cuda stream
void normalized_convert_hsv_to_rgb(const float* src, float* dst, size_t frame_res, const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_normalized_convert_hsv_to_rgb<<<blocks, threads, 0, stream>>>(src, dst, frame_res);
    cudaCheckError();
}

template <typename FUNC>
__global__ void kernel_compute_sum_depth(
    const hipComplex* gpu_input, float* gpu_output, size_t frame_res, size_t min_index, size_t max_index, FUNC func)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float res = 0.0f;

        for (size_t z = min_index; z <= max_index; ++z)
        {
            const hipComplex* current_p_frame = gpu_input + (z * frame_res);
            float input_elm = hypotf(current_p_frame[id].x, current_p_frame[id].y);

            res += input_elm * func(z);
        }

        const size_t range = max_index - min_index + 1;
        gpu_output[id] = (res / (float)range);
    }
}

/// @brief Compute the sum depth of a pixel : sum of input[z] * func(z) between z1 and z2
/// @param input The input hipComplex buffer
/// @param output The output float buffer
/// @param frame_res The total number of pixels in one frame
/// @param min_index z1
/// @param max_index z2
/// @param func the function to call on z
template <typename FUNC>
void compute_sum_depth(const hipComplex* input,
                       float* output,
                       size_t frame_res,
                       size_t min_index,
                       size_t max_index,
                       FUNC func,
                       const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_compute_sum_depth<<<blocks, threads, 0, stream>>>(input, output, frame_res, min_index, max_index, func);
    cudaCheckError();
}

__global__ void kernel_compute_and_fill_h(const hipComplex* gpu_input,
                                          float* gpu_output,
                                          const size_t frame_res,
                                          const uint min_h_index,
                                          const uint max_h_index)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float num = 0.0f;
        float denom = 0.0f;

        for (size_t z = min_h_index; z <= max_h_index; ++z)
        {
            const hipComplex* current_p_frame = gpu_input + (z * frame_res);
            float input_elm = hypotf(current_p_frame[id].x, current_p_frame[id].y);

            num += input_elm * z;
            denom += input_elm;
        }

        gpu_output[id] = (denom == 0.0f ? 0.0f : num / denom);
    }
}

void compute_and_fill_h(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream)
{
    const uint min_h_index = hsv_struct.h.frame_index.min;
    const uint max_h_index = hsv_struct.h.frame_index.max;

    float* gpu_h_output = gpu_output + HSV::H * frame_res;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    kernel_compute_and_fill_h<<<blocks, threads, 0, stream>>>(gpu_input,
                                                              gpu_h_output,
                                                              frame_res,
                                                              min_h_index,
                                                              max_h_index);
    cudaCheckError();
}

void compute_and_fill_s(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream)
{
    const uint min_s_index =
        hsv_struct.s.frame_index.activated ? hsv_struct.s.frame_index.min : hsv_struct.h.frame_index.min;
    const uint max_s_index =
        hsv_struct.s.frame_index.activated ? hsv_struct.s.frame_index.max : hsv_struct.h.frame_index.max;

    float* gpu_s_output = gpu_output + HSV::S * frame_res;

    // Saturation is the moment 2 (variance)
    auto func_moment_two = [] __device__(size_t z) -> size_t { return z * z; };

    compute_sum_depth(gpu_input, gpu_s_output, frame_res, min_s_index, max_s_index, func_moment_two, stream);
}

void compute_and_fill_v(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream)
{
    const uint min_v_index =
        hsv_struct.v.frame_index.activated ? hsv_struct.v.frame_index.min : hsv_struct.h.frame_index.min;
    const uint max_v_index =
        hsv_struct.v.frame_index.activated ? hsv_struct.v.frame_index.max : hsv_struct.h.frame_index.max;

    float* gpu_v_output = gpu_output + HSV::V * frame_res;

    // Value is the moment 0
    auto func_moment_zero = [] __device__(size_t z) -> size_t { return 1; };

    compute_sum_depth(gpu_input, gpu_v_output, frame_res, min_v_index, max_v_index, func_moment_zero, stream);
}

/// @brief Compute the hsv values of each pixel, each channel use his own lambda function that describe the calculus
/// done on z
void compute_and_fill_hsv(const hipComplex* gpu_input,
                          float* gpu_output,
                          const size_t frame_res,
                          const holovibes::CompositeHSV& hsv_struct,
                          const hipStream_t stream)
{
    compute_and_fill_h(gpu_input, gpu_output, frame_res, hsv_struct, stream);
    compute_and_fill_s(gpu_input, gpu_output, frame_res, hsv_struct, stream);
    compute_and_fill_v(gpu_input, gpu_output, frame_res, hsv_struct, stream);
}

// Apply a box blur on the specified array
void apply_blur(float* gpu_arr, uint height, uint width, float kernel_size, const hipStream_t stream)
{
    size_t frame_res = height * width;

    float* gpu_float_blur_matrix;
    cudaSafeCall(hipMalloc(&gpu_float_blur_matrix, frame_res * sizeof(float)));
    cudaSafeCall(hipMemsetAsync(gpu_float_blur_matrix, 0, frame_res * sizeof(float), stream));

    float* blur_matrix;
    cudaSafeCall(hipHostMalloc(&blur_matrix, kernel_size * sizeof(float)));
    float blur_value = 1.0f / (float)(kernel_size * kernel_size);
    unsigned min_pos_kernel_y = height / 2 - kernel_size / 2;
    unsigned min_pos_kernel_x = width / 2 - kernel_size / 2;
    for (size_t i = 0; i < kernel_size; i++)
        blur_matrix[i] = blur_value;

    for (size_t i = 0; i < kernel_size; i++)
    {
        cudaXMemcpyAsync(gpu_float_blur_matrix + min_pos_kernel_x + width * (i + min_pos_kernel_y),
                         blur_matrix,
                         kernel_size * sizeof(float),
                         hipMemcpyHostToDevice,
                         stream);
    }

    float* cpu_float_blur_matrix = new float[frame_res];
    cudaSafeCall(hipMemcpyAsync(cpu_float_blur_matrix,
                                 gpu_float_blur_matrix,
                                 frame_res * sizeof(float),
                                 hipMemcpyDeviceToHost,
                                 stream));

    hipComplex* gpu_complex_blur_matrix;
    cudaSafeCall(hipMalloc(&gpu_complex_blur_matrix, frame_res * sizeof(hipComplex)));
    cudaSafeCall(hipMemcpy2DAsync(gpu_complex_blur_matrix,
                                   sizeof(hipComplex),
                                   gpu_float_blur_matrix,
                                   sizeof(float),
                                   sizeof(float),
                                   frame_res,
                                   hipMemcpyDeviceToDevice,
                                   stream));

    shift_corners(gpu_complex_blur_matrix, 1, width, height, stream);

    CufftHandle handle{static_cast<int>(width), static_cast<int>(height), HIPFFT_C2C};
    cufftSafeCall(hipfftExecC2C(handle, gpu_complex_blur_matrix, gpu_complex_blur_matrix, HIPFFT_FORWARD));

    hipComplex* gpu_cuComplex_buffer;
    cudaSafeCall(hipMalloc(&gpu_cuComplex_buffer, frame_res * sizeof(hipComplex)));

    convolution_kernel(gpu_arr,
                       nullptr,
                       gpu_cuComplex_buffer,
                       &handle,
                       frame_res,
                       gpu_complex_blur_matrix,
                       false,
                       false,
                       stream);

    cudaXFree(gpu_cuComplex_buffer);
    cudaXFree(gpu_float_blur_matrix);
    cudaXFree(gpu_complex_blur_matrix);
}

void hsv_normalize(
    float* const gpu_arr, const uint frame_res, float* const gpu_min, float* const gpu_max, const hipStream_t stream)
{
    reduce_min(gpu_arr, gpu_min, frame_res, stream); // Get the minimum value
    reduce_max(gpu_arr, gpu_max, frame_res, stream); // Get the maximum value

    const auto lambda = [gpu_min, gpu_max] __device__(const float pixel)
    { return (pixel - *gpu_min) * (1 / (*gpu_max - *gpu_min)); };

    auto exec_policy = thrust::cuda::par.on(stream);
    thrust::transform(exec_policy, gpu_arr, gpu_arr + frame_res, gpu_arr, lambda);
}

void apply_operations(float* gpu_arr,
                      uint height,
                      uint width,
                      const holovibes::CompositeChannel& channel_struct,
                      HSV channel,
                      threshold_op op,
                      const hipStream_t stream)
{
    const uint frame_res = height * width;
    float* gpu_channel_arr = gpu_arr + frame_res * channel;
    auto exec_policy = thrust::cuda::par.on(stream);

    apply_percentile_and_threshold(gpu_channel_arr,
                                   frame_res,
                                   width,
                                   height,
                                   channel_struct.threshold.min,
                                   channel_struct.threshold.max,
                                   stream);

    if (op == CLAMP || op == CRUSH)
    {
        threshold_top_bottom(gpu_channel_arr,
                             channel_struct.slider_threshold.min,
                             channel_struct.slider_threshold.max,
                             frame_res,
                             stream);

        if (op == CRUSH)
        {
            auto min = channel_struct.slider_threshold.min;
            auto scale = 1.0f / (channel_struct.slider_threshold.max - min);
            const auto crush_op = [min, scale] __device__(const float pixel) { return (pixel - min) * scale; };

            thrust::transform(exec_policy, gpu_channel_arr, gpu_channel_arr + frame_res, gpu_channel_arr, crush_op);
        }
    }
    else if (op == ZOOM)
    {
        auto min = channel_struct.slider_threshold.min;
        auto diff = channel_struct.slider_threshold.max - min;
        const auto zoom_op = [min, diff] __device__(const float pixel) { return (pixel * diff) + min; };

        thrust::transform(exec_policy, gpu_channel_arr, gpu_channel_arr + frame_res, gpu_channel_arr, zoom_op);
    }
}

/// @brief Apply basic image processing operations on h,s and v (threshold, normalization, blur...)
void apply_operations_on_hsv(float* tmp_hsv_arr,
                             const uint height,
                             const uint width,
                             const holovibes::CompositeHSV& hsv_struct,
                             const hipStream_t stream)
{
    // To perform a renormalization, a single min buffer and single max buffer is needed gpu side
    holovibes::cuda_tools::CudaUniquePtr<float> gpu_min(1);
    holovibes::cuda_tools::CudaUniquePtr<float> gpu_max(1);

    // HUE
    apply_operations(tmp_hsv_arr, height, width, hsv_struct.h, HSV::H, threshold_op::CRUSH, stream);
    // H channel has a blur option
    if (hsv_struct.h.blur.enabled)
    {
        apply_blur(tmp_hsv_arr, height, width, hsv_struct.h.blur.kernel_size, stream);

        hsv_normalize(tmp_hsv_arr, height * width, gpu_min.get(), gpu_max.get(), stream);
    }

    // SATURATION
    apply_operations(tmp_hsv_arr, height, width, hsv_struct.s, HSV::S, threshold_op::CRUSH, stream);

    // VALUE
    apply_operations(tmp_hsv_arr, height, width, hsv_struct.v, HSV::V, threshold_op::CRUSH, stream);
}

/// @brief Create rgb color by using hsv computation and then converting to rgb
/// @param gpu_input complex input buffer, on gpu side, size = width * height * time_transformation_size
/// @param gpu_output float output buffer, on gpu side, size = width * height * 3
/// @param width Width of the frame
/// @param height Height of the frame
/// @param stream Cuda stream used
/// @param time_transformation_size Depth of the frame cube
/// @param hsv_struct Struct containing all the UI parameters
void hsv(const hipComplex* gpu_input,
         float* gpu_output,
         const uint width,
         const uint height,
         const hipStream_t stream,
         const int time_transformation_size,
         const holovibes::CompositeHSV& hsv_struct)
{
    const uint frame_res = height * width;

    float* tmp_hsv_arr = nullptr;
    cudaSafeCall(hipMalloc(&tmp_hsv_arr, frame_res * 3 * sizeof(float)));
    compute_and_fill_hsv(gpu_input, tmp_hsv_arr, frame_res, hsv_struct, stream);

    apply_operations_on_hsv(tmp_hsv_arr, height, width, hsv_struct, stream);

    normalized_convert_hsv_to_rgb(tmp_hsv_arr, gpu_output, frame_res, stream);

    cudaXFree(tmp_hsv_arr);
}

/*
__global__ void kernel_fill_hsv_xz_cut(const float* gpu_in_cut,
                                       float* gpu_hsv_cut,
                                       size_t width,
                                       uint depth,
                                       size_t frame_res,
                                       const holovibes::CompositeHSV& hsv_struct)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < width)
    {
        uint min_h_index = hsv_struct.h.frame_index.min;
        uint max_h_index = hsv_struct.h.frame_index.max;
        auto min = hsv_struct.h.slider_threshold.min;
        auto max = hsv_struct.h.slider_threshold.max;

        float num = 0.0f;
        float denom = 0.0f;
        for (size_t z = 0; z <= depth; ++z)
        {
            size_t index = width * z + id;
            float input_elm = gpu_in_cut[index];

            // For hsv cut colorization
            if (z >= min_h_index && z <= max_h_index)
            {
                float raw_hue = (float)(z - min_h_index) / (float)(max_h_index - min_h_index);

                if (raw_hue < min || raw_hue > max)
                {
                    // If the hue is outside the threshold, set the hue to red
                    gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                }
                else
                {
                    // Else, normalize the hue
                    auto scale = 1.0f / (max - min);
                    gpu_hsv_cut[HSV::H * frame_res + index] = (raw_hue - min) * scale;
                }

                // Not too saturated to see the average
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.5f;
            }
            else
            {
                // Greyscale
                gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.0f;
            }
            // Normalize the value to [0,1]
            gpu_hsv_cut[HSV::V * frame_res + index] = input_elm / max_ushort_value;

            // For average computation
            num += input_elm * z;
            denom += input_elm;
        }

        // Set the average index to a saturated red
        float avg = (denom == 0.0f ? 0.0f : num / denom);
        size_t avg_index = std::round(avg);
        size_t index = width * avg_index + id;
        gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
        gpu_hsv_cut[HSV::S * frame_res + index] = 1.0f;
        gpu_hsv_cut[HSV::V * frame_res + index] = 1.0f;
    }
}

void fill_hsv_xz_cut(const float* gpu_in_cut,
                     float* gpu_hsv_cut,
                     uint width,
                     uint depth,
                     const holovibes::CompositeHSV& hsv_struct,
                     const hipStream_t stream)
{
    size_t frame_res = width * depth;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(width, threads);
    kernel_fill_hsv_xz_cut<<<blocks, threads, 0, stream>>>(gpu_in_cut,
                                                           gpu_hsv_cut,
                                                           width,
                                                           depth,
                                                           frame_res,
                                                           hsv_struct);
    cudaCheckError();
}

__global__ void kernel_fill_hsv_gradient(float* gpu_hsv_cut, size_t width, uint height)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < width * height)
    {
        size_t x = id % width;
        size_t y = id / width;
        float x_ratio = (float)x / (float)width;
        float y_ratio = (float)y / (float)height;
        gpu_hsv_cut[HSV::H * width * height + id] = y_ratio;
        gpu_hsv_cut[HSV::S * width * height + id] = 1.0f;
        gpu_hsv_cut[HSV::V * width * height + id] = x_ratio;
    }
}

void fill_hsv_gradient(float* gpu_hsv_cut, uint width, uint height, const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(width, threads);
    kernel_fill_hsv_gradient<<<blocks, threads, 0, stream>>>(gpu_hsv_cut, width, height);
    cudaCheckError();
}

void fill_hsv_yz_cut(
    const float* gpu_in_cut, float* gpu_hsv_cut, uint height, uint depth, const holovibes::CompositeHSV& hsv_struct)
{
    size_t frame_res = height * depth;
    for (size_t i = 0; i < height; i++)
    {
        uint min_h_index = hsv_struct.h.frame_index.min;
        uint max_h_index = hsv_struct.h.frame_index.max;

        float num = 0.0f;
        float denom = 0.0f;
        for (size_t z = 0; z <= depth; ++z)
        {
            size_t index = depth * i + z;
            float input_elm = gpu_in_cut[index];

            // For hsv cut colorization
            if (z >= min_h_index && z <= max_h_index)
            {
                float raw_hue = (float)(z - min_h_index) / (float)(max_h_index - min_h_index);
                auto min = hsv_struct.h.slider_threshold.min;
                auto max = hsv_struct.h.slider_threshold.max;

                if (raw_hue < min || raw_hue > max)
                {
                    // If the hue is outside the threshold, set the hue to red
                    gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                }
                else
                {
                    // Else, normalize the hue
                    auto scale = 1.0f / (max - min);
                    gpu_hsv_cut[HSV::H * frame_res + index] = (raw_hue - min) * scale;
                }

                // Not too saturated to see the average
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.5f;
            }
            else
            {
                // Greyscale
                gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.0f;
            }
            // Value doesn't change
            gpu_hsv_cut[HSV::V * frame_res + index] = input_elm / max_ushort_value;

            // For average computation
            num += input_elm * z;
            denom += input_elm;
        }

        // Set the average index to a saturated red
        float avg = (denom == 0.0f ? 0.0f : num / denom);
        size_t avg_index = std::round(avg);
        size_t index = depth * i + avg_index;
        gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
        gpu_hsv_cut[HSV::S * frame_res + index] = 1.0f;
        gpu_hsv_cut[HSV::V * frame_res + index] = 1.0f;
    }
}

void hsv_cuts(const float* gpu_in_cut_xz,
              const float* gpu_in_cut_yz,
              float* gpu_out_cut_xz,
              float* gpu_out_cut_yz,
              uint width,
              uint height,
              int time_transformation_size,
              const holovibes::CompositeHSV& hsv_struct,
              const hipStream_t stream)
{
    float* tmp_xz_hsv_arr = nullptr;
    // float* tmp_yz_hsv_arr = nullptr;
    cudaSafeCall(hipMalloc(&tmp_xz_hsv_arr, width * time_transformation_size * 3 * sizeof(float)));
    // cudaSafeCall(hipMalloc(&tmp_yz_hsv_arr, height * time_transformation_size * 3 * sizeof(float)));
    //fill_hsv_xz_cut(gpu_in_cut_xz, tmp_xz_hsv_arr, width, time_transformation_size, hsv_struct, stream);
    fill_hsv_gradient(tmp_xz_hsv_arr, width, time_transformation_size, stream);
    // fill_hsv_yz_cut(gpu_in_cut_yz, tmp_yz_hsv_arr, height, time_transformation_size, hsv_struct, stream);

    normalized_convert_hsv_to_rgb(tmp_xz_hsv_arr, gpu_out_cut_xz, width * time_transformation_size, stream);
    // normalized_convert_hsv_to_rgb(tmp_yz_hsv_arr, gpu_out_cut_yz, height * time_transformation_size, stream);

    cudaXFree(tmp_xz_hsv_arr);
    // cudaXFree(tmp_yz_hsv_arr);
}
*/