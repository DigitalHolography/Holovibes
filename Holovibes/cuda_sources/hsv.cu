#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hsv.cuh"
#include "tools_hsv.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "percentile.cuh"
#include "cuda_memory.cuh"
#include "shift_corners.cuh"
#include "map.cuh"
#include "reduce.cuh"
#include "unique_ptr.hh"
#include "logger.hh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#define SAMPLING_FREQUENCY 1
static constexpr ushort max_ushort_value = (1 << (sizeof(ushort) * 8)) - 1;

__global__ void kernel_normalized_convert_hsv_to_rgb(const float* src, float* dst, size_t frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float nNormalizedH = src[frame_res * HSV::H + id];
        float nNormalizedS = src[frame_res * HSV::S + id];
        float nNormalizedV = src[frame_res * HSV::V + id];
        float nR;
        float nG;
        float nB;
        if (nNormalizedS == 0.0F)
        {
            nR = nG = nB = nNormalizedV;
        }
        else
        {
            if (nNormalizedH == 1.0F)
                nNormalizedH = 0.0F;
            else
                nNormalizedH = nNormalizedH * 6.0F; // / 0.1667F
        }
        float nI = floorf(nNormalizedH);
        float nF = nNormalizedH - nI;
        float nM = nNormalizedV * (1.0F - nNormalizedS);
        float nN = nNormalizedV * (1.0F - nNormalizedS * nF);
        float nK = nNormalizedV * (1.0F - nNormalizedS * (1.0F - nF));
        if (nI == 0.0F)
        {
            nR = nNormalizedV;
            nG = nK;
            nB = nM;
        }
        else if (nI == 1.0F)
        {
            nR = nN;
            nG = nNormalizedV;
            nB = nM;
        }
        else if (nI == 2.0F)
        {
            nR = nM;
            nG = nNormalizedV;
            nB = nK;
        }
        else if (nI == 3.0F)
        {
            nR = nM;
            nG = nN;
            nB = nNormalizedV;
        }
        else if (nI == 4.0F)
        {
            nR = nK;
            nG = nM;
            nB = nNormalizedV;
        }
        else if (nI == 5.0F)
        {
            nR = nNormalizedV;
            nG = nM;
            nB = nN;
        }
        dst[id * 3 + HSV::H] = nR * max_ushort_value;
        dst[id * 3 + HSV::S] = nG * max_ushort_value;
        dst[id * 3 + HSV::V] = nB * max_ushort_value;
    }
}

/// @brief Convert an array of HSV normalized float [0,1] to an array of RGB float [0,65536]
/// @param src Input hsv array (contiguous pixel on x: [h1,...,hn,s1,...,sn,v1,...,vn])
/// @param dst Output rgb array (contiguous rgb channels: [r1,g1,b1,...,rn,gn,bn])
/// @param frame_res Total number of pixels on one frame
/// @param stream The used cuda stream
void normalized_convert_hsv_to_rgb(const float* src, float* dst, size_t frame_res, const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_normalized_convert_hsv_to_rgb<<<blocks, threads, 0, stream>>>(src, dst, frame_res);
    cudaCheckError();
}

/// @brief get the real z value, because of the FFT frequency shift
__device__ float get_real_z(size_t z, int depth)
{
    if ((float)z < (float)depth / 2.0f)
        return (float)z;
    else if (z == depth / 2)
        return 0.0f;
    return (float)z - (float)depth;
}

/// @brief Convert the input complex number to a float
/// @param input_elm input complex number
/// @return a float, reprensenting the magnitude of the input complex number
__device__ float get_input_elm(hipComplex input_elm)
{
    return hypotf(input_elm.x, input_elm.y);
}

__global__ void kernel_compute_and_fill_h(const hipComplex* gpu_input,
                                          float* gpu_output,
                                          const size_t frame_res,
                                          const uint min_h_index,
                                          const uint max_h_index,
                                          int depth)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float num = 0.0f;
        float denom = 0.0f;

        // Compute the Average
        for (size_t z = min_h_index; z <= max_h_index; ++z)
        {
            const hipComplex* current_p_frame = gpu_input + (z * frame_res);
            float input_elm = get_input_elm(current_p_frame[id]);

            size_t real_z = get_real_z(z, depth);
            num += input_elm * real_z;
            denom += input_elm;
        }

        gpu_output[id] = (denom == 0.0f ? 0.0f : num / denom);
    }
}

__global__ void kernel_compute_and_fill_s(const hipComplex* gpu_input,
                                          float* gpu_output,
                                          const size_t frame_res,
                                          const uint min_s_index,
                                          const uint max_s_index,
                                          int depth)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        size_t z = min_s_index;
        float num = 0.0f;
        float denom = 0.0f;

        // Compute the Average
        for (z = min_s_index; z <= max_s_index; ++z)
        {
            const hipComplex* current_p_frame = gpu_input + (z * frame_res);
            float input_elm = get_input_elm(current_p_frame[id]);

            num += input_elm * get_real_z(z, depth);
            denom += input_elm;
        }
        float avg = (denom == 0.0f ? 0.0f : num / denom);

        // Compute the variance
        num = 0.0f;
        for (z = min_s_index; z <= max_s_index; ++z)
        {
            const hipComplex* current_p_frame = gpu_input + (z * frame_res);
            float input_elm = get_input_elm(current_p_frame[id]);

            float centered_z = get_real_z(z, depth) - avg;
            num += input_elm * centered_z * centered_z;
        }

        gpu_output[id] = (denom == 0.0f ? 0.0f : num / denom);
    }
}

__global__ void kernel_compute_and_fill_v(const hipComplex* gpu_input,
                                          float* gpu_output,
                                          const size_t frame_res,
                                          const uint min_v_index,
                                          const uint max_v_index,
                                          int depth)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float sum = 0.0f;

        // Compute the Sum
        for (size_t z = min_v_index; z <= max_v_index; ++z)
        {
            const hipComplex* current_p_frame = gpu_input + (z * frame_res);
            float input_elm = hypotf(current_p_frame[id].x, current_p_frame[id].y);
            sum += input_elm;
        }

        gpu_output[id] = sum;
    }
}

void compute_and_fill_h(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream,
                        int depth)
{
    const uint min_h_index = hsv_struct.h.frame_index.min;
    const uint max_h_index = hsv_struct.h.frame_index.max;

    float* gpu_h_output = gpu_output + HSV::H * frame_res;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    kernel_compute_and_fill_h<<<blocks, threads, 0, stream>>>(gpu_input,
                                                              gpu_h_output,
                                                              frame_res,
                                                              min_h_index,
                                                              max_h_index,
                                                              depth);
    cudaCheckError();
}

void compute_and_fill_s(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream,
                        int depth)
{
    const uint min_s_index =
        hsv_struct.s.frame_index.activated ? hsv_struct.s.frame_index.min : hsv_struct.h.frame_index.min;
    const uint max_s_index =
        hsv_struct.s.frame_index.activated ? hsv_struct.s.frame_index.max : hsv_struct.h.frame_index.max;

    float* gpu_s_output = gpu_output + HSV::S * frame_res;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    kernel_compute_and_fill_s<<<blocks, threads, 0, stream>>>(gpu_input,
                                                              gpu_s_output,
                                                              frame_res,
                                                              min_s_index,
                                                              max_s_index,
                                                              depth);
    cudaCheckError();
}

void compute_and_fill_v(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream,
                        int depth)
{
    const uint min_v_index =
        hsv_struct.v.frame_index.activated ? hsv_struct.v.frame_index.min : hsv_struct.h.frame_index.min;
    const uint max_v_index =
        hsv_struct.v.frame_index.activated ? hsv_struct.v.frame_index.max : hsv_struct.h.frame_index.max;

    float* gpu_v_output = gpu_output + HSV::V * frame_res;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    kernel_compute_and_fill_v<<<blocks, threads, 0, stream>>>(gpu_input,
                                                              gpu_v_output,
                                                              frame_res,
                                                              min_v_index,
                                                              max_v_index,
                                                              depth);
    cudaCheckError();
}

/// @brief Compute the hsv values of each pixel of the frame
void compute_and_fill_hsv(const hipComplex* gpu_input,
                          float* gpu_output,
                          const size_t frame_res,
                          const holovibes::CompositeHSV& hsv_struct,
                          const hipStream_t stream,
                          int depth)
{
    compute_and_fill_h(gpu_input, gpu_output, frame_res, hsv_struct, stream, depth);
    compute_and_fill_s(gpu_input, gpu_output, frame_res, hsv_struct, stream, depth);
    compute_and_fill_v(gpu_input, gpu_output, frame_res, hsv_struct, stream, depth);
}

// Apply a box blur on the specified array
void apply_blur(float* gpu_arr, uint height, uint width, float kernel_size, const hipStream_t stream)
{
    size_t frame_res = height * width;

    float* gpu_float_blur_matrix;
    cudaSafeCall(hipMalloc(&gpu_float_blur_matrix, frame_res * sizeof(float)));
    cudaSafeCall(hipMemsetAsync(gpu_float_blur_matrix, 0, frame_res * sizeof(float), stream));

    float* blur_matrix;
    cudaSafeCall(hipHostMalloc(&blur_matrix, kernel_size * sizeof(float)));
    float blur_value = 1.0f / (float)(kernel_size * kernel_size);
    unsigned min_pos_kernel_y = height / 2 - kernel_size / 2;
    unsigned min_pos_kernel_x = width / 2 - kernel_size / 2;
    for (size_t i = 0; i < kernel_size; i++)
        blur_matrix[i] = blur_value;

    for (size_t i = 0; i < kernel_size; i++)
    {
        cudaXMemcpyAsync(gpu_float_blur_matrix + min_pos_kernel_x + width * (i + min_pos_kernel_y),
                         blur_matrix,
                         kernel_size * sizeof(float),
                         hipMemcpyHostToDevice,
                         stream);
    }

    float* cpu_float_blur_matrix = new float[frame_res];
    cudaSafeCall(hipMemcpyAsync(cpu_float_blur_matrix,
                                 gpu_float_blur_matrix,
                                 frame_res * sizeof(float),
                                 hipMemcpyDeviceToHost,
                                 stream));

    hipComplex* gpu_complex_blur_matrix;
    cudaSafeCall(hipMalloc(&gpu_complex_blur_matrix, frame_res * sizeof(hipComplex)));
    cudaSafeCall(hipMemcpy2DAsync(gpu_complex_blur_matrix,
                                   sizeof(hipComplex),
                                   gpu_float_blur_matrix,
                                   sizeof(float),
                                   sizeof(float),
                                   frame_res,
                                   hipMemcpyDeviceToDevice,
                                   stream));

    shift_corners(gpu_complex_blur_matrix, 1, width, height, stream);

    CufftHandle handle{static_cast<int>(width), static_cast<int>(height), HIPFFT_C2C};
    cufftSafeCall(hipfftExecC2C(handle, gpu_complex_blur_matrix, gpu_complex_blur_matrix, HIPFFT_FORWARD));

    hipComplex* gpu_cuComplex_buffer;
    cudaSafeCall(hipMalloc(&gpu_cuComplex_buffer, frame_res * sizeof(hipComplex)));

    convolution_kernel(gpu_arr,
                       nullptr,
                       gpu_cuComplex_buffer,
                       &handle,
                       frame_res,
                       gpu_complex_blur_matrix,
                       false,
                       false,
                       stream);

    cudaXFree(gpu_cuComplex_buffer);
    cudaXFree(gpu_float_blur_matrix);
    cudaXFree(gpu_complex_blur_matrix);
}

void hsv_normalize(
    float* const gpu_arr, const uint frame_res, float* const gpu_min, float* const gpu_max, const hipStream_t stream)
{
    reduce_min(gpu_arr, gpu_min, frame_res, stream); // Get the minimum value
    reduce_max(gpu_arr, gpu_max, frame_res, stream); // Get the maximum value

    const auto lambda = [gpu_min, gpu_max] __device__(const float pixel)
    { return (pixel - *gpu_min) * (1 / (*gpu_max - *gpu_min)); };

    auto exec_policy = thrust::cuda::par.on(stream);
    thrust::transform(exec_policy, gpu_arr, gpu_arr + frame_res, gpu_arr, lambda);
}

/// @brief Basic operation on any specified channel, with any operation requested
void apply_operations(float* gpu_arr,
                      uint height,
                      uint width,
                      const holovibes::CompositeChannel& channel_struct,
                      HSV channel,
                      threshold_op op,
                      const hipStream_t stream)
{
    const uint frame_res = height * width;
    float* gpu_channel_arr = gpu_arr + frame_res * channel;
    auto exec_policy = thrust::cuda::par.on(stream);

    apply_percentile_and_threshold(gpu_channel_arr,
                                   frame_res,
                                   width,
                                   height,
                                   channel_struct.threshold.min,
                                   channel_struct.threshold.max,
                                   stream);

    if (op == CLAMP || op == CRUSH)
    {
        threshold_top_bottom(gpu_channel_arr,
                             channel_struct.slider_threshold.min,
                             channel_struct.slider_threshold.max,
                             frame_res,
                             stream);

        if (op == CRUSH)
        {
            auto min = channel_struct.slider_threshold.min;
            auto scale = 1.0f / (channel_struct.slider_threshold.max - min);
            const auto crush_op = [min, scale] __device__(const float pixel) { return (pixel - min) * scale; };

            thrust::transform(exec_policy, gpu_channel_arr, gpu_channel_arr + frame_res, gpu_channel_arr, crush_op);
        }
    }
    else if (op == ZOOM)
    {
        auto min = channel_struct.slider_threshold.min;
        auto diff = channel_struct.slider_threshold.max - min;
        const auto zoom_op = [min, diff] __device__(const float pixel) { return (pixel * diff) + min; };

        thrust::transform(exec_policy, gpu_channel_arr, gpu_channel_arr + frame_res, gpu_channel_arr, zoom_op);
    }
}

/// @brief Special function for hue channel because hue has two UI sliders and a blur option
void apply_operations_on_h(
    float* gpu_h_arr, uint height, uint width, const holovibes::CompositeH& h_struct, const hipStream_t stream)
{
    // To perform a renormalization, a single min buffer and single max buffer is needed gpu side
    holovibes::cuda_tools::CudaUniquePtr<float> gpu_min(1);
    holovibes::cuda_tools::CudaUniquePtr<float> gpu_max(1);
    const uint frame_res = height * width;
    auto exec_policy = thrust::cuda::par.on(stream);

    // H channel has a blur option
    if (h_struct.blur.enabled)
    {
        apply_blur(gpu_h_arr, height, width, h_struct.blur.kernel_size, stream);

        //hsv_normalize(gpu_h_arr, height * width, gpu_min.get(), gpu_max.get(), stream);
    }

    apply_percentile_and_threshold(gpu_h_arr,
                                   frame_res,
                                   width,
                                   height,
                                   h_struct.threshold.min,
                                   h_struct.threshold.max,
                                   stream);

    float range_min = h_struct.slider_threshold.min;
    float range_max = h_struct.slider_threshold.max;
    float shift_min = h_struct.slider_shift.min;
    float shift_max = h_struct.slider_shift.max;

    auto m = (range_max - range_min) / (shift_max - shift_min);
    auto p = range_min - m * shift_min;
    const auto op = [m, p, shift_min, shift_max, range_min, range_max] __device__(const float pixel)
    {
        if (pixel < shift_min)
            return range_min;
        else if (pixel > shift_max)
            return range_max;
        else
            return m * pixel + p;
    };
    thrust::transform(exec_policy, gpu_h_arr, gpu_h_arr + frame_res, gpu_h_arr, op);
}

/// @brief Apply basic image processing operations on h,s and v (threshold, normalization, blur...)
void apply_operations_on_hsv(float* tmp_hsv_arr,
                             const uint height,
                             const uint width,
                             const holovibes::CompositeHSV& hsv_struct,
                             const hipStream_t stream)
{
    // HUE
    apply_operations_on_h(tmp_hsv_arr, height, width, hsv_struct.h, stream);
    // SATURATION
    apply_operations(tmp_hsv_arr, height, width, hsv_struct.s, HSV::S, threshold_op::CRUSH, stream);
    // VALUE
    apply_operations(tmp_hsv_arr, height, width, hsv_struct.v, HSV::V, threshold_op::CRUSH, stream);
}

/// @brief Create rgb color by using hsv computation and then converting to rgb
/// @param gpu_input complex input buffer, on gpu side, size = width * height * time_transformation_size
/// @param gpu_output float output buffer, on gpu side, size = width * height * 3
/// @param width Width of the frame
/// @param height Height of the frame
/// @param stream Cuda stream used
/// @param time_transformation_size Depth of the frame cube
/// @param hsv_struct Struct containing all the UI parameters
void hsv(const hipComplex* gpu_input,
         float* gpu_output,
         const uint width,
         const uint height,
         const hipStream_t stream,
         const int time_transformation_size,
         const holovibes::CompositeHSV& hsv_struct)
{
    const uint frame_res = height * width;

    float* tmp_hsv_arr = nullptr;
    cudaSafeCall(hipMalloc(&tmp_hsv_arr, frame_res * 3 * sizeof(float)));
    compute_and_fill_hsv(gpu_input, tmp_hsv_arr, frame_res, hsv_struct, stream, time_transformation_size);

    apply_operations_on_hsv(tmp_hsv_arr, height, width, hsv_struct, stream);

    normalized_convert_hsv_to_rgb(tmp_hsv_arr, gpu_output, frame_res, stream);

    cudaXFree(tmp_hsv_arr);
}

/*
__global__ void kernel_fill_hsv_xz_cut(const float* gpu_in_cut,
                                       float* gpu_hsv_cut,
                                       size_t width,
                                       uint depth,
                                       size_t frame_res,
                                       const holovibes::CompositeHSV& hsv_struct)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < width)
    {
        uint min_h_index = hsv_struct.h.frame_index.min;
        uint max_h_index = hsv_struct.h.frame_index.max;
        auto min = hsv_struct.h.slider_threshold.min;
        auto max = hsv_struct.h.slider_threshold.max;

        float num = 0.0f;
        float denom = 0.0f;
        for (size_t z = 0; z <= depth; ++z)
        {
            size_t index = width * z + id;
            float input_elm = gpu_in_cut[index];

            // For hsv cut colorization
            if (z >= min_h_index && z <= max_h_index)
            {
                float raw_hue = (float)(z - min_h_index) / (float)(max_h_index - min_h_index);

                if (raw_hue < min || raw_hue > max)
                {
                    // If the hue is outside the threshold, set the hue to red
                    gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                }
                else
                {
                    // Else, normalize the hue
                    auto scale = 1.0f / (max - min);
                    gpu_hsv_cut[HSV::H * frame_res + index] = (raw_hue - min) * scale;
                }

                // Not too saturated to see the average
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.5f;
            }
            else
            {
                // Greyscale
                gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.0f;
            }
            // Normalize the value to [0,1]
            gpu_hsv_cut[HSV::V * frame_res + index] = input_elm / max_ushort_value;

            // For average computation
            num += input_elm * z;
            denom += input_elm;
        }

        // Set the average index to a saturated red
        float avg = (denom == 0.0f ? 0.0f : num / denom);
        size_t avg_index = std::round(avg);
        size_t index = width * avg_index + id;
        gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
        gpu_hsv_cut[HSV::S * frame_res + index] = 1.0f;
        gpu_hsv_cut[HSV::V * frame_res + index] = 1.0f;
    }
}

void fill_hsv_xz_cut(const float* gpu_in_cut,
                     float* gpu_hsv_cut,
                     uint width,
                     uint depth,
                     const holovibes::CompositeHSV& hsv_struct,
                     const hipStream_t stream)
{
    size_t frame_res = width * depth;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(width, threads);
    kernel_fill_hsv_xz_cut<<<blocks, threads, 0, stream>>>(gpu_in_cut,
                                                           gpu_hsv_cut,
                                                           width,
                                                           depth,
                                                           frame_res,
                                                           hsv_struct);
    cudaCheckError();
}

__global__ void kernel_fill_hsv_gradient(float* gpu_hsv_cut, size_t width, uint height)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < width * height)
    {
        size_t x = id % width;
        size_t y = id / width;
        float x_ratio = (float)x / (float)width;
        float y_ratio = (float)y / (float)height;
        gpu_hsv_cut[HSV::H * width * height + id] = y_ratio;
        gpu_hsv_cut[HSV::S * width * height + id] = 1.0f;
        gpu_hsv_cut[HSV::V * width * height + id] = x_ratio;
    }
}

void fill_hsv_gradient(float* gpu_hsv_cut, uint width, uint height, const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(width, threads);
    kernel_fill_hsv_gradient<<<blocks, threads, 0, stream>>>(gpu_hsv_cut, width, height);
    cudaCheckError();
}

void fill_hsv_yz_cut(
    const float* gpu_in_cut, float* gpu_hsv_cut, uint height, uint depth, const holovibes::CompositeHSV& hsv_struct)
{
    size_t frame_res = height * depth;
    for (size_t i = 0; i < height; i++)
    {
        uint min_h_index = hsv_struct.h.frame_index.min;
        uint max_h_index = hsv_struct.h.frame_index.max;

        float num = 0.0f;
        float denom = 0.0f;
        for (size_t z = 0; z <= depth; ++z)
        {
            size_t index = depth * i + z;
            float input_elm = gpu_in_cut[index];

            // For hsv cut colorization
            if (z >= min_h_index && z <= max_h_index)
            {
                float raw_hue = (float)(z - min_h_index) / (float)(max_h_index - min_h_index);
                auto min = hsv_struct.h.slider_threshold.min;
                auto max = hsv_struct.h.slider_threshold.max;

                if (raw_hue < min || raw_hue > max)
                {
                    // If the hue is outside the threshold, set the hue to red
                    gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                }
                else
                {
                    // Else, normalize the hue
                    auto scale = 1.0f / (max - min);
                    gpu_hsv_cut[HSV::H * frame_res + index] = (raw_hue - min) * scale;
                }

                // Not too saturated to see the average
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.5f;
            }
            else
            {
                // Greyscale
                gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
                gpu_hsv_cut[HSV::S * frame_res + index] = 0.0f;
            }
            // Value doesn't change
            gpu_hsv_cut[HSV::V * frame_res + index] = input_elm / max_ushort_value;

            // For average computation
            num += input_elm * z;
            denom += input_elm;
        }

        // Set the average index to a saturated red
        float avg = (denom == 0.0f ? 0.0f : num / denom);
        size_t avg_index = std::round(avg);
        size_t index = depth * i + avg_index;
        gpu_hsv_cut[HSV::H * frame_res + index] = 0.0f;
        gpu_hsv_cut[HSV::S * frame_res + index] = 1.0f;
        gpu_hsv_cut[HSV::V * frame_res + index] = 1.0f;
    }
}

void hsv_cuts(const float* gpu_in_cut_xz,
              const float* gpu_in_cut_yz,
              float* gpu_out_cut_xz,
              float* gpu_out_cut_yz,
              uint width,
              uint height,
              int time_transformation_size,
              const holovibes::CompositeHSV& hsv_struct,
              const hipStream_t stream)
{
    float* tmp_xz_hsv_arr = nullptr;
    // float* tmp_yz_hsv_arr = nullptr;
    cudaSafeCall(hipMalloc(&tmp_xz_hsv_arr, width * time_transformation_size * 3 * sizeof(float)));
    // cudaSafeCall(hipMalloc(&tmp_yz_hsv_arr, height * time_transformation_size * 3 * sizeof(float)));
    //fill_hsv_xz_cut(gpu_in_cut_xz, tmp_xz_hsv_arr, width, time_transformation_size, hsv_struct, stream);
    fill_hsv_gradient(tmp_xz_hsv_arr, width, time_transformation_size, stream);
    // fill_hsv_yz_cut(gpu_in_cut_yz, tmp_yz_hsv_arr, height, time_transformation_size, hsv_struct, stream);

    normalized_convert_hsv_to_rgb(tmp_xz_hsv_arr, gpu_out_cut_xz, width * time_transformation_size, stream);
    // normalized_convert_hsv_to_rgb(tmp_yz_hsv_arr, gpu_out_cut_yz, height * time_transformation_size, stream);

    cudaXFree(tmp_xz_hsv_arr);
    // cudaXFree(tmp_yz_hsv_arr);
}
*/