#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hsv.cuh"
#include "tools_hsv.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "percentile.cuh"
#include "cuda_memory.cuh"
#include "shift_corners.cuh"
#include "map.cuh"
#include "reduce.cuh"
#include "unique_ptr.hh"
#include "logger.hh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#define SAMPLING_FREQUENCY 1

__global__ void kernel_normalized_convert_hsv_to_rgb(const float* src, float* dst, size_t frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float nNormalizedH = src[frame_res * HSV::H + id];
        float nNormalizedS = src[frame_res * HSV::S + id];
        float nNormalizedV = src[frame_res * HSV::V + id];
        float nR;
        float nG;
        float nB;
        if (nNormalizedS == 0.0F)
        {
            nR = nG = nB = nNormalizedV;
        }
        else
        {
            if (nNormalizedH == 1.0F)
                nNormalizedH = 0.0F;
            else
                nNormalizedH = nNormalizedH * 6.0F; // / 0.1667F
        }
        float nI = floorf(nNormalizedH);
        float nF = nNormalizedH - nI;
        float nM = nNormalizedV * (1.0F - nNormalizedS);
        float nN = nNormalizedV * (1.0F - nNormalizedS * nF);
        float nK = nNormalizedV * (1.0F - nNormalizedS * (1.0F - nF));
        if (nI == 0.0F)
        {
            nR = nNormalizedV;
            nG = nK;
            nB = nM;
        }
        else if (nI == 1.0F)
        {
            nR = nN;
            nG = nNormalizedV;
            nB = nM;
        }
        else if (nI == 2.0F)
        {
            nR = nM;
            nG = nNormalizedV;
            nB = nK;
        }
        else if (nI == 3.0F)
        {
            nR = nM;
            nG = nN;
            nB = nNormalizedV;
        }
        else if (nI == 4.0F)
        {
            nR = nK;
            nG = nM;
            nB = nNormalizedV;
        }
        else if (nI == 5.0F)
        {
            nR = nNormalizedV;
            nG = nM;
            nB = nN;
        }
        dst[id * 3 + HSV::H] = nR * 65536;
        dst[id * 3 + HSV::S] = nG * 65536;
        dst[id * 3 + HSV::V] = nB * 65536;
    }
}

/// @brief Convert an array of HSV normalized float [0,1] to an array of RGB float [0,65536]
/// @param src Input hsv array (contiguous pixel on x: [h1,...,hn,s1,...,sn,v1,...,vn])
/// @param dst Output rgb array (contiguous rgb channels: [r1,g1,b1,...,rn,gn,bn])
/// @param frame_res Total number of pixels on one frame
/// @param stream The used cuda stream
void normalized_convert_hsv_to_rgb(const float* src, float* dst, size_t frame_res, const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_normalized_convert_hsv_to_rgb<<<blocks, threads, 0, stream>>>(src, dst, frame_res);
    cudaCheckError();
}

template <typename FUNC>
__global__ void kernel_compute_sum_depth(
    const hipComplex* input, float* output, size_t frame_res, size_t min_index, size_t max_index, FUNC func)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float res = 0.0f;

        for (size_t z = min_index; z <= max_index; ++z)
        {
            float input_elm = fabsf(input[z * frame_res + id].x);

            res += input_elm * func(z);
        }

        const size_t range = max_index - min_index + 1;
        output[id] = (res / (float)range);
    }
}

/// @brief Compute the sum depth of a pixel : sum of input[z] * func(z) between z1 and z2
/// @param input The input hipComplex buffer
/// @param output The output float buffer
/// @param frame_res The total number of pixels in one frame
/// @param min_index z1
/// @param max_index z2
/// @param func the function to call on z
template <typename FUNC>
void compute_sum_depth(const hipComplex* input,
                       float* output,
                       size_t frame_res,
                       size_t min_index,
                       size_t max_index,
                       FUNC func,
                       const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_compute_sum_depth<<<blocks, threads, 0, stream>>>(input, output, frame_res, min_index, max_index, func);
    cudaCheckError();
}

void compute_and_fill_h(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream)
{
    const uint min_h_index = hsv_struct.h.frame_index.min;
    const uint max_h_index = hsv_struct.h.frame_index.max;

    float* gpu_h_output = gpu_output + HSV::H * frame_res;

    // Hue is the moment 1 (average)
    auto func_moment_one = [] __device__(size_t z) -> size_t { return z; };

    compute_sum_depth(gpu_input, gpu_h_output, frame_res, min_h_index, max_h_index, func_moment_one, stream);
}

void compute_and_fill_s(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream)
{
    const uint min_s_index =
        hsv_struct.s.frame_index.activated ? hsv_struct.s.frame_index.min : hsv_struct.h.frame_index.min;
    const uint max_s_index =
        hsv_struct.s.frame_index.activated ? hsv_struct.s.frame_index.max : hsv_struct.h.frame_index.max;

    float* gpu_s_output = gpu_output + HSV::S * frame_res;

    // Saturation is the moment 2 (variance)
    auto func_moment_two = [] __device__(size_t z) -> size_t { return z * z; };

    compute_sum_depth(gpu_input, gpu_s_output, frame_res, min_s_index, max_s_index, func_moment_two, stream);
}

void compute_and_fill_v(const hipComplex* gpu_input,
                        float* gpu_output,
                        const size_t frame_res,
                        const holovibes::CompositeHSV& hsv_struct,
                        const hipStream_t stream)
{
    const uint min_v_index =
        hsv_struct.v.frame_index.activated ? hsv_struct.v.frame_index.min : hsv_struct.h.frame_index.min;
    const uint max_v_index =
        hsv_struct.v.frame_index.activated ? hsv_struct.v.frame_index.max : hsv_struct.h.frame_index.max;

    float* gpu_v_output = gpu_output + HSV::V * frame_res;

    // Value is the moment 0
    auto func_moment_zero = [] __device__(size_t z) -> size_t { return 1; };

    compute_sum_depth(gpu_input, gpu_v_output, frame_res, min_v_index, max_v_index, func_moment_zero, stream);
}

/// @brief Compute the hsv values of each pixel, each channel use his own lambda function that describe the calculus
/// done on z
void compute_and_fill_hsv(const hipComplex* gpu_input,
                          float* gpu_output,
                          const size_t frame_res,
                          const holovibes::CompositeHSV& hsv_struct,
                          const hipStream_t stream)
{
    compute_and_fill_h(gpu_input, gpu_output, frame_res, hsv_struct, stream);
    compute_and_fill_s(gpu_input, gpu_output, frame_res, hsv_struct, stream);
    compute_and_fill_v(gpu_input, gpu_output, frame_res, hsv_struct, stream);
}

// Apply a box blur on the specified array
void apply_blur(float* gpu_arr, uint height, uint width, float kernel_size, const hipStream_t stream)
{
    size_t frame_res = height * width;

    float* gpu_float_blur_matrix;
    cudaSafeCall(hipMalloc(&gpu_float_blur_matrix, frame_res * sizeof(float)));
    cudaSafeCall(hipMemsetAsync(gpu_float_blur_matrix, 0, frame_res * sizeof(float), stream));

    float* blur_matrix;
    cudaSafeCall(hipHostMalloc(&blur_matrix, kernel_size * sizeof(float)));
    float blur_value = 1.0f / (float)(kernel_size * kernel_size);
    unsigned min_pos_kernel_y = height / 2 - kernel_size / 2;
    unsigned min_pos_kernel_x = width / 2 - kernel_size / 2;
    for (size_t i = 0; i < kernel_size; i++)
        blur_matrix[i] = blur_value;

    for (size_t i = 0; i < kernel_size; i++)
    {
        cudaXMemcpyAsync(gpu_float_blur_matrix + min_pos_kernel_x + width * (i + min_pos_kernel_y),
                         blur_matrix,
                         kernel_size * sizeof(float),
                         hipMemcpyHostToDevice,
                         stream);
    }

    float* cpu_float_blur_matrix = new float[frame_res];
    cudaSafeCall(hipMemcpyAsync(cpu_float_blur_matrix,
                                 gpu_float_blur_matrix,
                                 frame_res * sizeof(float),
                                 hipMemcpyDeviceToHost,
                                 stream));

    hipComplex* gpu_complex_blur_matrix;
    cudaSafeCall(hipMalloc(&gpu_complex_blur_matrix, frame_res * sizeof(hipComplex)));
    cudaSafeCall(hipMemcpy2DAsync(gpu_complex_blur_matrix,
                                   sizeof(hipComplex),
                                   gpu_float_blur_matrix,
                                   sizeof(float),
                                   sizeof(float),
                                   frame_res,
                                   hipMemcpyDeviceToDevice,
                                   stream));

    shift_corners(gpu_complex_blur_matrix, 1, width, height, stream);

    CufftHandle handle{static_cast<int>(width), static_cast<int>(height), HIPFFT_C2C};
    cufftSafeCall(hipfftExecC2C(handle, gpu_complex_blur_matrix, gpu_complex_blur_matrix, HIPFFT_FORWARD));

    hipComplex* gpu_cuComplex_buffer;
    cudaSafeCall(hipMalloc(&gpu_cuComplex_buffer, frame_res * sizeof(hipComplex)));

    convolution_kernel(gpu_arr,
                       nullptr,
                       gpu_cuComplex_buffer,
                       &handle,
                       frame_res,
                       gpu_complex_blur_matrix,
                       false,
                       false,
                       stream);

    cudaXFree(gpu_cuComplex_buffer);
    cudaXFree(gpu_float_blur_matrix);
    cudaXFree(gpu_complex_blur_matrix);
}

void hsv_normalize(
    float* const gpu_arr, const uint frame_res, float* const gpu_min, float* const gpu_max, const hipStream_t stream)
{
    reduce_min(gpu_arr, gpu_min, frame_res, stream); // Get the minimum value
    reduce_max(gpu_arr, gpu_max, frame_res, stream); // Get the maximum value

    const auto lambda = [gpu_min, gpu_max] __device__(const float pixel)
    { return (pixel - *gpu_min) * (1 / (*gpu_max - *gpu_min)); };

    auto exec_policy = thrust::cuda::par.on(stream);
    thrust::transform(exec_policy, gpu_arr, gpu_arr + frame_res, gpu_arr, lambda);
}

void apply_operations(float* gpu_arr,
                      uint height,
                      uint width,
                      float* const gpu_min,
                      float* const gpu_max,
                      const holovibes::CompositeChannel& channel_struct,
                      HSV channel,
                      const hipStream_t stream)
{
    const uint frame_res = height * width;
    float* gpu_channel_arr = gpu_arr + frame_res * channel;

    apply_percentile_and_threshold(gpu_channel_arr,
                                   frame_res,
                                   width,
                                   height,
                                   channel_struct.threshold.min,
                                   channel_struct.threshold.max,
                                   stream);

    hsv_normalize(gpu_channel_arr, frame_res, gpu_min, gpu_max, stream);

    threshold_top_bottom(gpu_channel_arr,
                         channel_struct.slider_threshold.min,
                         channel_struct.slider_threshold.max,
                         frame_res,
                         stream);

    hsv_normalize(gpu_channel_arr, frame_res, gpu_min, gpu_max, stream);
}

void apply_operations_on_h(float* gpu_arr,
                           uint height,
                           uint width,
                           float* const gpu_min,
                           float* const gpu_max,
                           const holovibes::CompositeHSV& hsv_struct,
                           const hipStream_t stream)
{
    const uint frame_res = height * width;

    apply_operations(gpu_arr, height, width, gpu_min, gpu_max, hsv_struct.h, HSV::H, stream);

    // H channel has a blur option
    if (hsv_struct.h.blur.enabled)
    {
        apply_blur(gpu_arr, height, width, hsv_struct.h.blur.kernel_size, stream);
    }

    hsv_normalize(gpu_arr, frame_res, gpu_min, gpu_max, stream);
}

void apply_operations_on_s(float* gpu_arr,
                           uint height,
                           uint width,
                           float* const gpu_min,
                           float* const gpu_max,
                           const holovibes::CompositeHSV& hsv_struct,
                           const hipStream_t stream)
{
    apply_operations(gpu_arr, height, width, gpu_min, gpu_max, hsv_struct.s, HSV::S, stream);
}

void apply_operations_on_v(float* gpu_arr,
                           uint height,
                           uint width,
                           float* const gpu_min,
                           float* const gpu_max,
                           const holovibes::CompositeHSV& hsv_struct,
                           const hipStream_t stream)
{
    apply_operations(gpu_arr, height, width, gpu_min, gpu_max, hsv_struct.v, HSV::V, stream);
}

/// @brief Apply basic image processing operations on h,s and v (threshold, normalization, blur...)
void apply_operations_on_hsv(float* tmp_hsv_arr,
                             const uint height,
                             const uint width,
                             const holovibes::CompositeHSV& hsv_struct,
                             const hipStream_t stream)
{
    // To perform a renormalization, a single min buffer and single max buffer is needed gpu side
    holovibes::cuda_tools::CudaUniquePtr<float> gpu_min(1);
    holovibes::cuda_tools::CudaUniquePtr<float> gpu_max(1);

    apply_operations_on_h(tmp_hsv_arr, height, width, gpu_min.get(), gpu_max.get(), hsv_struct, stream);
    apply_operations_on_s(tmp_hsv_arr, height, width, gpu_min.get(), gpu_max.get(), hsv_struct, stream);
    apply_operations_on_v(tmp_hsv_arr, height, width, gpu_min.get(), gpu_max.get(), hsv_struct, stream);
}

/// @brief Create rgb color by using hsv computation and then converting to rgb
/// @param gpu_input complex input buffer, on gpu side, size = width * height * time_transformation_size
/// @param gpu_output float output buffer, on gpu side, size = width * height * 3
/// @param width Width of the frame
/// @param height Height of the frame
/// @param stream Cuda stream used
/// @param time_transformation_size Depth of the frame cube
/// @param hsv_struct Struct containing all the UI parameters
void hsv(const hipComplex* gpu_input,
         float* gpu_output,
         const uint width,
         const uint height,
         const hipStream_t stream,
         const int time_transformation_size,
         const holovibes::CompositeHSV& hsv_struct)
{
    const uint frame_res = height * width;

    float* tmp_hsv_arr = nullptr;
    cudaSafeCall(hipMalloc(&tmp_hsv_arr, frame_res * 3 * sizeof(float)));
    compute_and_fill_hsv(gpu_input, tmp_hsv_arr, frame_res, hsv_struct, stream);

    apply_operations_on_hsv(tmp_hsv_arr, height, width, hsv_struct, stream);

    normalized_convert_hsv_to_rgb(tmp_hsv_arr, gpu_output, frame_res, stream);

    cudaXFree(tmp_hsv_arr);
}