#include "hip/hip_runtime.h"
/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hsv.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "percentile.cuh"
#include "cuda_memory.cuh"
#include "shift_corners.cuh"
#include "map.cuh"
#include "reduce.cuh"
#include "unique_ptr.hh"

#define SAMPLING_FREQUENCY 1
/*
 * \brief Convert an array of HSV normalized float to an array of RGB normalized
 * float i.e.: with "[  ]" a pixel: [HSV][HSV][HSV][HSV] -> [RGB][RGB][RGB][RGB]
 * NVdia function
 */

__global__ void kernel_normalized_convert_hsv_to_rgb(const float* src,
                                                     float* dst,
                                                     size_t frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        float nNormalizedH = src[id * 3];
        float nNormalizedS = src[id * 3 + 1];
        float nNormalizedV = src[id * 3 + 2];
        float nR;
        float nG;
        float nB;
        if (nNormalizedS == 0.0F)
        {
            nR = nG = nB = nNormalizedV;
        }
        else
        {
            if (nNormalizedH == 1.0F)
                nNormalizedH = 0.0F;
            else
                nNormalizedH = nNormalizedH * 6.0F; // / 0.1667F
        }
        float nI = floorf(nNormalizedH);
        float nF = nNormalizedH - nI;
        float nM = nNormalizedV * (1.0F - nNormalizedS);
        float nN = nNormalizedV * (1.0F - nNormalizedS * nF);
        float nK = nNormalizedV * (1.0F - nNormalizedS * (1.0F - nF));
        if (nI == 0.0F)
        {
            nR = nNormalizedV;
            nG = nK;
            nB = nM;
        }
        else if (nI == 1.0F)
        {
            nR = nN;
            nG = nNormalizedV;
            nB = nM;
        }
        else if (nI == 2.0F)
        {
            nR = nM;
            nG = nNormalizedV;
            nB = nK;
        }
        else if (nI == 3.0F)
        {
            nR = nM;
            nG = nN;
            nB = nNormalizedV;
        }
        else if (nI == 4.0F)
        {
            nR = nK;
            nG = nM;
            nB = nNormalizedV;
        }
        else if (nI == 5.0F)
        {
            nR = nNormalizedV;
            nG = nM;
            nB = nN;
        }
        dst[id * 3] = nR;
        dst[id * 3 + 1] = nG;
        dst[id * 3 + 2] = nB;
    }
}

__global__ void kernel_fill_square_frequency_axis(const size_t length,
                                                  float* arr)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length)
    {
        arr[length + id] = arr[id] * arr[id];
    }
}

__global__ void kernel_fill_part_frequency_axis(const size_t min,
                                                const size_t max,
                                                const double step,
                                                const double origin,
                                                float* arr)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (min + id < max)
    {
        arr[min + id] = origin + id * step;
    }
}

void fill_frequencies_arrays(const holovibes::ComputeDescriptor& cd,
                             float* gpu_omega_arr,
                             size_t frame_res,
                             const hipStream_t stream)
{
    const int time_transformation_size = cd.time_transformation_size;
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    double step = SAMPLING_FREQUENCY / (double)time_transformation_size;
    size_t after_mid_index =
        time_transformation_size / (double)2.0 + (double)1.0;

    kernel_fill_part_frequency_axis<<<blocks, threads, 0, stream>>>(0,
                                                               after_mid_index,
                                                               step,
                                                               0,
                                                               gpu_omega_arr);
    double negative_origin = -SAMPLING_FREQUENCY / (double)2.0;
    negative_origin += time_transformation_size % 2 ? step / (double)2.0 : step;

    kernel_fill_part_frequency_axis<<<blocks, threads, 0, stream>>>(
        after_mid_index,
        time_transformation_size,
        step,
        negative_origin,
        gpu_omega_arr);
    kernel_fill_square_frequency_axis<<<blocks, threads, 0, stream>>>(
        time_transformation_size,
        gpu_omega_arr);
}

/*
** \brief Compute H component of hsv.
*/
__global__ void kernel_compute_and_fill_h(const hipComplex* input,
                                          float* output,
                                          const size_t frame_res,
                                          const size_t min_index,
                                          const size_t max_index,
                                          const size_t total_index,
                                          const size_t omega_size,
                                          const float* omega_arr)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        const size_t index_H = id * 3;
        output[index_H] = 0;
        float summ_p = 0;
        float min = FLT_MAX;

        for (size_t i = min_index; i <= max_index; ++i)
        {
            float input_elm = fabsf(input[i * frame_res + id].x);
            min = fminf(min, input_elm);
            output[index_H] += input_elm * omega_arr[i];
            summ_p += input_elm;
        }

        output[index_H] -= total_index * min;
        output[index_H] /= summ_p;
    }
}

/*
** \brief Compute S component of hsv.
** Could be factorized with H but I kept it like this for the clarity
*/
__global__ void kernel_compute_and_fill_s(const hipComplex* input,
                                          float* output,
                                          const size_t frame_res,
                                          const size_t min_index,
                                          const size_t max_index,
                                          const size_t total_index,
                                          const size_t omega_size,
                                          const float* omega_arr)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        const size_t index_S = id * 3 + 1;
        output[index_S] = 0;

        float summ_p = 0;
        float min = FLT_MAX;

        for (size_t i = min_index; i <= max_index; ++i)
        {
            float input_elm = fabsf(input[i * frame_res + id].x);
            min = fminf(min, input_elm);
            output[index_S] += input_elm * omega_arr[i];
            summ_p += input_elm;
        }

        output[index_S] -= total_index * min;
        output[index_S] /= summ_p;
    }
}

/*
** \brief Compute V component of hsv.
*/
__global__ void kernel_compute_and_fill_v(const hipComplex* input,
                                          float* output,
                                          const size_t frame_res,
                                          const size_t min_index,
                                          const size_t max_index)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        const size_t index_V = id * 3 + 2;
        output[index_V] = 0;
        for (size_t i = min_index; i <= max_index; ++i)
        {
            float input_elm = fabsf(input[i * frame_res + id].x);
            output[index_V] += input_elm;
        }
    }
}

void compute_and_fill_hsv(const hipComplex* gpu_input,
                          float* gpu_output,
                          const size_t frame_res,
                          const holovibes::ComputeDescriptor& cd,
                          float* gpu_omega_arr,
                          size_t omega_arr_size,
                          const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    const uint min_h_index = cd.composite_p_min_h;
    const uint max_h_index = cd.composite_p_max_h;
    const uint min_s_index = cd.composite_p_min_s;
    const uint max_s_index = cd.composite_p_max_s;
    const uint min_v_index = cd.composite_p_min_v;
    const uint max_v_index = cd.composite_p_max_v;

    kernel_compute_and_fill_h<<<blocks, threads, 0, stream>>>(gpu_input,
                                                         gpu_output,
                                                         frame_res,
                                                         min_h_index,
                                                         max_h_index,
                                                         max_h_index -
                                                             min_h_index + 1,
                                                         omega_arr_size,
                                                         gpu_omega_arr);

    if (cd.composite_p_activated_s)
        kernel_compute_and_fill_s<<<blocks, threads, 0, stream>>>(
            gpu_input,
            gpu_output,
            frame_res,
            min_s_index,
            max_s_index,
            max_s_index - min_s_index + 1,
            omega_arr_size,
            gpu_omega_arr + omega_arr_size);
    else
        kernel_compute_and_fill_s<<<blocks, threads, 0, stream>>>(
            gpu_input,
            gpu_output,
            frame_res,
            min_h_index,
            max_h_index,
            max_h_index - min_h_index + 1,
            omega_arr_size,
            gpu_omega_arr + omega_arr_size);

    if (cd.composite_p_activated_v)
        kernel_compute_and_fill_v<<<blocks, threads, 0, stream>>>(gpu_input,
                                                             gpu_output,
                                                             frame_res,
                                                             min_v_index,
                                                             max_v_index);
    else
        kernel_compute_and_fill_v<<<blocks, threads, 0, stream>>>(gpu_input,
                                                             gpu_output,
                                                             frame_res,
                                                             min_h_index,
                                                             max_h_index);

    cudaCheckError();
}

__global__ void threshold_top_bottom(float* output,
                                     const float tmin,
                                     const float tmax,
                                     const uint frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        output[id] = fminf(output[id], tmax);
        output[id] = fmaxf(output[id], tmin);
    }
}

__global__ void kernel_from_distinct_components_to_interweaved_components(
    const float* src, float* dst, size_t frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        dst[id * 3] = src[id];
        dst[id * 3 + 1] = src[id + frame_res];
        dst[id * 3 + 2] = src[id + frame_res * 2];
    }
}

void from_distinct_components_to_interweaved_components(const float* src,
                                                        float* dst,
                                                        size_t frame_res,
                                                        const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_from_distinct_components_to_interweaved_components
        <<<blocks, threads, 0, stream>>>(src, dst, frame_res);
}

__global__ void kernel_from_interweaved_components_to_distinct_components(
    const float* src, float* dst, size_t frame_res)
{
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        dst[id] = src[id * 3];
        dst[id + frame_res] = src[id * 3 + 1];
        dst[id + frame_res * 2] = src[id * 3 + 2];
    }
}

void from_interweaved_components_to_distinct_components(const float* src,
                                                        float* dst,
                                                        size_t frame_res,
                                                        const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_from_interweaved_components_to_distinct_components<<<blocks,
                                                                threads,
                                                                0,
                                                                stream>>>(src,
                                                                     dst,
                                                                     frame_res);
}

void apply_percentile_and_threshold(float* gpu_arr,
                                    uint frame_res,
                                    uint width,
                                    uint height,
                                    float low_threshold,
                                    float high_threshold,
                                    const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_res, threads);
    float percent_out[2];
    const float percent_in_h[2] = {low_threshold, high_threshold};

    compute_percentile_xy_view(gpu_arr,
                               width,
                               height,
                               percent_in_h,
                               percent_out,
                               2,
                               holovibes::units::RectFd(),
                               false,
                               stream);
    threshold_top_bottom<<<blocks, threads, 0, stream>>>(gpu_arr,
                                                    percent_out[0],
                                                    percent_out[1],
                                                    frame_res);
}

void apply_gaussian_blur(const holovibes::ComputeDescriptor& cd,
                         float* gpu_arr,
                         uint height,
                         uint width,
                         const hipStream_t stream)
{
    size_t frame_res = height * width;

    float* gpu_convolution_matrix;
    cudaXMalloc(&gpu_convolution_matrix, frame_res * sizeof(float));
    cudaXMemsetAsync(gpu_convolution_matrix, 0, frame_res * sizeof(float), stream);

    float* blur_matrix = new float[cd.h_blur_kernel_size];
    float blur_value =
        1.0f / (float)(cd.h_blur_kernel_size * cd.h_blur_kernel_size);
    unsigned min_pos_kernel = height / 2 - cd.h_blur_kernel_size / 2;
    for (size_t i = 0; i < cd.h_blur_kernel_size; i++)
    {
        blur_matrix[i] = blur_value;
    }

    // FIXME Might want to replace that with a hipMemcpy2D
    for (size_t i = 0; i < cd.h_blur_kernel_size; i++)
    {
        cudaXMemcpyAsync(gpu_convolution_matrix + min_pos_kernel +
                        width * (i + min_pos_kernel),
                    blur_matrix,
                    cd.h_blur_kernel_size * sizeof(float),
                    hipMemcpyHostToDevice, stream);
    }

    shift_corners(gpu_convolution_matrix, 1, width, height, stream);

    hipComplex* gpu_kernel;
    cudaXMalloc(&gpu_kernel, frame_res * sizeof(hipComplex));
    cudaXMemsetAsync(gpu_kernel, 0, frame_res * sizeof(hipComplex), stream);
    cudaSafeCall(hipMemcpy2DAsync(gpu_kernel,
                              sizeof(hipComplex),
                              gpu_convolution_matrix,
                              sizeof(float),
                              sizeof(float),
                              frame_res,
                              hipMemcpyDeviceToDevice,
                              stream));

    float* gpu_memory_space;
    hipComplex* gpu_cuComplex_buffer;
    cudaXMalloc(&gpu_memory_space, frame_res * sizeof(float));
    cudaXMalloc(&gpu_cuComplex_buffer, frame_res * sizeof(hipComplex));
    CufftHandle handle{static_cast<int>(width),
                       static_cast<int>(height),
                       HIPFFT_C2C};
    convolution_kernel(gpu_arr,
                       gpu_memory_space,
                       gpu_cuComplex_buffer,
                       &handle,
                       width * height,
                       gpu_kernel,
                       false,
                       false,
                       stream);
    cudaCheckError();

    delete[] blur_matrix;
    cudaXFree(gpu_memory_space);
    cudaXFree(gpu_cuComplex_buffer);
    cudaXFree(gpu_convolution_matrix);
    cudaXFree(gpu_kernel);
}

void hsv_normalize(float* const gpu_arr,
                   const uint frame_res,
                   float* const gpu_min,
                   float* const gpu_max,
                   const hipStream_t stream)
{
    reduce_min(gpu_arr, gpu_min, frame_res, stream); // Get the minimum value
    reduce_max(gpu_arr, gpu_max, frame_res, stream); // Get the maximum value

    const auto lambda = [gpu_min, gpu_max] __device__(const float pixel) {
        return (pixel - *gpu_min) * (1 / (*gpu_max - *gpu_min));
    };
    map_generic(gpu_arr, gpu_arr, frame_res, lambda, stream);
}

void apply_operations_on_h(const holovibes::ComputeDescriptor& cd,
                           float* gpu_arr,
                           uint height,
                           uint width,
                           float* const gpu_min,
                           float* const gpu_max,
                           const hipStream_t stream)
{
    const uint frame_res = height * width;
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    apply_percentile_and_threshold(gpu_arr,
                                   frame_res,
                                   width,
                                   height,
                                   cd.composite_low_h_threshold,
                                   cd.composite_high_h_threshold,
                                   stream);

    map_multiply(gpu_arr, gpu_arr, frame_res, -1.0f, stream);
    hsv_normalize(gpu_arr, frame_res, gpu_min, gpu_max, stream);

    threshold_top_bottom<<<blocks, threads, 0, stream>>>(gpu_arr,
                                                    cd.slider_h_threshold_min,
                                                    cd.slider_h_threshold_max,
                                                    frame_res);
    if (cd.h_blur_activated)
    {
        apply_gaussian_blur(cd, gpu_arr, height, width, stream);
    }

    hsv_normalize(gpu_arr, frame_res, gpu_min, gpu_max, stream);
    map_multiply(gpu_arr, gpu_arr, frame_res, 0.66f, stream);
}

void apply_operations_on_s(const holovibes::ComputeDescriptor& cd,
                           float* gpu_arr,
                           uint height,
                           uint width,
                           float* const gpu_min,
                           float* const gpu_max,
                           const hipStream_t stream)
{
    const uint frame_res = height * width;
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    float* gpu_arr_s = gpu_arr + frame_res;

    apply_percentile_and_threshold(gpu_arr_s,
                                   frame_res,
                                   width,
                                   height,
                                   cd.composite_low_s_threshold,
                                   cd.composite_high_s_threshold,
                                   stream);

    hsv_normalize(gpu_arr_s, frame_res, gpu_min, gpu_max, stream);

    threshold_top_bottom<<<blocks, threads, 0, stream>>>(gpu_arr_s,
                                                    cd.slider_s_threshold_min,
                                                    cd.slider_s_threshold_max,
                                                    frame_res);

    hsv_normalize(gpu_arr_s, frame_res, gpu_min, gpu_max, stream);
}

void apply_operations_on_v(const holovibes::ComputeDescriptor& cd,
                           float* gpu_arr,
                           uint height,
                           uint width,
                           float* const gpu_min,
                           float* const gpu_max,
                           const hipStream_t stream)
{
    const uint frame_res = height * width;
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    float* gpu_arr_v = gpu_arr + frame_res * 2;

    apply_percentile_and_threshold(gpu_arr_v,
                                   frame_res,
                                   width,
                                   height,
                                   cd.composite_low_v_threshold,
                                   cd.composite_high_v_threshold,
                                   stream);

    hsv_normalize(gpu_arr_v, frame_res, gpu_min, gpu_max, stream);

    threshold_top_bottom<<<blocks, threads, 0, stream>>>(gpu_arr_v,
                                                    cd.slider_v_threshold_min,
                                                    cd.slider_v_threshold_max,
                                                    frame_res);

    hsv_normalize(gpu_arr_v, frame_res, gpu_min, gpu_max, stream);
}

void hsv(const hipComplex* gpu_input,
         float* gpu_output,
         const uint width,
         const uint height,
         const holovibes::ComputeDescriptor& cd,
         const hipStream_t stream)
{
    const int time_transformation_size = cd.time_transformation_size;
    const uint frame_res = height * width;

    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    float* gpu_omega_arr = nullptr;
    cudaXMalloc(&gpu_omega_arr,
                sizeof(float) * time_transformation_size * 2); // w1[] && w2[]

    fill_frequencies_arrays(cd, gpu_omega_arr, frame_res, stream);

    float* tmp_hsv_arr;
    cudaXMalloc(&tmp_hsv_arr, sizeof(float) * frame_res * 3); // HSV temp array

    compute_and_fill_hsv(gpu_input,
                         gpu_output,
                         frame_res,
                         cd,
                         gpu_omega_arr,
                         time_transformation_size,
                         stream);

    kernel_from_interweaved_components_to_distinct_components<<<blocks,
                                                                threads,
                                                                0,
                                                                stream>>>(
        gpu_output,
        tmp_hsv_arr,
        frame_res);
    cudaCheckError();

    // To perform a renormalization, a single min buffer and single max buffer
    // is needed gpu side
    {
        holovibes::cuda_tools::UniquePtr<float> gpu_min(1);
        holovibes::cuda_tools::UniquePtr<float> gpu_max(1);
        apply_operations_on_h(cd,
                              tmp_hsv_arr,
                              height,
                              width,
                              gpu_min.get(),
                              gpu_max.get(),
                              stream);
        apply_operations_on_s(cd,
                              tmp_hsv_arr,
                              height,
                              width,
                              gpu_min.get(),
                              gpu_max.get(),
                              stream);
        apply_operations_on_v(cd,
                              tmp_hsv_arr,
                              height,
                              width,
                              gpu_min.get(),
                              gpu_max.get(),
                              stream);
    }

    kernel_from_distinct_components_to_interweaved_components<<<blocks,
                                                                threads,
                                                                0,
                                                                stream>>>(
        tmp_hsv_arr,
        gpu_output,
        frame_res);
    cudaCheckError();
    kernel_normalized_convert_hsv_to_rgb<<<blocks, threads, 0, stream>>>(gpu_output,
                                                                    gpu_output,
                                                                    frame_res);
    cudaCheckError();

    map_multiply(gpu_output, gpu_output, frame_res * 3, 65536, stream);

    cudaXFree(tmp_hsv_arr);
    cudaXFree(gpu_omega_arr);
}
