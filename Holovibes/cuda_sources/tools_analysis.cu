#include "cuda_memory.cuh"
#include "common.cuh"
#include "tools_analysis.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"

void load_kernel_in_GPU(hipComplex* output, const float* kernel, const size_t frame_res, hipStream_t stream)
{
   // Set the width of each element to `sizeof(float)` in bytes to copy the float data.
    // Set the pitch of the destination to `sizeof(hipComplex)` for correct alignment.
    hipMemcpy2DAsync(output,
                      sizeof(hipComplex),      // Pitch of destination memory (width of each row in bytes)
                      kernel,
                      sizeof(float),          // Pitch of source memory (width of each row in bytes)
                      frame_res * sizeof(float), // Width of data to transfer (in bytes)
                      1,                      // Height of data to transfer (1 row, since it’s 1D)
                      hipMemcpyHostToDevice,
                      stream);
}

float* kernel_add_padding(float* kernel, const int width, const int height, const int new_width, const int new_height) {
    // Check that new dimensions are greater than or equal to the original dimensions
    if (new_width < width || new_height < height) {
        std::cerr << "New dimensions must be greater than or equal to the original dimensions." << std::endl;
        return nullptr;
    }

    // Create a new array for the padded kernel, initialized to 0
    float* padded_kernel = new float[new_width * new_height];
    std::memset(padded_kernel, 0, new_width * new_height * sizeof(float));

    // Calculate the starting position (top-left corner) of the original kernel in the padded kernel
    int start_x = (new_width - width) / 2;
    int start_y = (new_height - height) / 2;

    // Copy the original kernel into the center of the new padded array
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            // Map original kernel indices to the padded array indices
            padded_kernel[(start_y + y) * new_width + (start_x + x)] = kernel[y * width + x];
        }
    }

    return padded_kernel;
}