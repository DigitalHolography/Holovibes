#include "hip/hip_runtime.h"
#include "cuda_memory.cuh"
#include "common.cuh"
#include "tools_analysis.cuh"
#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"

namespace
{
template <typename T>
__global__ void kernel_multiply_array_by_scalar(T* input_output, size_t size, const T scalar)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        input_output[index] *= scalar;
    }
}

template <typename T>
void multiply_array_by_scalar_caller(T* input_output, size_t size, T scalar, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_multiply_array_by_scalar<<<blocks, threads, 0, stream>>>(input_output, size, scalar);
}
}

void multiply_array_by_scalar(float* input_output, size_t size, float scalar, hipStream_t stream)
{
    multiply_array_by_scalar_caller<float>(input_output, size, scalar, stream);
}

// CUDA kernel to prepare H hessian matrices
__global__ void kernel_prepare_hessian(float* output, const float* ixx, const float* ixy, const float* iyy, const int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        // Prepare the 2x2 submatrix for point `index`
        output[index * 3 + 0] = ixx[index];
        output[index * 3 + 1] = ixy[index];
        output[index * 3 + 2] = iyy[index];
    }
}

void prepare_hessian(float* output, const float* ixx, const float* ixy, const float* iyy, const int size, hipStream_t stream)
{
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    kernel_prepare_hessian<<<numBlocks, blockSize, 0, stream>>>(output, ixx, ixy, iyy, size);
}

__global__ void kernel_compute_eigen(float* H, int size, float* lambda1, float* lambda2)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        double a = H[index * 3], b = H[index * 3 + 1], d = H[index * 3 + 2];
        double trace = a + d;
        double determinant = a * d - b * b;
        double discriminant = trace * trace - 4 * determinant;
        if (discriminant >= 0)
        {
            double eig1 = (trace + std::sqrt(discriminant)) / 2;
            double eig2 = (trace - std::sqrt(discriminant)) / 2;
            if (std::abs(eig1) < std::abs(eig2))
            {
                lambda1[index] = eig1;
                lambda2[index] = eig2;
            }
            else
            {
                lambda1[index] = eig2;
                lambda2[index] = eig1;
            }
        }
    }
}

void compute_eigen_values(float* H, int size, float* lambda1, float* lambda2, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_compute_eigen<<<blocks, threads, 0, stream>>>(H, size, lambda1, lambda2);
}

__global__ void kernel_padding(float* output, float* input, int height, int width, int new_width, int start_x, int start_y) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int y = idx / width;
    int x = idx % width;

    if (y < height && x < width) 
    {
        output[(start_y + y) * new_width + (start_x + x)] = input[y * width + x];
    }
}

void write1DFloatArrayToFile(const float* array, int rows, int cols, const std::string& filename)
{
    // Open the file in write mode
    std::ofstream outFile(filename);

    // Check if the file was opened successfully
    if (!outFile)
    {
        std::cerr << "Error: Unable to open the file " << filename << std::endl;
        return;
    }

    // Write the 1D array in row-major order to the file
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            outFile << array[i * cols + j]; // Calculate index in row-major order
            if (j < cols - 1)
            {
                outFile << " "; // Separate values in a row by a space
            }
        }
        outFile << std::endl; // New line after each row
    }

    // Close the file
    outFile.close();
    std::cout << "1D array written to the file " << filename << std::endl;
}

void print_in_file(float* input, uint size, std::string filename, hipStream_t stream)
{
    if (input == nullptr)
    {
        return;
    }
    float* result = new float[size];
    cudaXMemcpyAsync(result,
                        input,
                        size * sizeof(float),
                        hipMemcpyDeviceToHost,
                        stream);
    write1DFloatArrayToFile(result,
                            sqrt(size),
                            sqrt(size),
                            "test_" + filename + ".txt");
}

__global__ void
kernel_apply_diaphragm_mask(float* output, short width, short height, float center_X, float center_Y, float radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int index = y * width + x;

    if (x < width && y < height)
    {
        float distance_squared = (x - center_X) * (x - center_X) + (y - center_Y) * (y - center_Y);
        float radius_squared = radius * radius;

        // If the point is inside the circle set the value to 1.
        if (distance_squared > radius_squared)
            output[index] = 0;
    }
}

void apply_diaphragm_mask(float* output,
                       const float center_X,
                       const float center_Y,
                       const float radius,
                       const short width,
                       const short height,
                       const hipStream_t stream)
{
    // Setting up the parallelisation.
    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(1 + (width - 1) / threads_2d, 1 + (height - 1) / threads_2d);

    kernel_apply_diaphragm_mask<<<lblocks, lthreads, 0, stream>>>(output, width, height, center_X, center_Y, radius);

    cudaXStreamSynchronize(stream);
    cudaCheckError();
}

__global__ void
kernel_compute_circle_mask(float* output, short width, short height, float center_X, float center_Y, float radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int index = y * width + x;

    if (x < width && y < height)
    {
        float distance_squared = (x - center_X) * (x - center_X) + (y - center_Y) * (y - center_Y);
        float radius_squared = radius * radius;

        // If the point is inside the circle set the value to 1.
        if (distance_squared <= radius_squared)
            output[index] = 1;
        else
            output[index] = 0;
    }
}

void compute_circle_mask(float* output,
                       const float center_X,
                       const float center_Y,
                       const float radius,
                       const short width,
                       const short height,
                       const hipStream_t stream)
{
    // Setting up the parallelisation.
    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(1 + (width - 1) / threads_2d, 1 + (height - 1) / threads_2d);

    kernel_compute_circle_mask<<<lblocks, lthreads, 0, stream>>>(output, width, height, center_X, center_Y, radius);

    cudaXStreamSynchronize(stream);
    cudaCheckError();
}

__global__ void
kernel_apply_mask_and(float* output, const float* input, short width, short height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int index = y * width + x;

    if (x < width && y < height)
    {
        output[y * width + x] *= input[y * width + x];
    }
}

void apply_mask_and(float* output,
                       const float* input,
                       const short width,
                       const short height,
                       const hipStream_t stream)
{
    // Setting up the parallelisation.
    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(1 + (width - 1) / threads_2d, 1 + (height - 1) / threads_2d);

    kernel_apply_mask_and<<<lblocks, lthreads, 0, stream>>>(output, input, width, height);

    cudaXStreamSynchronize(stream);
    cudaCheckError();
}

__global__ void
kernel_apply_mask_or(float* output, const float* input, short width, short height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int index = y * width + x;

    if (x < width && y < height)
    {
        output[y * width + x] = (input[y * width + x] != 0.f) ? 1.f : output[y * width + x];
    }
}

void apply_mask_or(float* output,
                       const float* input,
                       const short width,
                       const short height,
                       const hipStream_t stream)
{
    // Setting up the parallelisation.
    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(1 + (width - 1) / threads_2d, 1 + (height - 1) / threads_2d);

    kernel_apply_mask_or<<<lblocks, lthreads, 0, stream>>>(output, input, width, height);

    cudaXStreamSynchronize(stream);
    cudaCheckError();
}