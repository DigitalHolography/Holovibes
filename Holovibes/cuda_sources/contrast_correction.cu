#include "hip/hip_runtime.h"
/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include <numeric>
#include "map.cuh"
#include "common.cuh"

void apply_contrast_correction(float* const input,
                               const uint size,
                               const ushort dynamic_range,
                               const float min,
                               const float max,
                               const hipStream_t stream)
{
    const float factor = dynamic_range / (max - min + FLT_EPSILON);
    const auto apply_contrast = [factor, min] __device__(float pixel) {
        return factor * (pixel - min);
    };

    map_generic(input, input, size, apply_contrast, stream);
    cudaCheckError();
}
