#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include <numeric>
#include "contrast_correction.cuh"
#include "tools_compute.cuh"
#include "min_max.cuh"
#include "percentile.cuh"

static __global__
void kernel_apply_contrast(float* const input,
					       const uint size,
						   const float factor,
						   const float min)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
		input[index] = factor * (input[index] - min);
}

void apply_contrast_correction(float* const input,
							   const uint size,
							   const ushort dynamic_range,
							   const float	min,
							   const float	max)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	const float factor = dynamic_range / (max - min + FLT_EPSILON);
	kernel_apply_contrast << <blocks, threads>> > (input, size, factor, min);
	cudaCheckError();
}
