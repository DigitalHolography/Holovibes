#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>

#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "tools_analysis.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"

// MIGHT NEED TO BE DELETED
// Useless for now !! Function to apply convolution with replicate padding
void applyConvolutionWithReplicatePadding(const float* image, float* output, int imgWidth, int imgHeight,
                                          const float* kernel, int kernelWidth, int kernelHeight,
                                          bool divideConvolution = false) {
    int padWidth = kernelWidth / 2;
    int padHeight = kernelHeight / 2;

    // Calculate the sum of the kernel for normalization
    float kernelSum = 0.0f;
    for (int i = 0; i < kernelHeight; ++i) {
        for (int j = 0; j < kernelWidth; ++j) {
            kernelSum += kernel[i * kernelWidth + j];
        }
    }

    // Iterate over each pixel in the output image
    for (int y = 0; y < imgHeight; ++y) {
        for (int x = 0; x < imgWidth; ++x) {
            float sum = 0.0f;

            // Convolution operation with replicate padding
            for (int ky = -padHeight; ky <= padHeight; ++ky) {
                for (int kx = -padWidth; kx <= padWidth; ++kx) {
                    int imgX = std::min(std::max(x + kx, 0), imgWidth - 1);
                    int imgY = std::min(std::max(y + ky, 0), imgHeight - 1);
                    int kernelIndex = (ky + padHeight) * kernelWidth + (kx + padWidth);
                    int imageIndex = imgY * imgWidth + imgX;

                    // Apply the kernel
                    sum += image[imageIndex] * kernel[kernelIndex];
                }
            }

            // If division is enabled, normalize the convolved value
            if (divideConvolution && kernelSum != 0.0f) {
                output[y * imgWidth + x] = sum / kernelSum;
            } else {
                output[y * imgWidth + x] = sum;
            }
        }
    }
}

__global__ void kernel_normalized_list(float* output, int lim, int size)
{
     const int index = blockIdx.x * blockDim.x + threadIdx.x;
     if (index < size)
     {
        output[index] = (int)index - lim;
     }
}

void normalized_list(float* output, int lim, int size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_normalized_list<<<blocks, threads, 0, stream>>>(output, lim, size);   
}

__device__ float comp_hermite_iter(int n, float x)
{
    if (n == 0)
        return 1.0f;
    if (n == 1)
        return 2.0f * x;
    if (n > 1)
        return (2.0f * x * comp_hermite_iter(n - 1, x)) - (2.0f * (n - 1) * comp_hermite_iter(n - 2, x));
    return 0.0f;
}

__device__ float comp_gaussian(float x, float sigma)
{
    return 1 / (sigma * (sqrt(2 * M_PI))) * exp((-1 * x * x) / (2 * sigma * sigma));
}

__device__ float device_comp_dgaussian(float x, float sigma, int n)
{
    float A = pow((-1 / (sigma * sqrt((float)2))), n);
    float B = comp_hermite_iter(n, x / (sigma * sqrt((float)2)));
    float C = comp_gaussian(x, sigma);
    return A * B * C;
}

__global__ void kernel_comp_dgaussian(float* output, float* input, size_t input_size, float sigma, int n)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] = device_comp_dgaussian(input[index], sigma, n);
    }
}


void comp_dgaussian(float* output, float* input, size_t input_size, float sigma, int n, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(input_size, threads);
    kernel_comp_dgaussian<<<blocks, threads, 0, stream>>>(output, input, input_size, sigma, n);   
}


__global__ void convolutionKernel(const float* image, const float* kernel, float* output, 
                                  int width, int height, int kWidth, int kHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return; // Ensure we don't go out of image bounds

    float result = 0.0f;
    int kHalfWidth = kWidth / 2;
    int kHalfHeight = kHeight / 2;

    // Apply the convolution with replicate boundary behavior
    for (int ky = -kHalfHeight; ky <= kHalfHeight; ++ky) {
        for (int kx = -kHalfWidth; kx <= kHalfWidth; ++kx) {
            // Calculate the coordinates for the image
            int ix = x + kx;
            int iy = y + ky;

            // Replicate boundary behavior
            if (ix < 0) ix = 0;
            if (ix >= width) ix = width - 1;
            if (iy < 0) iy = 0;
            if (iy >= height) iy = height - 1;

            float imageValue = image[iy * width + ix];
            float kernelValue = kernel[(ky + kHalfHeight) * kWidth + (kx + kHalfWidth)];
            result += imageValue * kernelValue;
        }
    }

    output[y * width + x] = result;
}

void applyConvolution(float* image, const float* kernel, 
                      int width, int height, int kWidth, int kHeight, hipStream_t stream)
{
    float * d_output;
    hipMalloc(&d_output, width * height * sizeof(float));


    // Définir la taille des blocs et de la grille
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);

    // Lancer le kernel
    convolutionKernel<<<gridSize, blockSize, 0, stream>>>(image, kernel, d_output, width, height, kWidth, kHeight);

    // Copier le résultat du GPU vers le CPU
    hipMemcpy(image, d_output, width * height * sizeof(float), hipMemcpyDeviceToDevice);

    // Libérer la mémoire sur le GPU
    hipFree(d_output);
}


void gaussian_imfilter_sep(float* input_output,
                            float* gpu_kernel_buffer,
                            int kernel_x_size,
                            int kernel_y_size,
                            const size_t frame_res,
                            float* convolution_buffer,
                            hipComplex* cuComplex_buffer,
                            CufftHandle* convolution_plan, 
                            hipStream_t stream)
{
    // This convolution method gives correct values compared to matlab
    applyConvolution(input_output,
                     gpu_kernel_buffer, 
                     std::sqrt(frame_res),
                     std::sqrt(frame_res),
                     kernel_x_size,
                     kernel_y_size,
                     stream);
}

__global__ void kernel_abs_lambda_division(float* output, float* lambda_1, float* lambda_2, size_t input_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] = abs(lambda_1[index]) / abs(lambda_2[index]);
    }
}

__global__ void kernel_normalize(float* output, float* lambda_1, float* lambda_2, size_t input_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] = sqrtf(powf(lambda_1[index], 2) + powf(lambda_2[index], 2));
    }
}

__global__ void kernel_If(float* output, size_t input_size, float* R_blob, float beta, float c, float *c_temp)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        float A = powf(R_blob[index], 2);
        float B = 2 * powf(beta, 2);
        float C = expf(-(A / B));
        float D = 2 * powf(c, 2);
        float E = c_temp[index] / D;
        float F = 1 - expf(-E);
        output[index] = C * F;

        //output[index] = expf(-(powf(R_blob[index], 2) / 2 * powf(beta, 2))) * (1 - expf(-(c_temp[index] / (2 * powf(*c, 2)))));
        //output[index] = exp(-(pow(R_blob[index], 2) / 2 * pow(beta, 2))) * (1 - exp(-(c_temp[index] / (2 * pow(1, 2)))));
    }
}

__global__ void kernel_lambda_2_logical(float* output, size_t input_size, float* lambda_2)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] *= (lambda_2[index] <= 0.f ? 1 : 0);
    }
}

// Debuging: return float* to print the result to the screen
void vesselness_filter(float* output,
                        float* input, 
                        float sigma, 
                        float* g_xx_mul, 
                        float* g_xy_mul, 
                        float* g_yy_mul,
                        int kernel_x_size,
                        int kernel_y_size,
                        int frame_res, 
                        float* convolution_buffer, 
                        hipComplex* cuComplex_buffer,
                        CufftHandle* convolution_plan,
                        hipblasHandle_t cublas_handler,
                        hipStream_t stream)
{
    int gamma = 1;
    float beta = 0.8f;

    float A = std::pow(sigma, gamma);


    float* Ixx;
    cudaXMalloc(&Ixx, frame_res * sizeof(float));
    cudaXMemcpyAsync(Ixx, input, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXStreamSynchronize(stream);

    gaussian_imfilter_sep(Ixx, g_xx_mul, kernel_x_size, kernel_y_size, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);
    cudaXStreamSynchronize(stream);

    multiply_array_by_scalar(Ixx, frame_res, A, stream);
    cudaXStreamSynchronize(stream);


    float* Ixy;
    cudaXMalloc(&Ixy, frame_res * sizeof(float));
    cudaXMemcpyAsync(Ixy, input, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXStreamSynchronize(stream);

    gaussian_imfilter_sep(Ixy, g_xy_mul, kernel_x_size, kernel_y_size, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);
    cudaXStreamSynchronize(stream);

    multiply_array_by_scalar(Ixy, frame_res, A, stream);
    cudaXStreamSynchronize(stream);


    // Iyx is the same as Ixy, we can simply copy it
    float* Iyx;
    cudaXMalloc(&Iyx, frame_res * sizeof(float));
    cudaXMemcpyAsync(Iyx, Ixy, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXStreamSynchronize(stream);


    float* Iyy;
    cudaXMalloc(&Iyy, frame_res * sizeof(float));
    cudaXMemcpyAsync(Iyy, input, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXStreamSynchronize(stream);

    gaussian_imfilter_sep(Iyy, g_yy_mul, kernel_x_size, kernel_y_size, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);
    cudaXStreamSynchronize(stream);

    multiply_array_by_scalar(Iyy, frame_res, A, stream);
    cudaXStreamSynchronize(stream);


    float* H;
    cudaXMalloc(&H, frame_res * sizeof(float) * 4);

    cudaXMemcpyAsync(H, Ixx, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXMemcpyAsync(H + frame_res, Ixy, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXMemcpyAsync(H + frame_res * 2, Iyx, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);
    cudaXMemcpyAsync(H + frame_res * 3, Iyy, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);

    cudaXStreamSynchronize(stream);


    print_in_file(Ixx, frame_res, "ixx", stream);
    print_in_file(Ixy, frame_res, "ixy", stream);
    print_in_file(Iyx, frame_res, "iyx", stream);
    print_in_file(Iyy, frame_res, "iyy", stream);

    cudaXFree(Ixx);
    cudaXFree(Ixy);
    cudaXFree(Iyx);
    cudaXFree(Iyy);


    float* lambda_1;
    cudaXMalloc(&lambda_1, frame_res * sizeof(float));
    cudaXMemset(lambda_1, 0, frame_res * sizeof(float));
    float* lambda_2;
    cudaXMalloc(&lambda_2, frame_res * sizeof(float));
    cudaXMemset(lambda_2, 0, frame_res * sizeof(float));

    cudaXStreamSynchronize(stream);


    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    dim3 blockSize(16, 16);
    dim3 gridSize((std::sqrt(frame_res) + blockSize.x - 1) / blockSize.x, (std::sqrt(frame_res) + blockSize.y - 1) / blockSize.y);

    kernel_4D_eigenvalues<<<gridSize, blockSize, 0, stream>>>(H, lambda_1, lambda_2, std::sqrt(frame_res), std::sqrt(frame_res));
    
    cudaXStreamSynchronize(stream);

    print_in_file(lambda_1, frame_res, "lambda_1", stream);
    print_in_file(lambda_2, frame_res, "lambda_2", stream);

    // cudaXFree(H);

    // float* R_blob;
    // cudaXMalloc(&R_blob, frame_res * sizeof(float));
    // threads = get_max_threads_1d();
    // blocks = map_blocks_to_problem(frame_res, threads);
    // kernel_abs_lambda_division<<<blocks, threads, 0, stream>>>(R_blob, lambda_1, lambda_2, frame_res);
    // cudaXStreamSynchronize(stream);


    // float *c_temp;
    // cudaXMalloc(&c_temp, frame_res * sizeof(float));
    // threads = get_max_threads_1d();
    // blocks = map_blocks_to_problem(frame_res, threads);
    // kernel_normalize<<<blocks, threads, 0, stream>>>(c_temp, lambda_1, lambda_2, frame_res);
    // cudaXStreamSynchronize(stream);


    // int c_index;
    // hipblasStatus_t status = hipblasIsamax(cublas_handler, frame_res, c_temp, 1, &c_index);
    // cudaXStreamSynchronize(stream);
    // float c;
    // hipMemcpy(&c, &c_temp[c_index - 1], sizeof(float), hipMemcpyDeviceToHost);
    // std::cout << "c : " << c << std::endl;

    // threads = get_max_threads_1d();
    // blocks = map_blocks_to_problem(frame_res, threads);
    // kernel_If<<<blocks, threads, 0, stream>>>(output, frame_res, R_blob, beta, c, c_temp);
    // cudaXStreamSynchronize(stream);

    // test_filter = new float[frame_res];
    // cudaXMemcpyAsync(test_filter,
    //     output,
    //     frame_res * sizeof(float),
    //     hipMemcpyDeviceToHost,
    //     stream);
    // write1DFloatArrayToFile(test_filter,
    //     sqrt(frame_res),
    //     sqrt(frame_res),
    //     "test_filter_output.txt");

    // cudaXFree(R_blob);
    // cudaXFree(c_temp);
    // cudaXFree(lambda_1);

    // threads = get_max_threads_1d();
    // blocks = map_blocks_to_problem(frame_res, threads);
    // kernel_lambda_2_logical<<<blocks, threads, 0, stream>>>(output, frame_res, lambda_2);
    // cudaXStreamSynchronize(stream);

    // cudaXFree(lambda_2);
}