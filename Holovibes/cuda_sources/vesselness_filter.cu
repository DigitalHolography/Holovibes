#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>

#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "tools_analysis.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"

__global__ void kernel_normalized_list(float* output, int lim, int size)
{
     const int index = blockIdx.x * blockDim.x + threadIdx.x;
     if (index < size)
     {
        output[index] = (int)index - lim;
     }
}

void normalized_list(float* output, int lim, int size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_normalized_list<<<blocks, threads, 0, stream>>>(output, lim, size);   
}

__device__ float comp_hermite_iter(int n, float x)
{
    if (n == 0)
        return 1.0f;
    if (n == 1)
        return 2.0f * x;
    if (n > 1)
        return (2.0f * x * comp_hermite_iter(n - 1, x)) - (2.0f * (n - 1) * comp_hermite_iter(n - 2, x));
    return 0.0f;
}

__device__ float comp_gaussian(float x, float sigma)
{
    return 1 / (sigma * (sqrt(2 * M_PI))) * exp((-1 * x * x) / (2 * sigma * sigma));
}

__device__ float device_comp_dgaussian(float x, float sigma, int n)
{
    float A = pow((-1 / (sigma * sqrt((float)2))), n);
    float B = comp_hermite_iter(n, x / (sigma * sqrt((float)2)));
    float C = comp_gaussian(x, sigma);
    return A * B * C;
}

__global__ void kernel_comp_dgaussian(float* output, float* input, size_t input_size, float sigma, int n)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] = device_comp_dgaussian(input[index], sigma, n);
    }
}


void comp_dgaussian(float* output, float* input, size_t input_size, float sigma, int n, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(input_size, threads);
    kernel_comp_dgaussian<<<blocks, threads, 0, stream>>>(output, input, input_size, sigma, n);   
}


__global__ void convolutionKernel(const float* image, const float* kernel, float* output, 
                                  int width, int height, int kWidth, int kHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return; // Ensure we don't go out of image bounds

    float result = 0.0f;
    int kHalfWidth = kWidth / 2;
    int kHalfHeight = kHeight / 2;

    // Apply the convolution with replicate boundary behavior
    for (int ky = -kHalfHeight; ky <= kHalfHeight; ++ky) {
        for (int kx = -kHalfWidth; kx <= kHalfWidth; ++kx) {
            // Calculate the coordinates for the image
            int ix = x + kx;
            int iy = y + ky;

            // Replicate boundary behavior
            if (ix < 0) ix = 0;
            if (ix >= width) ix = width - 1;
            if (iy < 0) iy = 0;
            if (iy >= height) iy = height - 1;

            float imageValue = image[iy * width + ix];
            float kernelValue = kernel[(ky + kHalfHeight) * kWidth + (kx + kHalfWidth)];
            result += imageValue * kernelValue;
        }
    }

    output[y * width + x] = result;
}

void applyConvolution(float* image, const float* kernel, 
                      int width, int height, int kWidth, int kHeight, hipStream_t stream)
{
    float * d_output;
    hipMalloc(&d_output, width * height * sizeof(float));


    // Définir la taille des blocs et de la grille
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);

    // Lancer le kernel
    convolutionKernel<<<gridSize, blockSize, 0, stream>>>(image, kernel, d_output, width, height, kWidth, kHeight);

    // Copier le résultat du GPU vers le CPU
    hipMemcpy(image, d_output, width * height * sizeof(float), hipMemcpyDeviceToDevice);

    // Libérer la mémoire sur le GPU
    hipFree(d_output);
}


void gaussian_imfilter_sep(float* input_output,
                            float* gpu_kernel_buffer,
                            int kernel_x_size,
                            int kernel_y_size,
                            const size_t frame_res,
                            float* convolution_buffer,
                            hipComplex* cuComplex_buffer,
                            CufftHandle* convolution_plan, 
                            hipStream_t stream)
{
    // This convolution method gives correct values compared to matlab
    applyConvolution(input_output,
                     gpu_kernel_buffer, 
                     std::sqrt(frame_res),
                     std::sqrt(frame_res),
                     kernel_x_size,
                     kernel_y_size,
                     stream);
}

__global__ void kernel_abs_lambda_division(float* output, float* lambda_1, float* lambda_2, size_t input_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] = abs(lambda_1[index]) / abs(lambda_2[index]);
    }
}

void abs_lambda_division(float* output, float* lambda_1, float* lambda_2, uint frame_res, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    
    kernel_abs_lambda_division<<<blocks, threads, 0, stream>>>(output, lambda_1, lambda_2, frame_res); 
}


__global__ void kernel_normalize(float* output, float* lambda_1, float* lambda_2, size_t input_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] = sqrtf(powf(lambda_1[index], 2) + powf(lambda_2[index], 2));
    }
}

void normalize(float* output, float* lambda_1, float* lambda_2, uint frame_res, hipStream_t stream) {
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    kernel_normalize<<<blocks, threads, 0, stream>>>(output, lambda_1, lambda_2, frame_res);
}

__global__ void kernel_If(float* output, size_t input_size, float* R_blob, float beta, float c, float *c_temp)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        float A = powf(R_blob[index], 2);
        float B = 2 * powf(beta, 2);
        float C = expf(-(A / B));
        float D = 2 * powf(c, 2);
        float E = c_temp[index] / D;
        float F = 1 - expf(-E);
        output[index] = C * F;
    }
}

void If(float* output, size_t input_size, float* R_blob, float beta, float c, float *c_temp, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(input_size, threads);

    kernel_If<<<blocks, threads, 0, stream>>>(output, input_size, R_blob, beta, c, c_temp);
}



__global__ void kernel_lambda_2_logical(float* output, size_t input_size, float* lambda_2)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        output[index] *= (lambda_2[index] <= 0.f ? 1 : 0);
    }
}

void lambda_2_logical(float* output, size_t input_size, float* lambda_2, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(input_size, threads);

    kernel_lambda_2_logical<<<blocks, threads, 0, stream>>>(output, input_size, lambda_2);
}

float* compute_I(float* input, float* g_mul, float A, uint frame_res, uint kernel_x_size, uint kernel_y_size, float* convolution_buffer, hipComplex* cuComplex_buffer,
                        CufftHandle* convolution_plan, hipStream_t stream) {
    float* I;
    cudaXMalloc(&I, frame_res * sizeof(float));
    cudaXMemcpyAsync(I, input, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);

    gaussian_imfilter_sep(I, g_mul, kernel_x_size, kernel_y_size, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);

    multiply_array_by_scalar(I, frame_res, A, stream);

    return I;
}

// Debuging: return float* to print the result to the screen
void vesselness_filter(float* output,
                        float* input, 
                        float sigma, 
                        float* g_xx_mul, 
                        float* g_xy_mul, 
                        float* g_yy_mul,
                        int kernel_x_size,
                        int kernel_y_size,
                        int frame_res, 
                        float* convolution_buffer, 
                        hipComplex* cuComplex_buffer,
                        CufftHandle* convolution_plan,
                        hipblasHandle_t cublas_handler,
                        hipStream_t stream)
{
    int gamma = 1;
    float beta = 0.8f;

    float A = std::pow(sigma, gamma);

    float* Ixx = compute_I(input, g_xx_mul, A, frame_res, kernel_x_size, kernel_y_size, convolution_buffer, cuComplex_buffer, convolution_plan, stream);

    float* Ixy = compute_I(input, g_xy_mul, A, frame_res, kernel_x_size, kernel_y_size, convolution_buffer, cuComplex_buffer, convolution_plan, stream);

    float* Iyy = compute_I(input, g_yy_mul, A, frame_res, kernel_x_size, kernel_y_size, convolution_buffer, cuComplex_buffer, convolution_plan, stream);



    float* H;
    hipMalloc(&H, frame_res * 3 * sizeof(float));


    int blockSize = 256;
    int numBlocks = (frame_res + blockSize - 1) / blockSize;
    prepareHessian<<<numBlocks, blockSize, 0, stream>>>(H, Ixx, Ixy, Iyy, frame_res);
    cudaXStreamSynchronize(stream);


    cudaXFree(Ixx);
    cudaXFree(Ixy);
    cudaXFree(Iyy);


    float* lambda_1 = new float[frame_res];
    cudaXMalloc(&lambda_1, frame_res * sizeof(float));
    cudaXMemset(lambda_1, 0, frame_res * sizeof(float));

    float* lambda_2 = new float[frame_res];
    cudaXMalloc(&lambda_2, frame_res * sizeof(float));
    cudaXMemset(lambda_2, 0, frame_res * sizeof(float));

    compute_eigen_values(H, frame_res, lambda_1, lambda_2, stream);


    cudaXFree(H);

    float* R_blob;
    cudaXMalloc(&R_blob, frame_res * sizeof(float));
    abs_lambda_division(R_blob, lambda_1, lambda_2, frame_res, stream);

    float *c_temp;
    cudaXMalloc(&c_temp, frame_res * sizeof(float));
    normalize(c_temp, lambda_1, lambda_2, frame_res, stream);

    int c_index;
    hipblasStatus_t status = hipblasIsamax(cublas_handler, frame_res, c_temp, 1, &c_index);

    float c;
    hipMemcpy(&c, &c_temp[c_index - 1], sizeof(float), hipMemcpyDeviceToHost);

    If(output, frame_res, R_blob, beta, c, c_temp, stream);

    cudaXFree(R_blob);
    cudaXFree(c_temp);
    cudaXFree(lambda_1);

    lambda_2_logical(output, frame_res, lambda_2, stream);

    cudaXFree(lambda_2);
}