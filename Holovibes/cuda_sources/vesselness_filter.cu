#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>

#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "tools_analysis.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"

// Function to apply convolution with replicate padding
void applyConvolutionWithReplicatePadding(const float* image, float* output, int imgWidth, int imgHeight,
                                          const float* kernel, int kernelWidth, int kernelHeight,
                                          bool divideConvolution = false) {
    int padWidth = kernelWidth / 2;
    int padHeight = kernelHeight / 2;

    // Calculate the sum of the kernel for normalization
    float kernelSum = 0.0f;
    for (int i = 0; i < kernelHeight; ++i) {
        for (int j = 0; j < kernelWidth; ++j) {
            kernelSum += kernel[i * kernelWidth + j];
        }
    }

    // Iterate over each pixel in the output image
    for (int y = 0; y < imgHeight; ++y) {
        for (int x = 0; x < imgWidth; ++x) {
            float sum = 0.0f;

            // Convolution operation with replicate padding
            for (int ky = -padHeight; ky <= padHeight; ++ky) {
                for (int kx = -padWidth; kx <= padWidth; ++kx) {
                    int imgX = std::min(std::max(x + kx, 0), imgWidth - 1);
                    int imgY = std::min(std::max(y + ky, 0), imgHeight - 1);
                    int kernelIndex = (ky + padHeight) * kernelWidth + (kx + padWidth);
                    int imageIndex = imgY * imgWidth + imgX;

                    // Apply the kernel
                    sum += image[imageIndex] * kernel[kernelIndex];
                }
            }

            // If division is enabled, normalize the convolved value
            if (divideConvolution && kernelSum != 0.0f) {
                output[y * imgWidth + x] = sum / kernelSum;
            } else {
                output[y * imgWidth + x] = sum;
            }
        }
    }
}

static void write1DFloatArrayToFile(const float* array, int rows, int cols, const std::string& filename)
{
    // Open the file in write mode
    std::ofstream outFile(filename);

    // Check if the file was opened successfully
    if (!outFile)
    {
        std::cerr << "Error: Unable to open the file " << filename << std::endl;
        return;
    }

    // Write the 1D array in row-major order to the file
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            outFile << array[i * cols + j]; // Calculate index in row-major order
            if (j < cols - 1)
            {
                outFile << " "; // Separate values in a row by a space
            }
        }
        outFile << std::endl; // New line after each row
    }

    // Close the file
    outFile.close();
    std::cout << "1D array written to the file " << filename << std::endl;
}

// OK
float comp_hermite_rec(int n, float x)
{
    if (n == 0)
        return 1.0f;
    if (n == 1)
        return 2.0f * x;
    if (n > 1)
        return (2.0f * x * comp_hermite_rec(n - 1, x)) - (2.0f * (n - 1) * comp_hermite_rec(n - 2, x));
    throw std::exception("comp_hermite_rec in velness_filter.cu : n can't be negative");
}

// OK
float comp_gaussian(float x, float sigma)
{
    return 1 / (sigma * (sqrt(2 * M_PI))) * std::exp((-1 * x * x) / (2 * sigma * sigma));
}

// OK
float comp_dgaussian(float x, float sigma, int n)
{
    float A = std::pow((-1 / (sigma * std::sqrt(2))), n);
    float B = comp_hermite_rec(n, x / (sigma * std::sqrt(2)));
    float C = comp_gaussian(x, sigma);
    return A * B * C;
}

// Overload for float array
float* comp_dgaussian(float* x, size_t x_size, float sigma, int n)
{
    float *res = new float[x_size];
    for (size_t i = 0; i < x_size; ++i)
    {
        res[i] = comp_dgaussian(x[i], sigma, n);
    }
    return res;
}

float* gaussian_imfilter_sep(float* input_img, 
                            float* kernel,
                            size_t kernel_height,
                            size_t kernel_width,
                            const size_t frame_res, 
                            float* convolution_buffer, 
                            hipComplex* cuComplex_buffer, 
                            CufftHandle* convolution_plan, 
                            hipStream_t stream)
{
   
    // float* gpu_output;
    // cudaXMalloc(&gpu_output, frame_res * sizeof(float));
    // cudaXMemcpyAsync(gpu_output, input_img, frame_res * sizeof(float), hipMemcpyDeviceToDevice, stream);

    // cudaXStreamSynchronize(stream);

    // hipComplex* output_complex;
    // cudaXMalloc(&output_complex, frame_res * sizeof(hipComplex));
    // cudaXMemset(output_complex, 0, frame_res * sizeof(hipComplex));
    // load_kernel_in_GPU(output_complex, kernel, frame_res, stream);
    // cudaXStreamSynchronize(stream);

    // convolution_kernel(gpu_output,
    //                     convolution_buffer,
    //                     cuComplex_buffer,
    //                     convolution_plan,
    //                     frame_res,
    //                     output_complex,
    //                     false,
    //                     stream);
    // cudaXStreamSynchronize(stream);

    float *input_copy = new float[frame_res];
    cudaXMemcpyAsync(input_copy, input_img, frame_res * sizeof(float), hipMemcpyDeviceToHost, stream);
    cudaXStreamSynchronize(stream);

    float *output = new float[frame_res];
    applyConvolutionWithReplicatePadding(input_copy, output, std::sqrt(frame_res), std::sqrt(frame_res), kernel, kernel_width, kernel_height, true);

    free(input_copy);

    // float *res = new float[frame_res];
    // cudaXMemcpyAsync(res, gpu_output, frame_res * sizeof(float), hipMemcpyDeviceToHost, stream);
    // cudaXStreamSynchronize(stream);
    
    return output;
}

void multiply_by_float(float* vect, float num, int frame_size)
{
    for (int i = 0; i < frame_size; i++)
    {
        vect[i] *= num;
    }
}


float* vesselness_filter(float* input, 
                        float sigma, 
                        float* g_xx_mul, 
                        float* g_xy_mul, 
                        float* g_yy_mul,
                        size_t kernel_height,
                        size_t kernel_width,
                        int frame_res, 
                        float* convolution_buffer, 
                        hipComplex* cuComplex_buffer,
                        CufftHandle* convolution_plan,
                        hipStream_t stream)
{
    int gamma = 1;

    float A = std::pow(sigma, gamma);

    float* Ixx = gaussian_imfilter_sep(input, g_xx_mul, kernel_height, kernel_width, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);
    multiply_by_float(Ixx, A, frame_res);

    // float* Ixy = gaussian_imfilter_sep(input, g_xy_mul, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);
    // multiply_by_float(Ixy, A, frame_res);

    // float* Iyx = new float[frame_res];
    // for (size_t i = 0; i < frame_res; ++i) {
    //     Iyx[i] = Ixy[i];
    // }

    // float* Iyy = gaussian_imfilter_sep(input, g_yy_mul, frame_res, convolution_buffer, cuComplex_buffer, convolution_plan, stream);
    // multiply_by_float(Iyy, A, frame_res);

    return Ixx;
}