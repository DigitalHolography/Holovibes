#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

# include "tools.hh"
# include "texture_update.cuh"

__global__
void kernelTextureUpdate(	unsigned short* frame,
							hipSurfaceObject_t cuSurface,
							dim3 texDim)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	/*unsigned short* p = &frame[y * texDim.x + x];
		ushort4  data = make_ushort4(p[0], 0x4000, p[1], 0xffff);*/

	surf2Dwrite(frame[y * texDim.x + x], cuSurface, x*4, y);
}

void textureUpdate(	hipSurfaceObject_t cuSurface,
					void *frame,
					unsigned short width,
					unsigned short height)
{
	dim3 threads(32, 32);
	dim3 blocks(width / threads.x, height / threads.y);

	kernelTextureUpdate << < blocks, threads >> >(reinterpret_cast<unsigned short*>(frame),
		cuSurface, dim3(width, height));
}
