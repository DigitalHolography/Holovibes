#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include "map.cuh"

#include <stdio.h>

#define AUTO_CONTRAST_COMPENSATOR 10000

__global__ void
kernel_complex_divide(hipComplex* image, const uint frame_res, const float divider, const uint batch_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < frame_res)
    {
        for (uint i = 0; i < batch_size; ++i)
        {
            const uint batch_index = index + i * frame_res;

            image[batch_index].x /= divider;
            image[batch_index].y /= divider;
        }
    }
}

__global__ void
kernel_multiply_frames_complex(const hipComplex* input1, const hipComplex* input2, hipComplex* output, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        const float new_x = (input1[index].x * input2[index].x) - (input1[index].y * input2[index].y);
        const float new_y = (input1[index].y * input2[index].x) + (input1[index].x * input2[index].y);
        output[index].x = new_x;
        output[index].y = new_y;
    }
}

__global__ void
kernel_divide_frames_float(const float* numerator, const float* denominator, float* output, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        const float new_x = numerator[index] / denominator[index];
        output[index] = new_x;
    }
}

void multiply_frames_complex(
    const hipComplex* input1, const hipComplex* input2, hipComplex* output, const uint size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_multiply_frames_complex<<<blocks, threads, 0, stream>>>(input1, input2, output, size);
    cudaCheckError();
}

void gpu_normalize(float* const input,
                   double* const result_reduce,
                   const size_t frame_res,
                   const uint norm_constant,
                   const hipStream_t stream)
{
    reduce_add(input, result_reduce, frame_res, stream);

    /* Let x be a pixel, after renormalization
    ** x = x * 2^(norm_constant) / mean
    ** x = x * 2^(norm_constant) * frame_res / reduce_result
    ** x = x * 2^(norm_constant) * (frame_res / reduce_result)
    */
    const float multiplier = (1 << norm_constant);
    auto map_function = [multiplier, frame_res, result_reduce] __device__(const float input_pixel) -> float
    {
        /* Computing on double is really slow on a GPU, in our case
         *result_reduce can never overflow
         ** Thus it can be casted to a float
         */
        return input_pixel * multiplier * (frame_res / static_cast<const float>(*result_reduce));
    };

    map_generic(input, input, frame_res, map_function, stream);
}