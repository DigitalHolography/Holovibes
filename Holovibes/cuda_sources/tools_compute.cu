#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include "map.cuh"

#include <stdio.h>

#define AUTO_CONTRAST_COMPENSATOR 10000

__global__ void
kernel_complex_divide(hipComplex* image, const uint frame_res, const float divider, const uint batch_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < frame_res)
    {
        for (uint i = 0; i < batch_size; ++i)
        {
            const uint batch_index = index + i * frame_res;

            image[batch_index].x /= divider;
            image[batch_index].y /= divider;
        }
    }
}

__global__ void
kernel_multiply_frames_complex(hipComplex* output, const hipComplex* input1, const hipComplex* input2, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        const float new_x = (input1[index].x * input2[index].x) - (input1[index].y * input2[index].y);
        const float new_y = (input1[index].y * input2[index].x) + (input1[index].x * input2[index].y);
        output[index].x = new_x;
        output[index].y = new_y;
    }
}

__global__ void
kernel_divide_frames_float(float* output, const float* numerator, const float* denominator, const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        const float new_x = numerator[index] / denominator[index];
        output[index] = new_x;
    }
}

__global__ void kernel_tensor_multiply_vector(float* output,
                                              const float* tensor,
                                              const float* vector,
                                              const size_t frame_res,
                                              const ushort f_start,
                                              const ushort f_end)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= frame_res)
        return;

    float val = 0.0f;
    for (uint i = f_start; i <= f_end; i++)
    {
        const float* current_frame = tensor + i * frame_res;
        val += current_frame[index] * vector[i];
    }

    output[index] = val;
}

void multiply_frames_complex(
    hipComplex* output, const hipComplex* input1, const hipComplex* input2, const uint size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_multiply_frames_complex<<<blocks, threads, 0, stream>>>(output, input1, input2, size);
    cudaCheckError();
}

void gpu_normalize(float* const input,
                   double* const result_reduce,
                   const size_t frame_res,
                   const uint norm_constant,
                   const hipStream_t stream)
{
    reduce_add(result_reduce, input, frame_res, stream);

    /* Let x be a pixel, after renormalization
    ** x = x * 2^(norm_constant) / mean
    ** x = x * 2^(norm_constant) * frame_res / reduce_result
    ** x = x * 2^(norm_constant) * (frame_res / reduce_result)
    */
    const float multiplier = (1 << norm_constant);
    auto map_function = [multiplier, frame_res, result_reduce] __device__(const float input_pixel) -> float
    {
        /* Computing on double is really slow on a GPU, in our case
         *result_reduce can never overflow
         ** Thus it can be casted to a float
         */
        return input_pixel * multiplier * (frame_res / static_cast<const float>(*result_reduce));
    };

    map_generic(input, input, frame_res, map_function, stream);
}

void tensor_multiply_vector(float* output,
                            const float* tensor,
                            const float* vector,
                            const size_t frame_res,
                            const ushort f_start,
                            const ushort f_end,
                            const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);
    kernel_tensor_multiply_vector<<<blocks, threads, 0, stream>>>(output, tensor, vector, frame_res, f_start, f_end);
    cudaCheckError();
}
