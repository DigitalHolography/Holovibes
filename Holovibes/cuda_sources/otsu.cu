#include "hip/hip_runtime.h"
#include "otsu.cuh"
#include "common.cuh"
#include "hip/hip_complex.h"
#include "hip/hip_runtime.h"
#include "hardware_limits.hh"
#include "cuda_memory.cuh"
using uint = unsigned int;

#define NUM_BINS 256

// Check if optimizable in future with `reduce.cuh` functions.
__global__ void histogram_kernel(float* image, uint* hist, int imgSize)
{
    extern __shared__ uint shared_hist[]; // Shared memory for histogram bins

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory histogram
    if (tid < NUM_BINS)
        shared_hist[tid] = 0;
    __syncthreads();

    // Populate shared histogram
    if (idx < imgSize)
    {
        int bin = static_cast<int>(image[idx] * NUM_BINS);
        atomicAdd(&shared_hist[bin], 1);
    }
    __syncthreads();

    // Merge shared histograms into global memory
    if (tid < NUM_BINS)
        atomicAdd(&hist[tid], shared_hist[tid]);
}

__global__ void global_threshold_kernel(float* input, int size, float globalThreshold)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
        input[idx] = (input[idx] > globalThreshold) * 1.0f;
}

__global__ void bradley_threshold_kernel(const float* image,
                                         float* output,
                                         int width,
                                         int height,
                                         int windowSize,
                                         float globalThreshold,
                                         float localThresholdFactor)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width || y < height)
    {
        int halfWindow = windowSize / 2;
        int startX = max(x - halfWindow, 0);
        int startY = max(y - halfWindow, 0);
        int endX = min(x + halfWindow, width - 1);
        int endY = min(y + halfWindow, height - 1);

        float localSum = 0;
        int count = 0;

        for (int i = startX; i <= endX; i++)
        {
            for (int j = startY; j <= endY; j++)
            {
                localSum += image[j * width + i];
                count++;
            }
        }

        float localMean = localSum / count;
        float localThreshold = localMean * (1 - localThresholdFactor * globalThreshold);
        output[y * width + x] = (image[y * width + x] > localThreshold) ? 1.0f : 0.0f;
    }
}

__global__ void otsu_threshold_kernel(uint* hist, int total, float* threshold_out)
{
    __shared__ float sum_shared;
    __shared__ float varMax_shared;
    __shared__ float threshold_shared;

    int tid = threadIdx.x;
    if (tid == 0)
    {
        sum_shared = 0;
        varMax_shared = 0;
        threshold_shared = 0;
    }
    __syncthreads();

    // Compute total sum in parallel
    __shared__ float partial_sum[NUM_BINS];
    partial_sum[tid] = (tid < NUM_BINS) ? tid * hist[tid] : 0;
    __syncthreads();

    // Reduce to get total sum
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (tid < offset)
            partial_sum[tid] += partial_sum[tid + offset];
        __syncthreads();
    }

    if (tid == 0)
        sum_shared = partial_sum[0];
    __syncthreads();

    // Variables for Otsu
    int wB = 0, wF = 0;
    float sumB = 0;
    float total_sum = sum_shared;

    for (int t = tid; t < NUM_BINS; t += blockDim.x)
    {
        wB += hist[t];
        if (wB == 0)
            continue;

        wF = total - wB;
        if (wF == 0)
            break;

        sumB += t * hist[t];
        float mB = sumB / wB;
        float mF = (total_sum - sumB) / wF;
        float varBetween = wB * wF * (mB - mF) * (mB - mF);

        atomicMax(reinterpret_cast<unsigned int*>(&varMax_shared), __float_as_uint(varBetween));

        if (varBetween == varMax_shared)
            atomicExch(reinterpret_cast<unsigned int*>(&threshold_shared), t);
    }

    __syncthreads();

    if (tid == 0)
        *threshold_out = threshold_shared / NUM_BINS;
}

float otsu_threshold(float* d_image, uint* histo_buffer_d, int size, const hipStream_t stream)
{
    uint threads = NUM_BINS; // 256;
    uint blocks = (size + threads - 1) / threads;

    // Step 1: Initialize histogram
    hipMemset(histo_buffer_d, 0, NUM_BINS * sizeof(uint));
    // Step 2: Compute histogram
    size_t shared_mem_size = NUM_BINS * sizeof(uint);
    histogram_kernel<<<blocks, threads, shared_mem_size, stream>>>(d_image, histo_buffer_d, size);

    // Step 3: Compute Otsu threshold
    float* d_threshold;
    hipMalloc(&d_threshold, sizeof(float));
    otsu_threshold_kernel<<<1, NUM_BINS, 0, stream>>>(histo_buffer_d, size, d_threshold);

    float threshold;
    hipMemcpy(&threshold, d_threshold, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_threshold);

    return threshold;
}

void compute_binarise_otsu(
    float* d_image, uint* histo_buffer_d, const size_t width, const size_t height, const hipStream_t stream)
{
    size_t img_size = width * height;

    float global_threshold = otsu_threshold(d_image, histo_buffer_d, img_size, stream);

    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(img_size, threads);

    global_threshold_kernel<<<blocks, threads, 0, stream>>>(d_image, img_size, global_threshold);
    cudaXStreamSynchronize(stream);
}

void compute_binarise_otsu_bradley(float* d_image,
                                   float*& d_output,
                                   uint* histo_buffer_d,
                                   const size_t width,
                                   const size_t height,
                                   const int window_size,
                                   const float local_threshold_factor,
                                   const hipStream_t stream)
{
    size_t img_size = width * height;

    float global_threshold = otsu_threshold(d_image, histo_buffer_d, img_size, stream);

    uint threads_2d = get_max_threads_2d();
    dim3 lthreads(threads_2d, threads_2d);
    dim3 lblocks(1 + (width - 1) / threads_2d, 1 + (height - 1) / threads_2d);

    bradley_threshold_kernel<<<lblocks, lthreads, 0, stream>>>(d_image,
                                                               d_output,
                                                               width,
                                                               height,
                                                               window_size,
                                                               global_threshold,
                                                               local_threshold_factor);
    cudaXStreamSynchronize(stream);
}
