#include "hip/hip_runtime.h"
/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include "cuda_memory.cuh"
#include "tools_conversion.cuh"
#include "unique_ptr.hh"
#include "composite.cuh"

struct rect
{
    int x;
    int y;
    int w;
    int h;
};

struct comp
{
    ushort p_min;
    ushort p_max;
    float weight;
};

namespace
{
void check_zone(rect& zone, const uint frame_res, const int line_size)
{
    const int lines = line_size ? frame_res / line_size : 0;
    if (!zone.h || !zone.w || zone.x + zone.w > line_size ||
        zone.y + zone.h > lines)
    {
        zone.x = 0;
        zone.y = 0;
        zone.w = line_size;
        zone.h = frame_res / line_size;
    }
}
} // namespace
__global__ static void kernel_composite(hipComplex* input,
                                        float* output,
                                        const uint frame_res,
                                        size_t min,
                                        size_t max,
                                        size_t range,
                                        const float* colors)
{
    const uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < frame_res)
    {
        double res_components[3] = {0};
        for (ushort p = min; p <= max; p++)
        {
            hipComplex* current_pframe = input + (frame_res * p);
            float intensity =
                hypotf(current_pframe[id].x, current_pframe[id].y);
            for (int i = 0; i < 3; i++)
                res_components[i] += colors[p * 3 + i] * intensity;
        }
        for (int i = 0; i < 3; i++)
            output[id * 3 + i] = res_components[i] / range;
    }
}

// ! Splits the image by nb_lines blocks and sums them
__global__ static void kernel_sum_one_line(float* input,
                                           const uint frame_res,
                                           const uchar pixel_depth,
                                           const uint line_size,
                                           const rect zone,
                                           float* sums_per_line)
{
    const uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < pixel_depth * zone.h)
    {
        uchar offset = id % pixel_depth;
        ushort line = id / pixel_depth;
        line += zone.y;
        uint index_begin = line_size * line + zone.x;
        uint index_end = index_begin + zone.w;
        if (index_end > frame_res)
            index_end = frame_res;
        float sum = 0;
        while (index_begin < index_end)
            sum += input[pixel_depth * (index_begin++) + offset];
        sums_per_line[id] = sum;
    }
}

// ! sums an array of size floats and put the result divided by nb_elements in
// *output
__global__ static void kernel_average_float_array(float* input,
                                                  uint size,
                                                  uint nb_elements,
                                                  uint offset_per_pixel,
                                                  float* output)
{
    const uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < offset_per_pixel)
    {
        input += id;
        float res = 0;
        while (size--)
        {
            res += *input;
            input += offset_per_pixel;
        }
        res /= static_cast<float>(nb_elements);
        output[id] = res;
    }
}

__global__ static void kernel_divide_by_weight(float* input,
                                               float weight_r,
                                               float weight_g,
                                               float weight_b)
{
    input[0] /= weight_r;
    input[1] /= weight_g;
    input[2] /= weight_b;
}
__global__ static void kernel_normalize_array(float* input,
                                              uint nb_pixels,
                                              uint pixel_depth,
                                              float* averages)
{
    const uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < pixel_depth * nb_pixels)
        input[id] /= averages[id % 3] / 1000;
    // The /1000 is used to have the result in [0;1000]
    // instead of [0;1] for a better contrast control
}

__global__ static void kernel_precompute_colors(float* colors,
                                                size_t red,
                                                size_t blue,
                                                size_t min,
                                                size_t max,
                                                size_t range,
                                                float weight_r,
                                                float weight_g,
                                                float weight_b)
{
    const uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < range)
    {
        ushort p = id + min;
        double hue = (p - min) / double(range);
        if (red > blue)
            hue = 1 - hue;
        if (hue < 0.25)
        {
            colors[p * 3 + 0] = weight_r;
            colors[p * 3 + 1] = (hue / 0.25) * weight_g;
            colors[p * 3 + 2] = 0;
        }
        else if (hue < 0.5)
        {
            colors[p * 3 + 0] = (1 - (hue - 0.25) / 0.25) * weight_r;
            colors[p * 3 + 1] = weight_g;
            colors[p * 3 + 2] = 0;
        }
        else if (hue < 0.75)
        {
            colors[p * 3 + 0] = 0;
            colors[p * 3 + 1] = weight_g;
            colors[p * 3 + 2] = ((hue - 0.5) / 0.25) * weight_b;
        }
        else
        {
            colors[p * 3 + 0] = 0;
            colors[p * 3 + 1] = (1 - (hue - 0.75) / 0.25) * weight_g;
            colors[p * 3 + 2] = weight_b;
        }
    }
}

void rgb(hipComplex* input,
         float* output,
         const uint frame_res,
         bool normalize,
         const ushort red,
         const ushort blue,
         const float weight_r,
         const float weight_g,
         const float weight_b)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    ushort min = std::min(red, blue);
    ushort max = std::max(red, blue);
    ushort range = std::abs(static_cast<short>(blue - red)) + 1;

    size_t colors_size = (max + 1) * 3;
    holovibes::cuda_tools::UniquePtr<float> colors(colors_size);

    if (normalize)
        kernel_precompute_colors<<<blocks, threads, 0, 0>>>(colors.get(),
                                                            red,
                                                            blue,
                                                            min,
                                                            max,
                                                            range,
                                                            1,
                                                            1,
                                                            1);
    else
        kernel_precompute_colors<<<blocks, threads, 0, 0>>>(colors.get(),
                                                            red,
                                                            blue,
                                                            min,
                                                            max,
                                                            range,
                                                            weight_r,
                                                            weight_g,
                                                            weight_b);

    kernel_composite<<<blocks, threads, 0, 0>>>(input,
                                                output,
                                                frame_res,
                                                min,
                                                max,
                                                range,
                                                colors.get());
    cudaCheckError();
    hipStreamSynchronize(0);
}

void postcolor_normalize(float* output,
                         const uint frame_res,
                         const uint real_line_size,
                         holovibes::units::RectFd selection,
                         const float weight_r,
                         const float weight_g,
                         const float weight_b)
{
    const uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(frame_res, threads);

    rect zone = {selection.x(),
                 selection.y(),
                 selection.unsigned_width(),
                 selection.unsigned_height()};
    check_zone(zone, frame_res, real_line_size);
    const ushort line_size = zone.w;
    const ushort lines = zone.h;
    float* averages = nullptr;
    float* sums_per_line = nullptr;
    const uchar pixel_depth = 3;
    cudaXMalloc(&averages, sizeof(float) * pixel_depth);
    cudaXMalloc(&sums_per_line, sizeof(float) * lines * pixel_depth);

    blocks = map_blocks_to_problem(lines * pixel_depth, threads);
    kernel_sum_one_line<<<blocks, threads, 0, 0>>>(output,
                                                   frame_res,
                                                   pixel_depth,
                                                   real_line_size,
                                                   zone,
                                                   sums_per_line);
    cudaCheckError();

    blocks = map_blocks_to_problem(pixel_depth, threads);
    kernel_average_float_array<<<blocks, threads, 0, 0>>>(sums_per_line,
                                                          lines,
                                                          lines * line_size,
                                                          pixel_depth,
                                                          averages);
    cudaCheckError();

    blocks = map_blocks_to_problem(frame_res * pixel_depth, threads);
    kernel_divide_by_weight<<<1, 1, 0, 0>>>(averages,
                                            weight_r,
                                            weight_g,
                                            weight_b);
    cudaCheckError();
    kernel_normalize_array<<<blocks, threads, 0, 0>>>(output,
                                                      frame_res,
                                                      pixel_depth,
                                                      averages);
    hipStreamSynchronize(0);
    cudaCheckError();
    cudaXFree(averages);
    cudaXFree(sums_per_line);
}
