#include "hip/hip_runtime.h"
#include "map.cuh"

#include "tools.hh"
#include "common.cuh"
#include "reduce.cuh"

/***** Overloaded and specific map implementations *****/
void map_log10(float* const output, const float* const input, const size_t size, const hipStream_t stream)
{
    static const auto log10 = [] __device__(const float input_pixel) { return log10f(input_pixel); };

    map_generic(output, input, size, log10, stream);
}

// It is mandatory to declare and implement these functions
// with float array parameters in order to be called from .cc

void map_divide(
    float* const output, const float* const input, const size_t size, const float value, const hipStream_t stream)
{
    // Call templated version map divide
    map_divide<float>(output, input, size, value, stream);
}

void map_multiply(
    float* const output, const float* const input, const size_t size, const float value, const hipStream_t stream)
{
    // Call templated version map multiply
    map_multiply<float>(output, input, size, value, stream);
}
