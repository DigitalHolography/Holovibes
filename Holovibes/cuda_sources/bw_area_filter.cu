#include "hip/hip_runtime.h"
#include "bw_area_filter.cuh"
#include "common.cuh"
#include "hip/hip_complex.h"
#include "hip/hip_runtime.h"
#include "cuda_memory.cuh"
#include "hardware_limits.hh"

#define IS_BACKGROUND(VALUE) ((VALUE) == 0.0f)

__device__ void lock(uint* mutex)
{
    while (atomicCAS(mutex, 0, 1) != 0)
        ;
}

__device__ void unlock(uint* mutex) { atomicExch(mutex, 0); }

__device__ void get_linked_label(uint* label, uint* linked_d)
{
    size_t pred = *label;
    while (*label != linked_d[*label])
    {
        *label = linked_d[*label];
    }
    linked_d[pred] = *label;
}

__global__ void first_pass_kernel1(const float* image_d,
                                   uint* labels_d,
                                   uint* linked_d,
                                   uint* lablels_sizes_d,
                                   const size_t width,
                                   const size_t height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int x = (idx / width) * 2;
    int y = (idx % width) * 2;
    idx = x * width + y;

    if (y >= 1 && x >= 1 && x < (width - 1) && (y < height - 1) && !IS_BACKGROUND(image_d[idx]))
    {
        linked_d[idx] = idx;
        labels_d[idx] = idx;
    }
}

__device__ void check_and_update_link(uint* labels_d,
                                      uint* linked_d,
                                      uint* labels_sizes_d,
                                      const size_t idx,
                                      uint* neighbors,
                                      const uint nb_neighbors,
                                      uint* mutex)
{
    if (nb_neighbors == 0)
    {
        linked_d[idx] = idx;
        labels_d[idx] = idx;
    }
    else
    {

        lock(mutex);
        for (size_t i = 0; i < nb_neighbors; i++)
        {
            uint tmp = labels_d[neighbors[i]];
            get_linked_label(&tmp, linked_d);
            neighbors[i] = tmp;
        }

        int min_l = 0;
        for (int k = 1; k < nb_neighbors; k++)
        {
            min_l = neighbors[k] < neighbors[min_l] ? k : min_l;
        }

        size_t label_min = neighbors[min_l];
        labels_d[idx] = label_min;

        for (int k = 1; k < nb_neighbors; k++)
        {
            linked_d[neighbors[(min_l + k) % nb_neighbors]] = label_min;
        }
        unlock(mutex);
    }
}

__global__ void first_pass_kernel2(const float* image_d,
                                   uint* labels_d,
                                   uint* linked_d,
                                   uint* labels_sizes_d,
                                   const size_t width,
                                   const size_t height,
                                   uint* mutex)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int x = (idx / width) * 2 + 1;
    int y = (idx % width) * 2;
    idx = x * width + y;

    if (y >= 1 && x >= 1 && x < (width - 1) && y < (height - 1) && !IS_BACKGROUND(image_d[idx]))
    {
        uint neighbors[2];
        uint nb_neighbors = 0;

        for (int k = -1; k <= 1; k += 2)
        {
            size_t jdx = (x + k) * width + y;
            if (labels_d[jdx])
                neighbors[nb_neighbors++] = jdx;
        }
        check_and_update_link(labels_d, linked_d, labels_sizes_d, idx, neighbors, nb_neighbors, mutex);
    }
}

__global__ void first_pass_kernel3(const float* image_d,
                                   uint* labels_d,
                                   uint* linked_d,
                                   uint* labels_sizes_d,
                                   const size_t width,
                                   const size_t height,
                                   uint* mutex)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int x = (idx / width) * 2;
    int y = (idx % width) * 2 + 1;
    idx = x * width + y;

    if (y >= 1 && x >= 1 && x < (width - 1) && y < (height - 1) && !IS_BACKGROUND(image_d[idx]))
    {
        uint neighbors[2];
        uint nb_neighbors = 0;

        for (int k = -1; k <= 1; k += 2)
        {
            size_t jdx = x * width + y + k;
            if (labels_d[jdx])
                neighbors[nb_neighbors++] = jdx;
        }
        check_and_update_link(labels_d, linked_d, labels_sizes_d, idx, neighbors, nb_neighbors, mutex);
    }
}

__global__ void first_pass_kernel4(const float* image_d,
                                   uint* labels_d,
                                   uint* linked_d,
                                   uint* labels_sizes_d,
                                   const size_t width,
                                   const size_t height,
                                   uint* mutex)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int x = (idx / width) * 2 + 1;
    int y = (idx % width) * 2 + 1;
    idx = x * width + y;

    if (y >= 1 && x >= 1 && x < (width - 1) && y < (height - 1) && !IS_BACKGROUND(image_d[idx]))
    {
        uint neighbors[4];
        uint nb_neighbors = 0;

        for (int k = -1; k <= 1; k += 2)
        {
            size_t jdx = x * width + y + k;
            if (labels_d[jdx])
                neighbors[nb_neighbors++] = jdx;
            jdx = (x + k) * width + y;
            if (labels_d[jdx])
                neighbors[nb_neighbors++] = jdx;
        }

        check_and_update_link(labels_d, linked_d, labels_sizes_d, idx, neighbors, nb_neighbors, mutex);
    }
}

void first_pass(const float* image_d,
                uint* labels_d,
                uint* linked_d,
                uint* labels_sizes_d,
                uint* size_t_gpu_,
                const size_t width,
                const size_t height,
                const hipStream_t stream)
{
    size_t size = width * height;
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size / 2, threads);
    cudaXMemset(linked_d, 0, sizeof(uint));
    cudaXMemset(size_t_gpu_, 0, sizeof(uint));

    first_pass_kernel1<<<blocks, threads, 0, stream>>>(image_d, labels_d, linked_d, labels_sizes_d, width, height);
    first_pass_kernel2<<<blocks, threads, 0, stream>>>(image_d,
                                                       labels_d,
                                                       linked_d,
                                                       labels_sizes_d,
                                                       width,
                                                       height,
                                                       size_t_gpu_);
    first_pass_kernel3<<<blocks, threads, 0, stream>>>(image_d,
                                                       labels_d,
                                                       linked_d,
                                                       labels_sizes_d,
                                                       width,
                                                       height,
                                                       size_t_gpu_);
    first_pass_kernel4<<<blocks, threads, 0, stream>>>(image_d,
                                                       labels_d,
                                                       linked_d,
                                                       labels_sizes_d,
                                                       width,
                                                       height,
                                                       size_t_gpu_);
}

__global__ void second_pass_kernel(uint* labels_d, size_t size, uint* linked_d, uint* labels_sizes_d)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && labels_d[idx] != 0)
    {
        uint l = labels_d[idx];
        get_linked_label(&l, linked_d);
        labels_d[idx] = l;
        atomicAdd(labels_sizes_d + l, 1);
    }
}

void get_connected_component(uint* labels_d,
                             uint* labels_sizes_d,
                             uint* linked_d,
                             uint* size_t_gpu_,
                             const float* image_d,
                             const size_t width,
                             const size_t height,
                             const hipStream_t stream)
{
    size_t size = width * height;
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);

    cudaXMemset(labels_d, 0, size * sizeof(uint));
    cudaXMemset(labels_sizes_d, 0, size * sizeof(uint));

    first_pass(image_d, labels_d, linked_d, labels_sizes_d, size_t_gpu_, width, height, stream);

    second_pass_kernel<<<blocks, threads, 0, stream>>>(labels_d, size, linked_d, labels_sizes_d);
}

__device__ void swap(uint* T, size_t i, size_t j)
{
    size_t tmp = T[i];
    T[i] = T[j];
    T[j] = tmp;
}

__global__ void _get_n_max_index(uint* labels_size_d, size_t nb_label, uint* labels_max_d, size_t n)
{
    for (size_t i = 0; i < n; i++)
    {
        size_t j = i;
        labels_max_d[j] = j;
        while (j > 0 && labels_size_d[labels_max_d[j - 1]] > labels_size_d[labels_max_d[j]])
        {
            swap(labels_max_d, j, j - 1);
            j--;
        }
    }
    for (size_t i = n; i < nb_label; i++)
    {
        if (labels_size_d[i] > labels_size_d[labels_max_d[0]])
        {
            labels_max_d[0] = i;
            size_t j = 1;
            while (j < n && labels_size_d[labels_max_d[j - 1]] > labels_size_d[labels_max_d[j]])
            {
                swap(labels_max_d, j, j - 1);
                j++;
            }
        }
    }
}

void get_n_max_index(uint* labels_sizes_d, size_t nb_label, uint* labels_max_d, size_t n, const hipStream_t stream)
{
    _get_n_max_index<<<1, 1, 0, stream>>>(labels_sizes_d, nb_label, labels_max_d, n);
}

__global__ void get_nb_label_kernel(uint* labels_sizes_d, size_t size, uint* res)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && labels_sizes_d[idx] != 0)
        atomicAdd(res, 1);
}

uint get_nb_label(uint* labels_sizes_d, size_t size, uint* size_t_gpu_, const hipStream_t stream)
{
    uint res;
    cudaXMemset(size_t_gpu_, 0, sizeof(uint));

    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    get_nb_label_kernel<<<blocks, threads, 0, stream>>>(labels_sizes_d, size, size_t_gpu_);
    hipDeviceSynchronize();

    cudaXMemcpy(&res, size_t_gpu_, sizeof(uint), hipMemcpyDeviceToHost);

    return res;
}

__global__ void area_filter_kernel(float* image_d, const uint* label_d, size_t size, uint* is_keep_d)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
        image_d[idx] = is_keep_d[label_d[idx]] ? 1.0f : 0.0f;
}

void area_filter(float* image_d, const uint* label_d, size_t size, uint* is_keep_d, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    area_filter_kernel<<<blocks, threads, 0, stream>>>(image_d, label_d, size, is_keep_d);
}

__global__ void
create_is_keep_in_label_size_kernel(uint* labels_sizes_d, size_t nb_labels, uint* labels_max_d, size_t n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        labels_sizes_d[labels_max_d[idx]] = 1;
}

void create_is_keep_in_label_size(
    uint* labels_sizes_d, size_t nb_labels, uint* labels_max_d, size_t n, const hipStream_t stream)
{
    cudaXMemsetAsync(labels_sizes_d, 0, nb_labels * sizeof(uint), stream);
    create_is_keep_in_label_size_kernel<<<1, n, 0, stream>>>(labels_sizes_d, nb_labels, labels_max_d, n);
}
