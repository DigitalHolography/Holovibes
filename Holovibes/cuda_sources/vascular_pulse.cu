#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <cmath>


#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "tools_analysis.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"
#include "vascular_pulse.cuh"

void compute_first_correlation()
{

}