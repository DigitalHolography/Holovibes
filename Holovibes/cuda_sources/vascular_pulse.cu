#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <cmath>

#include "convolution.cuh"
#include "tools_conversion.cuh"
#include "tools_analysis.cuh"
#include "unique_ptr.hh"
#include "tools_compute.cuh"
#include "cuda_memory.cuh"
#include "logger.hh"
#include "hip/hip_complex.h"
#include "cufft_handle.hh"
#include "vascular_pulse.cuh"

__global__ void kernel_divide_constant(float* vascular_pulse, int value, size_t size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        vascular_pulse[index] /= value;
}

void divide_constant(float* vascular_pulse, int value, size_t size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_divide_constant<<<blocks, threads, 0, stream>>>(vascular_pulse, value, size);
}

__global__ void kernel_divide(float* vascular_pulse, float* value, size_t size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        vascular_pulse[index] /= value[index];
}

void divide(float* vascular_pulse, float* value, size_t size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_divide<<<blocks, threads, 0, stream>>>(vascular_pulse, value, size);
}

__global__ void kernel_multiply_constant(float* vascular_pulse, float* value, size_t size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        vascular_pulse[index] *= (*value);
}

void multiply_constant(float* vascular_pulse, float* value, size_t size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_multiply_constant<<<blocks, threads, 0, stream>>>(vascular_pulse, value, size);
}

float compute_mean(float* vascular_pulse, size_t size)
{
    thrust::device_ptr<float> d_array(vascular_pulse);

    float sum = thrust::reduce(d_array, d_array + size, 0.0f, thrust::plus<float>());

    return sum / size;
}

__global__ void kernel_subtract_constant(float* output, float* input, float value, size_t size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        output[index] = input[index] - value;
}

void subtract_constant(float* output, float* input, float value, size_t size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_subtract_constant<<<blocks, threads, 0, stream>>>(output, input, value, size);
}

__global__ void kernel_multiply_three_vectors(float* output, float* input1, float* input2, float* input3, size_t size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        output[index] = input1[index] * input2[index] * input3[index];
}

void multiply_three_vectors(
    float* output, float* input1, float* input2, float* input3, size_t size, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_multiply_three_vectors<<<blocks, threads, 0, stream>>>(output, input1, input2, input3, size);
}

__global__ void kernel_computeMean(
    const float* M0, const float* vascularPulse, float* result, 
    int rows, int cols, int depth
) {
    // Calcul des indices globaux
    int x = blockIdx.x * blockDim.x + threadIdx.x; // index de ligne
    int y = blockIdx.y * blockDim.y + threadIdx.y; // index de colonne

    if (x < rows && y < cols) {
        float sum = 0.0f;

        // Somme sur la 3ème dimension
        for (int z = 0; z < depth; ++z) {
            int index3D = x * cols + y + z * rows * cols;
            sum += M0[index3D] * vascularPulse[z];
        }

        // Stocker la moyenne dans le tableau résultat
        result[x * cols + y] = sum / depth;
    }
}

void computeMean(const float* M0, const float* vascularPulse, float* result, 
    int rows, int cols, int depth, hipStream_t stream)
{
    dim3 blockSize(16, 16);
    dim3 gridSize((rows + blockSize.x - 1) / blockSize.x, 
                  (cols + blockSize.y - 1) / blockSize.y);

    // Lancer le kernel
    kernel_computeMean<<<gridSize, blockSize, 0, stream>>>(M0, vascularPulse, result, rows, cols, depth);
}

__global__ void kernel_compute_std(const float* input, float* output, int size, int depth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        float mean = 0.0f;
        float variance = 0.0f;

        // Compute mean along the third dimension
        for (int k = 0; k < depth; ++k) {
            mean += input[idx + depth * k];
        }
        mean /= depth;

        // Compute variance along the third dimension
        for (int k = 0; k < depth; ++k) {
            float diff = input[idx + depth * k] - mean;
            variance += diff * diff;
        }
        variance /= depth;

        // Store the standard deviation in the output array
        output[idx] = sqrt(variance);
    }
}

void compute_std(const float* input, float* output, int size, int depth, hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);
    kernel_compute_std<<<blocks, threads, 0, stream>>>(input, output, size, depth);
}



void compute_first_correlation(float* output,
                               float* M0_ff_video_centered,
                               float* vascular_pulse,
                               int nnz_mask_vesslness_clean,
                               size_t length_video,
                               size_t image_size,
                               hipStream_t stream) // Size here is future time window
{

    float* vascular_pulse_copy;
    cudaXMalloc(&vascular_pulse_copy, sizeof(float) * length_video);
    cudaXMemcpyAsync(vascular_pulse_copy,
                     vascular_pulse,
                     sizeof(float) * length_video,
                     hipMemcpyDeviceToDevice,
                     stream);

    divide_constant(vascular_pulse_copy, nnz_mask_vesslness_clean, length_video, stream);


    float* vascular_pulse_centered;
    cudaXMalloc(&vascular_pulse_centered,
                506 * sizeof(float)); // need to be replaced with time window (it's because csv)

    float vascular_mean = compute_mean(vascular_pulse_copy, length_video);
    subtract_constant(vascular_pulse_centered, vascular_pulse_copy, vascular_mean, length_video, stream);

    // TODO: la suite (le calcul de R_vascularPulse)
    computeMean(M0_ff_video_centered, vascular_pulse_centered, output, 512, 512, 506, stream);
    
    float *std_M0_ff_video_centered;
    cudaXMalloc(&std_M0_ff_video_centered, sizeof(float) * 512 * 512);
    compute_std(M0_ff_video_centered, std_M0_ff_video_centered, 512 * 512, 506, stream);

    float *std_vascular_pulse_centered;
    cudaXMalloc(&std_vascular_pulse_centered, sizeof(float));
    compute_std(vascular_pulse_centered, std_vascular_pulse_centered, 1, 506, stream);

    multiply_constant(std_M0_ff_video_centered, std_vascular_pulse_centered, 512 * 512, stream);

    divide(output, std_M0_ff_video_centered, 512 * 512, stream);
    
    cudaXFree(std_M0_ff_video_centered);
    cudaXFree(std_vascular_pulse_centered);
    cudaXFree(vascular_pulse_centered);
    cudaXFree(vascular_pulse_copy);
}