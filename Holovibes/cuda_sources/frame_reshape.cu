#include "hip/hip_runtime.h"
/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include "frame_reshape.cuh"

#include "tools.hh"
#include "common.cuh"

hipError_t embedded_frame_cpy(const char* input,
                               const uint input_width,
                               const uint input_height,
                               char* output,
                               const uint output_width,
                               const uint output_height,
                               const uint output_startx,
                               const uint output_starty,
                               const uint elm_size,
                               hipMemcpyKind kind,
                               const hipStream_t stream)
{
    assert(input_width + output_startx <= output_width);
    assert(input_height + output_starty <= output_height);

    char* output_write_start =
        output + elm_size * (output_starty * output_width + output_startx);
    return hipMemcpy2DAsync(output_write_start,
                             output_width * elm_size,
                             input,
                             input_width * elm_size,
                             input_width * elm_size,
                             input_height,
                             kind,
                             stream);
}

hipError_t embed_into_square(const char* input,
                              const uint input_width,
                              const uint input_height,
                              char* output,
                              const uint elm_size,
                              hipMemcpyKind kind,
                              const hipStream_t stream)
{
    uint output_startx;
    uint output_starty;
    uint square_side_len;

    if (input_width >= input_height) // Usually the case
    {
        square_side_len = input_width;
        output_startx = 0;
        output_starty = (square_side_len - input_height) / 2;
    }
    else
    {
        square_side_len = input_height;
        output_startx = (square_side_len - input_width) / 2;
        output_starty = 0;
    }
    return embedded_frame_cpy(input,
                              input_width,
                              input_height,
                              output,
                              square_side_len,
                              square_side_len,
                              output_startx,
                              output_starty,
                              elm_size,
                              kind,
                              stream);
}

static __global__ void
kernel_batched_embed_into_square(const char* input,
                                 const uint input_width,
                                 const uint input_height,
                                 char* output,
                                 const uint output_width,
                                 const uint output_height,
                                 const uint output_startx,
                                 const uint output_starty,
                                 const uint batch_size,
                                 const uint elm_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint x = index % output_width;
    const uint y = index / output_width;

    if (index < output_width * output_height)
    {
        for (uint i = 0; i < batch_size; i++)
        {
            const uint batch_index =
                index + i * input_width * input_height * elm_size;

            if (x < output_startx || x >= output_startx + input_width ||
                y < output_starty || y >= output_starty + input_height)
                output[batch_index] = 0;
            else
            {
                if (output_startx ==
                    0) // Horizontal black bands (top and bottom)
                    output[batch_index] =
                        input[batch_index -
                              output_starty * input_width * elm_size];
                else // Vertical black bands (left and right)
                    output[batch_index] =
                        input[batch_index -
                              (2 * y + 1) * output_startx * elm_size];
            }
        }
    }
}

void batched_embed_into_square(const char* input,
                               const uint input_width,
                               const uint input_height,
                               char* output,
                               const uint batch_size,
                               const uint elm_size,
                               const hipStream_t stream)
{
    uint output_startx;
    uint output_starty;
    uint square_side_len;

    if (input_width >= input_height) // Usually the case
    {
        square_side_len = input_width;
        output_startx = 0;
        output_starty = (square_side_len - input_height) / 2;
    }
    else
    {
        square_side_len = input_height;
        output_startx = (square_side_len - input_width) / 2;
        output_starty = 0;
    }

    size_t threads = get_max_threads_1d();
    size_t blocks =
        map_blocks_to_problem(square_side_len * square_side_len, threads);

    kernel_batched_embed_into_square<<<blocks, threads, 0, stream>>>
                                                         (input,
                                                          input_width,
                                                          input_height,
                                                          output,
                                                          square_side_len,
                                                          square_side_len,
                                                          output_startx,
                                                          output_starty,
                                                          batch_size,
                                                          elm_size);
    cudaCheckError();
}

hipError_t crop_frame(const char* input,
                       const uint input_width,
                       const uint input_height,
                       const uint crop_start_x,
                       const uint crop_start_y,
                       const uint crop_width,
                       const uint crop_height,
                       char* output,
                       const uint elm_size,
                       hipMemcpyKind kind,
                       const hipStream_t stream)
{
    assert(crop_start_x + crop_width <= input_width);
    assert(crop_start_y + crop_height <= input_height);

    const char* crop_start =
        input + elm_size * (crop_start_y * input_width + crop_start_x);
    return hipMemcpy2DAsync(output,
                             crop_width * elm_size,
                             crop_start,
                             input_width * elm_size,
                             crop_width * elm_size,
                             crop_height,
                             kind,
                             stream);
}

hipError_t crop_into_square(const char* input,
                             const uint input_width,
                             const uint input_height,
                             char* output,
                             const uint elm_size,
                             hipMemcpyKind kind,
                             const hipStream_t stream)
{
    uint crop_start_x;
    uint crop_start_y;
    uint square_side_len;

    if (input_width >= input_height)
    {
        square_side_len = input_height;
        crop_start_x = (input_width - square_side_len) / 2;
        crop_start_y = 0;
    }
    else
    {
        square_side_len = input_width;
        crop_start_x = 0;
        crop_start_y = (input_height - square_side_len) / 2;
    }

    return crop_frame(input,
                      input_width,
                      input_height,
                      crop_start_x,
                      crop_start_y,
                      square_side_len,
                      square_side_len,
                      output,
                      elm_size,
                      kind,
                      stream);
}

static __global__ void kernel_batched_crop_into_square(const char* input,
                                                       const uint input_width,
                                                       const uint input_height,
                                                       const uint crop_start_x,
                                                       const uint crop_start_y,
                                                       const uint crop_width,
                                                       const uint crop_height,
                                                       char* output,
                                                       const uint elm_size,
                                                       const uint batch_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = index / crop_width;

    if (index < crop_width * crop_height)
    {
        for (uint i = 0; i < batch_size; i++)
        {
            const uint batch_index =
                index + i * input_width * input_height * elm_size;

            if (crop_start_x == 0) // Horizontal black bands (top and bottom)
                output[batch_index] =
                    input[batch_index + crop_start_y * input_width * elm_size];
            else // Vertical black bands (left and right)
                output[batch_index] =
                    input[batch_index + (2 * y + 1) * crop_start_x * elm_size];
        }
    }
}

void batched_crop_into_square(const char* input,
                              const uint input_width,
                              const uint input_height,
                              char* output,
                              const uint elm_size,
                              const uint batch_size,
                              const hipStream_t stream)
{
    uint crop_start_x;
    uint crop_start_y;
    uint square_side_len;

    if (input_width >= input_height)
    {
        square_side_len = input_height;
        crop_start_x = (input_width - square_side_len) / 2;
        crop_start_y = 0;
    }
    else
    {
        square_side_len = input_width;
        crop_start_x = 0;
        crop_start_y = (input_height - square_side_len) / 2;
    }

    size_t threads = get_max_threads_1d();
    size_t blocks =
        map_blocks_to_problem(square_side_len * square_side_len, threads);

    kernel_batched_crop_into_square<<<blocks, threads, 0, stream>>>
                                                        (input,
                                                         input_width,
                                                         input_height,
                                                         crop_start_x,
                                                         crop_start_y,
                                                         square_side_len,
                                                         square_side_len,
                                                         output,
                                                         elm_size,
                                                         batch_size);
    cudaCheckError();
}