#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "filter2D.cuh"

using camera::FrameDescriptor;

__global__
void filter2D_roi(hipComplex	*input,
				const uint	tl_x,
				const uint	tl_y,
				const uint	br_x,
				const uint	br_y,
				const uint	width,
				const uint	size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	// In ROI
	if (index < size)
	{
		uint mod_index = index % width;
		if (!(index >= tl_y * width && index < br_y * width
			&& mod_index >= tl_x && mod_index < br_x))
		{
			input[index] = make_hipComplex(0, 0);
		}
	}
}


void filter2D(hipComplex				*input,
			hipComplex				*tmp_buffer,
			const hipfftHandle		plan2d,
			const holovibes::units::RectFd&	r,
			const FrameDescriptor&	desc,
			hipStream_t			stream)
{
	uint threads = THREADS_128;
	uint blocks = map_blocks_to_problem(desc.frame_res(), threads);
	uint size = desc.width * desc.height;

	hipfftExecC2C(plan2d, input, input, HIPFFT_FORWARD);
	hipStreamSynchronize(stream);

	if (!r.area())
		return;
	//int center_x = (r.x + r.bottom_right.x) >> 1;
	//int center_y = (r.top_left.y + r.bottom_right.y) >> 1;
	
	filter2D_roi << <blocks, threads, 0, stream >> >(
		input,
		r.x(),
		r.y(),
		r.bottomRight().x(),
		r.bottomRight().y(),
		desc.width,
		desc.width * desc.height);

	hipMemcpy(tmp_buffer, input, size * sizeof (hipComplex), hipMemcpyDeviceToDevice);

	circ_shift << <blocks, threads, 0, stream >> >(
		tmp_buffer,
		input,
		r.center().x(),
		r.center().y(),
		desc.width,
		desc.height,
		size);

	hipfftExecC2C(plan2d, input, input, HIPFFT_BACKWARD);
}
