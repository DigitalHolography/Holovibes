#include "input_filter.cuh"

#include <thrust/transform.h>
#include <thrust/execution_policy.h>

void apply_filter(float* gpu_filter, hipComplex* gpu_input, size_t frame_res, const hipStream_t stream)
{
    auto exec_policy = thrust::cuda::par.on(stream);

    auto mult_func = []  __device__ (hipComplex a, float b) {return make_hipComplex(a.x * b, a.y * b)};

    thrust::transform(exec_policy, gpu_input, gpu_input + frame_res, gpu_filter, mult_func);
}