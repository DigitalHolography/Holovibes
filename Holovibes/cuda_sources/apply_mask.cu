#include "hip/hip_runtime.h"
#include "apply_mask.cuh"

#include "hardware_limits.hh"
#include "common.cuh"

__host__ __device__ static hipComplex& operator*=(hipComplex& c, const float& r)
{
    c.x = c.x * r;
    c.y = c.y * r;
    return c;
}

__host__ __device__ static hipComplex operator*(const hipComplex& c, const float& r)
{
    hipComplex n;

    n.x = c.x * r;
    n.y = c.y * r;

    return n;
}

__host__ __device__ static hipComplex operator*(const hipComplex& c1, const hipComplex& c2) { return hipCmulf(c1, c2); }

__host__ __device__ static hipComplex& operator*=(hipComplex& c1, const hipComplex& c2)
{
    c1 = hipCmulf(c1, c2);
    return c1;
}

template <typename T, typename M>
__global__ static void kernel_apply_mask(T* in_out, const M* mask, const size_t size, const uint batch_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        for (uint i = 0; i < batch_size; ++i)
        {
            in_out[(size * i) + index] *= mask[index];
        }
    }
}

template <typename T, typename M>
__global__ static void
kernel_apply_mask(const T* input, const M* mask, T* output, const size_t size, const uint batch_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        for (uint i = 0; i < batch_size; ++i)
        {
            output[(size * i) + index] = input[(size * i) + index] * mask[index];
        }
    }
}

template <typename T, typename M>
static void
apply_mask_caller(T* in_out, const M* mask, const size_t size, const uint batch_size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);

    kernel_apply_mask<T, M><<<blocks, threads, 0, stream>>>(in_out, mask, size, batch_size);
    cudaCheckError();
}

template <typename T, typename M>
static void apply_mask_caller(
    const T* input, const M* mask, T* output, const size_t size, const uint batch_size, const hipStream_t stream)
{
    uint threads = get_max_threads_1d();
    uint blocks = map_blocks_to_problem(size, threads);

    kernel_apply_mask<T, M><<<blocks, threads, 0, stream>>>(input, mask, output, size, batch_size);
    cudaCheckError();
}

void apply_mask(
    hipComplex* in_out, const hipComplex* mask, const size_t size, const uint batch_size, const hipStream_t stream)
{
    apply_mask_caller<hipComplex, hipComplex>(in_out, mask, size, batch_size, stream);
}

void apply_mask(
    hipComplex* in_out, const float* mask, const size_t size, const uint batch_size, const hipStream_t stream)
{
    apply_mask_caller<hipComplex, float>(in_out, mask, size, batch_size, stream);
}

void apply_mask(float* in_out, const float* mask, const size_t size, const uint batch_size, const hipStream_t stream)
{
    apply_mask_caller<float, float>(in_out, mask, size, batch_size, stream);
}

void apply_mask(const hipComplex* input,
                const hipComplex* mask,
                hipComplex* output,
                const size_t size,
                const uint batch_size,
                const hipStream_t stream)
{
    apply_mask_caller<hipComplex, hipComplex>(input, mask, output, size, batch_size, stream);
}

void apply_mask(const hipComplex* input,
                const float* mask,
                hipComplex* output,
                const size_t size,
                const uint batch_size,
                const hipStream_t stream)
{
    apply_mask_caller<hipComplex, float>(input, mask, output, size, batch_size, stream);
}

void apply_mask(const float* input,
                const float* mask,
                float* output,
                const size_t size,
                const uint batch_size,
                const hipStream_t stream)
{
    apply_mask_caller<float, float>(input, mask, output, size, batch_size, stream);
}
