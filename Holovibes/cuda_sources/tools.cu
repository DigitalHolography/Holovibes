#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "tools.cuh"
#include "tools_compute.cuh"
#include "tools_unwrap.cuh"
#include "cuda_tools/unique_ptr.hh"
#include "cuda_tools/cufft_handle.hh"
#include "logger.hh"
#include "cuda_memory.cuh"
#include "Common.cuh"

#include <cassert>

using camera::FrameDescriptor;
using namespace holovibes;
using cuda_tools::UniquePtr;
using cuda_tools::CufftHandle;

__global__
void kernel_apply_lens(hipComplex		*input,
					hipComplex 			*output,
					const uint 			batch_size,
					const uint			input_size,
					const hipComplex		*lens,
					const uint			lens_size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < input_size)
	{
		for (uint i = 0; i < batch_size; ++i)
		{
			const uint batch_index = index + i * input_size;

			const float	tmp_x = input[batch_index].x;
			output[batch_index].x = input[batch_index].x * lens[index].x - input[batch_index].y * lens[index].y;
			output[batch_index].y = input[batch_index].y * lens[index].x + tmp_x * lens[index].y;
		}
	}
}

__global__
void kernel_complex_to_modulus(const hipComplex	*input,
							float				*output,
							const uint			size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
		output[index] = hypotf(input[index].x, input[index].y);
}

void frame_memcpy(const float				*input,
				const units::RectFd&	zone,
				const uint			input_width,
				float				*output,
				hipStream_t		stream)
{
	const float	*zone_ptr = input + (zone.topLeft().y() * input_width + zone.topLeft().x());
	cudaSafeCall(hipMemcpy2DAsync(output,
					  zone.width() * sizeof(float),
					  zone_ptr,
					  input_width * sizeof(float),
					  zone.width() * sizeof(float),
					  zone.height(),
					  hipMemcpyDeviceToDevice,
					  stream));
	hipStreamSynchronize(stream);
}

__global__
void circ_shift(const hipComplex	*input,
				hipComplex	*output,
				const uint 	batch_size,
				const int	i, // shift on x axis
				const int	j, // shift on y axis
				const uint	width,
				const uint	height,
				const uint	size)
{
	const uint	index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
	{
		int index_x = index % width;
		int index_y = index / width;
		int shift_x = index_x - i;
		int shift_y = index_y - j;
		shift_x = (shift_x < 0) ? (width + shift_x) : shift_x;
		shift_y = (shift_y < 0) ? (height + shift_y) : shift_y;

		for (uint i = 0; i < batch_size; ++i)
		{
			const uint batch_index = index + i * size;

			const hipComplex rhs = input[batch_index];

			output[((width * shift_y) + shift_x) + i * size] = rhs;
		}
	}
}

__global__
void circ_shift_float(const float		*input,
					float		*output,
					const uint 	batch_size,
					const int	i, // shift on x axis
					const int	j, // shift on y axis
					const uint	width,
					const uint	height,
					const uint	size)
{
	const uint	index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
	{
		int index_x = index % width;
		int index_y = index / width;
		int shift_x = index_x - i;
		int shift_y = index_y - j;
		shift_x = (shift_x < 0) ? (width + shift_x) : shift_x;
		shift_y = (shift_y < 0) ? (height + shift_y) : shift_y;

        for (uint i = 0; i < batch_size; ++i)
		{
			const uint batch_index = index + i * size;

			const float rhs = input[batch_index];

			output[((width * shift_y) + shift_x) + i * size] = rhs;
		}
	}
}

__global__
void kernel_translation(float		*input,
						float		*output,
						uint		width,
						uint		height,
						int			shift_x,
						int			shift_y)
{
	const uint	index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < width * height)
	{
		const int new_x = index % width;
		const int new_y = index / width;
		const int old_x = (new_x - shift_x + width) % width;
		const int old_y = (new_y - shift_y + height) % height;
		output[index] = input[old_y * width + old_x];
	}
}

