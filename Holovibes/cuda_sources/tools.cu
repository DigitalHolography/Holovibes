#include "hip/hip_runtime.h"
/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "tools.cuh"
#include "tools_compute.cuh"
#include "tools_unwrap.cuh"

__global__
void kernel_apply_lens(hipComplex		*input,
					const uint			input_size,
					const hipComplex		*lens,
					const uint			lens_size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	//while (index < input_size)
	{
		uint	index2 = index % lens_size; // necessary if more than one frame
		float	tmp_x = input[index].x;
		input[index].x = input[index].x * lens[index2].x - input[index].y * lens[index2].y;
		input[index].y = input[index].y * lens[index2].x + tmp_x * lens[index2].y;
		//index += blockDim.x * gridDim.x;
	}
}

static __global__
void kernel_shift_corners(float		*input,
						const uint	size_x,
						const uint	size_y)
{
	const uint	i = blockIdx.x * blockDim.x + threadIdx.x;
	const uint	j = blockIdx.y * blockDim.y + threadIdx.y;
	const uint	index = j * blockDim.x * gridDim.x + i;
	uint	ni = 0;
	uint	nj = 0;
	uint	nindex = 0;

	// Superior half of the matrix
	const uint size_x2 = size_x >> 1;
	const uint size_y2 = size_y >> 1;
	if (j >= size_y2)
	{
		// Left superior quarter of the matrix
		if (i < size_x2)
			ni = i + size_x2;
		else // Right superior quarter
			ni = i - size_x2;
		nj = j - size_y2;
		nindex = nj * size_x + ni;

		float tmp = input[nindex];
		input[nindex] = input[index];
		input[index] = tmp;
	}
}

void shift_corners(float		*input,
				const uint		size_x,
				const uint		size_y,
				hipStream_t	stream)
{
	uint threads_2d = get_max_threads_2d();
	dim3 lthreads(threads_2d, threads_2d);
	dim3 lblocks(size_x / threads_2d, size_y / threads_2d);

	kernel_shift_corners << < lblocks, lthreads, 0, stream >> >(input, size_x, size_y);
}

/* Kernel used in apply_log10 */
static __global__
void kernel_log10(float		*input,
				const uint	size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		input[index] = log10f(input[index]);
	//	index += blockDim.x * gridDim.x;
	}
}

void apply_log10(float			*input,
				const uint		size,
				hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_log10 << <blocks, threads, 0, stream >> >(input, size);
}

/* Kernel used in convolution_operator */
static __global__
void kernel_complex_to_modulus(const hipComplex	*input,
							float				*output,
							const uint			size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	//while (index < size)
	{
		output[index] = hypotf(input[index].x, input[index].y);
		//index += blockDim.x * gridDim.x;
	}
}

void demodulation(hipComplex			*input,
				const hipfftHandle	plan1d,
				hipStream_t		stream)
{
	// FFT 1D TEMPORAL
	hipfftExecC2C(plan1d, input, input, HIPFFT_FORWARD);
}


void convolution_operator(	const hipComplex		*x,
							const hipComplex		*k,
							float				*out,
							const uint			size,
							const hipfftHandle	plan2d_x,
							const hipfftHandle	plan2d_k,
							hipStream_t		stream)
{
	uint	threads = get_max_threads_1d();
	uint	blocks = map_blocks_to_problem(size, threads);

	/* The convolution operator is used only when using autofocus feature.
	 * It could be optimized but it's useless since it will be used occasionnally. */
	hipComplex *tmp_x;
	hipComplex *tmp_k;
	uint	complex_size = size * sizeof(hipComplex);
	if (hipMalloc<complex>(&tmp_x, complex_size) != hipSuccess)
		return;
	if (hipMalloc<complex>(&tmp_k, complex_size) != hipSuccess)
		return;

	hipfftExecC2C(plan2d_x, const_cast<hipComplex*>(x), tmp_x, HIPFFT_FORWARD);
	hipfftExecC2C(plan2d_k, const_cast<hipComplex*>(k), tmp_k, HIPFFT_FORWARD);

	hipStreamSynchronize(stream);

	kernel_multiply_frames_complex << <blocks, threads, 0, stream >> >(tmp_x, tmp_k, tmp_x, size);

	hipStreamSynchronize(stream);

	hipfftExecC2C(plan2d_x, tmp_x, tmp_x, HIPFFT_BACKWARD);

	hipStreamSynchronize(stream);

	kernel_complex_to_modulus << <blocks, threads, 0, stream >> >(tmp_x, out, size);

	hipFree(tmp_x);
	hipFree(tmp_k);
}

void frame_memcpy(float				*input,
				const Rectangle&	zone,
				const uint			input_width,
				float				*output,
				const uint			output_width,
				hipStream_t		stream)
{
	const float	*zone_ptr = input + (zone.topLeft().y() * input_width + zone.topLeft().x());
	const uint	output_width_float = output_width * sizeof(float);
	hipMemcpy2DAsync(	output,
						output_width_float,
						zone_ptr,
						input_width * sizeof(float),
						output_width_float,
						output_width,
						hipMemcpyDeviceToDevice,
						stream);
	hipStreamSynchronize(stream);
}

/* Kernel helper used in average.
 *
 * Sums up the *size* first elements of input and stores the result in sum.
 *
 * SpanSize is the number of elements processed by a single thread.
 * This way of doing things comes from the empirical fact that (at the point
 * of this writing) loop unrolling in CUDA kernels may prove more efficient,
 * when the operation is really small. */
template <uint SpanSize>

static __global__
void kernel_sum(const float* input, float* sum, const size_t size)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if ((index + SpanSize - 1) < size && (index % SpanSize) == 0)
	{
		float tmp_reduce = 0.0f;
		for (uint i = 0; i < SpanSize; ++i)
			tmp_reduce += input[index + i];
		// Atomic operation is needed here to guarantee a correct value.
		atomicAdd(sum, tmp_reduce);
	}
}

float average_operator(const float	*input,
					const uint		size,
					hipStream_t	stream)
{
	const uint	threads = THREADS_128;
	uint		blocks = map_blocks_to_problem(size, threads);
	float		*gpu_sum;
	float		cpu_sum = 0.0f;

	if (hipMalloc<float>(&gpu_sum, sizeof(float)) == hipSuccess)
		hipMemsetAsync(gpu_sum, 0, sizeof(float), stream);
	else
		return (0.f);
	hipStreamSynchronize(stream);

	// A SpanSize of 4 has been determined to be an optimal choice here.
	kernel_sum <4> << <blocks, threads, 0, stream >> >(
		input,
		gpu_sum,
		size);
	hipMemcpyAsync(&cpu_sum, gpu_sum, sizeof(float), hipMemcpyDeviceToHost);
	hipStreamSynchronize(stream);

	hipFree(gpu_sum);

	return cpu_sum /= static_cast<float>(size);
}

void phase_increase(const hipComplex			*cur,
					UnwrappingResources		*resources,
					const size_t			image_size)
{
	const uint	threads = THREADS_128; // 3072 cuda cores / 24 SMM = 128 Threads per SMM
	const uint	blocks = map_blocks_to_problem(image_size, threads);
	static bool first_time = true;
	if (first_time)
	{
		hipMemcpy(	resources->gpu_predecessor_,
					cur,
					sizeof(hipComplex)* image_size,
					hipMemcpyDeviceToDevice);
		first_time = false;
	}

	// Compute the newest phase image, not unwrapped yet
	kernel_compute_angle_mult << <blocks, threads >> >(	resources->gpu_predecessor_,
														cur,
														resources->gpu_angle_current_,
														image_size);
	// Updating predecessor (complex image) for the next iteration
	hipMemcpy(	resources->gpu_predecessor_,
				cur,
				sizeof(hipComplex) * image_size,
				hipMemcpyDeviceToDevice);

	/* Copying in order to later enqueue the (not summed up with values
	 * in gpu_unwrap_buffer_) phase image. */
	hipMemcpy(	resources->gpu_angle_copy_,
				resources->gpu_angle_current_,
				sizeof(float) * image_size,
				hipMemcpyDeviceToDevice);

	// Applying history on the latest phase image
	kernel_correct_angles << <blocks, threads >> >(	resources->gpu_angle_current_,
													resources->gpu_unwrap_buffer_,
													image_size,
													resources->size_);

	/* Store the new phase image in the next buffer position.
	* The buffer is handled as a circular buffer. */
	float	*next_unwrap = resources->gpu_unwrap_buffer_ + image_size * resources->next_index_;
	hipMemcpy(	next_unwrap,
				resources->gpu_angle_copy_,
				sizeof(float)* image_size,
				hipMemcpyDeviceToDevice);
	if (resources->size_ < resources->capacity_)
		++resources->size_;
	resources->next_index_ = (resources->next_index_ + 1) % resources->capacity_;
}

void unwrap_2d(	float*						input,
				const hipfftHandle			plan2d,
				UnwrappingResources_2d*		res,
				FrameDescriptor&	fd,
				float*						output,
				hipStream_t				stream)
{
	uint		threads_2d = get_max_threads_2d();
	dim3		lthreads(threads_2d, threads_2d);
	dim3		lblocks(fd.width / threads_2d, fd.height / threads_2d);
	const uint	threads = THREADS_128;
	const uint	blocks = map_blocks_to_problem(res->image_resolution_, threads);

	kernel_init_unwrap_2d << < lblocks, lthreads, 0, stream >> > (	fd.width,
																	fd.height,
																	fd.frame_res(),
																	input,
																	res->gpu_fx_,
																	res->gpu_fy_,
																	res->gpu_z_);
	ushort middlex = fd.width >> 1;
	ushort middley = fd.height >> 1;
	circ_shift_float << < blocks, threads, 0, stream >> > (	res->gpu_fx_,
															res->gpu_shift_fx_,
															middlex,
															middley,
															fd.width,
															fd.height,
															fd.frame_res());
	circ_shift_float << < blocks, threads, 0, stream >> > (	res->gpu_fy_,
															res->gpu_shift_fy_,
															middlex,
															middley,
															fd.width,
															fd.height,
															fd.frame_res());
	gradient_unwrap_2d(plan2d, res, fd, stream);
	eq_unwrap_2d(plan2d, res, fd, stream);
	phi_unwrap_2d(plan2d, res, fd, output, stream);
}

void gradient_unwrap_2d(const hipfftHandle			plan2d,
						UnwrappingResources_2d*		res,
						FrameDescriptor&			fd,
						hipStream_t				stream)
{
	const uint	threads = THREADS_128;
	const uint	blocks = map_blocks_to_problem(res->image_resolution_, threads);
	hipComplex	single_complex = make_hipComplex(0.f, static_cast<float>(M_2PI));

	hipfftExecC2C(plan2d, res->gpu_z_, res->gpu_grad_eq_x_, HIPFFT_FORWARD);
	hipfftExecC2C(plan2d, res->gpu_z_, res->gpu_grad_eq_y_, HIPFFT_FORWARD);
	kernel_multiply_complexes_by_floats_ << < blocks, threads, 0, stream >> >(	res->gpu_shift_fx_,
																				res->gpu_shift_fy_,
																				res->gpu_grad_eq_x_,
																				res->gpu_grad_eq_y_,
																				fd.frame_res());
	hipfftExecC2C(plan2d, res->gpu_grad_eq_x_, res->gpu_grad_eq_x_, HIPFFT_BACKWARD);
	hipfftExecC2C(plan2d, res->gpu_grad_eq_y_, res->gpu_grad_eq_y_, HIPFFT_BACKWARD);
	kernel_multiply_complexes_by_single_complex << < blocks, threads, 0, stream >> >(	res->gpu_grad_eq_x_,
																						res->gpu_grad_eq_y_,
																						single_complex,
																						fd.frame_res());
}

void eq_unwrap_2d(const hipfftHandle			plan2d,
				UnwrappingResources_2d*		res,
				FrameDescriptor&			fd,
				hipStream_t				stream)
{
	const uint	threads = THREADS_128;
	const uint	blocks = map_blocks_to_problem(res->image_resolution_, threads);
	hipComplex	single_complex = make_hipComplex(0, 1);

	kernel_multiply_complex_by_single_complex << < blocks, threads, 0, stream >> >(	res->gpu_z_,
																					single_complex,
																					fd.frame_res());
	kernel_conjugate_complex << < blocks, threads, 0, stream >> >(res->gpu_z_, fd.frame_res());
	kernel_multiply_complex_frames_by_complex_frame << < blocks, threads, 0, stream >> >(	res->gpu_grad_eq_x_,
																							res->gpu_grad_eq_y_,
																							res->gpu_z_,
																							fd.frame_res());
	hipfftExecC2C(plan2d, res->gpu_grad_eq_x_, res->gpu_grad_eq_x_, HIPFFT_FORWARD);
	hipfftExecC2C(plan2d, res->gpu_grad_eq_y_, res->gpu_grad_eq_y_, HIPFFT_FORWARD);
	kernel_norm_ratio << < blocks, threads, 0, stream >> >(	res->gpu_shift_fx_,
															res->gpu_shift_fy_,
															res->gpu_grad_eq_x_,
															res->gpu_grad_eq_y_,
															fd.frame_res());
}

void phi_unwrap_2d(	const hipfftHandle			plan2d,
					UnwrappingResources_2d*		res,
					FrameDescriptor&			fd,
					float*						output,
					hipStream_t				stream)
{
	const uint threads = THREADS_128;
	const uint blocks = map_blocks_to_problem(res->image_resolution_, threads);

	//	kernel_convergence << < 1, 1, 0, stream >> >(res->gpu_grad_eq_x_,
	//		res->gpu_grad_eq_y_);
	kernel_add_complex_frames << < blocks, threads, 0, stream >> >(res->gpu_grad_eq_x_, res->gpu_grad_eq_y_, fd.frame_res());
	hipfftExecC2C(plan2d, res->gpu_grad_eq_x_, res->gpu_grad_eq_x_, HIPFFT_BACKWARD);
	kernel_unwrap2d_last_step << < blocks, threads, 0, stream >> > (output, res->gpu_grad_eq_x_, fd.frame_res());
}

__global__
void circ_shift(hipComplex	*input,
				hipComplex	*output,
				const int	i, // shift on x axis
				const int	j, // shift on y axis
				const uint	width,
				const uint	height,
				const uint	size)
{
	const uint	index = blockIdx.x * blockDim.x + threadIdx.x;
	int		index_x = 0;
	int		index_y = 0;
	int		shift_x = 0;
	int		shift_y = 0;
	// In ROI
	//while (index < size)
	{
		index_x = index % width;
		index_y = index / height;
		shift_x = index_x - i;
		shift_y = index_y - j;
		shift_x = (shift_x < 0) ? (width + shift_x) : shift_x;
		shift_y = (shift_y < 0) ? (height + shift_y) : shift_y;
		output[(width * shift_y) + shift_x] = input[index];
		//index += blockDim.x * gridDim.x;
	}
}

__global__
void circ_shift_float(float		*input,
					float		*output,
					const int	i, // shift on x axis
					const int	j, // shift on y axis
					const uint	width,
					const uint	height,
					const uint	size)
{
	const uint	index = blockIdx.x * blockDim.x + threadIdx.x;
	int		index_x = 0;
	int		index_y = 0;
	int		shift_x = 0;
	int		shift_y = 0;
	// In ROI
	//while (index < size)
	{
		index_x = index % width;
		index_y = index / height;
		shift_x = index_x - i;
		shift_y = index_y - j;
		shift_x = (shift_x < 0) ? (width + shift_x) : shift_x;
		shift_y = (shift_y < 0) ? (height + shift_y) : shift_y;
		output[(width * shift_y) + shift_x] = input[index];
		//index += blockDim.x * gridDim.x;
	}
}
