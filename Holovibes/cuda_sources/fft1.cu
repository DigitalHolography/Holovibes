/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include "fft1.cuh"
#include "transforms.cuh"
#include "unique_ptr.hh"
#include "common.cuh"
#include "cuda_memory.cuh"

#include <hipfft/hipfftXt.h>

using camera::FrameDescriptor;

void fft1_lens(hipComplex* lens,
               const uint lens_side_size,
               const uint frame_height,
               const uint frame_width,
               const float lambda,
               const float z,
               const float pixel_size,
               const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks =
        map_blocks_to_problem(lens_side_size * lens_side_size, threads);

    hipComplex* square_lens;
    // In anamorphic mode, the lens is initally a square, it's then cropped to
    // be the same dimension as the frame
    if (frame_height != frame_width)
        cudaXMalloc(&square_lens,
                    lens_side_size * lens_side_size * sizeof(hipComplex));
    else
        square_lens = lens;

    kernel_quadratic_lens<<<blocks, threads, 0, stream>>>(square_lens,
                                                          lens_side_size,
                                                          lambda,
                                                          z,
                                                          pixel_size);
    cudaCheckError();

    if (frame_height != frame_width)
    {
        // Data is contiguous for a horizontal frame so a simple memcpy with an
        // offset and a limited size works
        if (frame_width > frame_height)
            cudaXMemcpyAsync(lens,
                        square_lens +
                            ((lens_side_size - frame_height) / 2) * frame_width,
                        frame_width * frame_height * sizeof(hipComplex), hipMemcpyDeviceToDevice, stream);
        else
        {
            // For a vertical frame we need memcpy 2d to copy row by row, taking
            // the offset into account every time
            cudaSafeCall(hipMemcpy2DAsync(
                lens,                            // Destination (frame)
                frame_width * sizeof(hipComplex), // Destination width in byte
                square_lens +
                    ((lens_side_size - frame_width) / 2), // Source (lens)
                lens_side_size * sizeof(hipComplex), // Source width in byte
                frame_width * sizeof(hipComplex),    // Destination width in byte
                                                    // (yes it's redoundant)
                frame_height, // Destination height (not in byte)
                hipMemcpyDeviceToDevice,
                stream));
        }
        cudaXFree(square_lens);
    }
}

void fft_1(hipComplex* input,
           hipComplex* output,
           const uint batch_size,
           const hipComplex* lens,
           const hipfftHandle plan2D,
           const uint frame_resolution,
           const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_resolution, threads);

    // Apply lens on multiple frames.
    kernel_apply_lens<<<blocks, threads, 0, stream>>>(input,
                                                      output,
                                                      batch_size,
                                                      frame_resolution,
                                                      lens,
                                                      frame_resolution);

    // No sync needed between kernel call and cufft call
    cudaCheckError();
    // FFT

    cufftSafeCall(hipfftXtExec(plan2D, input, output, HIPFFT_FORWARD));
    // Same, no sync needed since everything is executed on the stream 0

    cudaCheckError();
}
