/* **************************************************************************** */
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "fft1.cuh"
#include "preprocessing.cuh"
#include "transforms.cuh"

void fft1_lens(hipComplex*			lens,
			const FrameDescriptor&	fd,
			const float				lambda,
			const float				z,
			const float				pixel_size,
			hipStream_t			stream)
{
  uint threads = 128;
  uint blocks = map_blocks_to_problem(fd.frame_res(), threads);

  kernel_quadratic_lens << <blocks, threads, 0, stream >> >(lens, fd, lambda, z, pixel_size);
}

void fft_1(hipComplex*			input,
		const hipComplex*		lens,
		const hipfftHandle		plan1D,
		const hipfftHandle		plan2D,
		const uint				frame_resolution,
		const uint				nframes,
		const uint				p,
		const uint				q,
		hipStream_t			stream)
{
	uint threads = get_max_threads_1d();
	uint blocks = map_blocks_to_problem(frame_resolution, threads);
	
	hipComplex* pframe = input + frame_resolution * p;

	hipfftExecC2C(plan1D, input, input, HIPFFT_FORWARD);

	// Apply lens on multiple frames.
	kernel_apply_lens <<<blocks, threads, 0, stream>>>(pframe, frame_resolution, lens, frame_resolution);
	hipStreamSynchronize(stream);
	// FFT
    hipfftExecC2C(plan2D, pframe, pframe, HIPFFT_FORWARD);
	if (p != q)
	{
		hipComplex *qframe = input + frame_resolution * q;
		kernel_apply_lens <<<blocks, threads, 0, stream>>>(qframe, frame_resolution, lens, frame_resolution);
		hipfftExecC2C(plan2D, qframe, qframe, HIPFFT_FORWARD);
	}

	hipStreamSynchronize(stream);
}
