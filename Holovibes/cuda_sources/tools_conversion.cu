#include "hip/hip_runtime.h"
/* ________________________________________________________ */
/*                  _                _  _                   */
/*    /\  /\  ___  | |  ___  __   __(_)| |__    ___  ___    */
/*   / /_/ / / _ \ | | / _ \ \ \ / /| || '_ \  / _ \/ __|   */
/*  / __  / | (_) || || (_) | \ V / | || |_) ||  __/\__ \   */
/*  \/ /_/   \___/ |_| \___/   \_/  |_||_.__/  \___||___/   */
/* ________________________________________________________ */

#include "tools_conversion.cuh"
#include "cuda_memory.cuh"
#include "map.cuh"

using camera::FrameDescriptor;

static constexpr ushort max_ushort_value = (1 << (sizeof(ushort) * 8)) - 1;
static constexpr ushort max_ushort_value_to_float =
    static_cast<float>(max_ushort_value);

/* Kernel function wrapped by complex_to_modulus. */
static __global__ void kernel_complex_to_modulus_pacc(float* output,
                                                      const hipComplex* input,
                                                      const ushort pmin,
                                                      const ushort pmax,
                                                      const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        // We use a local variable so the global memory isn't read or written
        // everytime. Only written once at the end.
        float val = 0.0f;
        for (int i = pmin; i <= pmax; i++)
        {
            const hipComplex* current_p_frame = input + i * size;

            val += hypotf(current_p_frame[index].x, current_p_frame[index].y);
        }

        output[index] = val / (pmax - pmin + 1);
    }
}

void complex_to_modulus(float* output,
                        const hipComplex* input,
                        const ushort pmin,
                        const ushort pmax,
                        const uint size,
                        const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_complex_to_modulus_pacc<<<blocks, threads, 0, stream>>>(output,
                                                                   input,
                                                                   pmin,
                                                                   pmax,
                                                                   size);
    // No sync needed since everything is run on stream 0
    cudaCheckError();
}

/* Kernel function wrapped in complex_to_squared_modulus. */
static __global__ void kernel_complex_to_squared_modulus(float* output,
                                                         const hipComplex* input,
                                                         const ushort pmin,
                                                         const ushort pmax,
                                                         const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        // We use a local variable so the global memory isn't read or written
        // everytime. Only written once at the end.
        float val = 0.0f;
        for (int i = pmin; i <= pmax; i++)
        {
            const hipComplex* current_p_frame = input + i * size;
            // square of the square root of the sum of the squares of x and y
            float tmp =
                hypotf(current_p_frame[index].x, current_p_frame[index].y);
            val += tmp * tmp;
        }
        output[index] = val / (pmax - pmin + 1);
    }
}

static __device__ hipComplex device_float_to_complex(const float input)
{
    return hipComplex{input, 0.0f};
}

template <typename OTYPE, typename ITYPE, typename FUNC>
static __global__ void
kernel_input_queue_to_input_buffer(OTYPE* output,
                                   const ITYPE* const input,
                                   FUNC convert,
                                   const uint frame_res,
                                   const int batch_size,
                                   const uint current_queue_index,
                                   const uint queue_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < frame_res)
    {
        uint frame_copied = 0;
        // j swip through the queue from it's start index to either the end of
        // the queue or all of the batch
        for (int j = current_queue_index;
             j < queue_size && frame_copied < batch_size;
             ++j, ++frame_copied)
            output[index + frame_copied * frame_res] =
                device_float_to_complex(convert(input[index + j * frame_res]));

        // Copy might reach end of the queue so we copy the missing frames
        for (int j = 0; frame_copied < batch_size; ++frame_copied, ++j)
            output[index + frame_copied * frame_res] =
                device_float_to_complex(convert(input[index + j * frame_res]));
    }
}

void input_queue_to_input_buffer(void* output,
                                 void* input,
                                 const uint frame_res,
                                 const int batch_size,
                                 const uint current_queue_index,
                                 const uint queue_size,
                                 const uint depth)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_res, threads);

    /* Best way we found to pass function to kernels
     *  We can't declare the lambda outside this function for some reason
     * To pass lambda like that, we need to add the --extended-lambda  flag
     */
    static const auto convert_8_bit = [] __device__(const uchar input_pixel) {
        return static_cast<float>(input_pixel * 257);
    };
    static const auto convert_16_bit = [] __device__(const ushort input_pixel) {
        return static_cast<float>(input_pixel);
    };
    static const auto convert_32_bit = [] __device__(const float input_pixel) {
        return input_pixel;
    };

    switch (depth)
    {
    case 1:
        kernel_input_queue_to_input_buffer<hipComplex, uchar>
            <<<blocks, threads>>>(reinterpret_cast<hipComplex*>(output),
                                  reinterpret_cast<uchar*>(input),
                                  convert_8_bit,
                                  frame_res,
                                  batch_size,
                                  current_queue_index,
                                  queue_size);
        break;
    case 2:
        kernel_input_queue_to_input_buffer<hipComplex, ushort>
            <<<blocks, threads>>>(reinterpret_cast<hipComplex*>(output),
                                  reinterpret_cast<ushort*>(input),
                                  convert_16_bit,
                                  frame_res,
                                  batch_size,
                                  current_queue_index,
                                  queue_size);
        break;
    case 4:
        kernel_input_queue_to_input_buffer<hipComplex, float>
            <<<blocks, threads>>>(reinterpret_cast<hipComplex*>(output),
                                  reinterpret_cast<float*>(input),
                                  convert_32_bit,
                                  frame_res,
                                  batch_size,
                                  current_queue_index,
                                  queue_size);
        break;
    }
    // No sync needed since next call (fft1 is called on default main stream
    // (0))
    cudaCheckError();
}

void complex_to_squared_modulus(float* output,
                                const hipComplex* input,
                                const ushort pmin,
                                const ushort pmax,
                                const uint size,
                                const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_complex_to_squared_modulus<<<blocks, threads, 0, stream>>>(output,
                                                                      input,
                                                                      pmin,
                                                                      pmax,
                                                                      size);
    hipDeviceSynchronize();
    cudaCheckError();
}

/* Kernel function wrapped in complex_to_argument. */
static __global__ void kernel_complex_to_argument(float* output,
                                                  const hipComplex* input,
                                                  const ushort pmin,
                                                  const ushort pmax,
                                                  const uint size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        // We use a local variable so the global memory isn't read or written
        // everytime. Only written once at the end.
        float val = 0.0f;
        for (int i = pmin; i <= pmax; i++)
        {
            const hipComplex* current_p_frame = input + i * size;
            // Computes the arc tangent of y / x
            // We use std::atan2 in order to obtain results in [-pi; pi].
            val +=
                std::atan2(current_p_frame[index].y, current_p_frame[index].x);
        }
        output[index] = val / (pmax - pmin + 1);
    }
}

void complex_to_argument(float* output,
                         const hipComplex* input,
                         const ushort pmin,
                         const ushort pmax,
                         const uint size,
                         const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_complex_to_argument<<<blocks, threads, 0, stream>>>(output,
                                                               input,
                                                               pmin,
                                                               pmax,
                                                               size);
    hipDeviceSynchronize();
    cudaCheckError();
}

/* Find the minimum and the maximum of a floating-point array.
 *
 * The minimum and maximum can't be computed directly, because blocks
 * cannot communicate. Hence we compute local minima and maxima and
 * put them in two arrays.
 *
 * \param Size Number of threads in a block for this kernel.
 * Also, it's the size of min and max.
 * \param min Array of Size floats, which will contain local minima.
 * \param max Array of Size floats, which will contain local maxima.
 */
template <uint Size>
static __global__ void
kernel_minmax(const float* data, const size_t size, float* min, float* max)
{
    __shared__ float local_min[Size];
    __shared__ float local_max[Size];

    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index > size)
        return;
    local_min[threadIdx.x] = data[index];
    local_max[threadIdx.x] = data[index];

    __syncthreads();

    if (threadIdx.x == 0)
    {
        /* Accumulate the results of the neighbors, computing min-max values,
         * and store them in the first element of local arrays. */
        for (auto i = 1; i < Size; ++i)
        {
            if (local_min[i] < local_min[0])
                local_min[0] = local_min[i];
            if (local_max[i] > local_max[0])
                local_max[0] = local_max[i];
        }
        min[blockIdx.x] = local_min[0];
        max[blockIdx.x] = local_max[0];
    }
}

void rescale_float(const float* input,
                   float* output,
                   const uint size,
                   const hipStream_t stream)
{
    const uint threads = THREADS_128;
    const uint blocks = map_blocks_to_problem(size, threads);

    // TODO : See if gpu_postprocess_frame could be used directly.
    cudaXMemcpy(output, input, sizeof(float) * size, hipMemcpyDeviceToDevice);

    // Computing minimum and maximum values, in order to rescale properly.
    float* gpu_local_min;
    float* gpu_local_max;
    const uint float_blocks = sizeof(float) * blocks;
    cudaXMalloc(&gpu_local_min, float_blocks);
    cudaXMalloc(&gpu_local_max, float_blocks);

    /* We have to hardcode the template parameter, unfortunately.
     * It must be equal to the number of threads per block. */
    kernel_minmax<128>
        <<<blocks, threads, threads << 1, stream>>>(output,
                                                    size,
                                                    gpu_local_min,
                                                    gpu_local_max);
    hipDeviceSynchronize();
    cudaCheckError();

    float* cpu_local_min = new float[blocks];
    float* cpu_local_max = new float[blocks];
    cudaXMemcpy(cpu_local_min,
                gpu_local_min,
                float_blocks,
                hipMemcpyDeviceToHost);
    cudaXMemcpy(cpu_local_max,
                gpu_local_max,
                float_blocks,
                hipMemcpyDeviceToHost);

    constexpr float max_intensity = max_ushort_value_to_float;
    const float min_element =
        *(std::min_element(cpu_local_min, cpu_local_min + threads));
    const float max_element =
        *(std::max_element(cpu_local_max, cpu_local_max + threads));
    const auto lambda = [min_element, max_element, max_intensity] __device__(
                            const float in) -> float {
        return (in + fabsf(min_element)) * max_intensity /
               (fabsf(max_element) + fabsf(min_element));
    };

    map_generic<float>(output, output, size, lambda, stream);
    hipDeviceSynchronize();
    cudaCheckError();
    delete[] cpu_local_max;
    delete[] cpu_local_min;
    cudaXFree(gpu_local_min);
    cudaXFree(gpu_local_max);
}

void rescale_float_unwrap2d(float* input,
                            float* output,
                            float* cpu_buffer,
                            uint frame_res,
                            const hipStream_t stream)
{
    float min = 0;
    float max = 0;
    const uint threads = THREADS_128;
    const uint blocks = map_blocks_to_problem(frame_res, threads);
    uint float_frame_res = sizeof(float) * frame_res;
    cudaXMemcpy(cpu_buffer, input, float_frame_res, hipMemcpyDeviceToHost);
    auto minmax = std::minmax_element(cpu_buffer, cpu_buffer + frame_res);
    min = *minmax.first;
    max = *minmax.second;

    const auto lambda = [min, max] __device__(const float in) -> float {
        if (min < 0.f)
            return (in + fabs(min)) / (fabs(min) + max) *
                   max_ushort_value_to_float;
        else
            return (in - min) / (max - min) * max_ushort_value_to_float;
    };
    map_generic(input, output, frame_res, lambda, stream);
}

void endianness_conversion(const ushort* input,
                           ushort* output,
                           const uint batch_size,
                           const uint size,
                           const hipStream_t stream)
{
    static const auto lambda = [] __device__(const ushort in) -> ushort {
        return (in << 8) | (in >> 8);
    };
    map_generic(input, output, size * batch_size, lambda, stream);
}

/*
 * The input data shall be restricted first to the range [0; 2^16 - 1],
 * by forcing every negative  value to 0 and every positive one
 * greater than 2^16 - 1 to 2^16 - 1.
 * Then it is truncated to unsigned short data type.
 */
static __device__ ushort device_float_to_ushort(const float input,
                                                const uint shift = 0)
{
    if (input <= 0.0f) // Negative float
        return 0;
    // Cast in uint is needed to avoid overflow
    else if ((static_cast<uint>(input) << shift) > max_ushort_value_to_float)
        return max_ushort_value;
    else
        return static_cast<ushort>(input) << shift;
}

void complex_to_uint(const hipComplex* const input,
                     uint* const output,
                     const uint size,
                     const uint shift,
                     hipStream_t stream = 0)
{
    const auto lambda_complex_to_ushort =
        [shift] __device__(const hipComplex in) -> uint {
        /* hipComplex needs to be casted to a uint
        ** Each part (real & imaginary) are casted from float to ushort to then
        *be assembled into a uint
        ** The real part is on the left side of the uint, imaginary is on the
        *right one
        ** Here x & y are of type uint to avoid the overflow when shifting
        */
        constexpr uint size_half_uint = sizeof(uint) * 8 / 2;
        const uint x = device_float_to_ushort(in.x);
        const uint y = device_float_to_ushort(in.y);

        return ((x << size_half_uint) | y) << shift;
    };
    map_generic(input, output, size, lambda_complex_to_ushort, stream);
}

void float_to_ushort(const float* const input,
                     ushort* const output,
                     const uint size,
                     const uint shift,
                     hipStream_t stream)
{
    const auto lambda = [shift] __device__(const float in) -> ushort {
        return device_float_to_ushort(in, shift);
    };
    map_generic(input, output, size, lambda, stream);
}

void ushort_to_shifted_ushort(const ushort* const input,
                              ushort* const output,
                              const uint size,
                              const uint shift,
                              hipStream_t stream = 0)
{
    const auto lambda_shift_ushort =
        [shift] __device__(const ushort in) -> ushort { return in << shift; };
    map_generic(input, output, size, lambda_shift_ushort, stream);
}

void ushort_to_uchar(const ushort* input,
                     uchar* output,
                     const uint size,
                     const hipStream_t stream)
{
    static const auto lambda = [] __device__(const ushort in) -> uchar {
        return in >> (sizeof(uchar) * 8);
    };
    map_generic(input, output, size, lambda, stream);
}

void uchar_to_shifted_uchar(const uchar* input,
                            uchar* output,
                            const uint size,
                            const uint shift,
                            hipStream_t stream = 0)
{
    const auto lambda_shift_uchar =
        [shift] __device__(const uchar in) -> uchar { return in << shift; };
    map_generic(static_cast<const uchar* const>(input),
                static_cast<uchar* const>(output),
                size,
                lambda_shift_uchar,
                stream);
}

__global__ void kernel_accumulate_images(const float* input,
                                         float* output,
                                         const size_t end,
                                         const size_t max_elmt,
                                         const size_t nb_elmt,
                                         const size_t nb_pixel)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    long int pos = end; // end is excluded

    if (index < nb_pixel)
    {
        float val = 0;
        for (size_t i = 0; i < nb_elmt; i++)
        {
            // get last index when pos is out of range
            // reminder: the given input is from ciruclar queue
            pos--;
            if (pos < 0)
                pos = max_elmt - 1;

            val += input[index + pos * nb_pixel];
        }
        output[index] = val / nb_elmt;
    }
}

/*! \brief Kernel function wrapped in accumulate_images, making
** the call easier
**/
void accumulate_images(const float* input,
                       float* output,
                       const size_t start,
                       const size_t max_elmt,
                       const size_t nb_elmt,
                       const size_t nb_pixel,
                       const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(nb_pixel, threads);
    kernel_accumulate_images<<<blocks, threads, 0, stream>>>(input,
                                                             output,
                                                             start,
                                                             max_elmt,
                                                             nb_elmt,
                                                             nb_pixel);
    cudaCheckError();
}

void normalize_complex(hipComplex* image,
                       const uint size,
                       const hipStream_t stream)
{
    static const auto lambda = [] __device__(hipComplex in) -> hipComplex {
        in.x = (in.x + 1.0f) * (max_ushort_value_to_float / 2.0f);
        in.y = (in.y + 1.0f) * (max_ushort_value_to_float / 2.0f);
        return in;
    };
    map_generic(image, image, size, lambda, stream);
}

void convert_frame_for_display(const void* input,
                               void* output,
                               const uint size,
                               const uint depth,
                               const ushort shift)
{
    if (depth == 8)
    {
        // In depth 8 the output is encoded onto a uint (for the lens)
        complex_to_uint(static_cast<const hipComplex* const>(input),
                        static_cast<uint* const>(output),
                        size,
                        shift);
    }
    else if (depth == 4)
    {
        float_to_ushort(static_cast<const float* const>(input),
                        static_cast<ushort* const>(output),
                        size,
                        shift);
    }
    else if (depth == 2)
    {
        ushort_to_shifted_ushort(static_cast<const ushort* const>(input),
                                 static_cast<ushort* const>(output),
                                 size,
                                 shift);
    }
    else if (depth == 1)
    {
        uchar_to_shifted_uchar(static_cast<const uchar* const>(input),
                               static_cast<uchar* const>(output),
                               size,
                               shift);
    }
}