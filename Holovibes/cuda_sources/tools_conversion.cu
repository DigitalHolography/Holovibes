#include "hip/hip_runtime.h"
/* **************************************************************************** */  
/*                       ,,                     ,,  ,,                          */
/* `7MMF'  `7MMF'       `7MM       `7MMF'   `7MF'db *MM                         */
/*   MM      MM           MM         `MA     ,V      MM                         */
/*   MM      MM  ,pW"Wq.  MM  ,pW"Wq. VM:   ,V `7MM  MM,dMMb.   .gP"Ya  ,pP"Ybd */
/*   MMmmmmmmMM 6W'   `Wb MM 6W'   `Wb MM.  M'   MM  MM    `Mb ,M'   Yb 8I   `" */
/*   MM      MM 8M     M8 MM 8M     M8 `MM A'    MM  MM     M8 8M"""""" `YMMMa. */
/*   MM      MM YA.   ,A9 MM YA.   ,A9  :MM;     MM  MM.   ,M9 YM.    , L.   I8 */
/* .JMML.  .JMML.`Ybmd9'.JMML.`Ybmd9'    VF    .JMML.P^YbmdP'   `Mbmmd' M9mmmP' */
/*                                                                              */
/* **************************************************************************** */

#include "tools_conversion.cuh"
#include <thrust/device_vector.h>

__global__
void img8_to_complex(hipComplex		*output,
					const uchar		*input,
					const uint		size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		// Image rescaling on 2^16 colors (65535 / 255 = 257)
		const float val = static_cast<float>(input[index] * 257);
		output[index].x = val;
		output[index].y = 0;
	}
}

__global__
void img16_to_complex(hipComplex		*output,
					const ushort	*input,
					const uint		size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		float val = static_cast<float>(input[index]);
		output[index].x = val;
		output[index].y = 0;
	}
}

__global__
void float_to_complex(hipComplex	*output,
					const float	*input,
					const uint	size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		float val = input[index];
		output[index].x = val;
		output[index].y = 0;
	}
}

/* Kernel function wrapped by complex_to_modulus. */
static __global__
void kernel_complex_to_modulus(const hipComplex	*input,
							float				*output,
							const uint			size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		output[index] = hypotf(input[index].x, input[index].y);
	}
}

void complex_to_modulus(const hipComplex	*input,
						float			*output,
						const uint		size,
						hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_complex_to_modulus << <blocks, threads, 0, stream >> >(input, output, size);
}

/* Kernel function wrapped in complex_to_squared_modulus. */
static __global__
void kernel_complex_to_squared_modulus(const hipComplex	*input,
									float				*output,
									const uint			size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		output[index] = hypotf(input[index].x, input[index].y);
		output[index] *= output[index];
	}
}

void complex_to_squared_modulus(const hipComplex	*input,
								float			*output,
								const uint		size,
								hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_complex_to_squared_modulus << <blocks, threads, 0, stream >> >(input, output, size);
}

/* Kernel function wrapped in complex_to_argument. */
static __global__
void kernel_complex_to_argument(const hipComplex	*input,
								float			*output,
								const uint		size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		output[index] = (atanf(input[index].y / input[index].x) + M_PI_2);
	}
}

void complex_to_argument(const hipComplex	*input,
						float			*output,
						const uint		size,
						hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_complex_to_argument << <blocks, threads, 0, stream >> >(input, output, size);
}

/* Find the minimum and the maximum of a floating-point array.
 *
 * The minimum and maximum can't be computed directly, because blocks
 * cannot communicate. Hence we compute local minima and maxima and
 * put them in two arrays.
 *
 * \param Size Number of threads in a block for this kernel.
 * Also, it's the size of min and max.
 * \param min Array of Size floats, which will contain local minima.
 * \param max Array of Size floats, which will contain local maxima.
 */
template <uint Size>
static __global__
void kernel_minmax(const float	*data,
				const size_t	size,
				float			*min,
				float			*max)
{
	__shared__ float local_min[Size];
	__shared__ float local_max[Size];

	const uint index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index > size)
		return;
	local_min[threadIdx.x] = data[index];
	local_max[threadIdx.x] = data[index];

	__syncthreads();

	if (threadIdx.x == 0)
	{
		/* Accumulate the results of the neighbors, computing min-max values,
		 * and store them in the first element of local arrays. */
		for (auto i = 1; i < Size; ++i)
		{
			if (local_min[i] < local_min[0])
				local_min[0] = local_min[i];
			if (local_max[i] > local_max[0])
				local_max[0] = local_max[i];
		}
		min[blockIdx.x] = local_min[0];
		max[blockIdx.x] = local_max[0];
	}
}

template <typename T>
static __global__
void kernel_rescale(T				*data,
					const size_t	size,
					const T			min,
					const T			max,
					const T			new_max)
{
	const uint index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index > size)
		return;

	data[index] = (data[index] + fabsf(min)) * new_max / (fabsf(max) + fabsf(min));
}

void rescale_float(const float	*input,
				float			*output,
				const uint		size,
				hipStream_t	stream)
{
	const uint threads = THREADS_128;
	const uint blocks = map_blocks_to_problem(size, threads);

	// TODO : See if gpu_float_buffer_ could be used directly.
	hipMemcpy(output, input, sizeof(float) * size, hipMemcpyDeviceToDevice);

	// Computing minimum and maximum values, in order to rescale properly.
	float* gpu_local_min;
	float* gpu_local_max;
	const uint float_blocks = sizeof(float) * blocks;
	if (hipMalloc(&gpu_local_min, float_blocks) != hipSuccess)
		return;
	if (hipMalloc(&gpu_local_max, float_blocks) != hipSuccess)
	{
		hipFree(gpu_local_min);
		return;
	}

	/* We have to hardcode the template parameter, unfortunately.
	 * It must be equal to the number of threads per block. */
	kernel_minmax <128> << <blocks, threads, threads << 1, stream >> > (output, size, gpu_local_min, gpu_local_max);

	float	*cpu_local_min = new float[blocks];
	float	*cpu_local_max = new float[blocks];
	hipMemcpy(cpu_local_min, gpu_local_min, float_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_local_max, gpu_local_max, float_blocks, hipMemcpyDeviceToHost);

	const float max_intensity = 65535.f;
	kernel_rescale << <blocks, threads, 0, stream >> >(	output,
														size,
														*(std::min_element(cpu_local_min, cpu_local_min + threads)),
														*(std::max_element(cpu_local_max, cpu_local_max + threads)),
														max_intensity);
	delete[] cpu_local_max;
	delete[] cpu_local_min;
	hipFree(gpu_local_min);
	hipFree(gpu_local_max);
}

void rescale_float_unwrap2d(float			*input,
							float			*output,
							float			*cpu_buffer,
							uint			frame_res,
							hipStream_t	stream)
{
	float		min = 0;
	float		max = 0;
	const uint	threads = THREADS_128;
	const uint	blocks = map_blocks_to_problem(frame_res, threads);
	uint float_frame_res = sizeof(float)* frame_res;
	hipMemcpy(cpu_buffer, input, float_frame_res, hipMemcpyDeviceToHost);
	auto minmax = std::minmax_element(cpu_buffer, cpu_buffer + frame_res);
	min = *minmax.first;
	max = *minmax.second;

	hipMemcpy(output, input, float_frame_res, hipMemcpyDeviceToDevice);

	kernel_normalize_images << < blocks, threads, 0, stream >> > (
		output,
		max,
		min,
		frame_res);
}

__global__
void kernel_rescale_argument(float		*input,
							const uint	size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		input[index] *= 65535.0f / M_PI;
	}
}

void rescale_argument(float			*input,
					const uint		frame_res,
					hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(frame_res, threads);

	kernel_rescale_argument << <blocks, threads, 0, stream >> >(input, frame_res);
}

/*! \brief Kernel function wrapped in endianness_conversion, making
 ** the call easier
 **/
static __global__
void kernel_endianness_conversion(const ushort	*input,
								ushort			*output,
								const uint		size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		output[index] = (input[index] << 8) | (input[index] >> 8);
	}
}

void endianness_conversion(const ushort	*input,
						ushort			*output,
						const uint		size,
						hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_endianness_conversion << <blocks, threads, 0, stream >> >(input, output, size);
}



__global__
static void kernel_composite(hipComplex			*input,
							float				*output,
							const uint			frame_res,
							ushort				pmin_r,
							ushort				pmax_r,
							ushort				pmin_g,
							ushort				pmax_g,
							ushort				pmin_b,
							ushort				pmax_b)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	ushort pmin[] = { pmin_r, pmin_g, pmin_b };
	ushort pmax[] = { pmax_r, pmax_g, pmax_b };
	if (id < frame_res)
	{
		for (int i = 0; i < 3; i++)
		{
			float res = 0;
			for (ushort p = pmin[i]; p <= pmax[i]; p++)
			{
				hipComplex *current_pframe = input + (frame_res * p);
				res += hypotf(current_pframe[id].x, current_pframe[id].y);
			}
			output[id * 3 + i] = res / (pmax[i] - pmin[i] + 1);
		}
	}
}

// ! Splits the image by nb_lines blocks and sums them
__global__
static void kernel_sum_one_line(float			*input,
							const uint			frame_res,
							const uchar			pixel_depth,
							const uint			nb_lines,
							const uint			line_size,
							float				*sums_per_line)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < pixel_depth * nb_lines)
	{
		uchar offset = id % pixel_depth;
		ushort line = id / pixel_depth;
		uint index_begin = line_size * line;
		uint index_end = line_size * (line + 1);
		if (index_end > frame_res)
			index_end = frame_res;
		float sum = 0;
		while(index_begin < index_end)
			sum += input[pixel_depth * (index_begin++) + offset];
		sums_per_line[id * pixel_depth + offset] = sum;
	}
}

// ! sums an array of size floats and put the result divided by nb_elements in *output
__global__
static void kernel_average_float_array(float		*input,
								uint				size,
								uint				nb_elements,
								uint				offset_per_pixel,
								float				*output)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < offset_per_pixel)
	{
		input += id;
		float res = 0;
		while (size--)
		{
			res += *input;
			input += offset_per_pixel;
		}
		res /= static_cast<float>(nb_elements);
		output[id] = res;
	}
}

__global__
static void kernel_divide_by_weight(float		*input,
							float				weight_r,
							float				weight_g,
							float				weight_b)
{
	input[0] /= weight_r;
	input[1] /= weight_g;
	input[2] /= weight_b;
}
__global__
static void kernel_normalize_array(float			*input,
								uint				nb_pixels,
								uint				pixel_depth,
								float				*averages)
{
	const uint	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < pixel_depth * nb_pixels)
		input[id] /= averages[id % 3];
}

void composite(hipComplex	*input,
			float			*output,
			const uint		frame_res,
			bool			normalize,
			ushort			pmin_r,
			ushort			pmax_r,
			float			weight_r,
			ushort			pmin_g,
			ushort			pmax_g,
			float			weight_g,
			ushort			pmin_b,
			ushort			pmax_b,
			float			weight_b)
{
	const uint threads = get_max_threads_1d();
	uint blocks = map_blocks_to_problem(frame_res, threads);

	kernel_composite << <blocks, threads, 0, 0 >> > (input,
		output,
		frame_res,
		pmin_r,
		pmax_r,
		pmin_g,
		pmax_g,
		pmin_b,
		pmax_b);
	if (normalize)
	{
		const ushort line_size = 1024;
		const ushort lines = frame_res / line_size + 1;
		float *averages = nullptr;
		float *sums_per_line = nullptr;
		const uchar pixel_depth = 3;
		hipMalloc(&averages, sizeof(float) * pixel_depth);
		hipMalloc(&sums_per_line, sizeof(float) * lines);
		blocks = map_blocks_to_problem(lines * pixel_depth, threads);
		kernel_sum_one_line << <blocks, threads, 0, 0 >> > (output,
			frame_res,
			pixel_depth,
			lines,
			line_size,
			sums_per_line);
		blocks = map_blocks_to_problem(pixel_depth, threads);
		kernel_average_float_array << <blocks, threads, 0, 0 >> > (output,
			lines,
			frame_res,
			pixel_depth,
			averages);
		blocks = map_blocks_to_problem(frame_res * pixel_depth, threads);
		kernel_divide_by_weight << <1, 1, 0, 0 >> > (averages, weight_r, weight_g, weight_b);
		kernel_normalize_array << <blocks, threads, 0, 0 >> > (output,
			frame_res,
			pixel_depth,
			averages);
		hipFree(averages);
		hipFree(sums_per_line);
	}
}

/*! \brief Kernel function wrapped in float_to_ushort, making
 ** the call easier
 **/
static __global__
void kernel_float_to_ushort(const float	*input,
							void		*output,
							const uint	size,
							const float	depth)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		if (depth != 1.f)
		{
			ushort *out = reinterpret_cast<ushort *>(output);
			if (input[index] > 65535.f)
				out[index] = 65535;
			else if (input[index] < 0.f)
				out[index] = 0;
			else
				out[index] = static_cast<ushort>(input[index]);
		}
		else
		{
			uchar *out = reinterpret_cast<uchar *>(output);
			out[index] = static_cast<uchar>(input[index]);
		}
	}
}

void float_to_ushort(const float	*input,
					void			*output,
					const uint		size,
					const float		depth,
					hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_float_to_ushort << <blocks, threads, 0, stream >> >(input, output, size, depth);
}

static __global__
void kernel_complex_to_ushort(const hipComplex	*input,
							uint				*output,
							const uint			size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		ushort x = 0;
		ushort y = 0;
		if (input[index].x > 65535.0f)
			x = 65535;
		else if (input[index].x >= 1.0f)
			x = static_cast<ushort>(input[index].x * input[index].x);

		if (input[index].y > 65535.0f)
			y = 65535;
		else if (input[index].y >= 0.0f)
			y = static_cast<ushort>(input[index].y * input[index].x);

		auto& res = output[index];
		res ^= res;
		res = x << 16;
		res += y;
	}
}

void complex_to_ushort(const hipComplex	*input,
					uint				*output,
					const uint			size,
					hipStream_t		stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(size, threads);

	kernel_complex_to_ushort << <blocks, threads, 0 >> >(input, output, size);
}

/*! \brief Memcpy of a complex sized frame into another buffer */
void complex_to_complex(const hipComplex	*input,
						ushort*			output,
						const uint		size,
						hipStream_t	stream)
{
	hipMemcpy(output, input, size, hipMemcpyDeviceToDevice);
}

__global__
void kernel_buffer_size_conversion(char			*real_buffer,
								const char		*buffer,
								const size_t	frame_desc_width,
								const size_t	frame_desc_height,
								const size_t	real_frame_desc_width,
								const size_t	area)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < area)
	{
		uint x = index % real_frame_desc_width;
		uint y = index / real_frame_desc_width;
		if (y < frame_desc_height && x < frame_desc_width)
			real_buffer[index] = buffer[y * frame_desc_width + x];
	}
}

void buffer_size_conversion(char*					real_buffer,
							const char*				buffer,
							const FrameDescriptor	real_frame_desc,
							const FrameDescriptor	frame_desc)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem((frame_desc.height * real_frame_desc.width * static_cast<size_t>(frame_desc.depth)), threads);

	kernel_buffer_size_conversion << <blocks, threads, 0 >> >(	real_buffer,
																buffer,
																frame_desc.width * static_cast<uint>(frame_desc.depth),
																frame_desc.height * static_cast<uint>(frame_desc.depth),
																real_frame_desc.width * static_cast<uint>(frame_desc.depth),
																frame_desc.height * real_frame_desc.width * static_cast<size_t>(frame_desc.depth));
}

__global__
void kernel_accumulate_images(const float	*input,
							float			*output,
							const size_t	start,
							const size_t	max_elmt,
							const size_t	nb_elmt,
							const size_t	nb_pixel)
{
	const uint	index = blockIdx.x * blockDim.x + threadIdx.x;
	size_t	i = 0;
	int		pos = start;

	if (index < nb_pixel)
	{
		output[index] = 0;
		while (i++ < nb_elmt)
		{
			output[index] += input[index + pos * nb_pixel];
			if (--pos < 0)
				pos = max_elmt - 1;
		}
		output[index] /= nb_elmt;
	}
}

/*! \brief Kernel function wrapped in accumulate_images, making
** the call easier
**/
void accumulate_images(const float	*input,
					float			*output,
					const size_t	start,
					const size_t	max_elmt,
					const size_t	nb_elmt,
					const size_t	nb_pixel,
					hipStream_t	stream)
{
	const uint threads = get_max_threads_1d();
	const uint blocks = map_blocks_to_problem(nb_pixel, threads);

	kernel_accumulate_images << <blocks, threads, 0, stream >> >(input, output, start, max_elmt, nb_elmt, nb_pixel);
}

__global__
void kernel_normalize_images(float		*image,
							const float	max,
							const float	min,
							const uint	size)
{
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		if (min < 0.f)
			image[index] = (image[index] + fabs(min)) / (fabs(min) + max) * 65535.0f;
		else
			image[index] = (image[index] - min) / (max - min) * 65535.0f;
	}
}
