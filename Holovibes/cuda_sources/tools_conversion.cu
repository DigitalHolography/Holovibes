#include "hip/hip_runtime.h"
#include "tools_conversion.cuh"
#include "cuda_memory.cuh"
#include "map.cuh"

using camera::FrameDescriptor;

static constexpr ushort max_ushort_value = (1 << (sizeof(ushort) * 8)) - 1;
static constexpr ushort max_ushort_value_to_float = static_cast<float>(max_ushort_value);

/* Kernel function wrapped by complex_to_modulus. */
static __global__ void kernel_complex_to_modulus_pacc(
    float* output, const hipComplex* input, const ushort pmin, const ushort pmax, const size_t size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        // We use a local variable so the global memory isn't read or written
        // everytime. Only written once at the end.
        float val = 0.0f;
        for (int i = pmin; i <= pmax; i++)
        {
            const hipComplex* current_p_frame = input + i * size;

            val += hypotf(current_p_frame[index].x, current_p_frame[index].y);
        }

        output[index] = val / (pmax - pmin + 1);
    }
}

void complex_to_modulus(float* output,
                        const hipComplex* input,
                        const ushort pmin,
                        const ushort pmax,
                        const size_t size,
                        const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_complex_to_modulus_pacc<<<blocks, threads, 0, stream>>>(output, input, pmin, pmax, size);
    // No sync needed since everything is run on stream 0
    cudaCheckError();
}

/* Kernel function wrapped in complex_to_squared_modulus. */
static __global__ void kernel_complex_to_squared_modulus(
    float* output, const hipComplex* input, const ushort pmin, const ushort pmax, const size_t size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        // We use a local variable so the global memory isn't read or written
        // everytime. Only written once at the end.
        float val = 0.0f;
        for (int i = pmin; i <= pmax; i++)
        {
            const hipComplex* current_p_frame = input + i * size;
            // square of the square root of the sum of the squares of x and y
            float tmp = hypotf(current_p_frame[index].x, current_p_frame[index].y);
            val += tmp * tmp;
        }
        output[index] = val / (pmax - pmin + 1);
    }
}

static __device__ hipComplex device_float_to_complex(const float input) { return hipComplex{input, 0.0f}; }

template <typename OTYPE, typename ITYPE, typename FUNC>
static __global__ void kernel_input_queue_to_input_buffer(
    OTYPE* output, const ITYPE* const input, FUNC convert, const uint frame_res, const int batch_size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < frame_res)
    {
        for (int i = 0; i < batch_size; i++)
            output[index + i * frame_res] = device_float_to_complex(convert(input[index + i * frame_res]));
    }
}

void input_queue_to_input_buffer(void* const output,
                                 const void* const input,
                                 const size_t frame_res,
                                 const int batch_size,
                                 const uint depth,
                                 const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(frame_res, threads);

    /* Best way we found to pass function to kernels
     *  We can't declare the lambda outside this function for some reason
     * To pass lambda like that, we need to add the --extended-lambda  flag
     */
    static const auto convert_8_bit = [] __device__(const uchar input_pixel)
    {
        // max uchar value is 255, multiplied by 257 you have 65535 which is max
        // ushort
        return static_cast<float>(input_pixel) * 257;
    };
    static const auto convert_16_bit = [] __device__(const ushort input_pixel)
    { return static_cast<float>(input_pixel); };
    static const auto convert_32_bit = [] __device__(const float input_pixel) { return input_pixel; };

    switch (depth)
    {
    case 1:
        kernel_input_queue_to_input_buffer<hipComplex, uchar>
            <<<blocks, threads, 0, stream>>>(reinterpret_cast<hipComplex* const>(output),
                                             reinterpret_cast<const uchar* const>(input),
                                             convert_8_bit,
                                             frame_res,
                                             batch_size);
        break;
    case 2:
        kernel_input_queue_to_input_buffer<hipComplex, ushort>
            <<<blocks, threads, 0, stream>>>(reinterpret_cast<hipComplex* const>(output),
                                             reinterpret_cast<const ushort* const>(input),
                                             convert_16_bit,
                                             frame_res,
                                             batch_size);
        break;
    case 4:
        kernel_input_queue_to_input_buffer<hipComplex, float>
            <<<blocks, threads, 0, stream>>>(reinterpret_cast<hipComplex* const>(output),
                                             reinterpret_cast<const float* const>(input),
                                             convert_32_bit,
                                             frame_res,
                                             batch_size);
        break;
    }
    cudaCheckError();
}

void complex_to_squared_modulus(float* output,
                                const hipComplex* input,
                                const ushort pmin,
                                const ushort pmax,
                                const size_t size,
                                const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_complex_to_squared_modulus<<<blocks, threads, 0, stream>>>(output, input, pmin, pmax, size);
    cudaCheckError();
}

/* Kernel function wrapped in complex_to_argument. */
static __global__ void kernel_complex_to_argument(
    float* output, const hipComplex* input, const ushort pmin, const ushort pmax, const size_t size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        // We use a local variable so the global memory isn't read or written
        // everytime. Only written once at the end.
        float val = 0.0f;
        for (int i = pmin; i <= pmax; i++)
        {
            const hipComplex* current_p_frame = input + i * size;
            // Computes the arc tangent of y / x
            // We use std::atan2 in order to obtain results in [-pi; pi].
            val += std::atan2(current_p_frame[index].y, current_p_frame[index].x);
        }
        output[index] = val / (pmax - pmin + 1);
    }
}

void complex_to_argument(float* output,
                         const hipComplex* input,
                         const ushort pmin,
                         const ushort pmax,
                         const size_t size,
                         const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_complex_to_argument<<<blocks, threads, 0, stream>>>(output, input, pmin, pmax, size);
    cudaCheckError();
}

/* Find the minimum and the maximum of a floating-point array.
 *
 * The minimum and maximum can't be computed directly, because blocks
 * cannot communicate. Hence we compute local minima and maxima and
 * put them in two arrays.
 *
 * \param Size Number of threads in a block for this kernel.
 * Also, it's the size of min and max.
 * \param min Array of Size floats, which will contain local minima.
 * \param max Array of Size floats, which will contain local maxima.
 */
template <size_t Size>
static __global__ void kernel_minmax(const float* data, const size_t size, float* min, float* max)
{
    __shared__ float local_min[Size];
    __shared__ float local_max[Size];

    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index > size)
        return;
    local_min[threadIdx.x] = data[index];
    local_max[threadIdx.x] = data[index];

    __syncthreads();

    if (threadIdx.x == 0)
    {
        /* Accumulate the results of the neighbors, computing min-max values,
         * and store them in the first element of local arrays. */
        for (auto i = 1; i < Size; ++i)
        {
            if (local_min[i] < local_min[0])
                local_min[0] = local_min[i];
            if (local_max[i] > local_max[0])
                local_max[0] = local_max[i];
        }
        min[blockIdx.x] = local_min[0];
        max[blockIdx.x] = local_max[0];
    }
}

void rescale_float(const float* input, float* output, const size_t size, const hipStream_t stream)
{
    const uint threads = THREADS_128;
    const uint blocks = map_blocks_to_problem(size, threads);

    // TODO : See if gpu_postprocess_frame could be used directly.
    cudaXMemcpyAsync(output, input, sizeof(float) * size, hipMemcpyDeviceToDevice, stream);

    // Computing minimum and maximum values, in order to rescale properly.
    float* gpu_local_min;
    float* gpu_local_max;
    const uint float_blocks = sizeof(float) * blocks;
    cudaXMalloc(&gpu_local_min, float_blocks);
    cudaXMalloc(&gpu_local_max, float_blocks);

    kernel_minmax<threads><<<blocks, threads, threads << 1, stream>>>(output, size, gpu_local_min, gpu_local_max);
    cudaCheckError();

    float* cpu_local_min;
    float* cpu_local_max;
    cudaXMallocHost(&cpu_local_min, sizeof(float) * blocks);
    cudaXMallocHost(&cpu_local_max, sizeof(float) * blocks);
    cudaXMemcpyAsync(cpu_local_min, gpu_local_min, float_blocks, hipMemcpyDeviceToHost, stream);
    cudaXMemcpyAsync(cpu_local_max, gpu_local_max, float_blocks, hipMemcpyDeviceToHost, stream);
    cudaXStreamSynchronize(stream);

    constexpr float max_intensity = max_ushort_value_to_float;
    const float min_element = *(std::min_element(cpu_local_min, cpu_local_min + threads));
    const float max_element = *(std::max_element(cpu_local_max, cpu_local_max + threads));
    const auto lambda = [min_element, max_element, max_intensity] __device__(const float in) -> float
    { return (in + fabsf(min_element)) * max_intensity / (fabsf(max_element) + fabsf(min_element)); };

    map_generic<float>(output, output, size, lambda, stream);
    cudaCheckError();
    cudaXFreeHost(cpu_local_max);
    cudaXFreeHost(cpu_local_min);
    cudaXFree(gpu_local_min);
    cudaXFree(gpu_local_max);
}

void rescale_float_unwrap2d(float* input, float* output, float* cpu_buffer, size_t frame_res, const hipStream_t stream)
{
    float min = 0;
    float max = 0;

    uint float_frame_res = sizeof(float) * frame_res;
    cudaXMemcpyAsync(cpu_buffer, input, float_frame_res, hipMemcpyDeviceToHost, stream);
    cudaXStreamSynchronize(stream);
    auto minmax = std::minmax_element(cpu_buffer, cpu_buffer + frame_res);
    min = *minmax.first;
    max = *minmax.second;

    const auto lambda = [min, max] __device__(const float in) -> float
    {
        if (min < 0.f)
            return (in + fabs(min)) / (fabs(min) + max) * max_ushort_value_to_float;
        else
            return (in - min) / (max - min) * max_ushort_value_to_float;
    };
    map_generic(input, output, frame_res, lambda, stream);
}

void endianness_conversion(
    const ushort* input, ushort* output, const uint batch_size, const size_t frame_res, const hipStream_t stream)
{
    static const auto lambda = [] __device__(const ushort in) -> ushort { return (in << 8) | (in >> 8); };
    map_generic(input, output, frame_res * batch_size, lambda, stream);
}

/*
 * The input data shall be restricted first to the range [0; 2^16 - 1],
 * by forcing every negative  value to 0 and every positive one
 * greater than 2^16 - 1 to 2^16 - 1.
 * Then it is truncated to unsigned short data type.
 */
static __device__ ushort device_float_to_ushort(const float input, const uint shift = 0)
{
    if (input <= 0.0f) // Negative float
        return 0;
    // Cast in uint is needed to avoid overflow
    else if ((static_cast<uint>(input) << shift) > max_ushort_value_to_float)
        return max_ushort_value;
    else
        return static_cast<ushort>(input) << shift;
}

void complex_to_uint(
    const hipComplex* const input, uint* const output, const size_t size, hipStream_t stream, const uint shift)
{
    const auto lambda_complex_to_ushort = [shift] __device__(const hipComplex in) -> uint
    {
        /* hipComplex needs to be casted to a uint
        ** Each part (real & imaginary) are casted from float to ushort to then
        *be assembled into a uint
        ** The real part is on the left side of the uint, imaginary is on the
        *right one
        ** Here x & y are of type uint to avoid the overflow when shifting
        */
        constexpr uint size_half_uint = sizeof(uint) * 8 / 2;
        const uint x = device_float_to_ushort(in.x);
        const uint y = device_float_to_ushort(in.y);

        return ((x << size_half_uint) | y) << shift;
    };
    map_generic(input, output, size, lambda_complex_to_ushort, stream);
}

void float_to_ushort(
    const float* const input, ushort* const output, const size_t size, hipStream_t stream, const uint shift)
{
    const auto lambda = [shift] __device__(const float in) -> ushort { return device_float_to_ushort(in, shift); };
    map_generic(input, output, size, lambda, stream);
}

void float_to_ushort_normalized(const float* const input, ushort* const output, const size_t size, hipStream_t stream)
{
    const auto lambda = [] __device__(const float in) -> ushort {return in * max_ushort_value; };
    map_generic(input, output, size, lambda, stream);
}

void ushort_to_shifted_ushort(
    const ushort* const input, ushort* const output, const size_t size, hipStream_t stream, const uint shift)
{
    const auto lambda_shift_ushort = [shift] __device__(const ushort in) -> ushort { return in << shift; };
    map_generic(input, output, size, lambda_shift_ushort, stream);
}

void ushort_to_uchar(const ushort* input, uchar* output, const size_t size, const hipStream_t stream)
{
    static const auto lambda = [] __device__(const ushort in) -> uchar { return in >> (sizeof(uchar) * 8); };
    map_generic(input, output, size, lambda, stream);
}

void uchar_to_shifted_uchar(const uchar* input, uchar* output, const size_t size, hipStream_t stream, const uint shift)
{
    const auto lambda_shift_uchar = [shift] __device__(const uchar in) -> uchar { return in << shift; };
    map_generic(static_cast<const uchar* const>(input),
                static_cast<uchar* const>(output),
                size,
                lambda_shift_uchar,
                stream);
}

__global__ void kernel_accumulate_images(const float* input,
                                         float* output,
                                         const size_t end,
                                         const size_t max_elmt,
                                         const size_t nb_elmt,
                                         const size_t nb_pixel)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    long int pos = end; // end is excluded

    if (index < nb_pixel)
    {
        float val = 0;
        for (size_t i = 0; i < nb_elmt; i++)
        {
            // get last index when pos is out of range
            // reminder: the given input is from ciruclar queue
            pos--;
            if (pos < 0)
                pos = max_elmt - 1;

            val += input[index + pos * nb_pixel];
        }
        output[index] = val / nb_elmt;
    }
}

/*! \brief Kernel function wrapped in accumulate_images, making
** the call easier
**/
void accumulate_images(const float* input,
                       float* output,
                       const size_t start,
                       const size_t max_elmt,
                       const size_t nb_elmt,
                       const size_t nb_pixel,
                       const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(nb_pixel, threads);
    kernel_accumulate_images<<<blocks, threads, 0, stream>>>(input, output, start, max_elmt, nb_elmt, nb_pixel);
    cudaCheckError();
}

void normalize_complex(hipComplex* image, const size_t size, const hipStream_t stream)
{
    static const auto lambda = [] __device__(hipComplex in) -> hipComplex
    {
        in.x = (in.x + 1.0f) * (max_ushort_value_to_float / 2.0f);
        in.y = (in.y + 1.0f) * (max_ushort_value_to_float / 2.0f);
        return in;
    };
    map_generic(image, image, size, lambda, stream);
}

void convert_frame_for_display(
    const void* input, void* output, const size_t size, const uint depth, const ushort shift, const hipStream_t stream)
{
    if (depth == 8)
    {
        // In depth 8 the output is encoded onto a uint (for the lens)
        complex_to_uint(static_cast<const hipComplex* const>(input),
                        static_cast<uint* const>(output),
                        size,
                        stream,
                        shift);
    }
    else if (depth == 4)
    {
        float_to_ushort(static_cast<const float* const>(input),
                        static_cast<ushort* const>(output),
                        size,
                        stream,
                        shift);
    }
    else if (depth == 2)
    {
        ushort_to_shifted_ushort(static_cast<const ushort* const>(input),
                                 static_cast<ushort* const>(output),
                                 size,
                                 stream,
                                 shift);
    }
    else if (depth == 1)
    {
        uchar_to_shifted_uchar(static_cast<const uchar* const>(input),
                               static_cast<uchar* const>(output),
                               size,
                               stream,
                               shift);
    }
}

/* Simply transfers values from float buffer to hipComplex buffer */
static __global__ void kernel_float_to_complex(hipComplex* output, const float* input, size_t size)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        output[index] = hipComplex{input[index], 0.0f};
    }
}

void float_to_complex(hipComplex* output, const float* input, size_t size, const hipStream_t stream)
{
    const uint threads = get_max_threads_1d();
    const uint blocks = map_blocks_to_problem(size, threads);

    kernel_float_to_complex<<<blocks, threads, 0, stream>>>(output, input, size);
    cudaCheckError();
}
