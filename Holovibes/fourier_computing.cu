#include "hip/hip_runtime.h"
#include "fourier_computing.cuh"

hipfftComplex *do_cufft_3d(hipfftComplex * input, int nbimages, int size_x, int size_y)
{
  hipfftHandle plan;
  hipfftPlan3d(&plan, size_x, size_y, nbimages, HIPFFT_C2C);
  hipfftComplex *output;
  hipError_t er = hipMalloc(&output, size_x * size_y * nbimages * sizeof (hipfftComplex));
  hipfftExecC2C(plan, input, output, HIPFFT_FORWARD);
  return output;
}

hipfftComplex *fft_3d(holovibes::Queue *q, int nbimages)
{
  int threads = 512;
  int blocks = (q->get_pixels() * nbimages + 511) / 512;
  hipfftComplex *input = make_contigous_complex(q, nbimages);  // sqrt applied here
  dim3 lthreads(16, 16);
  dim3 lblocks(1600 / 16, 1200 / 16); // width / eight
  hipfftComplex *lens;
  hipMalloc(&lens, q->get_size() * sizeof (hipfftComplex));
  kernel_quadratic_lens <<<lblocks, lthreads >>>(lens, (unsigned int) q->get_pixels(), 600.0e-9f, 2.5f);
  apply_quadratic_lens <<<blocks, threads >>>(input, q->get_pixels() * nbimages, lens, q->get_pixels());
  hipFree(lens);
  hipfftComplex *result = do_cufft_3d(input, nbimages, q->get_frame_desc().width, q->get_frame_desc().height);
  return result;
}
