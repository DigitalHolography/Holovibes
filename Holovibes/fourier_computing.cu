#include "fourier_computing.cuh"

hipfftComplex *do_cufft_3d(hipfftComplex *input, int nbimages, int size_x, int size_y)
{
  hipfftHandle plan;
  hipfftPlan3d(&plan, size_x, size_y, nbimages, HIPFFT_C2C);
  hipfftExecC2C(plan, input, input, HIPFFT_FORWARD);
  return input;
}
