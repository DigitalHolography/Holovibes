#include "hip/hip_runtime.h"
#include "fourier_computing.cuh"

hipfftComplex *do_cufft_3d(hipfftComplex * input, int nbimages, int size_x, int size_y)
{
  hipfftHandle plan;
  hipfftPlan3d(&plan, size_x, size_y, nbimages, HIPFFT_C2C);
  hipfftComplex *output;
  hipError_t er = hipMalloc(&output, size_x * size_y * nbimages * sizeof (hipfftComplex));
  hipfftExecC2C(plan, input, output, HIPFFT_FORWARD);
  return output;
}

hipfftComplex *fft_3d(holovibes::Queue *q, int nbimages)
{
  int threads = 512;
  int blocks = (q->get_size() * nbimages + 511) / 512;
  hipfftComplex *input = make_contigous_complex(q, nbimages);  // sqrt applied here
  dim3 lthreads(16, 16);
  dim3 lblocks(1600 / 16, 1200 / 16); // width / eight
  hipfftComplex *lens;
  hipMalloc(&lens, q->get_size() * sizeof (hipfftComplex));
  kernel_quadratic_lens << <lblocks, lthreads >>>(lens, (unsigned int) q->get_size(), 600.0e-9f, 2.5f);
  apply_quadratic_lens << <blocks, threads >> >(input, q->get_size() * nbimages, lens, q->get_size());
  hipFree(lens);
  hipfftComplex *result = do_cufft_3d(input, nbimages, 1600, 1200);
  return result;
}
