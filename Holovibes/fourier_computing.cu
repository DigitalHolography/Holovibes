#include "hip/hip_runtime.h"
#include "fourier_computing.cuh"

hipfftComplex *do_cufft_3d(hipfftComplex *input, int nbimages, int size_x, int size_y)
{
  hipfftHandle plan;
  hipfftPlan3d(&plan, size_x, size_y, nbimages, HIPFFT_C2C);
  hipfftComplex *output;
  hipMalloc(&output, size_x * size_y * nbimages * sizeof (hipfftComplex));
  hipfftExecC2C(plan, input, output, HIPFFT_FORWARD);
  return output;
}

hipfftComplex *fft_3d(holovibes::Queue *q, int nbimages)
{
  int threads = 512;
  int blocks = (q->get_pixels() * nbimages + threads - 1) / threads;

  if (blocks > 65536)
    blocks = 65536;

  hipfftComplex *input = make_contigous_complex(q, nbimages);  // sqrt applied here

  // Constructing lens
  dim3 lthreads(16, 16);
  dim3 lblocks(q->get_frame_desc().width / 16, q->get_frame_desc().height / 16); // width / height
  hipfftComplex *lens;
  hipMalloc(&lens, q->get_pixels() * sizeof (hipfftComplex));
  kernel_quadratic_lens << <lblocks, lthreads >> >(lens, q->get_frame_desc().width, q->get_frame_desc().height, 532.0e-9f, 1.36f);

  // Applying lens
  apply_quadratic_lens <<<blocks, threads >>>(input, q->get_pixels() * nbimages, lens, q->get_pixels());
  hipFree(lens);

  // Applying FFT
  hipfftComplex *result = do_cufft_3d(input, nbimages, q->get_frame_desc().width, q->get_frame_desc().height);
  return result;
}
