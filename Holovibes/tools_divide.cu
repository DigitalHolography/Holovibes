#include "hip/hip_runtime.h"
#include "tools_divide.cuh"

#include <>
#include "hardware_limits.hh"

__global__ void kernel_complex_divide(
  hipfftComplex* image,
  unsigned int size,
  float divider)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    image[index].x = image[index].x / divider;
    image[index].y = image[index].y / divider;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_float_divide(
  float* input,
  unsigned int size,
  float divider)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    input[index] /= divider;
    index += blockDim.x * gridDim.x;
  }
}