#include "hip/hip_runtime.h"
#include "tools.cuh"

#include <>
#include "hardware_limits.hh"

// CONVERSION FUNCTIONS

__global__ void img8_to_complex(
  hipfftComplex* output,
  unsigned char* input,
  unsigned int size,
  const float* sqrt_array)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    // Image rescaling on 2^16 colors (65535 / 255 = 257)
    unsigned int val = sqrt_array[input[index] * 257];
    output[index].x = val;
    output[index].y = val;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void img16_to_complex(
  hipfftComplex* output,
  unsigned short* input,
  unsigned int size,
  const float* sqrt_array)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index].x = sqrt_array[input[index]];
    output[index].y = sqrt_array[input[index]];
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_complex_to_modulus(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = sqrtf(input[index].x * input[index].x + input[index].y * input[index].y);

    index += blockDim.x * gridDim.x;
  }
}

void complex_to_modulus(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  kernel_complex_to_modulus<<<blocks, threads>>>(input, output, size);
}

static __global__ void kernel_complex_to_squared_modulus(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input[index].x * input[index].x + input[index].y * input[index].y;

    index += blockDim.x * gridDim.x;
  }
}

void complex_to_squared_modulus(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  kernel_complex_to_squared_modulus<<<blocks, threads>>>(input, output, size);
}

static __global__ void kernel_complex_to_argument(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  float pi_div_2 = M_PI / 2.0f;
  float c = 65535.0f / M_PI;

  while (index < size)
  {
    output[index] = (atanf(input[index].y / input[index].x) + pi_div_2) * c;

    index += blockDim.x * gridDim.x;
  }
}

void complex_to_argument(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  kernel_complex_to_argument<<<blocks, threads>>>(input, output, size);
}

__global__ void kernel_apply_lens(
  hipfftComplex *input,
  unsigned int input_size,
  hipfftComplex *lens,
  unsigned int lens_size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < input_size)
  {
    input[index].x = input[index].x * lens[index % lens_size].x;
    input[index].y = input[index].y * lens[index % lens_size].y;
    index += blockDim.x * gridDim.x;
  }
}

static __global__ void kernel_shift_corners(
  float* input,
  unsigned int size_x,
  unsigned int size_y)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = j * blockDim.x * gridDim.x + i;
  unsigned int ni = 0;
  unsigned int nj = 0;
  unsigned int nindex = 0;
  float tmp = 0.0f;

  // Superior half of the matrix
  if (j >= size_y / 2)
  {
    // Left superior quarter of the matrix
    if (i < size_x / 2)
    {
      ni = i + size_x / 2;
      nj = j - size_y / 2;
    }
    // Right superior quarter
    else
    {
      ni = i - size_x / 2;
      nj = j - size_y / 2;
    }

    nindex = nj * size_x + ni;

    tmp = input[nindex];
    input[nindex] = input[index];
    input[index] = tmp;
  }
}

void shift_corners(
  float* input,
  unsigned int size_x,
  unsigned int size_y)
{
  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(size_x / threads_2d, size_y / threads_2d);

  kernel_shift_corners <<< lblocks, lthreads >>>(input, size_x, size_y);
}

static __global__ void kernel_endianness_conversion(
  unsigned short* input,
  unsigned short* output,
  size_t size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = (input[index] << 8) | (input[index] >> 8);

    index += blockDim.x * gridDim.x;
  }
}

void endianness_conversion(
  unsigned short* input,
  unsigned short* output,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks - 1;

  kernel_endianness_conversion <<<blocks, threads >>>(input, output, size);
}

__global__ void kernel_divide(
  hipfftComplex* image,
  unsigned int size,
  float divider)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    image[index].x = image[index].x / divider;
    image[index].y = image[index].y / divider;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_log10(
  float* input,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    input[index] = log10f(input[index]);

    index += blockDim.x * gridDim.x;
  }
}

void apply_log10(
  float* input,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  kernel_log10<<<blocks, threads>>>(input, size);
}

static __global__ void kernel_float_to_ushort(
  float* input,
  unsigned short* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    if (input[index] > 65535.0f)
      output[index] = 65535;
    else if (input[index] < 0.0f)
      output[index] = 0;
    else
      output[index] = static_cast<unsigned short>(input[index]);

    index += blockDim.x * gridDim.x;
  }
}

void float_to_ushort(
  float* input,
  unsigned short* output,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  kernel_float_to_ushort<<<blocks, threads>>>(input, output, size);
}

__global__ void kernel_multiply_frames_complex(
  const hipfftComplex* input1,
  const hipfftComplex* input2,
  hipfftComplex* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index].x = input1[index].x * input2[index].x;
    output[index].y = input1[index].y * input2[index].y;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_multiply_frames_float(
  const float* input1,
  const float* input2,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input1[index] * input2[index];
    index += blockDim.x * gridDim.x;
  }
}

void convolution_operator(
  const hipfftComplex* x,
  const hipfftComplex* k,
  float* out,
  unsigned int size,
  hipfftHandle plan2d_x,
  hipfftHandle plan2d_k)
{
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* The convolution operator is used only when using autofocus feature.
   * It could be optimized but it's useless since it will be used sometimes. */
  hipfftComplex* tmp_x;
  hipfftComplex* tmp_k;
  hipMalloc<hipfftComplex>(&tmp_x, size * sizeof(hipfftComplex));
  hipMalloc<hipfftComplex>(&tmp_k, size * sizeof(hipfftComplex));

  hipfftExecC2C(plan2d_x, const_cast<hipfftComplex*>(x), tmp_x, HIPFFT_FORWARD);
  hipfftExecC2C(plan2d_k, const_cast<hipfftComplex*>(k), tmp_k, HIPFFT_FORWARD);
  
  hipDeviceSynchronize();

  kernel_multiply_frames_complex <<<blocks, threads>>>(tmp_x, tmp_k, tmp_x, size);

  hipDeviceSynchronize();

  hipfftExecC2C(plan2d_x, tmp_x, tmp_x, HIPFFT_BACKWARD);

  hipDeviceSynchronize();

  kernel_complex_to_modulus <<<blocks, threads>>>(tmp_x, out, size);

  hipFree(tmp_x);
  hipFree(tmp_k);
}

void frame_memcpy(
  const float* input,
  const holovibes::Rectangle& zone,
  const unsigned int input_width,
  float* output,
  const unsigned int output_width)
{
  const unsigned int zone_width = abs(zone.top_right.x - zone.top_left.x);
  const unsigned int zone_height = abs(zone.bottom_left.y - zone.top_left.y);

  const float* zone_ptr = input + (zone.top_left.y * input_width + zone.top_left.x);

  hipMemcpy2D(
    output,
    output_width,
    zone_ptr,
    input_width,
    zone_width,
    zone_height,
    hipMemcpyDeviceToDevice);
#if 0
  for (unsigned int y = 0; y < zone_height; ++y)
  {
    hipMemcpy(
      output + y * output_width,
      zone_ptr + y * input_width,
      zone_width,
      hipMemcpyDeviceToDevice);
  }
#endif
}

static __global__ void kernel_sum(
  const float* input,
  float* sum,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (index < size)
  {
    atomicAdd(sum, input[index]);
    index += blockDim.x * gridDim.x;
  }
}

float average_operator(
  const float* input,
  const unsigned int size)
{
  const unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  float* gpu_sum;
  hipMalloc<float>(&gpu_sum, sizeof(float));
  hipMemset(gpu_sum, 0, sizeof(float));

  kernel_sum <<<blocks, threads>>>(
    input,
    gpu_sum,
    size);

  float cpu_sum;
  hipMemcpy(&cpu_sum, gpu_sum, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpu_sum);

  cpu_sum /= float(size);

  return cpu_sum;
}
