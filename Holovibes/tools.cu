#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "tools.cuh"


// CONVERSION FUNCTIONS

__global__ void image_2_float(hipfftReal* res, unsigned char* data, int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    res[index] = (float)data[index];
    index += blockDim.x * gridDim.x;
  }
}

__global__ void image_2_complex(hipfftComplex* res, unsigned char* data, int size, float *sqrt_tab)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    res[index].x = data[index];//sqrt_tab[data[index]];
    res[index].y = data[index];//sqrt_tab[data[index]];
    index += blockDim.x * gridDim.x;
  }
}

__global__ void complex_2_module(hipfftComplex* input, unsigned char* output, int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = sqrtf(input[index].x * input[index].x + input[index].y * input[index].y);
    index += blockDim.x * gridDim.x;
  }
}