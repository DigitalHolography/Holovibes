#include "hip/hip_runtime.h"
#include "tools.cuh"
#include "tools_multiply.cuh"

#include <>
#include "hardware_limits.hh"

/*! \brief  Apply a previously computed lens to image(s).
*
* The image(s) to treat, seen as input, should be contigous, the input_size is the total number of pixels to
* treat with the function.
*/

__global__ void kernel_apply_lens(
  hipfftComplex *input,
  unsigned int input_size,
  hipfftComplex *lens,
  unsigned int lens_size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < input_size)
  {
    input[index].x = input[index].x * lens[index % lens_size].x;
    input[index].y = input[index].y * lens[index % lens_size].y;
    index += blockDim.x * gridDim.x;
  }
}

/*! \brief  Permits to shift the corners of an image.
*
* This function shift zero-frequency component to center of spectrum
* as explaines in the matlab documentation(http://fr.mathworks.com/help/matlab/ref/fftshift.html).
* The transformation happens in-place.
*/
static __global__ void kernel_shift_corners(
  float* input,
  unsigned int size_x,
  unsigned int size_y)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = j * blockDim.x * gridDim.x + i;
  unsigned int ni = 0;
  unsigned int nj = 0;
  unsigned int nindex = 0;
  float tmp = 0.0f;

  // Superior half of the matrix
  if (j >= size_y / 2)
  {
    // Left superior quarter of the matrix
    if (i < size_x / 2)
    {
      ni = i + size_x / 2;
      nj = j - size_y / 2;
    }
    // Right superior quarter
    else
    {
      ni = i - size_x / 2;
      nj = j - size_y / 2;
    }

    nindex = nj * size_x + ni;

    tmp = input[nindex];
    input[nindex] = input[index];
    input[index] = tmp;
  }
}

/*! \brief  Permits to shift the corners of an image.
*
* This function shift zero-frequency component to center of spectrum
* as explaines in the matlab documentation(http://fr.mathworks.com/help/matlab/ref/fftshift.html).
* This function makes the Kernel call for the user in order to make the usage of the previous function easier.
*/
void shift_corners(
  float* input,
  unsigned int size_x,
  unsigned int size_y)
{
  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(size_x / threads_2d, size_y / threads_2d);

  kernel_shift_corners << < lblocks, lthreads >> >(input, size_x, size_y);
}



/*! \brief  compute the log of all the pixels of input image(s).
*
* The image(s) to treat should be contigous, the size is the total number of pixels to
* convert with the function.
* The value of pixels is replaced by their log10 value
*/

__global__ void kernel_log10(
  float* input,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    input[index] = log10f(input[index]);

    index += blockDim.x * gridDim.x;
  }
}

/*! \brief  compute the log of all the pixels of input image(s).
*
* The image(s) to treat should be contigous, the size is the total number of pixels to
* convert with the function.
* The value of pixels is replaced by their log10 value
* This function makes the Kernel call for the user in order to make the usage of the previous function easier.
*/
void apply_log10(
  float* input,
  unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  kernel_log10 << <blocks, threads >> >(input, size);
}



/*! \brief Kernel function used in convolution_operator
*/
static __global__ void kernel_complex_to_modulus(
  hipfftComplex* input,
  float* output,
  unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = sqrtf(input[index].x * input[index].x + input[index].y * input[index].y);

    index += blockDim.x * gridDim.x;
  }
}

/*! \brief  apply the convolution operator to 2 complex images (x,k).
*
* The 2 images should have the same size.
* The result value is given is out.
* The 2 used planes should be externally prepared (for performance reasons).
* For further informations: Autofocus of holograms based on image sharpness.
*/
void convolution_operator(
  const hipfftComplex* x,
  const hipfftComplex* k,
  float* out,
  unsigned int size,
  hipfftHandle plan2d_x,
  hipfftHandle plan2d_k)
{
  unsigned int threads = get_max_threads_1d();
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  /* The convolution operator is used only when using autofocus feature.
   * It could be optimized but it's useless since it will be used sometimes. */
  hipfftComplex* tmp_x;
  hipfftComplex* tmp_k;
  hipMalloc<hipfftComplex>(&tmp_x, size * sizeof(hipfftComplex));
  hipMalloc<hipfftComplex>(&tmp_k, size * sizeof(hipfftComplex));

  hipfftExecC2C(plan2d_x, const_cast<hipfftComplex*>(x), tmp_x, HIPFFT_FORWARD);
  hipfftExecC2C(plan2d_k, const_cast<hipfftComplex*>(k), tmp_k, HIPFFT_FORWARD);

  hipDeviceSynchronize();

  kernel_multiply_frames_complex << <blocks, threads >> >(tmp_x, tmp_k, tmp_x, size);

  hipDeviceSynchronize();

  hipfftExecC2C(plan2d_x, tmp_x, tmp_x, HIPFFT_BACKWARD);

  hipDeviceSynchronize();

  kernel_complex_to_modulus << <blocks, threads >> >(tmp_x, out, size);

  hipFree(tmp_x);
  hipFree(tmp_k);
}

/*! \brief  Extract a part of the input image to the output.
*
* The exracted aera should be less Than the input image.
* The result extracted image given is contained in output, the output should be preallocated.
* Coordonates of the extracted area are specified into the zone.
*/
void frame_memcpy(
  const float* input,
  const holovibes::Rectangle& zone,
  const unsigned int input_width,
  float* output,
  const unsigned int output_width)
{
  const unsigned int zone_width = abs(zone.top_right.x - zone.top_left.x);
  const unsigned int zone_height = abs(zone.bottom_left.y - zone.top_left.y);

  const float* zone_ptr = input + (zone.top_left.y * input_width + zone.top_left.x);

  hipMemcpy2D(
    output,
    output_width * sizeof(float),
    zone_ptr,
    input_width * sizeof(float),
    zone_width * sizeof(float),
    zone_height,
    hipMemcpyDeviceToDevice);
}

/*! \brief  Sum all the pixels of the input image.
**
** The result of the summation is contained in the parameted sum,
** The size parameter represent the number of pixels to sum,
** it should be equal to the number of pixels of the image.
** \param SpanSize Number of values to sum up serially before
** calling atomicAdd.
*/
template <unsigned SpanSize>
static __global__ void kernel_sum(const float* input, float* sum, size_t size)
{
  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  if ((index + SpanSize - 1) < size && (index % SpanSize) == 0)
  {
    float tmp_reduce = 0.0f;
    for (unsigned i = 0; i < SpanSize; ++i)
      tmp_reduce += input[index + i];
    atomicAdd(sum, tmp_reduce);
  }
}

/*! \brief   Make the average of all pixels contained into the input image
*
* The size parameter is the number of pixels of the input image
*/
float average_operator(
  const float* input,
  const unsigned int size)
{
  const unsigned int threads = 128;
  const unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks;

  float* gpu_sum;
  hipMalloc<float>(&gpu_sum, sizeof(float));
  hipMemset(gpu_sum, 0, sizeof(float));

  // SpanSize pf 4 has been determined to be an optimal choice here.
  kernel_sum <4> << <blocks, threads >> >(
    input,
    gpu_sum,
    size);

  float cpu_sum = 0.0f;
  hipMemcpy(&cpu_sum, gpu_sum, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpu_sum);

  cpu_sum /= float(size);

  return cpu_sum;
}