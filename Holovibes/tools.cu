#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "tools.cuh"


// CONVERSION FUNCTIONS

__global__ void image_2_complex8(hipfftComplex* res, unsigned char* data, int size, float *sqrt_tab)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    res[index].x = sqrt_tab[data[index]];
    res[index].y = sqrt_tab[data[index]];
    index += blockDim.x * gridDim.x;
  }
}

__global__ void image_2_complex16(hipfftComplex* res, unsigned short* data, int size, float *sqrt_tab)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    res[index].x = sqrt_tab[data[index]]; 
    res[index].y = sqrt_tab[data[index]]; 
    index += blockDim.x * gridDim.x;
  }
}

__global__ void complex_2_module(hipfftComplex* input, unsigned short* output, int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    float m = sqrtf(input[index].x * input[index].x + input[index].y * input[index].y);

    if (m > 65535.0f)
      output[index] = 65535;
    else
      output[index] = m;

    index += blockDim.x * gridDim.x;
  }
}

__global__ void apply_quadratic_lens(hipfftComplex *input, int input_size, hipfftComplex *lens, int lens_size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < input_size)
  {
    input[index].x = input[index].x * lens[index % lens_size].x;
    input[index].y = input[index].y * lens[index % lens_size].y;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void shift_corners(unsigned short *input, unsigned short *output, int size_x, int size_y)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size_x * size_y)
  {
    int x = index % size_x;
    int y = index / size_y;
    int n_x;
    int n_y;
    if (x < size_x / 2) // zone 1/3
    {
      if (y < size_y / 2) //zone 1
      {
        n_x = x + size_x / 2;
        n_y = y + size_y / 2;
      }
      else // zone 3
      {
        n_x = x + size_x / 2;
        n_y = y - size_y / 2;
      }
    }
    else // zone 2/4
    {
      if (y < size_y / 2) //zone 2
      {
        n_x = x - size_x / 2;
        n_y = y + size_y / 2;
      }
      else // zone 4
      {
        n_x = x - size_x / 2;
        n_y = y - size_y / 2;
      }
    }
    output[n_y * size_x + n_x] = input[index];
    index += blockDim.x * gridDim.x;
  }
}

void shift_corners(unsigned short **input, int size_x, int size_y)
{
  unsigned short *output;
  unsigned int size = size_x * size_y * sizeof(unsigned short);
  hipMalloc(&output, size);
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = ((size_x * size_x) + threads - 1) / threads;
  if (blocks > get_max_blocks())
    blocks = get_max_blocks();
  shift_corners << <blocks, threads >> >(*input, output, size_x, size_y);
  hipMemcpy(*input, output, size, hipMemcpyDeviceToDevice);
  hipFree(output);
}

__global__ void kernel_endianness_conversion(unsigned short* input, unsigned short* output, unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = (input[index] << 8) | (input[index] >> 8);

    index += blockDim.x * gridDim.x;
  }
}

void endianness_conversion(unsigned short* input, unsigned short* output, unsigned int size)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int max_blocks = get_max_blocks();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > max_blocks)
    blocks = max_blocks - 1;

  kernel_endianness_conversion << <blocks, threads >> >(input, output, size);
}