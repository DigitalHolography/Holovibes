#include "fft2.cuh"

hipfftComplex *create_spectral(float lambda, float distance, int size_x, int size_y, float pasx, float pasy, camera::FrameDescriptor fd)
{
  hipfftComplex *output;
  hipMalloc(&output, size_x * size_y * sizeof(hipfftComplex));
  hipMemset(output, 0, size_x * size_y * sizeof(hipfftComplex));

  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(size_x / threads_2d, size_y / threads_2d);
  kernel_spectral_lens<<<lblocks, lthreads>>>(output, fd, lambda, distance);

  return output;
}

void fft_2(int nbimages, holovibes::Queue *q, hipfftComplex *lens, float *sqrt_vect, unsigned short *result_buffer, hipfftHandle plan3d, unsigned int p, hipfftHandle plan2d)
{
  // Sizes
  unsigned int pixel_size = q->get_pixels() * nbimages;
  unsigned int complex_size = pixel_size * sizeof(hipfftComplex);
  unsigned int image_pixel = q->get_pixels();
  unsigned int complex_image_size = image_pixel * sizeof(hipfftComplex);
  unsigned int short_size = pixel_size * sizeof(unsigned short);
  unsigned short size_x = q->get_frame_desc().width;
  unsigned short size_y = q->get_frame_desc().height;

  // Loaded images --> complex
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (image_pixel + threads - 1) / threads; //one image blocks

  // Hardware limit !!
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;

  //get contigous images
  hipfftComplex* complex_input = make_contigous_complex(q, nbimages, sqrt_vect);

  //3d fft
  hipfftExecC2C(plan3d, complex_input, complex_input, HIPFFT_FORWARD);

  // extratct the (p) image
  hipfftComplex* pimage;
  hipMalloc(&pimage, complex_image_size);
  hipfftComplex *image = complex_input + p * image_pixel;
  hipMemcpy(pimage, image, complex_image_size, hipMemcpyDeviceToDevice);

  // apply lens
  apply_quadratic_lens << <blocks, threads >> >(pimage, image_pixel, lens, image_pixel);

  if (hipfftExecC2C(plan2d, pimage, pimage, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
    std::cout << "fail fft 2" << std::endl;
  hipDeviceSynchronize();

  divide<<<blocks, threads >>>(pimage, size_x, size_y, nbimages);

  //back to real
  complex_2_module <<<blocks, threads >> >(pimage, result_buffer, image_pixel); // one image

  hipFree(pimage);
  hipFree(complex_input);
}