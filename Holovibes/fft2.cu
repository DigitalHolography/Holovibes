#include "fft2.cuh"
#include "hardware_limits.hh"
#include "frame_desc.hh"
#include "tools.hh"
#include "transforms.cuh"
#include "preprocessing.cuh"
#include "tools_divide.cuh"
#include "tools.cuh"

void fft2_lens(
  hipfftComplex* lens,
  const camera::FrameDescriptor& fd,
  const float lambda,
  const float z,
  hipStream_t stream)
{
  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);
  float test = fd.pixel_size;
  kernel_spectral_lens << <lblocks, lthreads, 0, stream >> >(lens, fd, lambda, z);
}

void fft_2(
  hipfftComplex* input,
  const hipfftComplex* lens,
  const hipfftHandle plan3d,
  const hipfftHandle plan2d,
  const unsigned int frame_resolution,
  const unsigned int nframes,
  const unsigned int p,
  const unsigned int q,
  hipStream_t stream)
{
  const unsigned int n_frame_resolution = frame_resolution * nframes;

  unsigned int threads = 128;
  unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);

  hipfftExecC2C(plan3d, input, input, HIPFFT_FORWARD);
  hipStreamSynchronize(stream);
  hipfftComplex* pframe = input + frame_resolution * p;
  hipfftComplex* qframe = input + frame_resolution * q;

  kernel_apply_lens << <blocks, threads, 0, stream >> >(
	  pframe,
	  frame_resolution,
	  lens,
	  frame_resolution);
	  

  hipStreamSynchronize(stream);

  hipfftExecC2C(plan2d, pframe, pframe, HIPFFT_BACKWARD);

  kernel_complex_divide << <blocks, threads, 0, stream >> >(pframe, frame_resolution, static_cast<float>(n_frame_resolution));

  if (p != q)
  {
    hipfftExecC2C(plan2d, qframe, qframe, HIPFFT_BACKWARD);
    kernel_complex_divide << <blocks, threads, 0, stream >> >(qframe, frame_resolution, static_cast<float>(n_frame_resolution));
  }
  hipStreamSynchronize(stream);
}