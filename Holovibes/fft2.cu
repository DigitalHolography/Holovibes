#include "fft2.cuh"

#include <hip/hip_runtime.h>

#include "hardware_limits.hh"
#include "transforms.cuh"
#include "preprocessing.cuh"
#include "tools.cuh"

void fft2_lens(
  hipfftComplex* lens,
  const camera::FrameDescriptor& fd,
  float lambda,
  float z)
{
  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(fd.width / threads_2d, fd.height / threads_2d);

  kernel_spectral_lens<<<lblocks, lthreads>>>(lens, fd, lambda, z);
}

void fft_2(
  hipfftComplex* input,
  hipfftComplex* lens,
  hipfftHandle plan3d,
  hipfftHandle plan2d,
  unsigned int frame_resolution,
  unsigned int nframes,
  unsigned int p,
  unsigned int q)
{
  const unsigned int n_frame_resolution = frame_resolution * nframes;

  unsigned int threads = 128;
  unsigned int blocks = n_frame_resolution / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  hipfftExecC2C(plan3d, input, input, HIPFFT_FORWARD);

  hipfftComplex* pframe = input + frame_resolution * p;
  hipfftComplex* qframe = input + frame_resolution * q;

  hipDeviceSynchronize();

  kernel_apply_lens<<<blocks, threads>>>(
    input,
    n_frame_resolution,
    lens,
    frame_resolution);

  hipDeviceSynchronize();

  hipfftExecC2C(plan2d, pframe, pframe, HIPFFT_BACKWARD);
  kernel_complex_divide<<<blocks, threads >>>(pframe, frame_resolution, static_cast<float>(n_frame_resolution));
  if (p != q)
  {
    hipfftExecC2C(plan2d, qframe, qframe, HIPFFT_BACKWARD);
    kernel_complex_divide <<<blocks, threads>>>(qframe, frame_resolution, static_cast<float>(n_frame_resolution));
  }

  hipDeviceSynchronize();
}
