#include "fft2.cuh"


void create_spectral(hipfftComplex *output, float lambda, float distance, int size_x, int size_y, int pasu, int pasv)
{
  float *u;
  float *v;
  hipMalloc(&u, size_x * sizeof(float));
  hipMalloc(&v, size_y * sizeof(float));

  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size_x + threads - 1) / threads;

  // Hardware limit !!
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;

  fft2_make_u_v<<<blocks, threads>>>(pasu, pasv, u, v, size_x, size_y);

  int mesh_size_u = size_x * size_x;
  int mesh_size_v = size_y * size_y;
  float* u_mesh;
  float* v_mesh;
  hipMalloc(&u_mesh, mesh_size_u * sizeof(float));
  hipMalloc(&v_mesh, mesh_size_v * sizeof(float));

  blocks = ((size_x * size_y) + threads - 1) / threads;
  // Hardware limit !!
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;

  meshgrind_square<<<blocks,threads>>>(u, v, u_mesh, v_mesh, size_x, size_y);
  hipFree(u);
  hipFree(v);

  spectral <<<blocks,threads>>>(u_mesh, v_mesh, output, size_x * size_y, lambda, distance);
  hipFree(u_mesh);
  hipFree(v_mesh);
}