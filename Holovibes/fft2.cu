#include "fft2.cuh"

#include <hip/hip_runtime.h>

#include "hardware_limits.hh"
#include "transforms.cuh"
#include "preprocessing.cuh"
#include "tools.cuh"

hipfftComplex *create_spectral(
  float lambda,
  float distance,
  int size_x,
  int size_y,
  const camera::FrameDescriptor& fd)
{
  hipfftComplex *output;
  hipMalloc(&output, size_x * size_y * sizeof(hipfftComplex));
  hipMemset(output, 0, size_x * size_y * sizeof(hipfftComplex));

  unsigned int threads_2d = get_max_threads_2d();
  dim3 lthreads(threads_2d, threads_2d);
  dim3 lblocks(size_x / threads_2d, size_y / threads_2d);
  kernel_spectral_lens<<<lblocks, lthreads>>>(output, fd, lambda, distance);

  return output;
}

void fft_2(
  unsigned short* result_buffer,
  holovibes::Queue& q,
  hipfftComplex *lens,
  float *sqrt_vect,
  hipfftHandle plan3d,
  hipfftHandle plan2d,
  unsigned int nbimages,
  unsigned int p)
{
  // Sizes
  unsigned int pixel_size = q.get_pixels() * nbimages;
  unsigned int image_pixel = q.get_pixels();
  unsigned int complex_image_size = image_pixel * sizeof(hipfftComplex);
  unsigned short size_x = q.get_frame_desc().width;
  unsigned short size_y = q.get_frame_desc().height;

  // Loaded images --> complex
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (image_pixel + threads - 1) / threads; //one image blocks

  // Hardware limit !!
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;

  //get contigous images
  hipfftComplex* complex_input = make_contiguous_complex(q, nbimages, sqrt_vect);

  //3d fft
  hipfftExecC2C(plan3d, complex_input, complex_input, HIPFFT_FORWARD);

  // extratct the (p) image
  hipfftComplex* pimage;
  hipMalloc(&pimage, complex_image_size);
  hipfftComplex *image = complex_input + p * image_pixel;
  hipMemcpy(pimage, image, complex_image_size, hipMemcpyDeviceToDevice);

  // apply lens
  apply_quadratic_lens <<<blocks, threads >>>(pimage, image_pixel, lens, image_pixel);

  if (hipfftExecC2C(plan2d, pimage, pimage, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
    std::cout << "fail fft 2" << std::endl;
  hipDeviceSynchronize();

  divide<<<blocks, threads >>>(pimage, size_x, size_y, nbimages);

  //back to real
  complex_2_module <<<blocks, threads >> >(pimage, result_buffer, image_pixel); // one image

  hipFree(pimage);
  hipFree(complex_input);
}