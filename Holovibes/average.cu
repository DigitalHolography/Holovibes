#include "hip/hip_runtime.h"
#include <>
#include <cmath>

#include "average.cuh"
#include "geometry.hh"
#include "tools.hh"
#include "tools.cuh"
#include "tools_conversion.cuh"
#include "hardware_limits.hh"

#define THREADS 256

/*! \brief  Sume 2 zone of input image
*
* \param input The image from where zones should be summed.
* \param width The width of the input image.
* \param height The height of the input image.
*
*/
static __global__ void kernel_zone_sum(
	float* input,
	const unsigned int width,
	float* output,
	const unsigned int zone_start_x,
	const unsigned int zone_start_y,
	const unsigned int zone_width,
	const unsigned int zone_height)
{
	const unsigned int size = zone_width * zone_height;
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x * blockDim.x + tid;
	extern __shared__ float  sdata[];

	// INIT
	sdata[tid] = 0.0f;

	// SUM input in sdata
	while (index < size)
	{
		int x = index % zone_width + zone_start_x;
		int y = index / zone_width + zone_start_y;
		int index2 = y * width + x;

		sdata[tid] += input[index2];
		index += blockDim.x * gridDim.x;
	}

	// Sum sdata in sdata[0]
	__syncthreads();
	for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1)
	{
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}

	// Return result
	__syncthreads();
	if (tid == 0)
		*output = sdata[0];
}

std::tuple<float, float, float, float> make_average_plot(
	float *input,
	const unsigned int width,
	const unsigned int height,
	const holovibes::Rectangle& signal,
	const holovibes::Rectangle& noise,
	hipStream_t stream)
{
	//const unsigned int size = width * height;
	unsigned int threads = THREADS;
	//unsigned int blocks = map_blocks_to_problem(size, threads);

	float* gpu_s;
	float* gpu_n;

	hipMalloc(&gpu_s, sizeof(float));
	hipMalloc(&gpu_n, sizeof(float));

	hipMemsetAsync(gpu_s, 0, sizeof(float), stream);
	hipMemsetAsync(gpu_n, 0, sizeof(float), stream);

	unsigned int signal_width = abs(signal.top_right.x - signal.top_left.x);
	unsigned int signal_height = abs(signal.top_left.y - signal.bottom_left.y);
	unsigned int noise_width = abs(noise.top_right.x - noise.top_left.x);
	unsigned int noise_height = abs(noise.top_left.y - noise.bottom_left.y);

	kernel_zone_sum << <1, threads, threads * sizeof(float), stream >> >(input, width, gpu_n,
		noise.top_left.x, noise.top_left.y, noise_width, noise_height);
	kernel_zone_sum << <1, threads, threads * sizeof(float), stream >> >(input, width, gpu_s,
		signal.top_left.x, signal.top_left.y, signal_width, signal_height);

	float cpu_s;
	float cpu_n;

	hipMemcpyAsync(&cpu_s, gpu_s, sizeof(float), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(&cpu_n, gpu_n, sizeof(float), hipMemcpyDeviceToHost, stream);

	cpu_s /= float(signal_width * signal_height);
	cpu_n /= float(noise_width * noise_height);

	float moy = cpu_s / cpu_n;

	hipFree(gpu_n);
	hipFree(gpu_s);

	return std::tuple < float, float, float, float > { cpu_s, cpu_n, moy, 10 * log10f(moy)};
}

std::tuple<float, float, float, float> make_average_stft_plot(
	hipfftComplex*          cbuf,
	float*                 fbuf,
	hipfftComplex*          stft_buffer,
	const unsigned int     width,
	const unsigned int     height,
	const unsigned int     width_roi,
	const unsigned int     height_roi,
	holovibes::Rectangle&  signal_zone,
	holovibes::Rectangle&  noise_zone,
	const unsigned int     pindex,
	const unsigned int     nsamples,
	hipStream_t stream)
{
	std::tuple<float, float, float, float> res;

	const unsigned int size = width * height;
	//unsigned int threads = 128;
	//unsigned int blocks = map_blocks_to_problem(size, threads);

	// Reconstruct Roi
	/*kernel_reconstruct_roi << <blocks, threads, 0, stream >> >(
	  stft_buffer,
	  cbuf,
	  width_roi,
	  height_roi,
	  width,
	  width,
	  height,
	  pindex,
	  nsamples);*/

	complex_to_modulus(cbuf, fbuf, size, stream);

	return make_average_plot(fbuf, width, height, signal_zone, noise_zone, stream);
}