#include "hip/hip_runtime.h"

#include "transforms.cuh"

__global__ void kernel_quadratic_lens(hipfftComplex* output,
  unsigned int xsize,
  unsigned int ysize,
  float lambda,
  float dist)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = j * blockDim.x * gridDim.x + i;

  float c = M_PI / (lambda * dist);
  float csquare;
  float dx = 5.5e-6f;
  float dy = 5.5e-6f;

  float x = (i - ((float)xsize / 2)) * dx;
  float y = (j - ((float)ysize / 2)) * dy;

  if (index < xsize * ysize)
  {
    csquare = c * (x * x + y * y);
    output[index].x = cosf(csquare);
    output[index].y = sinf(csquare);
  }
}

__global__ void spectral(float *u_square, float *v_square, hipfftComplex *output, unsigned int output_size, float lambda, float distance)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < output_size)
  {
    float lambda_square = lambda * lambda;
    float thetha = 2 * M_PI * distance / lambda * sqrt(1 - (lambda_square * u_square[index]) - (lambda_square * v_square[index]));
    output[index].x = cosf(thetha);// fix me
    output[index].y = sinf(thetha );// fix me
      index += blockDim.x * gridDim.x;
  }

}