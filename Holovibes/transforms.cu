#include "hip/hip_runtime.h"
#include "transforms.cuh"

#include <>

#ifndef _USE_MATH_DEFINES
/* Enables math constants. */
# define _USE_MATH_DEFINES
#endif /* !_USE_MATH_DEFINES */
#include <math.h>

/*! \brief Compute a lens to apply to an image 
*
*
* \param n output The lens computed by the function.
* The output should have the same caracteristics of 
* of the images on wich the lens will be applied.
* \param fd File descriptor of the images on wich the lens will be applied.
*/
__global__ void kernel_quadratic_lens(
  hipfftComplex* output,
  const camera::FrameDescriptor fd,
  float lambda,
  float dist)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = j * blockDim.x * gridDim.x + i;

  float c = M_PI / (lambda * dist);
  float csquare;
  float dx = fd.pixel_size * 1.0e-6f;
  float dy = fd.pixel_size * 1.0e-6f;

  float x = (i - ((float)fd.width / 2)) * dx;
  float y = (j - ((float)fd.height / 2)) * dy;

  if (index < fd.width * fd.height)
  {
    csquare = c * (x * x + y * y);
    output[index].x = cosf(csquare);
    output[index].y = sinf(csquare);
  }
}

/*! \brief Compute a lens to apply to an image
*
*
* \param n output The lens computed by the function.
* The output should have the same caracteristics of
* of the images on wich the lens will be applied.
* \param fd File descriptor of the images on wich the lens will be applied.
*/
__global__ void kernel_spectral_lens(
  hipfftComplex* output,
  const camera::FrameDescriptor fd,
  float lambda,
  float distance)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = j * blockDim.x * gridDim.x + i;

  float c = 2 * M_PI * distance / lambda;
  float csquare;

  float dx = fd.pixel_size * 1.0e-6f;
  float dy = fd.pixel_size * 1.0e-6f;

  float du = 1 / (((float)fd.width) * dx);
  float dv = 1 / (((float)fd.height) * dy);

  float u = (i - (float)(lrintf((float)fd.width / 2))) * du;
  float v = (j - (float)(lrintf((float)fd.height / 2))) * dv;

  if (index < fd.width * fd.height)
  {
    csquare = c * sqrtf(1.0f - lambda * lambda * u * u - lambda * lambda * v * v);
    output[index].x = cosf(csquare);
    output[index].y = sinf(csquare);
  }
}