#include "hip/hip_runtime.h"
#include "contrast_correction.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <cstdlib>

#include "hardware_limits.hh"

static __global__ void kernel_histogram(
  float* input,
  unsigned int input_size,
  unsigned int* histogram,
  unsigned int histogram_size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < input_size)
  {
    unsigned int pixel_value = __float2_uint_rz(input[index]);

    if (pixel_value >= histogram_size)
      pixel_value = histogram_size - 1;

    atomicAdd(histogram[pixel_value], 1);

    index += blockDim.x * gridDim.x;
  }
}

static void find_min_max(
  unsigned int *min,
  unsigned int *max,
  int *histo,
  int bytedepth,
  int percent,
  unsigned int nbpixels)
{
  int acceptable = (percent / 100) * nbpixels;
  if (bytedepth == 1)
  {
    *min = 255;
    *max = 0;
    for (int i = 0; i < 255; i++)
    {
      if (histo[i] > acceptable)
      {
        if (i > *max)
          *max = i;
        if (i < *min)
          *min = i;
      }
    }
  }
  else
  {
    *min = 65535;
    *max = 0;
    for (int i = 0; i < 65535; i++)
    {
      if (histo[i] > acceptable)
      {
        if (i > *max)
          *max = i;
        if (i < *min)
          *min = i;
      }
    }
  }
}

// Fix this
#if 0
void auto_contrast_correction(
  float* input,
  unsigned int size,
  unsigned int* min,
  unsigned int* max,
  float threshold) // percent
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  int *histo;
  int *histo_cpu = (int*)calloc(sizeof(int)* tons, 1);
  hipMalloc(&histo, tons * sizeof(int));
  hipMemset(histo, 0, tons * sizeof(int));
  make_histo << <blocks, threads >> >(histo, img, img_size, bytedepth);
  hipMemcpy(histo_cpu, histo, tons * sizeof(int), hipMemcpyDeviceToHost);
  find_min_max(min, max, histo_cpu, bytedepth, percent, img_size);
  float factor = tons / (*max - *min);
  hipFree(histo);
  free(histo_cpu);
}
#endif

static __global__ void apply_contrast(
  float* input,
  unsigned int size,
  float factor,
  unsigned short min)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    input[index] = factor * (input[index] - static_cast<float>(min));
    index += blockDim.x * gridDim.x;
  }
}

void manual_contrast_correction(
  float* input,
  unsigned int size,
  unsigned short dynamic_range,
  unsigned short min,
  unsigned short max)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  const float factor = static_cast<float>(dynamic_range) / static_cast<float>(max - min);
  apply_contrast<<<blocks, threads>>>(input, size, factor, min);
}
