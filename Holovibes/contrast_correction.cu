#include "hip/hip_runtime.h"
#include "contrast_correction.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <cstdlib>

#include "hardware_limits.hh"

__global__ static void make_histo(int *histo, unsigned char *img, int img_size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < img_size)
  {
    atomicAdd(&histo[img[index]], 1);
  }
}

__global__ static void apply_correction(int *sum_histo, unsigned char *img, int img_size, int tons)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < img_size)
  {
    img[index] = ((tons - 1) * sum_histo[img[index]] / img_size);;
    index += blockDim.x * gridDim.x;
  }
}

void sum_histo_c(int *histo, int *summed_histo, int bytedepth)
{
  int tons = 65536;
  if (bytedepth == 1)
    tons = 256;
  summed_histo[0] = histo[0];
  for (int i = 1; i < tons; i++)
  {
    summed_histo[i] += summed_histo[i - 1] + histo[i];
  }
}

void correct_contrast(unsigned char *img, int img_size, int bytedepth)
{
  int tons = 65536;
  if (bytedepth == 1)
    tons = 256;
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (img_size + threads - 1) / threads;
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;

  int *histo;
  int *sum_histo;
  int *histo_cpu = (int*)calloc(sizeof(int) * tons, 1);
  int *sum_histo_cpu = (int*)calloc(1, sizeof(int)* tons);
  hipMalloc(&sum_histo, tons * sizeof(int));
  hipMalloc(&histo, tons * sizeof(int));
  hipMemset(histo, 0, tons * sizeof(int));
  make_histo<<<blocks, threads>>>(histo, img, img_size);
  hipMemcpy(histo_cpu, histo, tons * sizeof(int), hipMemcpyDeviceToHost);
  sum_histo_c(histo_cpu, sum_histo_cpu, bytedepth);
  hipMemcpy(sum_histo, sum_histo_cpu, tons * sizeof(int), hipMemcpyHostToDevice);
  apply_correction <<<blocks, threads>>>(sum_histo, img, img_size, tons);
  hipFree(histo);
  hipFree(sum_histo);
  free(histo_cpu);
  free(sum_histo_cpu);
}