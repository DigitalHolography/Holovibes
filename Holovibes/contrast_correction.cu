#include "hip/hip_runtime.h"
#include "contrast_correction.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <cstdlib>

#include "hardware_limits.hh"

__global__ void make_histo(
  int *histo,
  void *img,
  int img_size,
  int bytedepth)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < img_size)
  {
    if (bytedepth == 1)
      atomicAdd(&histo[((unsigned char*)img)[index]], 1);
    else
      atomicAdd(&histo[((unsigned short*)img)[index]], 1);
  }
}

__global__ void apply_contrast(
  unsigned int min,
  float factor,
  void *img,
  unsigned int size,
  int bytedepth)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    if (bytedepth == 1)
    {
      int value = factor * (float)(((unsigned char*)img)[index] - min);
      if (value > 255)
        value = 255;
      else if (value < 0)
        value = 0;
      ((unsigned char*)img)[index] = value;
    }
    else
    {
      int value = factor * (float)(((unsigned short*)img)[index] - min);
      if (value > 65535)
        value = 65535;
      else if (value < 0)
        value = 0;
      ((unsigned short*)img)[index] = value;
    }
    index += blockDim.x * gridDim.x;
  }
}

void find_min_max(
  unsigned int *min,
  unsigned int *max,
  int *histo,
  int bytedepth,
  int percent,
  unsigned int nbpixels)
{
  int acceptable = (percent / 100) * nbpixels;
  if (bytedepth == 1)
  {
    *min = 255;
    *max = 0;
    for (int i = 0; i < 255; i++)
    {
      if (histo[i] > acceptable)
      {
        if (i > *max)
          *max = i;
        if (i < *min)
          *min = i;
      }
    }
  }
  else
  {
    *min = 65535;
    *max = 0;
    for (int i = 0; i < 65535; i++)
    {
      if (histo[i] > acceptable)
      {
        if (i > *max)
          *max = i;
        if (i < *min)
          *min = i;
      }
    }
  }
}

void manual_contrast_correction(
  void *img,
  unsigned int img_size,
  int bytedepth,
  unsigned int manual_min,
  unsigned int manual_max)
{
  int tons = 65536;
  if (bytedepth == 1)
    tons = 256;
  int threads = get_max_threads_1d();
  int blocks = (img_size + threads - 1) / threads;
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;
  float factor = tons / (manual_max - manual_min);
  apply_contrast <<<blocks, threads>>>(manual_min, factor, img, img_size, bytedepth);
}

void auto_contrast_correction(
  unsigned int *min,
  unsigned int *max,
  void *img,
  unsigned int img_size,
  unsigned int bytedepth,
  unsigned int percent)
{
  int tons = 65536;
  if (bytedepth == 1)
    tons = 256;
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (img_size + threads - 1) / threads;
  if (blocks > get_max_blocks())
    blocks = get_max_blocks() - 1;
  int *histo;
  int *histo_cpu = (int*)calloc(sizeof(int)* tons, 1);
  hipMalloc(&histo, tons * sizeof(int));
  hipMemset(histo, 0, tons * sizeof(int));
  make_histo << <blocks, threads >> >(histo, img, img_size, bytedepth);
  hipMemcpy(histo_cpu, histo, tons * sizeof(int), hipMemcpyDeviceToHost);
  find_min_max(min, max, histo_cpu, bytedepth, percent, img_size);
  float factor = tons / (*max - *min);
  hipFree(histo);
  free(histo_cpu);
}
