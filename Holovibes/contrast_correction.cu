#include "hip/hip_runtime.h"
#include "contrast_correction.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <cstdlib>

#include "hardware_limits.hh"

static __global__ void make_histo(
  int *histo,
  void *img,
  int img_size,
  int bytedepth)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < img_size)
  {
    if (bytedepth == 1)
      atomicAdd(&histo[((unsigned char*)img)[index]], 1);
    else
      atomicAdd(&histo[((unsigned short*)img)[index]], 1);
  }
}

static void find_min_max(
  unsigned int *min,
  unsigned int *max,
  int *histo,
  int bytedepth,
  int percent,
  unsigned int nbpixels)
{
  int acceptable = (percent / 100) * nbpixels;
  if (bytedepth == 1)
  {
    *min = 255;
    *max = 0;
    for (int i = 0; i < 255; i++)
    {
      if (histo[i] > acceptable)
      {
        if (i > *max)
          *max = i;
        if (i < *min)
          *min = i;
      }
    }
  }
  else
  {
    *min = 65535;
    *max = 0;
    for (int i = 0; i < 65535; i++)
    {
      if (histo[i] > acceptable)
      {
        if (i > *max)
          *max = i;
        if (i < *min)
          *min = i;
      }
    }
  }
}

// Fix this
#if 0
void auto_contrast_correction(
  float* input,
  unsigned int size,
  unsigned int* min,
  unsigned int* max,
  float threshold) // percent
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  int *histo;
  int *histo_cpu = (int*)calloc(sizeof(int)* tons, 1);
  hipMalloc(&histo, tons * sizeof(int));
  hipMemset(histo, 0, tons * sizeof(int));
  make_histo << <blocks, threads >> >(histo, img, img_size, bytedepth);
  hipMemcpy(histo_cpu, histo, tons * sizeof(int), hipMemcpyDeviceToHost);
  find_min_max(min, max, histo_cpu, bytedepth, percent, img_size);
  float factor = tons / (*max - *min);
  hipFree(histo);
  free(histo_cpu);
}
#endif

static __global__ void apply_contrast(
  float* input,
  unsigned int size,
  float factor,
  unsigned short min)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    input[index] = factor * (input[index] - static_cast<float>(min));
    index += blockDim.x * gridDim.x;
  }
}

void manual_contrast_correction(
  float* input,
  unsigned int size,
  unsigned short dynamic_range,
  unsigned short min,
  unsigned short max)
{
  unsigned int threads = get_max_threads_1d();
  unsigned int blocks = (size + threads - 1) / threads;

  if (blocks > get_max_blocks())
    blocks = get_max_blocks();

  const float factor = static_cast<float>(dynamic_range) / static_cast<float>(max - min);
  apply_contrast<<<blocks, threads>>>(input, size, factor, min);
}
