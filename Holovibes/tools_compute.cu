#include "hip/hip_runtime.h"
#include <>

#include "tools_compute.cuh"
#include "hardware_limits.hh"
#include "tools.hh"

__global__ void kernel_complex_divide(
  hipfftComplex* image,
  const unsigned int size,
  const float divider)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    image[index].x = image[index].x / divider;
    image[index].y = image[index].y / divider;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_float_divide(
  float* input,
  const unsigned int size,
  const float divider)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  while (index < size)
  {
    input[index] /= divider;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_multiply_frames_complex(
	const hipfftComplex* input1,
	const hipfftComplex* input2,
	hipfftComplex* output,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		output[index].x = input1[index].x * input2[index].x;
		output[index].y = input1[index].y * input2[index].y;
		index += blockDim.x * gridDim.x;
	}
}

__global__ void kernel_multiply_frames_float(
	const float* input1,
	const float* input2,
	float* output,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		output[index] = input1[index] * input2[index];
		index += blockDim.x * gridDim.x;
	}
}

__global__ void kernel_substract_ref(
	hipfftComplex*      input,
	hipfftComplex*      reference,
	const unsigned int size,
	const unsigned int frame_size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < size)
	{
		input[index].x -= reference[index % frame_size].x;
		index += blockDim.x * gridDim.x;
	}
}

void substract_ref(
	hipfftComplex* input,
	hipfftComplex* reference,
	const unsigned int frame_resolution,
	const unsigned int nframes,
	hipStream_t stream)
{
	const unsigned int n_frame_resolution = frame_resolution * nframes;
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(frame_resolution, threads);
    kernel_substract_ref << <blocks, threads, 0, stream >> >(input, reference, n_frame_resolution, frame_resolution);
}

__global__ void kernel_mean_images(
	hipfftComplex *input,
	hipfftComplex *output,
	unsigned int n,
	unsigned int frame_size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < frame_size)
	{
		float tmp = 0;
		for (int i = 0; i < n; i++)
			tmp += input[index + i * frame_size].x;
		tmp /= n;
		output[index].x = tmp;
		index += blockDim.x * gridDim.x;
	}
}

void mean_images(
	 hipfftComplex* input,
	 hipfftComplex* output,
	unsigned int n,
	unsigned int frame_size,
	hipStream_t stream)
{
	unsigned int threads = get_max_threads_1d();
	unsigned int blocks = map_blocks_to_problem(frame_size, threads);

	kernel_mean_images << <blocks, threads, 0, stream >> >(input, output, n, frame_size);
}