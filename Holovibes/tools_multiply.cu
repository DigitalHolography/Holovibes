#include "hip/hip_runtime.h"
#include <>

#include "tools_multiply.cuh"
#include "hardware_limits.hh"

__global__ void kernel_multiply_frames_complex(
  const hipfftComplex* input1,
  const hipfftComplex* input2,
  hipfftComplex* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index].x = input1[index].x * input2[index].x;
    output[index].y = input1[index].y * input2[index].y;
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_multiply_frames_float(
  const float* input1,
  const float* input2,
  float* output,
  const unsigned int size)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < size)
  {
    output[index] = input1[index] * input2[index];
    index += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_multiply_complexes_by_floats_(
	const float* input1,
	const float* input2,
	hipfftComplex* output1,
	hipfftComplex* output2,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		output1[index].x = output1[index].x * input1[index];
		output1[index].y = output1[index].y * input1[index];
		output2[index].x = output2[index].x * input2[index];
		output2[index].y = output2[index].y * input2[index];
		index += blockDim.x * gridDim.x;
	}
}

__global__ void kernel_multiply_complexes_by_single_complex(
	hipfftComplex* output1,
	hipfftComplex* output2,
	const hipfftComplex input,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		const hipfftComplex cpy_o1 = output1[index];
		const hipfftComplex cpy_o2 = output2[index];

		output1[index].x = cpy_o1.x * input.x - cpy_o1.y * input.y;
		output1[index].y = cpy_o1.x * input.y + cpy_o1.y * input.x;
		output2[index].x = cpy_o2.x * input.x - cpy_o2.y * input.y;
		output2[index].y = cpy_o2.x * input.y + cpy_o2.y * input.x;
		index += blockDim.x * gridDim.x;
	}
}

__global__ void kernel_multiply_complex_by_single_complex(
	hipfftComplex* output,
	const hipfftComplex input,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		const hipfftComplex cpy_o1 = output[index];

		output[index].x = cpy_o1.x * input.x - cpy_o1.y * input.y;
		output[index].y = cpy_o1.x * input.y + cpy_o1.y * input.x;
		index += blockDim.x * gridDim.x;
	}
}

__global__ void kernel_conjugate_complex(
	hipfftComplex* output,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		output[index].y = -output[index].y;
		index += blockDim.x * gridDim.x;
	}
}

__global__ void kernel_multiply_complex_frames_by_complex_frame(
	hipfftComplex* output1,
	hipfftComplex* output2,
	const hipfftComplex* input,
	const unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < size)
	{
		const hipfftComplex cpy_o1 = output1[index];
		const hipfftComplex cpy_o2 = output2[index];

		output1[index].x = cpy_o1.x * input[index].x - cpy_o1.y * input[index].y;
		output1[index].y = cpy_o1.x * input[index].y + cpy_o1.y * input[index].x;
		output2[index].x = cpy_o2.x * input[index].x - cpy_o2.y * input[index].y;
		output2[index].y = cpy_o2.x * input[index].y + cpy_o2.y * input[index].x;
		index += blockDim.x * gridDim.x;
	}
}