#include "hip/hip_runtime.h"
#include "preprocessing.cuh"

float *make_contigous_float(holovibes::Queue *q, int nbimages)
{
  int threads = 512;
  int blocks = (q->get_size() * nbimages + 511) / 512;
  int contigous_elts = 0;
  hipfftReal *output;
  hipMalloc(&output, q->get_size() * nbimages * sizeof(float));
  int index = (q->get_start_index() + q->get_current_elts() - nbimages) % q->get_max_elts();

  if (q->get_max_elts() - index < nbimages)
    contigous_elts = q->get_max_elts() - nbimages;
  else
    contigous_elts = nbimages;

  std::cout << "contigous: " << contigous_elts << std::endl;

  if (contigous_elts < nbimages)
  {
    unsigned char *contigous;
    hipMalloc(&contigous, q->get_size() * nbimages); //modify for 16bit
    hipMemcpy(contigous, q->get_last_images(nbimages), contigous_elts * q->get_size(), hipMemcpyDeviceToDevice);
    hipMemcpy(contigous + contigous_elts * q->get_size(), q->get_buffer(), (nbimages - contigous_elts) * q->get_size(), hipMemcpyDeviceToDevice);
    image_2_float << <blocks, threads >> >(output, contigous, q->get_size() * nbimages);
    return output;
  }
  else
  {
    std::cout << "hey" << std::endl;
    image_2_float << <blocks, threads >> >(output, (unsigned char*)q->get_last_images(nbimages), q->get_size() * nbimages);
    return output;
  }
}

float *make_sqrt_vec(int vec_size)
{
  float *vec = (float*)malloc(sizeof(float)* vec_size);
  for (int i = 0; i < vec_size; i++)
    vec[i] = sqrt(i);
  float *vec_gpu;
  hipMalloc(&vec_gpu, sizeof(float)* vec_size);
  hipMemcpy(vec_gpu, vec, sizeof(float)* vec_size, hipMemcpyHostToDevice);
  free(vec);
  return vec_gpu;
}

hipfftComplex *make_contigous_complex(holovibes::Queue *q, int nbimages)
{
  float *sqrt_vec = make_sqrt_vec(256); //think about free it
  int threads = 512;
  int blocks = (q->get_size() * nbimages + 511) / 512;
  int contigous_elts = 0;
  hipfftComplex *output;
  hipMalloc(&output, q->get_size() * nbimages * sizeof(hipfftComplex));
  int index = (q->get_start_index() + q->get_current_elts() - nbimages) % q->get_max_elts();

  if (q->get_max_elts() - index < nbimages)
    contigous_elts = q->get_max_elts() - nbimages;
  else
    contigous_elts = nbimages;
  std::cout << "contigous: " << contigous_elts << std::endl;
  if (contigous_elts < nbimages)
  {
    unsigned char *contigous;
    hipMalloc(&contigous, q->get_size() * nbimages); //modify for 16bit
    hipMemcpy(contigous, q->get_last_images(nbimages), contigous_elts * q->get_size(), hipMemcpyDeviceToDevice);
    hipMemcpy(contigous + contigous_elts * q->get_size(), q->get_buffer(), (nbimages - contigous_elts) * q->get_size(), hipMemcpyDeviceToDevice);
    image_2_complex << <blocks, threads >> >(output, contigous, q->get_size() * nbimages, sqrt_vec);
    hipFree(contigous);
    return output;
  }
  else
  {
    image_2_complex << <blocks, threads >> >(output, (unsigned char*)q->get_last_images(nbimages), q->get_size() * nbimages, sqrt_vec);
    return output;
  }
}
